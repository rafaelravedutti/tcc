
#include <hip/hip_runtime.h>
extern "C" {
typedef struct struct_12604 {
    int e0;
    char* e1;
} struct_12604;
typedef struct struct_12602 {
    float* e0;
    struct_12604 e1;
    struct_12604 e2;
    struct_12604 e3;
    struct_12604 e4;
    int e5;
    int e6;
} struct_12602;
typedef struct array_12689 {
    int e[2];
} array_12689;
typedef struct array_12690 {
    array_12689 e[7];
} array_12690;

__device__ inline int threadIdx_x() { return threadIdx.x; }
__device__ inline int threadIdx_y() { return threadIdx.y; }
__device__ inline int threadIdx_z() { return threadIdx.z; }
__device__ inline int blockIdx_x() { return blockIdx.x; }
__device__ inline int blockIdx_y() { return blockIdx.y; }
__device__ inline int blockIdx_z() { return blockIdx.z; }
__device__ inline int blockDim_x() { return blockDim.x; }
__device__ inline int blockDim_y() { return blockDim.y; }
__device__ inline int blockDim_z() { return blockDim.z; }
__device__ inline int gridDim_x() { return gridDim.x; }
__device__ inline int gridDim_y() { return gridDim.y; }
__device__ inline int gridDim_z() { return gridDim.z; }
__global__ void lambda_58814(float*, float*, float*, struct_12602);
__global__ void lambda_58390();
__global__ void lambda_58645(float*, float*, float*, struct_12602);
__global__ void lambda_58978(float*, float*, float*, float*, struct_12602);
__global__ void lambda_59217(float*, float*, float*, float*, struct_12602);

__global__ void lambda_58814(float* _58817_60498, float* _58818_60499, float* _58819_60500, struct_12602 _58820_60501) {
    int  _60504;
    int p_60504;
    int  _60507;
    int p_60507;
    int  _60510;
    int p_60510;
    int  _60513;
    int p_60513;
    int  _60516;
    int p_60516;
    int  _60519;
    int p_60519;
    #line 1 "/home/rafael/Utilities/new_anydsl/anydsl/runtime/platforms/intrinsics_cuda.impala"
    _60504 = blockIdx_x();
    p_60504 = _60504;
    l60502: ;
        _60504 = p_60504;
        #line 1 "/home/rafael/Utilities/new_anydsl/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _60507 = blockDim_x();
        p_60507 = _60507;
    l60505: ;
        _60507 = p_60507;
        #line 1 "/home/rafael/Utilities/new_anydsl/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _60510 = threadIdx_x();
        p_60510 = _60510;
    l60508: ;
        _60510 = p_60510;
        #line 1 "/home/rafael/Utilities/new_anydsl/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _60513 = blockIdx_y();
        p_60513 = _60513;
    l60511: ;
        _60513 = p_60513;
        #line 1 "/home/rafael/Utilities/new_anydsl/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _60516 = blockDim_y();
        p_60516 = _60516;
    l60514: ;
        _60516 = p_60516;
        #line 1 "/home/rafael/Utilities/new_anydsl/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _60519 = threadIdx_y();
        p_60519 = _60519;
    l60517: ;
        _60519 = p_60519;
        #line 88 "gpu_device.impala"
        int _60520;
        _60520 = _60504 * _60507;
        #line 88 "gpu_device.impala"
        int x_60521;
        x_60521 = _60520 + _60510;
        #line 91 "gpu_device.impala"
        bool _60522;
        _60522 = 1 < x_60521;
        #line 91 "gpu_device.impala"
        if (_60522) goto l60523; else goto l60620;
    l60620: ;
        #line 93 "gpu_device.impala"
        goto l60617;
    l60523: ;
        #line 410 "dsl.impala"
        int _60524;
        _60524 = _58820_60501.e5;
        #line 91 "gpu_device.impala"
        int _60525;
        _60525 = _60524 - 2;
        #line 91 "gpu_device.impala"
        bool _60526;
        _60526 = x_60521 < _60525;
        #line 91 "gpu_device.impala"
        if (_60526) goto l60527; else goto l60619;
    l60619: ;
        #line 93 "gpu_device.impala"
        goto l60617;
    l60527: ;
        #line 89 "gpu_device.impala"
        int _60528;
        _60528 = _60513 * _60516;
        #line 89 "gpu_device.impala"
        int y_60529;
        y_60529 = _60528 + _60519;
        #line 91 "gpu_device.impala"
        bool _60530;
        _60530 = 1 < y_60529;
        #line 91 "gpu_device.impala"
        if (_60530) goto l60531; else goto l60618;
    l60618: ;
        #line 93 "gpu_device.impala"
        goto l60617;
    l60531: ;
        #line 495 "dsl.impala"
        int _60532;
        _60532 = _58820_60501.e6;
        #line 322 "dsl.impala"
        int _60533;
        _60533 = _60532 - 2;
        #line 91 "gpu_device.impala"
        bool _60534;
        _60534 = y_60529 < _60533;
        #line 91 "gpu_device.impala"
        if (_60534) goto l60535; else goto l60616;
    l60616: ;
        #line 93 "gpu_device.impala"
        goto l60617;
    l60617: ;
        return ;
    l60535: ;
        #line 190 "dsl.impala"
        int _60563;
        _60563 = -1 + y_60529;
        #line 9 "gpu_device.impala"
        float* i_60575;
        i_60575 = _58817_60498 + 2;
        #line 185 "dsl.impala"
        int _60536;
        _60536 = y_60529 * _60524;
        #line 9 "gpu_device.impala"
        float* i_60559;
        i_60559 = _58817_60498 + 1;
        #line 190 "dsl.impala"
        int _60589;
        _60589 = 1 + y_60529;
        #line 9 "gpu_device.impala"
        float* i_60586;
        i_60586 = _58817_60498 + 3;
        #line 9 "gpu_device.impala"
        float* i_60601;
        i_60601 = _58817_60498 + 4;
        #line 492 "dsl.impala"
        int _60537;
        _60537 = 2 * _60524;
        #line 9 "gpu_device.impala"
        float* i_60543;
        i_60543 = _58817_60498 + 0;
        #line 190 "dsl.impala"
        int _60604;
        _60604 = 2 + y_60529;
        #line 190 "dsl.impala"
        int _60564;
        _60564 = _60563 * _60524;
        #line 92 "gpu_device.impala"
        int _60538;
        _60538 = x_60521 + _60537;
        #line 190 "dsl.impala"
        int _60547;
        _60547 = -2 + y_60529;
        #line 190 "dsl.impala"
        int _60548;
        _60548 = _60547 * _60524;
        #line 185 "dsl.impala"
        int _60539;
        _60539 = _60536 + _60538;
        #line 190 "dsl.impala"
        int _60590;
        _60590 = _60589 * _60524;
        #line 190 "dsl.impala"
        int _60605;
        _60605 = _60604 * _60524;
        #line 190 "dsl.impala"
        int _60565;
        _60565 = _60564 + _60538;
        #line 190 "dsl.impala"
        int _60549;
        _60549 = _60548 + _60538;
        #line 190 "dsl.impala"
        int _60606;
        _60606 = _60605 + _60538;
        #line 190 "dsl.impala"
        int _60591;
        _60591 = _60590 + _60538;
        #line 185 "dsl.impala"
        int _60540;
        _60540 = 5 + _60539;
        #line 190 "dsl.impala"
        int _60566;
        _60566 = 5 + _60565;
        #line 190 "dsl.impala"
        int _60550;
        _60550 = 5 + _60549;
        #line 190 "dsl.impala"
        int _60607;
        _60607 = 5 + _60606;
        #line 190 "dsl.impala"
        int _60592;
        _60592 = 5 + _60591;
        #line 13 "gpu_device.impala"
        float* i_60541;
        i_60541 = _58818_60499 + _60540;
        #line 9 "gpu_device.impala"
        float* i_60578;
        i_60578 = _58819_60500 + _60540;
        #line 9 "gpu_device.impala"
        float* i_60567;
        i_60567 = _58819_60500 + _60566;
        #line 9 "gpu_device.impala"
        float* i_60551;
        i_60551 = _58819_60500 + _60550;
        #line 9 "gpu_device.impala"
        float* i_60608;
        i_60608 = _58819_60500 + _60607;
        #line 9 "gpu_device.impala"
        float* i_60593;
        i_60593 = _58819_60500 + _60592;
        #line 14 "gpu_device.impala"
        *i_60541 = 0.000000e+00f;
        #line 10 "gpu_device.impala"
        float _60544;
        _60544 = *i_60543;
        #line 10 "gpu_device.impala"
        float _60554;
        _60554 = _60544;
        #line 10 "gpu_device.impala"
        float _60552;
        _60552 = *i_60551;
        #line 10 "gpu_device.impala"
        float _60555;
        _60555 = _60552;
        #line 190 "dsl.impala"
        float _60556;
        _60556 = _60554 * _60555;
        #line 189 "dsl.impala"
        float _60557;
        _60557 = 0.000000e+00f + _60556;
        #line 14 "gpu_device.impala"
        *i_60541 = _60557;
        #line 10 "gpu_device.impala"
        float _60560;
        _60560 = *i_60559;
        #line 10 "gpu_device.impala"
        float _60570;
        _60570 = _60560;
        #line 10 "gpu_device.impala"
        float _60568;
        _60568 = *i_60567;
        #line 10 "gpu_device.impala"
        float _60571;
        _60571 = _60568;
        #line 190 "dsl.impala"
        float _60572;
        _60572 = _60570 * _60571;
        #line 189 "dsl.impala"
        float _60573;
        _60573 = _60557 + _60572;
        #line 14 "gpu_device.impala"
        *i_60541 = _60573;
        #line 10 "gpu_device.impala"
        float _60576;
        _60576 = *i_60575;
        #line 10 "gpu_device.impala"
        float _60581;
        _60581 = _60576;
        #line 10 "gpu_device.impala"
        float _60579;
        _60579 = *i_60578;
        #line 10 "gpu_device.impala"
        float _60582;
        _60582 = _60579;
        #line 190 "dsl.impala"
        float _60583;
        _60583 = _60581 * _60582;
        #line 189 "dsl.impala"
        float _60584;
        _60584 = _60573 + _60583;
        #line 14 "gpu_device.impala"
        *i_60541 = _60584;
        #line 10 "gpu_device.impala"
        float _60587;
        _60587 = *i_60586;
        #line 10 "gpu_device.impala"
        float _60596;
        _60596 = _60587;
        #line 10 "gpu_device.impala"
        float _60594;
        _60594 = *i_60593;
        #line 10 "gpu_device.impala"
        float _60597;
        _60597 = _60594;
        #line 190 "dsl.impala"
        float _60598;
        _60598 = _60596 * _60597;
        #line 189 "dsl.impala"
        float _60599;
        _60599 = _60584 + _60598;
        #line 14 "gpu_device.impala"
        *i_60541 = _60599;
        #line 10 "gpu_device.impala"
        float _60602;
        _60602 = *i_60601;
        #line 10 "gpu_device.impala"
        float _60611;
        _60611 = _60602;
        #line 10 "gpu_device.impala"
        float _60609;
        _60609 = *i_60608;
        #line 10 "gpu_device.impala"
        float _60612;
        _60612 = _60609;
        #line 190 "dsl.impala"
        float _60613;
        _60613 = _60611 * _60612;
        #line 189 "dsl.impala"
        float _60614;
        _60614 = _60599 + _60613;
        #line 14 "gpu_device.impala"
        *i_60541 = _60614;
        return ;
}

__global__ void lambda_58390() {
    int  _60339;
    int p_60339;
    int  _60345;
    int p_60345;
    int  _60351;
    int p_60351;
    int  _60357;
    int p_60357;
    int  _60363;
    int p_60363;
    int  _60369;
    int p_60369;
    #line 1 "/home/rafael/Utilities/new_anydsl/anydsl/runtime/platforms/intrinsics_cuda.impala"
    _60339 = blockIdx_x();
    p_60339 = _60339;
    l60337: ;
        _60339 = p_60339;
        #line 1 "/home/rafael/Utilities/new_anydsl/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _60345 = blockDim_x();
        p_60345 = _60345;
    l60343: ;
        _60345 = p_60345;
        #line 1 "/home/rafael/Utilities/new_anydsl/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _60351 = threadIdx_x();
        p_60351 = _60351;
    l60349: ;
        _60351 = p_60351;
        #line 1 "/home/rafael/Utilities/new_anydsl/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _60357 = blockIdx_y();
        p_60357 = _60357;
    l60355: ;
        _60357 = p_60357;
        #line 1 "/home/rafael/Utilities/new_anydsl/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _60363 = blockDim_y();
        p_60363 = _60363;
    l60361: ;
        _60363 = p_60363;
        #line 1 "/home/rafael/Utilities/new_anydsl/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _60369 = threadIdx_y();
        p_60369 = _60369;
    l60367: ;
        _60369 = p_60369;
        return ;
}

__global__ void lambda_58645(float* _58648_60373, float* _58649_60374, float* _58650_60375, struct_12602 _58651_60376) {
    int  _60379;
    int p_60379;
    int  _60382;
    int p_60382;
    int  _60385;
    int p_60385;
    int  _60388;
    int p_60388;
    int  _60391;
    int p_60391;
    int  _60394;
    int p_60394;
    #line 1 "/home/rafael/Utilities/new_anydsl/anydsl/runtime/platforms/intrinsics_cuda.impala"
    _60379 = blockIdx_x();
    p_60379 = _60379;
    l60377: ;
        _60379 = p_60379;
        #line 1 "/home/rafael/Utilities/new_anydsl/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _60382 = blockDim_x();
        p_60382 = _60382;
    l60380: ;
        _60382 = p_60382;
        #line 1 "/home/rafael/Utilities/new_anydsl/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _60385 = threadIdx_x();
        p_60385 = _60385;
    l60383: ;
        _60385 = p_60385;
        #line 1 "/home/rafael/Utilities/new_anydsl/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _60388 = blockIdx_y();
        p_60388 = _60388;
    l60386: ;
        _60388 = p_60388;
        #line 1 "/home/rafael/Utilities/new_anydsl/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _60391 = blockDim_y();
        p_60391 = _60391;
    l60389: ;
        _60391 = p_60391;
        #line 1 "/home/rafael/Utilities/new_anydsl/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _60394 = threadIdx_y();
        p_60394 = _60394;
    l60392: ;
        _60394 = p_60394;
        #line 88 "gpu_device.impala"
        int _60396;
        _60396 = _60379 * _60382;
        #line 88 "gpu_device.impala"
        int x_60397;
        x_60397 = _60396 + _60385;
        #line 91 "gpu_device.impala"
        bool _60398;
        _60398 = 1 < x_60397;
        #line 91 "gpu_device.impala"
        if (_60398) goto l60399; else goto l60494;
    l60494: ;
        #line 93 "gpu_device.impala"
        goto l60491;
    l60399: ;
        #line 410 "dsl.impala"
        int _60401;
        _60401 = _58651_60376.e5;
        #line 91 "gpu_device.impala"
        int _60403;
        _60403 = _60401 - 2;
        #line 91 "gpu_device.impala"
        bool _60404;
        _60404 = x_60397 < _60403;
        #line 91 "gpu_device.impala"
        if (_60404) goto l60405; else goto l60493;
    l60493: ;
        #line 93 "gpu_device.impala"
        goto l60491;
    l60405: ;
        #line 89 "gpu_device.impala"
        int _60406;
        _60406 = _60388 * _60391;
        #line 89 "gpu_device.impala"
        int y_60407;
        y_60407 = _60406 + _60394;
        #line 91 "gpu_device.impala"
        bool _60408;
        _60408 = 1 < y_60407;
        #line 91 "gpu_device.impala"
        if (_60408) goto l60409; else goto l60492;
    l60492: ;
        #line 93 "gpu_device.impala"
        goto l60491;
    l60409: ;
        #line 495 "dsl.impala"
        int _60411;
        _60411 = _58651_60376.e6;
        #line 322 "dsl.impala"
        int _60412;
        _60412 = _60411 - 2;
        #line 91 "gpu_device.impala"
        bool _60413;
        _60413 = y_60407 < _60412;
        #line 91 "gpu_device.impala"
        if (_60413) goto l60414; else goto l60490;
    l60490: ;
        #line 93 "gpu_device.impala"
        goto l60491;
    l60491: ;
        return ;
    l60414: ;
        #line 170 "dsl.impala"
        int _60416;
        _60416 = y_60407 * _60401;
        #line 9 "gpu_device.impala"
        float* i_60425;
        i_60425 = _58650_60375 + 0;
        #line 9 "gpu_device.impala"
        float* i_60440;
        i_60440 = _58650_60375 + 1;
        #line 9 "gpu_device.impala"
        float* i_60453;
        i_60453 = _58650_60375 + 2;
        #line 9 "gpu_device.impala"
        float* i_60477;
        i_60477 = _58650_60375 + 4;
        #line 9 "gpu_device.impala"
        float* i_60464;
        i_60464 = _58650_60375 + 3;
        #line 492 "dsl.impala"
        int _60417;
        _60417 = 2 * _60401;
        #line 92 "gpu_device.impala"
        int _60418;
        _60418 = x_60397 + _60417;
        #line 170 "dsl.impala"
        int _60419;
        _60419 = _60416 + _60418;
        #line 175 "dsl.impala"
        int _60468;
        _60468 = 6 + _60419;
        #line 170 "dsl.impala"
        int _60420;
        _60420 = 5 + _60419;
        #line 175 "dsl.impala"
        int _60481;
        _60481 = 7 + _60419;
        #line 175 "dsl.impala"
        int _60444;
        _60444 = 4 + _60419;
        #line 175 "dsl.impala"
        int _60430;
        _60430 = 3 + _60419;
        #line 9 "gpu_device.impala"
        float* i_60469;
        i_60469 = _58648_60373 + _60468;
        #line 13 "gpu_device.impala"
        float* i_60421;
        i_60421 = _58649_60374 + _60420;
        #line 9 "gpu_device.impala"
        float* i_60456;
        i_60456 = _58648_60373 + _60420;
        #line 9 "gpu_device.impala"
        float* i_60482;
        i_60482 = _58648_60373 + _60481;
        #line 9 "gpu_device.impala"
        float* i_60445;
        i_60445 = _58648_60373 + _60444;
        #line 9 "gpu_device.impala"
        float* i_60431;
        i_60431 = _58648_60373 + _60430;
        #line 14 "gpu_device.impala"
        *i_60421 = 0.000000e+00f;
        #line 10 "gpu_device.impala"
        float _60426;
        _60426 = *i_60425;
        #line 10 "gpu_device.impala"
        float _60435;
        _60435 = _60426;
        #line 10 "gpu_device.impala"
        float _60432;
        _60432 = *i_60431;
        #line 10 "gpu_device.impala"
        float _60436;
        _60436 = _60432;
        #line 175 "dsl.impala"
        float _60437;
        _60437 = _60435 * _60436;
        #line 174 "dsl.impala"
        float _60438;
        _60438 = 0.000000e+00f + _60437;
        #line 14 "gpu_device.impala"
        *i_60421 = _60438;
        #line 10 "gpu_device.impala"
        float _60441;
        _60441 = *i_60440;
        #line 10 "gpu_device.impala"
        float _60448;
        _60448 = _60441;
        #line 10 "gpu_device.impala"
        float _60446;
        _60446 = *i_60445;
        #line 10 "gpu_device.impala"
        float _60449;
        _60449 = _60446;
        #line 175 "dsl.impala"
        float _60450;
        _60450 = _60448 * _60449;
        #line 174 "dsl.impala"
        float _60451;
        _60451 = _60438 + _60450;
        #line 14 "gpu_device.impala"
        *i_60421 = _60451;
        #line 10 "gpu_device.impala"
        float _60454;
        _60454 = *i_60453;
        #line 10 "gpu_device.impala"
        float _60459;
        _60459 = _60454;
        #line 10 "gpu_device.impala"
        float _60457;
        _60457 = *i_60456;
        #line 10 "gpu_device.impala"
        float _60460;
        _60460 = _60457;
        #line 175 "dsl.impala"
        float _60461;
        _60461 = _60459 * _60460;
        #line 174 "dsl.impala"
        float _60462;
        _60462 = _60451 + _60461;
        #line 14 "gpu_device.impala"
        *i_60421 = _60462;
        #line 10 "gpu_device.impala"
        float _60465;
        _60465 = *i_60464;
        #line 10 "gpu_device.impala"
        float _60472;
        _60472 = _60465;
        #line 10 "gpu_device.impala"
        float _60470;
        _60470 = *i_60469;
        #line 10 "gpu_device.impala"
        float _60473;
        _60473 = _60470;
        #line 175 "dsl.impala"
        float _60474;
        _60474 = _60472 * _60473;
        #line 174 "dsl.impala"
        float _60475;
        _60475 = _60462 + _60474;
        #line 14 "gpu_device.impala"
        *i_60421 = _60475;
        #line 10 "gpu_device.impala"
        float _60478;
        _60478 = *i_60477;
        #line 10 "gpu_device.impala"
        float _60485;
        _60485 = _60478;
        #line 10 "gpu_device.impala"
        float _60483;
        _60483 = *i_60482;
        #line 10 "gpu_device.impala"
        float _60486;
        _60486 = _60483;
        #line 175 "dsl.impala"
        float _60487;
        _60487 = _60485 * _60486;
        #line 174 "dsl.impala"
        float _60488;
        _60488 = _60475 + _60487;
        #line 14 "gpu_device.impala"
        *i_60421 = _60488;
        return ;
}

__global__ void lambda_58978(float* _58981_60624, float* _58982_60625, float* _58983_60626, float* _58984_60627, struct_12602 _58985_60628) {
    int  _60631;
    int p_60631;
    int  _60634;
    int p_60634;
    int  _60637;
    int p_60637;
    int  _60640;
    int p_60640;
    int  _60643;
    int p_60643;
    int  _60646;
    int p_60646;
    bool  converge_60675;
    bool pconverge_60675;
    bool  converge_60677;
    bool pconverge_60677;
    #line 1 "/home/rafael/Utilities/new_anydsl/anydsl/runtime/platforms/intrinsics_cuda.impala"
    _60631 = blockIdx_x();
    p_60631 = _60631;
    l60629: ;
        _60631 = p_60631;
        #line 1 "/home/rafael/Utilities/new_anydsl/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _60634 = blockDim_x();
        p_60634 = _60634;
    l60632: ;
        _60634 = p_60634;
        #line 1 "/home/rafael/Utilities/new_anydsl/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _60637 = threadIdx_x();
        p_60637 = _60637;
    l60635: ;
        _60637 = p_60637;
        #line 1 "/home/rafael/Utilities/new_anydsl/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _60640 = blockIdx_y();
        p_60640 = _60640;
    l60638: ;
        _60640 = p_60640;
        #line 1 "/home/rafael/Utilities/new_anydsl/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _60643 = blockDim_y();
        p_60643 = _60643;
    l60641: ;
        _60643 = p_60643;
        #line 1 "/home/rafael/Utilities/new_anydsl/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _60646 = threadIdx_y();
        p_60646 = _60646;
    l60644: ;
        _60646 = p_60646;
        #line 135 "gpu_device.impala"
        int _60647;
        _60647 = _60631 * _60634;
        #line 135 "gpu_device.impala"
        int x_60648;
        x_60648 = _60647 + _60637;
        #line 138 "gpu_device.impala"
        bool _60649;
        _60649 = 1 < x_60648;
        #line 138 "gpu_device.impala"
        if (_60649) goto l60650; else goto l60804;
    l60804: ;
        #line 140 "gpu_device.impala"
        goto l60801;
    l60650: ;
        #line 410 "dsl.impala"
        int _60651;
        _60651 = _58985_60628.e5;
        #line 91 "gpu_device.impala"
        int _60652;
        _60652 = _60651 - 2;
        #line 138 "gpu_device.impala"
        bool _60653;
        _60653 = x_60648 < _60652;
        #line 138 "gpu_device.impala"
        if (_60653) goto l60654; else goto l60803;
    l60803: ;
        #line 140 "gpu_device.impala"
        goto l60801;
    l60654: ;
        #line 136 "gpu_device.impala"
        int _60655;
        _60655 = _60640 * _60643;
        #line 136 "gpu_device.impala"
        int y_60656;
        y_60656 = _60655 + _60646;
        #line 138 "gpu_device.impala"
        bool _60657;
        _60657 = 1 < y_60656;
        #line 138 "gpu_device.impala"
        if (_60657) goto l60658; else goto l60802;
    l60802: ;
        #line 140 "gpu_device.impala"
        goto l60801;
    l60658: ;
        #line 495 "dsl.impala"
        int _60659;
        _60659 = _58985_60628.e6;
        #line 322 "dsl.impala"
        int _60660;
        _60660 = _60659 - 2;
        #line 138 "gpu_device.impala"
        bool _60661;
        _60661 = y_60656 < _60660;
        #line 138 "gpu_device.impala"
        if (_60661) goto l60662; else goto l60800;
    l60800: ;
        #line 140 "gpu_device.impala"
        goto l60801;
    l60801: ;
        return ;
    l60662: ;
        #line 492 "dsl.impala"
        int _60663;
        _60663 = 2 * _60651;
        #line 321 "dsl.impala"
        int _60664;
        _60664 = 5 + _60663;
        #line 139 "gpu_device.impala"
        int _60665;
        _60665 = x_60648 + _60663;
        #line 139 "gpu_device.impala"
        int _60666;
        _60666 = 5 + _60665;
        #line 321 "dsl.impala"
        bool _60667;
        _60667 = _60664 < _60666;
        #line 321 "dsl.impala"
        if (_60667) goto l60668; else goto l60799;
    l60799: ;
        #line 322 "dsl.impala"
        goto l60798;
    l60668: ;
        #line 321 "dsl.impala"
        int _60669;
        _60669 = 3 * _60651;
        #line 321 "dsl.impala"
        int _60670;
        _60670 = 4 + _60669;
        #line 321 "dsl.impala"
        bool _60671;
        _60671 = _60666 < _60670;
        #line 321 "dsl.impala"
        if (_60671) goto l60672; else goto l60797;
    l60797: ;
        #line 322 "dsl.impala"
        goto l60798;
    l60798: ;
        #line 322 "dsl.impala"
        pconverge_60677 = false;
        goto l60676;
    l60672: ;
        #line 138 "gpu_device.impala"
        if (_60657) goto l60673; else goto l60795;
    l60795: ;
        #line 322 "dsl.impala"
        pconverge_60675 = false;
        goto l60674;
    l60673: ;
        #line 322 "dsl.impala"
        pconverge_60675 = _60661;
        goto l60674;
    l60674: ;
        converge_60675 = pconverge_60675;
        #line 322 "dsl.impala"
        pconverge_60677 = converge_60675;
        goto l60676;
    l60676: ;
        converge_60677 = pconverge_60677;
        #line 247 "dsl.impala"
        int _60683;
        _60683 = y_60656 - 1;
        #line 242 "dsl.impala"
        int _60678;
        _60678 = y_60656 * _60651;
        #line 247 "dsl.impala"
        int _60685;
        _60685 = _60666 - 1;
        #line 242 "dsl.impala"
        int _60679;
        _60679 = _60678 + _60665;
        #line 271 "dsl.impala"
        int _60719;
        _60719 = 6 + _60679;
        #line 253 "dsl.impala"
        int _60693;
        _60693 = _60678 + _60685;
        #line 320 "dsl.impala"
        float condition_60776;
        condition_60776 = (float)converge_60677;
        #line 259 "dsl.impala"
        int _60702;
        _60702 = 1 + y_60656;
        #line 247 "dsl.impala"
        int _60684;
        _60684 = _60683 * _60651;
        #line 259 "dsl.impala"
        int _60703;
        _60703 = _60702 * _60651;
        #line 259 "dsl.impala"
        int _60704;
        _60704 = _60703 + _60685;
        #line 247 "dsl.impala"
        int _60686;
        _60686 = _60684 + _60685;
        #line 242 "dsl.impala"
        int _60680;
        _60680 = 5 + _60679;
        #line 9 "gpu_device.impala"
        float* i_60720;
        i_60720 = _58984_60627 + _60719;
        #line 9 "gpu_device.impala"
        float* i_60694;
        i_60694 = _58984_60627 + _60693;
        #line 265 "dsl.impala"
        int _60711;
        _60711 = _60684 + _60665;
        #line 277 "dsl.impala"
        int _60727;
        _60727 = _60703 + _60665;
        #line 9 "gpu_device.impala"
        float* i_60705;
        i_60705 = _58984_60627 + _60704;
        #line 9 "gpu_device.impala"
        float* i_60687;
        i_60687 = _58984_60627 + _60686;
        #line 13 "gpu_device.impala"
        float* i_60788;
        i_60788 = _58981_60624 + _60680;
        #line 13 "gpu_device.impala"
        float* i_60735;
        i_60735 = _58983_60626 + _60680;
        #line 13 "gpu_device.impala"
        float* i_60681;
        i_60681 = _58982_60625 + _60680;
        #line 265 "dsl.impala"
        int _60712;
        _60712 = 6 + _60711;
        #line 265 "dsl.impala"
        int _60742;
        _60742 = 5 + _60711;
        #line 277 "dsl.impala"
        int _60728;
        _60728 = 6 + _60727;
        #line 277 "dsl.impala"
        int _60760;
        _60760 = 5 + _60727;
        #line 14 "gpu_device.impala"
        *i_60681 = 0.000000e+00f;
        #line 9 "gpu_device.impala"
        float* i_60713;
        i_60713 = _58984_60627 + _60712;
        #line 9 "gpu_device.impala"
        float* i_60743;
        i_60743 = _58984_60627 + _60742;
        #line 9 "gpu_device.impala"
        float* i_60729;
        i_60729 = _58984_60627 + _60728;
        #line 9 "gpu_device.impala"
        float* i_60761;
        i_60761 = _58984_60627 + _60760;
        #line 10 "gpu_device.impala"
        float _60688;
        _60688 = *i_60687;
        #line 10 "gpu_device.impala"
        float _60690;
        _60690 = _60688;
        #line 246 "dsl.impala"
        float _60691;
        _60691 = 0.000000e+00f - _60690;
        #line 14 "gpu_device.impala"
        *i_60681 = _60691;
        #line 10 "gpu_device.impala"
        float _60695;
        _60695 = *i_60694;
        #line 10 "gpu_device.impala"
        float _60698;
        _60698 = _60695;
        #line 253 "dsl.impala"
        float _60699;
        _60699 = 2.000000e+00f * _60698;
        #line 252 "dsl.impala"
        float _60700;
        _60700 = _60691 - _60699;
        #line 14 "gpu_device.impala"
        *i_60681 = _60700;
        #line 10 "gpu_device.impala"
        float _60706;
        _60706 = *i_60705;
        #line 10 "gpu_device.impala"
        float _60708;
        _60708 = _60706;
        #line 258 "dsl.impala"
        float _60709;
        _60709 = _60700 - _60708;
        #line 14 "gpu_device.impala"
        *i_60681 = _60709;
        #line 10 "gpu_device.impala"
        float _60714;
        _60714 = *i_60713;
        #line 10 "gpu_device.impala"
        float _60716;
        _60716 = _60714;
        #line 264 "dsl.impala"
        float _60717;
        _60717 = _60709 + _60716;
        #line 14 "gpu_device.impala"
        *i_60681 = _60717;
        #line 10 "gpu_device.impala"
        float _60721;
        _60721 = *i_60720;
        #line 10 "gpu_device.impala"
        float _60723;
        _60723 = _60721;
        #line 271 "dsl.impala"
        float _60724;
        _60724 = 2.000000e+00f * _60723;
        #line 270 "dsl.impala"
        float _60725;
        _60725 = _60717 + _60724;
        #line 14 "gpu_device.impala"
        *i_60681 = _60725;
        #line 10 "gpu_device.impala"
        float _60730;
        _60730 = *i_60729;
        #line 10 "gpu_device.impala"
        float _60732;
        _60732 = _60730;
        #line 276 "dsl.impala"
        float _60733;
        _60733 = _60725 + _60732;
        #line 14 "gpu_device.impala"
        *i_60681 = _60733;
        #line 14 "gpu_device.impala"
        *i_60735 = 0.000000e+00f;
        #line 10 "gpu_device.impala"
        float _60737;
        _60737 = *i_60687;
        #line 10 "gpu_device.impala"
        float _60739;
        _60739 = _60737;
        #line 285 "dsl.impala"
        float _60740;
        _60740 = 0.000000e+00f - _60739;
        #line 14 "gpu_device.impala"
        *i_60735 = _60740;
        #line 10 "gpu_device.impala"
        float _60744;
        _60744 = *i_60743;
        #line 10 "gpu_device.impala"
        float _60746;
        _60746 = _60744;
        #line 292 "dsl.impala"
        float _60747;
        _60747 = 2.000000e+00f * _60746;
        #line 291 "dsl.impala"
        float _60748;
        _60748 = _60740 - _60747;
        #line 14 "gpu_device.impala"
        *i_60735 = _60748;
        #line 10 "gpu_device.impala"
        float _60750;
        _60750 = *i_60713;
        #line 10 "gpu_device.impala"
        float _60752;
        _60752 = _60750;
        #line 297 "dsl.impala"
        float _60753;
        _60753 = _60748 - _60752;
        #line 14 "gpu_device.impala"
        *i_60735 = _60753;
        #line 10 "gpu_device.impala"
        float _60755;
        _60755 = *i_60705;
        #line 10 "gpu_device.impala"
        float _60757;
        _60757 = _60755;
        #line 303 "dsl.impala"
        float _60758;
        _60758 = _60753 + _60757;
        #line 14 "gpu_device.impala"
        *i_60735 = _60758;
        #line 10 "gpu_device.impala"
        float _60762;
        _60762 = *i_60761;
        #line 10 "gpu_device.impala"
        float _60764;
        _60764 = _60762;
        #line 310 "dsl.impala"
        float _60765;
        _60765 = 2.000000e+00f * _60764;
        #line 309 "dsl.impala"
        float _60766;
        _60766 = _60758 + _60765;
        #line 14 "gpu_device.impala"
        *i_60735 = _60766;
        #line 10 "gpu_device.impala"
        float _60768;
        _60768 = *i_60729;
        #line 10 "gpu_device.impala"
        float _60770;
        _60770 = _60768;
        #line 315 "dsl.impala"
        float _60771;
        _60771 = _60766 + _60770;
        #line 14 "gpu_device.impala"
        *i_60735 = _60771;
        #line 10 "gpu_device.impala"
        float _60773;
        _60773 = *i_60681;
        #line 10 "gpu_device.impala"
        float _60775;
        _60775 = _60773;
        #line 327 "dsl.impala"
        float _60777;
        _60777 = _60775 * condition_60776;
        #line 14 "gpu_device.impala"
        *i_60681 = _60777;
        #line 10 "gpu_device.impala"
        float _60779;
        _60779 = *i_60735;
        #line 10 "gpu_device.impala"
        float _60781;
        _60781 = _60779;
        #line 332 "dsl.impala"
        float _60782;
        _60782 = _60781 * condition_60776;
        #line 14 "gpu_device.impala"
        *i_60735 = _60782;
        #line 10 "gpu_device.impala"
        float _60784;
        _60784 = *i_60681;
        #line 10 "gpu_device.impala"
        float _60789;
        _60789 = _60784;
        #line 10 "gpu_device.impala"
        float _60786;
        _60786 = *i_60735;
        #line 337 "dsl.impala"
        float _60790;
        _60790 = _60789 * _60789;
        #line 10 "gpu_device.impala"
        float _60791;
        _60791 = _60786;
        #line 338 "dsl.impala"
        float _60792;
        _60792 = _60791 * _60791;
        #line 337 "dsl.impala"
        float _60793;
        _60793 = _60790 + _60792;
        #line 14 "gpu_device.impala"
        *i_60788 = _60793;
        return ;
}

__global__ void lambda_59217(float* _59220_60808, float* _59221_60809, float* _59222_60810, float* _59223_60811, struct_12602 _59224_60812) {
    int  _60815;
    int p_60815;
    int  _60818;
    int p_60818;
    int  _60821;
    int p_60821;
    int  _60824;
    int p_60824;
    int  _60827;
    int p_60827;
    int  _60830;
    int p_60830;
    #line 1 "/home/rafael/Utilities/new_anydsl/anydsl/runtime/platforms/intrinsics_cuda.impala"
    _60815 = blockIdx_x();
    p_60815 = _60815;
    l60813: ;
        _60815 = p_60815;
        #line 1 "/home/rafael/Utilities/new_anydsl/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _60818 = blockDim_x();
        p_60818 = _60818;
    l60816: ;
        _60818 = p_60818;
        #line 1 "/home/rafael/Utilities/new_anydsl/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _60821 = threadIdx_x();
        p_60821 = _60821;
    l60819: ;
        _60821 = p_60821;
        #line 1 "/home/rafael/Utilities/new_anydsl/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _60824 = blockIdx_y();
        p_60824 = _60824;
    l60822: ;
        _60824 = p_60824;
        #line 1 "/home/rafael/Utilities/new_anydsl/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _60827 = blockDim_y();
        p_60827 = _60827;
    l60825: ;
        _60827 = p_60827;
        #line 1 "/home/rafael/Utilities/new_anydsl/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _60830 = threadIdx_y();
        p_60830 = _60830;
    l60828: ;
        _60830 = p_60830;
        #line 135 "gpu_device.impala"
        int _60831;
        _60831 = _60815 * _60818;
        #line 135 "gpu_device.impala"
        int x_60832;
        x_60832 = _60831 + _60821;
        #line 138 "gpu_device.impala"
        bool _60833;
        _60833 = 1 < x_60832;
        #line 138 "gpu_device.impala"
        if (_60833) goto l60834; else goto l60955;
    l60955: ;
        #line 140 "gpu_device.impala"
        goto l60952;
    l60834: ;
        #line 410 "dsl.impala"
        int _60835;
        _60835 = _59224_60812.e5;
        #line 91 "gpu_device.impala"
        int _60836;
        _60836 = _60835 - 2;
        #line 138 "gpu_device.impala"
        bool _60837;
        _60837 = x_60832 < _60836;
        #line 138 "gpu_device.impala"
        if (_60837) goto l60838; else goto l60954;
    l60954: ;
        #line 140 "gpu_device.impala"
        goto l60952;
    l60838: ;
        #line 136 "gpu_device.impala"
        int _60839;
        _60839 = _60824 * _60827;
        #line 136 "gpu_device.impala"
        int y_60840;
        y_60840 = _60839 + _60830;
        #line 138 "gpu_device.impala"
        bool _60841;
        _60841 = 1 < y_60840;
        #line 138 "gpu_device.impala"
        if (_60841) goto l60842; else goto l60953;
    l60953: ;
        #line 140 "gpu_device.impala"
        goto l60952;
    l60842: ;
        #line 495 "dsl.impala"
        int _60843;
        _60843 = _59224_60812.e6;
        #line 322 "dsl.impala"
        int _60844;
        _60844 = _60843 - 2;
        #line 138 "gpu_device.impala"
        bool _60845;
        _60845 = y_60840 < _60844;
        #line 138 "gpu_device.impala"
        if (_60845) goto l60846; else goto l60951;
    l60951: ;
        #line 140 "gpu_device.impala"
        goto l60952;
    l60952: ;
        return ;
    l60846: ;
        #line 492 "dsl.impala"
        int _60848;
        _60848 = 2 * _60835;
        #line 354 "dsl.impala"
        int _60847;
        _60847 = y_60840 * _60835;
        #line 139 "gpu_device.impala"
        int _60849;
        _60849 = x_60832 + _60848;
        #line 139 "gpu_device.impala"
        int _60919;
        _60919 = 5 + _60849;
        #line 354 "dsl.impala"
        int _60850;
        _60850 = _60847 + _60849;
        #line 354 "dsl.impala"
        int _60851;
        _60851 = 5 + _60850;
        #line 13 "gpu_device.impala"
        float* i_60927;
        i_60927 = _59220_60808 + _60851;
        #line 9 "gpu_device.impala"
        float* i_60858;
        i_60858 = _59223_60811 + _60851;
        #line 9 "gpu_device.impala"
        float* i_60855;
        i_60855 = _59222_60810 + _60851;
        #line 9 "gpu_device.impala"
        float* i_60852;
        i_60852 = _59221_60809 + _60851;
        #line 10 "gpu_device.impala"
        float _60853;
        _60853 = *i_60852;
        #line 10 "gpu_device.impala"
        float _60879;
        _60879 = _60853;
        #line 10 "gpu_device.impala"
        float _60856;
        _60856 = *i_60855;
        #line 354 "dsl.impala"
        int xs_60880;
        xs_60880 = (int)_60879;
        #line 10 "gpu_device.impala"
        float _60866;
        _60866 = _60856;
        #line 146 "dsl.impala"
        bool _60881;
        _60881 = 0 <= xs_60880;
        #line 147 "dsl.impala"
        int _60884;
        _60884 = 0 - xs_60880;
        #line 147 "dsl.impala"
        bool _60885;
        _60885 = xs_60880 < 0;
        #line 10 "gpu_device.impala"
        float _60859;
        _60859 = *i_60858;
        #line 355 "dsl.impala"
        int ys_60867;
        ys_60867 = (int)_60866;
        #line 146 "dsl.impala"
        int _60882;
        _60882 = (int)_60881;
        #line 147 "dsl.impala"
        int _60886;
        _60886 = (int)_60885;
        #line 10 "gpu_device.impala"
        float _60941;
        _60941 = _60859;
        #line 147 "dsl.impala"
        bool _60872;
        _60872 = ys_60867 < 0;
        #line 364 "dsl.impala"
        int _60900;
        _60900 = xs_60880 ^ ys_60867;
        #line 146 "dsl.impala"
        bool _60868;
        _60868 = 0 <= ys_60867;
        #line 147 "dsl.impala"
        int _60871;
        _60871 = 0 - ys_60867;
        #line 146 "dsl.impala"
        int _60883;
        _60883 = xs_60880 * _60882;
        #line 147 "dsl.impala"
        int _60887;
        _60887 = _60884 * _60886;
        #line 147 "dsl.impala"
        int _60873;
        _60873 = (int)_60872;
        #line 364 "dsl.impala"
        bool _60901;
        _60901 = _60900 < 0;
        #line 146 "dsl.impala"
        int _60869;
        _60869 = (int)_60868;
        #line 147 "dsl.impala"
        int _60874;
        _60874 = _60871 * _60873;
        #line 146 "dsl.impala"
        int _60888;
        _60888 = _60883 + _60887;
        #line 364 "dsl.impala"
        int cond3_60902;
        cond3_60902 = (int)_60901;
        #line 146 "dsl.impala"
        int _60870;
        _60870 = ys_60867 * _60869;
        #line 146 "dsl.impala"
        int _60875;
        _60875 = _60870 + _60874;
        #line 359 "dsl.impala"
        int tg22x_60889;
        tg22x_60889 = 13573 * _60888;
        #line 360 "dsl.impala"
        int _60894;
        _60894 = _60888 << 16;
        #line 357 "dsl.impala"
        int my_60877;
        my_60877 = _60875 << 15;
        #line 362 "dsl.impala"
        bool _60890;
        _60890 = my_60877 < tg22x_60889;
        #line 360 "dsl.impala"
        int tg67x_60895;
        tg67x_60895 = tg22x_60889 + _60894;
        #line 363 "dsl.impala"
        bool _60896;
        _60896 = tg67x_60895 < my_60877;
        #line 362 "dsl.impala"
        int cond1_60891;
        cond1_60891 = (int)_60890;
        #line 363 "dsl.impala"
        int cond2_60897;
        cond2_60897 = (int)_60896;
        #line 366 "dsl.impala"
        int _60892;
        _60892 = cond1_60891 << 2;
        #line 366 "dsl.impala"
        int _60898;
        _60898 = cond2_60897 << 1;
        #line 366 "dsl.impala"
        int _60899;
        _60899 = _60892 + _60898;
        #line 366 "dsl.impala"
        int index_60903;
        index_60903 = _60899 + cond3_60902;
        #line 367 "dsl.impala"
        array_12689 _60861_93;
        _60861_93.e[0] = 1;
        _60861_93.e[1] = 1;
        array_12689 _60862_96;
        _60862_96.e[0] = -1;
        _60862_96.e[1] = 1;
        array_12689 _60863_99;
        _60863_99.e[0] = 0;
        _60863_99.e[1] = 1;
        array_12689 _60864_102;
        _60864_102.e[0] = 1;
        _60864_102.e[1] = 0;
        array_12690 offsets_60865_105;
        offsets_60865_105.e[0] = _60861_93;
        offsets_60865_105.e[1] = _60862_96;
        offsets_60865_105.e[2] = _60863_99;
        offsets_60865_105.e[3] = _60863_99;
        offsets_60865_105.e[4] = _60864_102;
        offsets_60865_105.e[5] = _60864_102;
        offsets_60865_105.e[6] = _60864_102;
        array_12689 _60904;
        _60904 = offsets_60865_105.e[index_60903];
        #line 367 "dsl.impala"
        int _60908;
        _60908 = _60904.e[0];
        #line 368 "dsl.impala"
        int _60905;
        _60905 = _60904.e[1];
        #line 369 "dsl.impala"
        int nb2_x_60920;
        nb2_x_60920 = _60919 - _60908;
        #line 367 "dsl.impala"
        int nb1_x_60909;
        nb1_x_60909 = _60849 + _60908;
        #line 370 "dsl.impala"
        int nb2_y_60917;
        nb2_y_60917 = y_60840 - _60905;
        #line 368 "dsl.impala"
        int nb1_y_60906;
        nb1_y_60906 = y_60840 + _60905;
        #line 377 "dsl.impala"
        int _60918;
        _60918 = nb2_y_60917 * _60835;
        #line 373 "dsl.impala"
        int _60907;
        _60907 = nb1_y_60906 * _60835;
        #line 377 "dsl.impala"
        int _60921;
        _60921 = _60918 + nb2_x_60920;
        #line 373 "dsl.impala"
        int _60910;
        _60910 = _60907 + nb1_x_60909;
        #line 9 "gpu_device.impala"
        float* i_60922;
        i_60922 = _59223_60811 + _60921;
        #line 373 "dsl.impala"
        int _60911;
        _60911 = 5 + _60910;
        #line 9 "gpu_device.impala"
        float* i_60912;
        i_60912 = _59223_60811 + _60911;
        #line 10 "gpu_device.impala"
        float _60913;
        _60913 = *i_60912;
        #line 10 "gpu_device.impala"
        float _60940;
        _60940 = _60913;
        #line 373 "dsl.impala"
        bool _60942;
        _60942 = _60940 < _60941;
        #line 10 "gpu_device.impala"
        float _60915;
        _60915 = *i_60858;
        #line 372 "dsl.impala"
        float nb1_cond_60943;
        nb1_cond_60943 = (float)_60942;
        #line 10 "gpu_device.impala"
        float _60946;
        _60946 = _60915;
        #line 10 "gpu_device.impala"
        float _60923;
        _60923 = *i_60922;
        #line 10 "gpu_device.impala"
        float _60945;
        _60945 = _60923;
        #line 10 "gpu_device.impala"
        float _60925;
        _60925 = *i_60858;
        #line 377 "dsl.impala"
        bool _60947;
        _60947 = _60945 < _60946;
        #line 10 "gpu_device.impala"
        float _60930;
        _60930 = _60925;
        #line 376 "dsl.impala"
        float nb2_cond_60948;
        nb2_cond_60948 = (float)_60947;
        #line 397 "dsl.impala"
        bool _60931;
        _60931 = 1.600000e+03f < _60930;
        #line 398 "dsl.impala"
        bool _60936;
        _60936 = 1.440000e+04f < _60930;
        #line 397 "dsl.impala"
        float _60932;
        _60932 = (float)_60931;
        #line 398 "dsl.impala"
        float _60937;
        _60937 = (float)_60936;
        #line 397 "dsl.impala"
        float _60933;
        _60933 = 1.000000e+00f * _60932;
        #line 398 "dsl.impala"
        float _60938;
        _60938 = 1.400000e+01f * _60937;
        #line 397 "dsl.impala"
        float _60939;
        _60939 = _60933 + _60938;
        #line 396 "dsl.impala"
        float _60944;
        _60944 = _60939 * nb1_cond_60943;
        #line 396 "dsl.impala"
        float _60949;
        _60949 = _60944 * nb2_cond_60948;
        #line 14 "gpu_device.impala"
        *i_60927 = _60949;
        return ;
}

}