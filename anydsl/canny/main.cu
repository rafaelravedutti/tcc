
#include <hip/hip_runtime.h>
extern "C" {
typedef struct struct_10057 {
    int e0;
    char* e1;
} struct_10057;
typedef struct struct_10055 {
    float* e0;
    int e1;
    int e2;
    struct_10057 e3;
    struct_10057 e4;
    struct_10057 e5;
    struct_10057 e6;
    struct_10057 e7;
    struct_10057 e8;
    struct_10057 e9;
    struct_10057 e10;
    struct_10057 e11;
} struct_10055;

__device__ inline int threadIdx_x() { return threadIdx.x; }
__device__ inline int threadIdx_y() { return threadIdx.y; }
__device__ inline int threadIdx_z() { return threadIdx.z; }
__device__ inline int blockIdx_x() { return blockIdx.x; }
__device__ inline int blockIdx_y() { return blockIdx.y; }
__device__ inline int blockIdx_z() { return blockIdx.z; }
__device__ inline int blockDim_x() { return blockDim.x; }
__device__ inline int blockDim_y() { return blockDim.y; }
__device__ inline int blockDim_z() { return blockDim.z; }
__device__ inline int gridDim_x() { return gridDim.x; }
__device__ inline int gridDim_y() { return gridDim.y; }
__device__ inline int gridDim_z() { return gridDim.z; }
__global__ void lambda_49095(float*, float*, struct_10055, float*);
__global__ void lambda_49587(float*, float*, struct_10055, float*, float*, float*);
__global__ void lambda_49385(float*, float*, struct_10055, float*, float*);
__global__ void lambda_48861();
__global__ void lambda_49241(float*, float*, struct_10055, float*);

__global__ void lambda_49095(float* _49098_50457, float* _49099_50458, struct_10055 _49100_50459, float* _49101_50460) {
    int  _50463;
    int p_50463;
    int  _50466;
    int p_50466;
    int  _50469;
    int p_50469;
    int  _50472;
    int p_50472;
    int  _50475;
    int p_50475;
    int  _50478;
    int p_50478;
    #line 1 "/home/rafael/Utilities/new_anydsl/anydsl/runtime/platforms/intrinsics_cuda.impala"
    _50463 = blockIdx_x();
    p_50463 = _50463;
    l50461: ;
        _50463 = p_50463;
        #line 1 "/home/rafael/Utilities/new_anydsl/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _50466 = blockDim_x();
        p_50466 = _50466;
    l50464: ;
        _50466 = p_50466;
        #line 1 "/home/rafael/Utilities/new_anydsl/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _50469 = threadIdx_x();
        p_50469 = _50469;
    l50467: ;
        _50469 = p_50469;
        #line 1 "/home/rafael/Utilities/new_anydsl/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _50472 = blockIdx_y();
        p_50472 = _50472;
    l50470: ;
        _50472 = p_50472;
        #line 1 "/home/rafael/Utilities/new_anydsl/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _50475 = blockDim_y();
        p_50475 = _50475;
    l50473: ;
        _50475 = p_50475;
        #line 1 "/home/rafael/Utilities/new_anydsl/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _50478 = threadIdx_y();
        p_50478 = _50478;
    l50476: ;
        _50478 = p_50478;
        #line 120 "gpu_device.impala"
        int _50480;
        _50480 = _50463 * _50466;
        #line 120 "gpu_device.impala"
        int x_50481;
        x_50481 = _50480 + _50469;
        #line 123 "gpu_device.impala"
        bool _50482;
        _50482 = 1 < x_50481;
        #line 123 "gpu_device.impala"
        if (_50482) goto l50483; else goto l50577;
    l50577: ;
        #line 125 "gpu_device.impala"
        goto l50574;
    l50483: ;
        #line 427 "dsl.impala"
        int _50485;
        _50485 = _49100_50459.e1;
        #line 123 "gpu_device.impala"
        int _50487;
        _50487 = _50485 - 2;
        #line 123 "gpu_device.impala"
        bool _50488;
        _50488 = x_50481 < _50487;
        #line 123 "gpu_device.impala"
        if (_50488) goto l50489; else goto l50576;
    l50576: ;
        #line 125 "gpu_device.impala"
        goto l50574;
    l50489: ;
        #line 121 "gpu_device.impala"
        int _50490;
        _50490 = _50472 * _50475;
        #line 121 "gpu_device.impala"
        int y_50491;
        y_50491 = _50490 + _50478;
        #line 123 "gpu_device.impala"
        bool _50492;
        _50492 = 1 < y_50491;
        #line 123 "gpu_device.impala"
        if (_50492) goto l50493; else goto l50575;
    l50575: ;
        #line 125 "gpu_device.impala"
        goto l50574;
    l50493: ;
        #line 511 "dsl.impala"
        int _50495;
        _50495 = _49100_50459.e2;
        #line 341 "dsl.impala"
        int _50496;
        _50496 = _50495 - 2;
        #line 123 "gpu_device.impala"
        bool _50497;
        _50497 = y_50491 < _50496;
        #line 123 "gpu_device.impala"
        if (_50497) goto l50498; else goto l50573;
    l50573: ;
        #line 125 "gpu_device.impala"
        goto l50574;
    l50574: ;
        return ;
    l50498: ;
        #line 11 "gpu_device.impala"
        float* i_50547;
        i_50547 = _49099_50458 + 3;
        #line 11 "gpu_device.impala"
        float* i_50560;
        i_50560 = _49099_50458 + 4;
        #line 189 "dsl.impala"
        int _50500;
        _50500 = y_50491 * _50485;
        #line 11 "gpu_device.impala"
        float* i_50536;
        i_50536 = _49099_50458 + 2;
        #line 11 "gpu_device.impala"
        float* i_50523;
        i_50523 = _49099_50458 + 1;
        #line 508 "dsl.impala"
        int _50501;
        _50501 = 2 * _50485;
        #line 11 "gpu_device.impala"
        float* i_50509;
        i_50509 = _49099_50458 + 0;
        #line 124 "gpu_device.impala"
        int _50502;
        _50502 = x_50481 + _50501;
        #line 189 "dsl.impala"
        int _50503;
        _50503 = _50500 + _50502;
        #line 194 "dsl.impala"
        int _50551;
        _50551 = 6 + _50503;
        #line 194 "dsl.impala"
        int _50514;
        _50514 = 3 + _50503;
        #line 194 "dsl.impala"
        int _50564;
        _50564 = 7 + _50503;
        #line 194 "dsl.impala"
        int _50527;
        _50527 = 4 + _50503;
        #line 189 "dsl.impala"
        int _50504;
        _50504 = 5 + _50503;
        #line 11 "gpu_device.impala"
        float* i_50552;
        i_50552 = _49098_50457 + _50551;
        #line 11 "gpu_device.impala"
        float* i_50515;
        i_50515 = _49098_50457 + _50514;
        #line 11 "gpu_device.impala"
        float* i_50565;
        i_50565 = _49098_50457 + _50564;
        #line 11 "gpu_device.impala"
        float* i_50528;
        i_50528 = _49098_50457 + _50527;
        #line 15 "gpu_device.impala"
        float* i_50505;
        i_50505 = _49101_50460 + _50504;
        #line 11 "gpu_device.impala"
        float* i_50539;
        i_50539 = _49098_50457 + _50504;
        #line 16 "gpu_device.impala"
        *i_50505 = 0.000000e+00f;
        #line 12 "gpu_device.impala"
        float _50510;
        _50510 = *i_50509;
        #line 12 "gpu_device.impala"
        float _50518;
        _50518 = _50510;
        #line 12 "gpu_device.impala"
        float _50516;
        _50516 = *i_50515;
        #line 12 "gpu_device.impala"
        float _50519;
        _50519 = _50516;
        #line 194 "dsl.impala"
        float _50520;
        _50520 = _50518 * _50519;
        #line 193 "dsl.impala"
        float _50521;
        _50521 = 0.000000e+00f + _50520;
        #line 16 "gpu_device.impala"
        *i_50505 = _50521;
        #line 12 "gpu_device.impala"
        float _50524;
        _50524 = *i_50523;
        #line 12 "gpu_device.impala"
        float _50531;
        _50531 = _50524;
        #line 12 "gpu_device.impala"
        float _50529;
        _50529 = *i_50528;
        #line 12 "gpu_device.impala"
        float _50532;
        _50532 = _50529;
        #line 194 "dsl.impala"
        float _50533;
        _50533 = _50531 * _50532;
        #line 193 "dsl.impala"
        float _50534;
        _50534 = _50521 + _50533;
        #line 16 "gpu_device.impala"
        *i_50505 = _50534;
        #line 12 "gpu_device.impala"
        float _50537;
        _50537 = *i_50536;
        #line 12 "gpu_device.impala"
        float _50542;
        _50542 = _50537;
        #line 12 "gpu_device.impala"
        float _50540;
        _50540 = *i_50539;
        #line 12 "gpu_device.impala"
        float _50543;
        _50543 = _50540;
        #line 194 "dsl.impala"
        float _50544;
        _50544 = _50542 * _50543;
        #line 193 "dsl.impala"
        float _50545;
        _50545 = _50534 + _50544;
        #line 16 "gpu_device.impala"
        *i_50505 = _50545;
        #line 12 "gpu_device.impala"
        float _50548;
        _50548 = *i_50547;
        #line 12 "gpu_device.impala"
        float _50555;
        _50555 = _50548;
        #line 12 "gpu_device.impala"
        float _50553;
        _50553 = *i_50552;
        #line 12 "gpu_device.impala"
        float _50556;
        _50556 = _50553;
        #line 194 "dsl.impala"
        float _50557;
        _50557 = _50555 * _50556;
        #line 193 "dsl.impala"
        float _50558;
        _50558 = _50545 + _50557;
        #line 16 "gpu_device.impala"
        *i_50505 = _50558;
        #line 12 "gpu_device.impala"
        float _50561;
        _50561 = *i_50560;
        #line 12 "gpu_device.impala"
        float _50568;
        _50568 = _50561;
        #line 12 "gpu_device.impala"
        float _50566;
        _50566 = *i_50565;
        #line 12 "gpu_device.impala"
        float _50569;
        _50569 = _50566;
        #line 194 "dsl.impala"
        float _50570;
        _50570 = _50568 * _50569;
        #line 193 "dsl.impala"
        float _50571;
        _50571 = _50558 + _50570;
        #line 16 "gpu_device.impala"
        *i_50505 = _50571;
        return ;
}

__global__ void lambda_49587(float* _49590_50891, float* _49591_50892, struct_10055 _49592_50893, float* _49593_50894, float* _49594_50895, float* _49595_50896) {
    int  _50899;
    int p_50899;
    int  _50902;
    int p_50902;
    int  _50905;
    int p_50905;
    int  _50908;
    int p_50908;
    int  _50911;
    int p_50911;
    int  _50914;
    int p_50914;
    #line 1 "/home/rafael/Utilities/new_anydsl/anydsl/runtime/platforms/intrinsics_cuda.impala"
    _50899 = blockIdx_x();
    p_50899 = _50899;
    l50897: ;
        _50899 = p_50899;
        #line 1 "/home/rafael/Utilities/new_anydsl/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _50902 = blockDim_x();
        p_50902 = _50902;
    l50900: ;
        _50902 = p_50902;
        #line 1 "/home/rafael/Utilities/new_anydsl/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _50905 = threadIdx_x();
        p_50905 = _50905;
    l50903: ;
        _50905 = p_50905;
        #line 1 "/home/rafael/Utilities/new_anydsl/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _50908 = blockIdx_y();
        p_50908 = _50908;
    l50906: ;
        _50908 = p_50908;
        #line 1 "/home/rafael/Utilities/new_anydsl/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _50911 = blockDim_y();
        p_50911 = _50911;
    l50909: ;
        _50911 = p_50911;
        #line 1 "/home/rafael/Utilities/new_anydsl/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _50914 = threadIdx_y();
        p_50914 = _50914;
    l50912: ;
        _50914 = p_50914;
        #line 153 "gpu_device.impala"
        int _50915;
        _50915 = _50899 * _50902;
        #line 153 "gpu_device.impala"
        int x_50916;
        x_50916 = _50915 + _50905;
        #line 156 "gpu_device.impala"
        bool _50917;
        _50917 = 1 < x_50916;
        #line 156 "gpu_device.impala"
        if (_50917) goto l50918; else goto l51051;
    l51051: ;
        #line 158 "gpu_device.impala"
        goto l51048;
    l50918: ;
        #line 427 "dsl.impala"
        int _50919;
        _50919 = _49592_50893.e1;
        #line 123 "gpu_device.impala"
        int _50920;
        _50920 = _50919 - 2;
        #line 156 "gpu_device.impala"
        bool _50921;
        _50921 = x_50916 < _50920;
        #line 156 "gpu_device.impala"
        if (_50921) goto l50922; else goto l51050;
    l51050: ;
        #line 158 "gpu_device.impala"
        goto l51048;
    l50922: ;
        #line 154 "gpu_device.impala"
        int _50923;
        _50923 = _50908 * _50911;
        #line 154 "gpu_device.impala"
        int y_50924;
        y_50924 = _50923 + _50914;
        #line 156 "gpu_device.impala"
        bool _50925;
        _50925 = 1 < y_50924;
        #line 156 "gpu_device.impala"
        if (_50925) goto l50926; else goto l51049;
    l51049: ;
        #line 158 "gpu_device.impala"
        goto l51048;
    l50926: ;
        #line 511 "dsl.impala"
        int _50927;
        _50927 = _49592_50893.e2;
        #line 341 "dsl.impala"
        int _50928;
        _50928 = _50927 - 2;
        #line 156 "gpu_device.impala"
        bool _50929;
        _50929 = y_50924 < _50928;
        #line 156 "gpu_device.impala"
        if (_50929) goto l50930; else goto l51047;
    l51047: ;
        #line 158 "gpu_device.impala"
        goto l51048;
    l51048: ;
        return ;
    l50930: ;
        #line 508 "dsl.impala"
        int _50932;
        _50932 = 2 * _50919;
        #line 370 "dsl.impala"
        int _50931;
        _50931 = y_50924 * _50919;
        #line 157 "gpu_device.impala"
        int _50933;
        _50933 = x_50916 + _50932;
        #line 370 "dsl.impala"
        int pos_50934;
        pos_50934 = _50931 + _50933;
        #line 157 "gpu_device.impala"
        int _51013;
        _51013 = 5 + _50933;
        #line 370 "dsl.impala"
        int pos_50935;
        pos_50935 = 5 + pos_50934;
        #line 15 "gpu_device.impala"
        float* i_51023;
        i_51023 = _49594_50895 + pos_50935;
        #line 11 "gpu_device.impala"
        float* i_50992;
        i_50992 = _49590_50891 + pos_50935;
        #line 11 "gpu_device.impala"
        float* i_50939;
        i_50939 = _49593_50894 + pos_50935;
        #line 11 "gpu_device.impala"
        float* i_50936;
        i_50936 = _49595_50896 + pos_50935;
        #line 12 "gpu_device.impala"
        float _50937;
        _50937 = *i_50936;
        #line 12 "gpu_device.impala"
        float _50955;
        _50955 = _50937;
        #line 12 "gpu_device.impala"
        float _50940;
        _50940 = *i_50939;
        #line 371 "dsl.impala"
        int xs_50956;
        xs_50956 = (int)_50955;
        #line 12 "gpu_device.impala"
        float _50942;
        _50942 = _50940;
        #line 166 "dsl.impala"
        bool _50961;
        _50961 = xs_50956 < 0;
        #line 165 "dsl.impala"
        bool _50957;
        _50957 = 0 <= xs_50956;
        #line 166 "dsl.impala"
        int _50960;
        _50960 = 0 - xs_50956;
        #line 372 "dsl.impala"
        int ys_50943;
        ys_50943 = (int)_50942;
        #line 166 "dsl.impala"
        int _50962;
        _50962 = (int)_50961;
        #line 165 "dsl.impala"
        int _50958;
        _50958 = (int)_50957;
        #line 166 "dsl.impala"
        int _50963;
        _50963 = _50960 * _50962;
        #line 166 "dsl.impala"
        bool _50948;
        _50948 = ys_50943 < 0;
        #line 166 "dsl.impala"
        int _50947;
        _50947 = 0 - ys_50943;
        #line 381 "dsl.impala"
        int _50976;
        _50976 = xs_50956 ^ ys_50943;
        #line 165 "dsl.impala"
        bool _50944;
        _50944 = 0 <= ys_50943;
        #line 165 "dsl.impala"
        int _50959;
        _50959 = xs_50956 * _50958;
        #line 165 "dsl.impala"
        int _50964;
        _50964 = _50959 + _50963;
        #line 166 "dsl.impala"
        int _50949;
        _50949 = (int)_50948;
        #line 166 "dsl.impala"
        int _50950;
        _50950 = _50947 * _50949;
        #line 381 "dsl.impala"
        bool _50977;
        _50977 = _50976 < 0;
        #line 165 "dsl.impala"
        int _50945;
        _50945 = (int)_50944;
        #line 376 "dsl.impala"
        int tg22x_50965;
        tg22x_50965 = 13573 * _50964;
        #line 377 "dsl.impala"
        int _50970;
        _50970 = _50964 << 16;
        #line 381 "dsl.impala"
        int cond3_50978;
        cond3_50978 = (int)_50977;
        #line 165 "dsl.impala"
        int _50946;
        _50946 = ys_50943 * _50945;
        #line 377 "dsl.impala"
        int tg67x_50971;
        tg67x_50971 = tg22x_50965 + _50970;
        #line 165 "dsl.impala"
        int _50951;
        _50951 = _50946 + _50950;
        #line 374 "dsl.impala"
        int my_50953;
        my_50953 = _50951 << 15;
        #line 379 "dsl.impala"
        bool _50966;
        _50966 = my_50953 < tg22x_50965;
        #line 380 "dsl.impala"
        bool _50972;
        _50972 = tg67x_50971 < my_50953;
        #line 379 "dsl.impala"
        int cond1_50967;
        cond1_50967 = (int)_50966;
        #line 380 "dsl.impala"
        int cond2_50973;
        cond2_50973 = (int)_50972;
        #line 383 "dsl.impala"
        int _50968;
        _50968 = cond1_50967 << 2;
        #line 383 "dsl.impala"
        int _50974;
        _50974 = cond2_50973 << 1;
        #line 383 "dsl.impala"
        int _50975;
        _50975 = _50968 + _50974;
        #line 383 "dsl.impala"
        int index_50979;
        index_50979 = _50975 + cond3_50978;
        #line 384 "dsl.impala"
        int _50980;
        _50980 = 2 * index_50979;
        #line 385 "dsl.impala"
        int _50984;
        _50984 = 1 + _50980;
        #line 11 "gpu_device.impala"
        float* i_50981;
        i_50981 = _49591_50892 + _50980;
        #line 11 "gpu_device.impala"
        float* i_50985;
        i_50985 = _49591_50892 + _50984;
        #line 12 "gpu_device.impala"
        float _50982;
        _50982 = *i_50981;
        #line 12 "gpu_device.impala"
        float _50999;
        _50999 = _50982;
        #line 384 "dsl.impala"
        int _51000;
        _51000 = (int)_50999;
        #line 12 "gpu_device.impala"
        float _50986;
        _50986 = *i_50985;
        #line 384 "dsl.impala"
        int nb1_x_51001;
        nb1_x_51001 = _50933 + _51000;
        #line 12 "gpu_device.impala"
        float _50995;
        _50995 = _50986;
        #line 12 "gpu_device.impala"
        float _50988;
        _50988 = *i_50981;
        #line 385 "dsl.impala"
        int _50996;
        _50996 = (int)_50995;
        #line 12 "gpu_device.impala"
        float _51014;
        _51014 = _50988;
        #line 385 "dsl.impala"
        int nb1_y_50997;
        nb1_y_50997 = y_50924 + _50996;
        #line 386 "dsl.impala"
        int _51015;
        _51015 = (int)_51014;
        #line 12 "gpu_device.impala"
        float _50990;
        _50990 = *i_50985;
        #line 390 "dsl.impala"
        int _50998;
        _50998 = nb1_y_50997 * _50919;
        #line 386 "dsl.impala"
        int nb2_x_51016;
        nb2_x_51016 = _51013 - _51015;
        #line 12 "gpu_device.impala"
        float _51009;
        _51009 = _50990;
        #line 390 "dsl.impala"
        int _51002;
        _51002 = _50998 + nb1_x_51001;
        #line 12 "gpu_device.impala"
        float _50993;
        _50993 = *i_50992;
        #line 387 "dsl.impala"
        int _51010;
        _51010 = (int)_51009;
        #line 390 "dsl.impala"
        int _51003;
        _51003 = 5 + _51002;
        #line 12 "gpu_device.impala"
        float _51037;
        _51037 = _50993;
        #line 387 "dsl.impala"
        int nb2_y_51011;
        nb2_y_51011 = y_50924 - _51010;
        #line 11 "gpu_device.impala"
        float* i_51004;
        i_51004 = _49590_50891 + _51003;
        #line 12 "gpu_device.impala"
        float _51005;
        _51005 = *i_51004;
        #line 394 "dsl.impala"
        int _51012;
        _51012 = nb2_y_51011 * _50919;
        #line 12 "gpu_device.impala"
        float _51036;
        _51036 = _51005;
        #line 394 "dsl.impala"
        int _51017;
        _51017 = _51012 + nb2_x_51016;
        #line 390 "dsl.impala"
        bool _51038;
        _51038 = _51036 < _51037;
        #line 12 "gpu_device.impala"
        float _51007;
        _51007 = *i_50992;
        #line 11 "gpu_device.impala"
        float* i_51018;
        i_51018 = _49590_50891 + _51017;
        #line 389 "dsl.impala"
        float nb1_cond_51039;
        nb1_cond_51039 = (float)_51038;
        #line 12 "gpu_device.impala"
        float _51042;
        _51042 = _51007;
        #line 12 "gpu_device.impala"
        float _51019;
        _51019 = *i_51018;
        #line 12 "gpu_device.impala"
        float _51041;
        _51041 = _51019;
        #line 12 "gpu_device.impala"
        float _51021;
        _51021 = *i_50992;
        #line 394 "dsl.impala"
        bool _51043;
        _51043 = _51041 < _51042;
        #line 12 "gpu_device.impala"
        float _51026;
        _51026 = _51021;
        #line 393 "dsl.impala"
        float nb2_cond_51044;
        nb2_cond_51044 = (float)_51043;
        #line 414 "dsl.impala"
        bool _51027;
        _51027 = 1.600000e+03f < _51026;
        #line 415 "dsl.impala"
        bool _51032;
        _51032 = 1.440000e+04f < _51026;
        #line 414 "dsl.impala"
        float _51028;
        _51028 = (float)_51027;
        #line 415 "dsl.impala"
        float _51033;
        _51033 = (float)_51032;
        #line 414 "dsl.impala"
        float _51029;
        _51029 = 1.000000e+00f * _51028;
        #line 415 "dsl.impala"
        float _51034;
        _51034 = 1.400000e+01f * _51033;
        #line 414 "dsl.impala"
        float _51035;
        _51035 = _51029 + _51034;
        #line 413 "dsl.impala"
        float _51040;
        _51040 = _51035 * nb1_cond_51039;
        #line 413 "dsl.impala"
        float _51045;
        _51045 = _51040 * nb2_cond_51044;
        #line 16 "gpu_device.impala"
        *i_51023 = _51045;
        return ;
}

__global__ void lambda_49385(float* _49388_50707, float* _49389_50708, struct_10055 _49390_50709, float* _49391_50710, float* _49392_50711) {
    int  _50714;
    int p_50714;
    int  _50717;
    int p_50717;
    int  _50720;
    int p_50720;
    int  _50723;
    int p_50723;
    int  _50726;
    int p_50726;
    int  _50729;
    int p_50729;
    bool  converge_50758;
    bool pconverge_50758;
    bool  converge_50760;
    bool pconverge_50760;
    #line 1 "/home/rafael/Utilities/new_anydsl/anydsl/runtime/platforms/intrinsics_cuda.impala"
    _50714 = blockIdx_x();
    p_50714 = _50714;
    l50712: ;
        _50714 = p_50714;
        #line 1 "/home/rafael/Utilities/new_anydsl/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _50717 = blockDim_x();
        p_50717 = _50717;
    l50715: ;
        _50717 = p_50717;
        #line 1 "/home/rafael/Utilities/new_anydsl/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _50720 = threadIdx_x();
        p_50720 = _50720;
    l50718: ;
        _50720 = p_50720;
        #line 1 "/home/rafael/Utilities/new_anydsl/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _50723 = blockIdx_y();
        p_50723 = _50723;
    l50721: ;
        _50723 = p_50723;
        #line 1 "/home/rafael/Utilities/new_anydsl/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _50726 = blockDim_y();
        p_50726 = _50726;
    l50724: ;
        _50726 = p_50726;
        #line 1 "/home/rafael/Utilities/new_anydsl/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _50729 = threadIdx_y();
        p_50729 = _50729;
    l50727: ;
        _50729 = p_50729;
        #line 153 "gpu_device.impala"
        int _50730;
        _50730 = _50714 * _50717;
        #line 153 "gpu_device.impala"
        int x_50731;
        x_50731 = _50730 + _50720;
        #line 156 "gpu_device.impala"
        bool _50732;
        _50732 = 1 < x_50731;
        #line 156 "gpu_device.impala"
        if (_50732) goto l50733; else goto l50887;
    l50887: ;
        #line 158 "gpu_device.impala"
        goto l50884;
    l50733: ;
        #line 427 "dsl.impala"
        int _50734;
        _50734 = _49390_50709.e1;
        #line 123 "gpu_device.impala"
        int _50735;
        _50735 = _50734 - 2;
        #line 156 "gpu_device.impala"
        bool _50736;
        _50736 = x_50731 < _50735;
        #line 156 "gpu_device.impala"
        if (_50736) goto l50737; else goto l50886;
    l50886: ;
        #line 158 "gpu_device.impala"
        goto l50884;
    l50737: ;
        #line 154 "gpu_device.impala"
        int _50738;
        _50738 = _50723 * _50726;
        #line 154 "gpu_device.impala"
        int y_50739;
        y_50739 = _50738 + _50729;
        #line 156 "gpu_device.impala"
        bool _50740;
        _50740 = 1 < y_50739;
        #line 156 "gpu_device.impala"
        if (_50740) goto l50741; else goto l50885;
    l50885: ;
        #line 158 "gpu_device.impala"
        goto l50884;
    l50741: ;
        #line 511 "dsl.impala"
        int _50742;
        _50742 = _49390_50709.e2;
        #line 341 "dsl.impala"
        int _50743;
        _50743 = _50742 - 2;
        #line 156 "gpu_device.impala"
        bool _50744;
        _50744 = y_50739 < _50743;
        #line 156 "gpu_device.impala"
        if (_50744) goto l50745; else goto l50883;
    l50883: ;
        #line 158 "gpu_device.impala"
        goto l50884;
    l50884: ;
        return ;
    l50745: ;
        #line 508 "dsl.impala"
        int _50746;
        _50746 = 2 * _50734;
        #line 340 "dsl.impala"
        int _50747;
        _50747 = 5 + _50746;
        #line 157 "gpu_device.impala"
        int _50748;
        _50748 = x_50731 + _50746;
        #line 157 "gpu_device.impala"
        int _50749;
        _50749 = 5 + _50748;
        #line 340 "dsl.impala"
        bool _50750;
        _50750 = _50747 < _50749;
        #line 340 "dsl.impala"
        if (_50750) goto l50751; else goto l50882;
    l50882: ;
        #line 341 "dsl.impala"
        goto l50881;
    l50751: ;
        #line 340 "dsl.impala"
        int _50752;
        _50752 = 3 * _50734;
        #line 340 "dsl.impala"
        int _50753;
        _50753 = 4 + _50752;
        #line 340 "dsl.impala"
        bool _50754;
        _50754 = _50749 < _50753;
        #line 340 "dsl.impala"
        if (_50754) goto l50755; else goto l50880;
    l50880: ;
        #line 341 "dsl.impala"
        goto l50881;
    l50881: ;
        #line 341 "dsl.impala"
        pconverge_50760 = false;
        goto l50759;
    l50755: ;
        #line 156 "gpu_device.impala"
        if (_50740) goto l50756; else goto l50878;
    l50878: ;
        #line 341 "dsl.impala"
        pconverge_50758 = false;
        goto l50757;
    l50756: ;
        #line 341 "dsl.impala"
        pconverge_50758 = _50744;
        goto l50757;
    l50757: ;
        converge_50758 = pconverge_50758;
        #line 341 "dsl.impala"
        pconverge_50760 = converge_50758;
        goto l50759;
    l50759: ;
        converge_50760 = pconverge_50760;
        #line 266 "dsl.impala"
        int _50766;
        _50766 = y_50739 - 1;
        #line 278 "dsl.impala"
        int _50785;
        _50785 = 1 + y_50739;
        #line 339 "dsl.impala"
        float condition_50859;
        condition_50859 = (float)converge_50760;
        #line 266 "dsl.impala"
        int _50768;
        _50768 = _50749 - 1;
        #line 261 "dsl.impala"
        int _50761;
        _50761 = y_50739 * _50734;
        #line 272 "dsl.impala"
        int _50776;
        _50776 = _50761 + _50768;
        #line 278 "dsl.impala"
        int _50786;
        _50786 = _50785 * _50734;
        #line 261 "dsl.impala"
        int _50762;
        _50762 = _50761 + _50748;
        #line 296 "dsl.impala"
        int _50810;
        _50810 = _50786 + _50748;
        #line 266 "dsl.impala"
        int _50767;
        _50767 = _50766 * _50734;
        #line 11 "gpu_device.impala"
        float* i_50777;
        i_50777 = _49389_50708 + _50776;
        #line 278 "dsl.impala"
        int _50787;
        _50787 = _50786 + _50768;
        #line 266 "dsl.impala"
        int _50769;
        _50769 = _50767 + _50768;
        #line 261 "dsl.impala"
        int _50763;
        _50763 = 5 + _50762;
        #line 290 "dsl.impala"
        int _50802;
        _50802 = 6 + _50762;
        #line 296 "dsl.impala"
        int _50811;
        _50811 = 6 + _50810;
        #line 296 "dsl.impala"
        int _50843;
        _50843 = 5 + _50810;
        #line 284 "dsl.impala"
        int _50794;
        _50794 = _50767 + _50748;
        #line 11 "gpu_device.impala"
        float* i_50788;
        i_50788 = _49389_50708 + _50787;
        #line 11 "gpu_device.impala"
        float* i_50770;
        i_50770 = _49389_50708 + _50769;
        #line 15 "gpu_device.impala"
        float* i_50871;
        i_50871 = _49392_50711 + _50763;
        #line 15 "gpu_device.impala"
        float* i_50818;
        i_50818 = _49388_50707 + _50763;
        #line 15 "gpu_device.impala"
        float* i_50764;
        i_50764 = _49391_50710 + _50763;
        #line 11 "gpu_device.impala"
        float* i_50803;
        i_50803 = _49389_50708 + _50802;
        #line 11 "gpu_device.impala"
        float* i_50812;
        i_50812 = _49389_50708 + _50811;
        #line 11 "gpu_device.impala"
        float* i_50844;
        i_50844 = _49389_50708 + _50843;
        #line 284 "dsl.impala"
        int _50795;
        _50795 = 6 + _50794;
        #line 284 "dsl.impala"
        int _50825;
        _50825 = 5 + _50794;
        #line 16 "gpu_device.impala"
        *i_50764 = 0.000000e+00f;
        #line 11 "gpu_device.impala"
        float* i_50796;
        i_50796 = _49389_50708 + _50795;
        #line 11 "gpu_device.impala"
        float* i_50826;
        i_50826 = _49389_50708 + _50825;
        #line 12 "gpu_device.impala"
        float _50771;
        _50771 = *i_50770;
        #line 12 "gpu_device.impala"
        float _50773;
        _50773 = _50771;
        #line 265 "dsl.impala"
        float _50774;
        _50774 = 0.000000e+00f - _50773;
        #line 16 "gpu_device.impala"
        *i_50764 = _50774;
        #line 12 "gpu_device.impala"
        float _50778;
        _50778 = *i_50777;
        #line 12 "gpu_device.impala"
        float _50781;
        _50781 = _50778;
        #line 272 "dsl.impala"
        float _50782;
        _50782 = 2.000000e+00f * _50781;
        #line 271 "dsl.impala"
        float _50783;
        _50783 = _50774 - _50782;
        #line 16 "gpu_device.impala"
        *i_50764 = _50783;
        #line 12 "gpu_device.impala"
        float _50789;
        _50789 = *i_50788;
        #line 12 "gpu_device.impala"
        float _50791;
        _50791 = _50789;
        #line 277 "dsl.impala"
        float _50792;
        _50792 = _50783 - _50791;
        #line 16 "gpu_device.impala"
        *i_50764 = _50792;
        #line 12 "gpu_device.impala"
        float _50797;
        _50797 = *i_50796;
        #line 12 "gpu_device.impala"
        float _50799;
        _50799 = _50797;
        #line 283 "dsl.impala"
        float _50800;
        _50800 = _50792 + _50799;
        #line 16 "gpu_device.impala"
        *i_50764 = _50800;
        #line 12 "gpu_device.impala"
        float _50804;
        _50804 = *i_50803;
        #line 12 "gpu_device.impala"
        float _50806;
        _50806 = _50804;
        #line 290 "dsl.impala"
        float _50807;
        _50807 = 2.000000e+00f * _50806;
        #line 289 "dsl.impala"
        float _50808;
        _50808 = _50800 + _50807;
        #line 16 "gpu_device.impala"
        *i_50764 = _50808;
        #line 12 "gpu_device.impala"
        float _50813;
        _50813 = *i_50812;
        #line 12 "gpu_device.impala"
        float _50815;
        _50815 = _50813;
        #line 295 "dsl.impala"
        float _50816;
        _50816 = _50808 + _50815;
        #line 16 "gpu_device.impala"
        *i_50764 = _50816;
        #line 16 "gpu_device.impala"
        *i_50818 = 0.000000e+00f;
        #line 12 "gpu_device.impala"
        float _50820;
        _50820 = *i_50770;
        #line 12 "gpu_device.impala"
        float _50822;
        _50822 = _50820;
        #line 304 "dsl.impala"
        float _50823;
        _50823 = 0.000000e+00f - _50822;
        #line 16 "gpu_device.impala"
        *i_50818 = _50823;
        #line 12 "gpu_device.impala"
        float _50827;
        _50827 = *i_50826;
        #line 12 "gpu_device.impala"
        float _50829;
        _50829 = _50827;
        #line 311 "dsl.impala"
        float _50830;
        _50830 = 2.000000e+00f * _50829;
        #line 310 "dsl.impala"
        float _50831;
        _50831 = _50823 - _50830;
        #line 16 "gpu_device.impala"
        *i_50818 = _50831;
        #line 12 "gpu_device.impala"
        float _50833;
        _50833 = *i_50796;
        #line 12 "gpu_device.impala"
        float _50835;
        _50835 = _50833;
        #line 316 "dsl.impala"
        float _50836;
        _50836 = _50831 - _50835;
        #line 16 "gpu_device.impala"
        *i_50818 = _50836;
        #line 12 "gpu_device.impala"
        float _50838;
        _50838 = *i_50788;
        #line 12 "gpu_device.impala"
        float _50840;
        _50840 = _50838;
        #line 322 "dsl.impala"
        float _50841;
        _50841 = _50836 + _50840;
        #line 16 "gpu_device.impala"
        *i_50818 = _50841;
        #line 12 "gpu_device.impala"
        float _50845;
        _50845 = *i_50844;
        #line 12 "gpu_device.impala"
        float _50847;
        _50847 = _50845;
        #line 329 "dsl.impala"
        float _50848;
        _50848 = 2.000000e+00f * _50847;
        #line 328 "dsl.impala"
        float _50849;
        _50849 = _50841 + _50848;
        #line 16 "gpu_device.impala"
        *i_50818 = _50849;
        #line 12 "gpu_device.impala"
        float _50851;
        _50851 = *i_50812;
        #line 12 "gpu_device.impala"
        float _50853;
        _50853 = _50851;
        #line 334 "dsl.impala"
        float _50854;
        _50854 = _50849 + _50853;
        #line 16 "gpu_device.impala"
        *i_50818 = _50854;
        #line 12 "gpu_device.impala"
        float _50856;
        _50856 = *i_50764;
        #line 12 "gpu_device.impala"
        float _50858;
        _50858 = _50856;
        #line 346 "dsl.impala"
        float _50860;
        _50860 = _50858 * condition_50859;
        #line 16 "gpu_device.impala"
        *i_50764 = _50860;
        #line 12 "gpu_device.impala"
        float _50862;
        _50862 = *i_50818;
        #line 12 "gpu_device.impala"
        float _50864;
        _50864 = _50862;
        #line 351 "dsl.impala"
        float _50865;
        _50865 = _50864 * condition_50859;
        #line 16 "gpu_device.impala"
        *i_50818 = _50865;
        #line 12 "gpu_device.impala"
        float _50867;
        _50867 = *i_50764;
        #line 12 "gpu_device.impala"
        float _50872;
        _50872 = _50867;
        #line 12 "gpu_device.impala"
        float _50869;
        _50869 = *i_50818;
        #line 356 "dsl.impala"
        float _50873;
        _50873 = _50872 * _50872;
        #line 12 "gpu_device.impala"
        float _50874;
        _50874 = _50869;
        #line 357 "dsl.impala"
        float _50875;
        _50875 = _50874 * _50874;
        #line 356 "dsl.impala"
        float _50876;
        _50876 = _50873 + _50875;
        #line 16 "gpu_device.impala"
        *i_50871 = _50876;
        return ;
}

__global__ void lambda_48861() {
    int  _50423;
    int p_50423;
    int  _50429;
    int p_50429;
    int  _50435;
    int p_50435;
    int  _50441;
    int p_50441;
    int  _50447;
    int p_50447;
    int  _50453;
    int p_50453;
    #line 1 "/home/rafael/Utilities/new_anydsl/anydsl/runtime/platforms/intrinsics_cuda.impala"
    _50423 = blockIdx_x();
    p_50423 = _50423;
    l50421: ;
        _50423 = p_50423;
        #line 1 "/home/rafael/Utilities/new_anydsl/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _50429 = blockDim_x();
        p_50429 = _50429;
    l50427: ;
        _50429 = p_50429;
        #line 1 "/home/rafael/Utilities/new_anydsl/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _50435 = threadIdx_x();
        p_50435 = _50435;
    l50433: ;
        _50435 = p_50435;
        #line 1 "/home/rafael/Utilities/new_anydsl/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _50441 = blockIdx_y();
        p_50441 = _50441;
    l50439: ;
        _50441 = p_50441;
        #line 1 "/home/rafael/Utilities/new_anydsl/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _50447 = blockDim_y();
        p_50447 = _50447;
    l50445: ;
        _50447 = p_50447;
        #line 1 "/home/rafael/Utilities/new_anydsl/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _50453 = threadIdx_y();
        p_50453 = _50453;
    l50451: ;
        _50453 = p_50453;
        return ;
}

__global__ void lambda_49241(float* _49244_50581, float* _49245_50582, struct_10055 _49246_50583, float* _49247_50584) {
    int  _50587;
    int p_50587;
    int  _50590;
    int p_50590;
    int  _50593;
    int p_50593;
    int  _50596;
    int p_50596;
    int  _50599;
    int p_50599;
    int  _50602;
    int p_50602;
    #line 1 "/home/rafael/Utilities/new_anydsl/anydsl/runtime/platforms/intrinsics_cuda.impala"
    _50587 = blockIdx_x();
    p_50587 = _50587;
    l50585: ;
        _50587 = p_50587;
        #line 1 "/home/rafael/Utilities/new_anydsl/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _50590 = blockDim_x();
        p_50590 = _50590;
    l50588: ;
        _50590 = p_50590;
        #line 1 "/home/rafael/Utilities/new_anydsl/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _50593 = threadIdx_x();
        p_50593 = _50593;
    l50591: ;
        _50593 = p_50593;
        #line 1 "/home/rafael/Utilities/new_anydsl/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _50596 = blockIdx_y();
        p_50596 = _50596;
    l50594: ;
        _50596 = p_50596;
        #line 1 "/home/rafael/Utilities/new_anydsl/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _50599 = blockDim_y();
        p_50599 = _50599;
    l50597: ;
        _50599 = p_50599;
        #line 1 "/home/rafael/Utilities/new_anydsl/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _50602 = threadIdx_y();
        p_50602 = _50602;
    l50600: ;
        _50602 = p_50602;
        #line 120 "gpu_device.impala"
        int _50603;
        _50603 = _50587 * _50590;
        #line 120 "gpu_device.impala"
        int x_50604;
        x_50604 = _50603 + _50593;
        #line 123 "gpu_device.impala"
        bool _50605;
        _50605 = 1 < x_50604;
        #line 123 "gpu_device.impala"
        if (_50605) goto l50606; else goto l50703;
    l50703: ;
        #line 125 "gpu_device.impala"
        goto l50700;
    l50606: ;
        #line 427 "dsl.impala"
        int _50607;
        _50607 = _49246_50583.e1;
        #line 123 "gpu_device.impala"
        int _50608;
        _50608 = _50607 - 2;
        #line 123 "gpu_device.impala"
        bool _50609;
        _50609 = x_50604 < _50608;
        #line 123 "gpu_device.impala"
        if (_50609) goto l50610; else goto l50702;
    l50702: ;
        #line 125 "gpu_device.impala"
        goto l50700;
    l50610: ;
        #line 121 "gpu_device.impala"
        int _50611;
        _50611 = _50596 * _50599;
        #line 121 "gpu_device.impala"
        int y_50612;
        y_50612 = _50611 + _50602;
        #line 123 "gpu_device.impala"
        bool _50613;
        _50613 = 1 < y_50612;
        #line 123 "gpu_device.impala"
        if (_50613) goto l50614; else goto l50701;
    l50701: ;
        #line 125 "gpu_device.impala"
        goto l50700;
    l50614: ;
        #line 511 "dsl.impala"
        int _50615;
        _50615 = _49246_50583.e2;
        #line 341 "dsl.impala"
        int _50616;
        _50616 = _50615 - 2;
        #line 123 "gpu_device.impala"
        bool _50617;
        _50617 = y_50612 < _50616;
        #line 123 "gpu_device.impala"
        if (_50617) goto l50618; else goto l50699;
    l50699: ;
        #line 125 "gpu_device.impala"
        goto l50700;
    l50700: ;
        return ;
    l50618: ;
        #line 209 "dsl.impala"
        int _50687;
        _50687 = 2 + y_50612;
        #line 209 "dsl.impala"
        int _50646;
        _50646 = -1 + y_50612;
        #line 209 "dsl.impala"
        int _50672;
        _50672 = 1 + y_50612;
        #line 11 "gpu_device.impala"
        float* i_50626;
        i_50626 = _49244_50581 + 0;
        #line 11 "gpu_device.impala"
        float* i_50684;
        i_50684 = _49244_50581 + 4;
        #line 209 "dsl.impala"
        int _50688;
        _50688 = _50687 * _50607;
        #line 11 "gpu_device.impala"
        float* i_50669;
        i_50669 = _49244_50581 + 3;
        #line 508 "dsl.impala"
        int _50620;
        _50620 = 2 * _50607;
        #line 209 "dsl.impala"
        int _50630;
        _50630 = -2 + y_50612;
        #line 11 "gpu_device.impala"
        float* i_50642;
        i_50642 = _49244_50581 + 1;
        #line 11 "gpu_device.impala"
        float* i_50658;
        i_50658 = _49244_50581 + 2;
        #line 204 "dsl.impala"
        int _50619;
        _50619 = y_50612 * _50607;
        #line 209 "dsl.impala"
        int _50647;
        _50647 = _50646 * _50607;
        #line 209 "dsl.impala"
        int _50673;
        _50673 = _50672 * _50607;
        #line 209 "dsl.impala"
        int _50631;
        _50631 = _50630 * _50607;
        #line 124 "gpu_device.impala"
        int _50621;
        _50621 = x_50604 + _50620;
        #line 204 "dsl.impala"
        int _50622;
        _50622 = _50619 + _50621;
        #line 209 "dsl.impala"
        int _50648;
        _50648 = _50647 + _50621;
        #line 209 "dsl.impala"
        int _50674;
        _50674 = _50673 + _50621;
        #line 209 "dsl.impala"
        int _50632;
        _50632 = _50631 + _50621;
        #line 209 "dsl.impala"
        int _50689;
        _50689 = _50688 + _50621;
        #line 204 "dsl.impala"
        int _50623;
        _50623 = 5 + _50622;
        #line 209 "dsl.impala"
        int _50649;
        _50649 = 5 + _50648;
        #line 209 "dsl.impala"
        int _50675;
        _50675 = 5 + _50674;
        #line 209 "dsl.impala"
        int _50633;
        _50633 = 5 + _50632;
        #line 209 "dsl.impala"
        int _50690;
        _50690 = 5 + _50689;
        #line 15 "gpu_device.impala"
        float* i_50624;
        i_50624 = _49245_50582 + _50623;
        #line 11 "gpu_device.impala"
        float* i_50661;
        i_50661 = _49247_50584 + _50623;
        #line 11 "gpu_device.impala"
        float* i_50650;
        i_50650 = _49247_50584 + _50649;
        #line 11 "gpu_device.impala"
        float* i_50676;
        i_50676 = _49247_50584 + _50675;
        #line 11 "gpu_device.impala"
        float* i_50634;
        i_50634 = _49247_50584 + _50633;
        #line 11 "gpu_device.impala"
        float* i_50691;
        i_50691 = _49247_50584 + _50690;
        #line 16 "gpu_device.impala"
        *i_50624 = 0.000000e+00f;
        #line 12 "gpu_device.impala"
        float _50627;
        _50627 = *i_50626;
        #line 12 "gpu_device.impala"
        float _50637;
        _50637 = _50627;
        #line 12 "gpu_device.impala"
        float _50635;
        _50635 = *i_50634;
        #line 12 "gpu_device.impala"
        float _50638;
        _50638 = _50635;
        #line 209 "dsl.impala"
        float _50639;
        _50639 = _50637 * _50638;
        #line 208 "dsl.impala"
        float _50640;
        _50640 = 0.000000e+00f + _50639;
        #line 16 "gpu_device.impala"
        *i_50624 = _50640;
        #line 12 "gpu_device.impala"
        float _50643;
        _50643 = *i_50642;
        #line 12 "gpu_device.impala"
        float _50653;
        _50653 = _50643;
        #line 12 "gpu_device.impala"
        float _50651;
        _50651 = *i_50650;
        #line 12 "gpu_device.impala"
        float _50654;
        _50654 = _50651;
        #line 209 "dsl.impala"
        float _50655;
        _50655 = _50653 * _50654;
        #line 208 "dsl.impala"
        float _50656;
        _50656 = _50640 + _50655;
        #line 16 "gpu_device.impala"
        *i_50624 = _50656;
        #line 12 "gpu_device.impala"
        float _50659;
        _50659 = *i_50658;
        #line 12 "gpu_device.impala"
        float _50664;
        _50664 = _50659;
        #line 12 "gpu_device.impala"
        float _50662;
        _50662 = *i_50661;
        #line 12 "gpu_device.impala"
        float _50665;
        _50665 = _50662;
        #line 209 "dsl.impala"
        float _50666;
        _50666 = _50664 * _50665;
        #line 208 "dsl.impala"
        float _50667;
        _50667 = _50656 + _50666;
        #line 16 "gpu_device.impala"
        *i_50624 = _50667;
        #line 12 "gpu_device.impala"
        float _50670;
        _50670 = *i_50669;
        #line 12 "gpu_device.impala"
        float _50679;
        _50679 = _50670;
        #line 12 "gpu_device.impala"
        float _50677;
        _50677 = *i_50676;
        #line 12 "gpu_device.impala"
        float _50680;
        _50680 = _50677;
        #line 209 "dsl.impala"
        float _50681;
        _50681 = _50679 * _50680;
        #line 208 "dsl.impala"
        float _50682;
        _50682 = _50667 + _50681;
        #line 16 "gpu_device.impala"
        *i_50624 = _50682;
        #line 12 "gpu_device.impala"
        float _50685;
        _50685 = *i_50684;
        #line 12 "gpu_device.impala"
        float _50694;
        _50694 = _50685;
        #line 12 "gpu_device.impala"
        float _50692;
        _50692 = *i_50691;
        #line 12 "gpu_device.impala"
        float _50695;
        _50695 = _50692;
        #line 209 "dsl.impala"
        float _50696;
        _50696 = _50694 * _50695;
        #line 208 "dsl.impala"
        float _50697;
        _50697 = _50682 + _50696;
        #line 16 "gpu_device.impala"
        *i_50624 = _50697;
        return ;
}

}