
#include <hip/hip_runtime.h>
extern "C" {
typedef struct struct_14391 {
    int e0;
    char* e1;
} struct_14391;
typedef struct struct_14389 {
    float* e0;
    struct_14391 e1;
    struct_14391 e2;
    struct_14391 e3;
    struct_14391 e4;
    int e5;
    int e6;
} struct_14389;
typedef struct array_14454 {
    int e[2];
} array_14454;
typedef struct array_14455 {
    array_14454 e[7];
} array_14455;

__device__ inline int threadIdx_x() { return threadIdx.x; }
__device__ inline int threadIdx_y() { return threadIdx.y; }
__device__ inline int threadIdx_z() { return threadIdx.z; }
__device__ inline int blockIdx_x() { return blockIdx.x; }
__device__ inline int blockIdx_y() { return blockIdx.y; }
__device__ inline int blockIdx_z() { return blockIdx.z; }
__device__ inline int blockDim_x() { return blockDim.x; }
__device__ inline int blockDim_y() { return blockDim.y; }
__device__ inline int blockDim_z() { return blockDim.z; }
__device__ inline int gridDim_x() { return gridDim.x; }
__device__ inline int gridDim_y() { return gridDim.y; }
__device__ inline int gridDim_z() { return gridDim.z; }
__global__ void lambda_crit_37781(float, char*, char*, char*, float, struct_14389, char*);
__global__ void lambda_crit_37249(float*, float*, struct_14389, char*, float*);
__global__ void lambda_crit_37068(float*, float*, struct_14389, float*, char*);
__global__ void lambda_crit_37448(char*, char*, struct_14389, char*, char*);
__global__ void lambda_36802();
__device__ float read_39282(struct_14391, int);
__device__ int abs_39538(int);
__device__ void write_38998(struct_14391, int, float);

__global__ void lambda_crit_37781(float _37784_39481, char* _37785_39482, char* _37786_39483, char* _37787_39484, float _37788_39485, struct_14389 _37789_39486, char* _37790_39487) {
    int  _39490;
    int p_39490;
    int  _39493;
    int p_39493;
    int  _39496;
    int p_39496;
    int  _39499;
    int p_39499;
    int  _39502;
    int p_39502;
    int  _39505;
    int p_39505;
    float  read_39533;
    float pread_39533;
    float  read_39537;
    float pread_39537;
    int  mx_39553;
    int pmx_39553;
    int  abs_39557;
    int pabs_39557;
    float  read_39561;
    float pread_39561;
    float  read_39595;
    float pread_39595;
    float  read_39598;
    float pread_39598;
    float  read_39605;
    float pread_39605;
    float  read_39608;
    float pread_39608;
    float  read_39611;
    float pread_39611;
    #line 1 "/home/rafael/Utilities/anydsl/runtime/platforms/intrinsics_cuda.impala"
    _39490 = blockIdx_x();
    p_39490 = _39490;
    l39488: ;
        _39490 = p_39490;
        #line 1 "/home/rafael/Utilities/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _39493 = blockDim_x();
        p_39493 = _39493;
    l39491: ;
        _39493 = p_39493;
        #line 1 "/home/rafael/Utilities/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _39496 = threadIdx_x();
        p_39496 = _39496;
    l39494: ;
        _39496 = p_39496;
        #line 1 "/home/rafael/Utilities/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _39499 = blockIdx_y();
        p_39499 = _39499;
    l39497: ;
        _39499 = p_39499;
        #line 1 "/home/rafael/Utilities/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _39502 = blockDim_y();
        p_39502 = _39502;
    l39500: ;
        _39502 = p_39502;
        #line 1 "/home/rafael/Utilities/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _39505 = threadIdx_y();
        p_39505 = _39505;
    l39503: ;
        _39505 = p_39505;
        #line 354 "dsl.impala"
        int _39506;
        _39506 = _37789_39486.e5;
        #line 135 "gpu_device.impala"
        int _39509;
        _39509 = _39490 * _39493;
        #line 116 "gpu_device.impala"
        int _39507;
        _39507 = 2 * _39506;
        #line 135 "gpu_device.impala"
        int _39510;
        _39510 = _39509 + _39496;
        #line 116 "gpu_device.impala"
        int offset_39508;
        offset_39508 = 5 + _39507;
        #line 135 "gpu_device.impala"
        int _39511;
        _39511 = _39510 + _39507;
        #line 135 "gpu_device.impala"
        int x_39512;
        x_39512 = 5 + _39511;
        #line 138 "gpu_device.impala"
        bool _39513;
        _39513 = offset_39508 < x_39512;
        #line 138 "gpu_device.impala"
        if (_39513) goto l39514; else goto l39636;
    l39636: ;
        #line 140 "gpu_device.impala"
        goto l39633;
    l39514: ;
        #line 138 "gpu_device.impala"
        int _39515;
        _39515 = _39506 + _39507;
        #line 138 "gpu_device.impala"
        int _39516;
        _39516 = 5 + _39515;
        #line 138 "gpu_device.impala"
        bool _39517;
        _39517 = x_39512 < _39516;
        #line 138 "gpu_device.impala"
        if (_39517) goto l39518; else goto l39635;
    l39635: ;
        #line 140 "gpu_device.impala"
        goto l39633;
    l39518: ;
        #line 136 "gpu_device.impala"
        int _39519;
        _39519 = _39499 * _39502;
        #line 136 "gpu_device.impala"
        int y_39520;
        y_39520 = _39519 + _39505;
        #line 138 "gpu_device.impala"
        bool _39521;
        _39521 = 1 < y_39520;
        #line 138 "gpu_device.impala"
        if (_39521) goto l39522; else goto l39634;
    l39634: ;
        #line 140 "gpu_device.impala"
        goto l39633;
    l39522: ;
        #line 115 "gpu_device.impala"
        int _39523;
        _39523 = _37789_39486.e6;
        #line 138 "gpu_device.impala"
        int _39524;
        _39524 = _39523 - 2;
        #line 138 "gpu_device.impala"
        bool _39525;
        _39525 = y_39520 < _39524;
        #line 138 "gpu_device.impala"
        if (_39525) goto l39526; else goto l39632;
    l39632: ;
        #line 140 "gpu_device.impala"
        goto l39633;
    l39633: ;
        #line 140 "gpu_device.impala"
        goto l39630;
    l39526: ;
        #line 354 "dsl.impala"
        int _39528;
        _39528 = y_39520 * _39506;
        #line 36 "/home/rafael/Utilities/anydsl/runtime/src/runtime.impala"
        struct_14391 _39527;
        _39527.e0 = 1;
        _39527.e1 = _37790_39487;
        #line 354 "dsl.impala"
        int _39529;
        _39529 = _39528 + _39511;
        #line 354 "dsl.impala"
        int _39530;
        _39530 = 5 + _39529;
        #line 9 "gpu_device.impala"
        read_39533 = read_39282(_39527, _39530);
        pread_39533 = read_39533;
    l39531: ;
        read_39533 = pread_39533;
        #line 36 "/home/rafael/Utilities/anydsl/runtime/src/runtime.impala"
        struct_14391 _39534;
        _39534.e0 = 1;
        _39534.e1 = _37785_39482;
        #line 9 "gpu_device.impala"
        read_39537 = read_39282(_39534, _39530);
        pread_39537 = read_39537;
    l39535: ;
        read_39537 = pread_39537;
        #line 354 "dsl.impala"
        int xs_39550;
        xs_39550 = (int)read_39533;
        #line 145 "dsl.impala"
        mx_39553 = abs_39538(xs_39550);
        pmx_39553 = mx_39553;
    l39551: ;
        mx_39553 = pmx_39553;
        #line 355 "dsl.impala"
        int ys_39554;
        ys_39554 = (int)read_39537;
        #line 145 "dsl.impala"
        abs_39557 = abs_39538(ys_39554);
        pabs_39557 = abs_39557;
    l39555: ;
        abs_39557 = pabs_39557;
        #line 36 "/home/rafael/Utilities/anydsl/runtime/src/runtime.impala"
        struct_14391 _39558;
        _39558.e0 = 1;
        _39558.e1 = _37787_39484;
        #line 9 "gpu_device.impala"
        read_39561 = read_39282(_39558, _39530);
        pread_39561 = read_39561;
    l39559: ;
        read_39561 = pread_39561;
        #line 357 "dsl.impala"
        int my_39568;
        my_39568 = abs_39557 << 15;
        #line 359 "dsl.impala"
        int tg22x_39570;
        tg22x_39570 = 13573 * mx_39553;
        #line 360 "dsl.impala"
        int _39575;
        _39575 = mx_39553 << 16;
        #line 364 "dsl.impala"
        int _39581;
        _39581 = xs_39550 ^ ys_39554;
        #line 360 "dsl.impala"
        int tg67x_39576;
        tg67x_39576 = tg22x_39570 + _39575;
        #line 362 "dsl.impala"
        bool _39571;
        _39571 = my_39568 < tg22x_39570;
        #line 362 "dsl.impala"
        int cond1_39572;
        cond1_39572 = (int)_39571;
        #line 363 "dsl.impala"
        bool _39577;
        _39577 = tg67x_39576 < my_39568;
        #line 364 "dsl.impala"
        bool _39582;
        _39582 = _39581 < 0;
        #line 366 "dsl.impala"
        int _39573;
        _39573 = cond1_39572 << 2;
        #line 363 "dsl.impala"
        int cond2_39578;
        cond2_39578 = (int)_39577;
        #line 364 "dsl.impala"
        int cond3_39583;
        cond3_39583 = (int)_39582;
        #line 366 "dsl.impala"
        int _39579;
        _39579 = cond2_39578 << 1;
        #line 366 "dsl.impala"
        int _39580;
        _39580 = _39573 + _39579;
        #line 366 "dsl.impala"
        int index_39584;
        index_39584 = _39580 + cond3_39583;
        #line 367 "dsl.impala"
        array_14454 _39562_24;
        _39562_24.e[0] = 1;
        _39562_24.e[1] = 1;
        array_14454 _39563_27;
        _39563_27.e[0] = -1;
        _39563_27.e[1] = 1;
        array_14454 _39564_30;
        _39564_30.e[0] = 0;
        _39564_30.e[1] = 1;
        array_14454 _39565_33;
        _39565_33.e[0] = 1;
        _39565_33.e[1] = 0;
        array_14455 offsets_39566_36;
        offsets_39566_36.e[0] = _39562_24;
        offsets_39566_36.e[1] = _39563_27;
        offsets_39566_36.e[2] = _39564_30;
        offsets_39566_36.e[3] = _39564_30;
        offsets_39566_36.e[4] = _39565_33;
        offsets_39566_36.e[5] = _39565_33;
        offsets_39566_36.e[6] = _39565_33;
        array_14454 _39585;
        _39585 = offsets_39566_36.e[index_39584];
        #line 367 "dsl.impala"
        int _39589;
        _39589 = _39585.e[0];
        #line 368 "dsl.impala"
        int _39586;
        _39586 = _39585.e[1];
        #line 367 "dsl.impala"
        int nb1_x_39590;
        nb1_x_39590 = _39511 + _39589;
        #line 368 "dsl.impala"
        int nb1_y_39587;
        nb1_y_39587 = y_39520 + _39586;
        #line 373 "dsl.impala"
        int _39588;
        _39588 = nb1_y_39587 * _39506;
        #line 373 "dsl.impala"
        int _39591;
        _39591 = _39588 + nb1_x_39590;
        #line 373 "dsl.impala"
        int _39592;
        _39592 = 5 + _39591;
        #line 9 "gpu_device.impala"
        read_39595 = read_39282(_39558, _39592);
        pread_39595 = read_39595;
    l39593: ;
        read_39595 = pread_39595;
        #line 9 "gpu_device.impala"
        read_39598 = read_39282(_39558, _39530);
        pread_39598 = read_39598;
    l39596: ;
        read_39598 = pread_39598;
        #line 369 "dsl.impala"
        int nb2_x_39601;
        nb2_x_39601 = x_39512 - _39589;
        #line 370 "dsl.impala"
        int nb2_y_39599;
        nb2_y_39599 = y_39520 - _39586;
        #line 377 "dsl.impala"
        int _39600;
        _39600 = nb2_y_39599 * _39506;
        #line 377 "dsl.impala"
        int _39602;
        _39602 = _39600 + nb2_x_39601;
        #line 9 "gpu_device.impala"
        read_39605 = read_39282(_39558, _39602);
        pread_39605 = read_39605;
    l39603: ;
        read_39605 = pread_39605;
        #line 9 "gpu_device.impala"
        read_39608 = read_39282(_39558, _39530);
        pread_39608 = read_39608;
    l39606: ;
        read_39608 = pread_39608;
        #line 9 "gpu_device.impala"
        read_39611 = read_39282(_39558, _39530);
        pread_39611 = read_39611;
    l39609: ;
        read_39611 = pread_39611;
        #line 36 "/home/rafael/Utilities/anydsl/runtime/src/runtime.impala"
        struct_14391 _39612;
        _39612.e0 = 1;
        _39612.e1 = _37786_39483;
        #line 377 "dsl.impala"
        bool _39625;
        _39625 = read_39605 < read_39598;
        #line 373 "dsl.impala"
        bool _39622;
        _39622 = read_39595 < read_39561;
        #line 376 "dsl.impala"
        float nb2_cond_39626;
        nb2_cond_39626 = (float)_39625;
        #line 398 "dsl.impala"
        bool _39618;
        _39618 = _37788_39485 < read_39611;
        #line 397 "dsl.impala"
        bool _39614;
        _39614 = _37784_39481 < read_39608;
        #line 372 "dsl.impala"
        float nb1_cond_39623;
        nb1_cond_39623 = (float)_39622;
        #line 398 "dsl.impala"
        float _39619;
        _39619 = (float)_39618;
        #line 397 "dsl.impala"
        float _39615;
        _39615 = (float)_39614;
        #line 398 "dsl.impala"
        float _39620;
        _39620 = 1.400000e+01f * _39619;
        #line 397 "dsl.impala"
        float _39616;
        _39616 = 1.000000e+00f * _39615;
        #line 397 "dsl.impala"
        float _39621;
        _39621 = _39616 + _39620;
        #line 396 "dsl.impala"
        float _39624;
        _39624 = _39621 * nb1_cond_39623;
        #line 396 "dsl.impala"
        float _39627;
        _39627 = _39624 * nb2_cond_39626;
        #line 13 "gpu_device.impala"
        write_38998(_39612, _39530, _39627);
    l39628: ;
        #line 140 "gpu_device.impala"
        goto l39630;
    l39630: ;
        return ;
}

__global__ void lambda_crit_37249(float* _37252_39095, float* _37253_39096, struct_14389 _37254_39097, char* _37255_39098, float* _37256_39099) {
    int  _39102;
    int p_39102;
    int  _39105;
    int p_39105;
    int  _39108;
    int p_39108;
    int  _39111;
    int p_39111;
    int  _39114;
    int p_39114;
    int  _39117;
    int p_39117;
    #line 1 "/home/rafael/Utilities/anydsl/runtime/platforms/intrinsics_cuda.impala"
    _39102 = blockIdx_x();
    p_39102 = _39102;
    l39100: ;
        _39102 = p_39102;
        #line 1 "/home/rafael/Utilities/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _39105 = blockDim_x();
        p_39105 = _39105;
    l39103: ;
        _39105 = p_39105;
        #line 1 "/home/rafael/Utilities/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _39108 = threadIdx_x();
        p_39108 = _39108;
    l39106: ;
        _39108 = p_39108;
        #line 1 "/home/rafael/Utilities/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _39111 = blockIdx_y();
        p_39111 = _39111;
    l39109: ;
        _39111 = p_39111;
        #line 1 "/home/rafael/Utilities/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _39114 = blockDim_y();
        p_39114 = _39114;
    l39112: ;
        _39114 = p_39114;
        #line 1 "/home/rafael/Utilities/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _39117 = threadIdx_y();
        p_39117 = _39117;
    l39115: ;
        _39117 = p_39117;
        #line 88 "gpu_device.impala"
        int _39121;
        _39121 = _39102 * _39105;
        #line 88 "gpu_device.impala"
        int _39122;
        _39122 = _39121 + _39108;
        #line 185 "dsl.impala"
        int _39118;
        _39118 = _37254_39097.e5;
        #line 73 "gpu_device.impala"
        int _39119;
        _39119 = 2 * _39118;
        #line 73 "gpu_device.impala"
        int offset_39120;
        offset_39120 = 5 + _39119;
        #line 88 "gpu_device.impala"
        int _39123;
        _39123 = _39122 + _39119;
        #line 88 "gpu_device.impala"
        int x_39124;
        x_39124 = 5 + _39123;
        #line 91 "gpu_device.impala"
        bool _39125;
        _39125 = offset_39120 < x_39124;
        #line 91 "gpu_device.impala"
        if (_39125) goto l39126; else goto l39228;
    l39228: ;
        #line 93 "gpu_device.impala"
        goto l39225;
    l39126: ;
        #line 91 "gpu_device.impala"
        int _39127;
        _39127 = _39118 + _39119;
        #line 91 "gpu_device.impala"
        int _39128;
        _39128 = 5 + _39127;
        #line 91 "gpu_device.impala"
        bool _39129;
        _39129 = x_39124 < _39128;
        #line 91 "gpu_device.impala"
        if (_39129) goto l39130; else goto l39227;
    l39227: ;
        #line 93 "gpu_device.impala"
        goto l39225;
    l39130: ;
        #line 89 "gpu_device.impala"
        int _39131;
        _39131 = _39111 * _39114;
        #line 89 "gpu_device.impala"
        int y_39132;
        y_39132 = _39131 + _39117;
        #line 91 "gpu_device.impala"
        bool _39133;
        _39133 = 1 < y_39132;
        #line 91 "gpu_device.impala"
        if (_39133) goto l39134; else goto l39226;
    l39226: ;
        #line 93 "gpu_device.impala"
        goto l39225;
    l39134: ;
        #line 72 "gpu_device.impala"
        int _39135;
        _39135 = _37254_39097.e6;
        #line 91 "gpu_device.impala"
        int _39136;
        _39136 = _39135 - 2;
        #line 91 "gpu_device.impala"
        bool _39137;
        _39137 = y_39132 < _39136;
        #line 91 "gpu_device.impala"
        if (_39137) goto l39138; else goto l39224;
    l39224: ;
        #line 93 "gpu_device.impala"
        goto l39225;
    l39225: ;
        #line 93 "gpu_device.impala"
        goto l39145;
    l39138: ;
        #line 185 "dsl.impala"
        int _39140;
        _39140 = y_39132 * _39118;
        #line 185 "dsl.impala"
        int _39141;
        _39141 = _39140 + _39123;
        #line 36 "/home/rafael/Utilities/anydsl/runtime/src/runtime.impala"
        struct_14391 _39139;
        _39139.e0 = 1;
        _39139.e1 = _37255_39098;
        #line 185 "dsl.impala"
        int _39142;
        _39142 = 5 + _39141;
        #line 13 "gpu_device.impala"
        write_38998(_39139, _39142, 0.000000e+00f);
    l39143: ;
        #line 190 "dsl.impala"
        int _39171;
        _39171 = -1 + y_39132;
        #line 9 "gpu_device.impala"
        float* i_39183;
        i_39183 = _37252_39095 + 2;
        #line 9 "gpu_device.impala"
        float* i_39186;
        i_39186 = _37253_39096 + _39142;
        #line 9 "gpu_device.impala"
        float* i_39150;
        i_39150 = _37252_39095 + 0;
        #line 190 "dsl.impala"
        int _39172;
        _39172 = _39171 * _39118;
        #line 9 "gpu_device.impala"
        float* i_39147;
        i_39147 = _37256_39099 + _39142;
        #line 9 "gpu_device.impala"
        float* i_39209;
        i_39209 = _37252_39095 + 4;
        #line 190 "dsl.impala"
        int _39212;
        _39212 = 2 + y_39132;
        #line 9 "gpu_device.impala"
        float* i_39194;
        i_39194 = _37252_39095 + 3;
        #line 10 "gpu_device.impala"
        float _39148;
        _39148 = *i_39147;
        #line 9 "gpu_device.impala"
        float* i_39167;
        i_39167 = _37252_39095 + 1;
        #line 10 "gpu_device.impala"
        float _39161;
        _39161 = _39148;
        #line 190 "dsl.impala"
        int _39197;
        _39197 = 1 + y_39132;
        #line 190 "dsl.impala"
        int _39173;
        _39173 = _39172 + _39123;
        #line 190 "dsl.impala"
        int _39154;
        _39154 = -2 + y_39132;
        #line 10 "gpu_device.impala"
        float _39151;
        _39151 = *i_39150;
        #line 190 "dsl.impala"
        int _39213;
        _39213 = _39212 * _39118;
        #line 190 "dsl.impala"
        int _39198;
        _39198 = _39197 * _39118;
        #line 190 "dsl.impala"
        int _39174;
        _39174 = 5 + _39173;
        #line 190 "dsl.impala"
        int _39155;
        _39155 = _39154 * _39118;
        #line 10 "gpu_device.impala"
        float _39162;
        _39162 = _39151;
        #line 190 "dsl.impala"
        int _39214;
        _39214 = _39213 + _39123;
        #line 190 "dsl.impala"
        int _39199;
        _39199 = _39198 + _39123;
        #line 9 "gpu_device.impala"
        float* i_39175;
        i_39175 = _37253_39096 + _39174;
        #line 190 "dsl.impala"
        int _39156;
        _39156 = _39155 + _39123;
        #line 190 "dsl.impala"
        int _39215;
        _39215 = 5 + _39214;
        #line 190 "dsl.impala"
        int _39200;
        _39200 = 5 + _39199;
        #line 190 "dsl.impala"
        int _39157;
        _39157 = 5 + _39156;
        #line 9 "gpu_device.impala"
        float* i_39216;
        i_39216 = _37253_39096 + _39215;
        #line 9 "gpu_device.impala"
        float* i_39201;
        i_39201 = _37253_39096 + _39200;
        #line 9 "gpu_device.impala"
        float* i_39158;
        i_39158 = _37253_39096 + _39157;
        #line 10 "gpu_device.impala"
        float _39159;
        _39159 = *i_39158;
        #line 10 "gpu_device.impala"
        float _39163;
        _39163 = _39159;
        #line 190 "dsl.impala"
        float _39164;
        _39164 = _39162 * _39163;
        #line 189 "dsl.impala"
        float _39165;
        _39165 = _39161 + _39164;
        #line 14 "gpu_device.impala"
        *i_39147 = _39165;
        #line 10 "gpu_device.impala"
        float _39168;
        _39168 = *i_39167;
        #line 10 "gpu_device.impala"
        float _39178;
        _39178 = _39168;
        #line 10 "gpu_device.impala"
        float _39176;
        _39176 = *i_39175;
        #line 10 "gpu_device.impala"
        float _39179;
        _39179 = _39176;
        #line 190 "dsl.impala"
        float _39180;
        _39180 = _39178 * _39179;
        #line 189 "dsl.impala"
        float _39181;
        _39181 = _39165 + _39180;
        #line 14 "gpu_device.impala"
        *i_39147 = _39181;
        #line 10 "gpu_device.impala"
        float _39184;
        _39184 = *i_39183;
        #line 10 "gpu_device.impala"
        float _39189;
        _39189 = _39184;
        #line 10 "gpu_device.impala"
        float _39187;
        _39187 = *i_39186;
        #line 10 "gpu_device.impala"
        float _39190;
        _39190 = _39187;
        #line 190 "dsl.impala"
        float _39191;
        _39191 = _39189 * _39190;
        #line 189 "dsl.impala"
        float _39192;
        _39192 = _39181 + _39191;
        #line 14 "gpu_device.impala"
        *i_39147 = _39192;
        #line 10 "gpu_device.impala"
        float _39195;
        _39195 = *i_39194;
        #line 10 "gpu_device.impala"
        float _39204;
        _39204 = _39195;
        #line 10 "gpu_device.impala"
        float _39202;
        _39202 = *i_39201;
        #line 10 "gpu_device.impala"
        float _39205;
        _39205 = _39202;
        #line 190 "dsl.impala"
        float _39206;
        _39206 = _39204 * _39205;
        #line 189 "dsl.impala"
        float _39207;
        _39207 = _39192 + _39206;
        #line 14 "gpu_device.impala"
        *i_39147 = _39207;
        #line 10 "gpu_device.impala"
        float _39210;
        _39210 = *i_39209;
        #line 10 "gpu_device.impala"
        float _39219;
        _39219 = _39210;
        #line 10 "gpu_device.impala"
        float _39217;
        _39217 = *i_39216;
        #line 10 "gpu_device.impala"
        float _39220;
        _39220 = _39217;
        #line 190 "dsl.impala"
        float _39221;
        _39221 = _39219 * _39220;
        #line 189 "dsl.impala"
        float _39222;
        _39222 = _39207 + _39221;
        #line 14 "gpu_device.impala"
        *i_39147 = _39222;
        #line 93 "gpu_device.impala"
        goto l39145;
    l39145: ;
        return ;
}

__global__ void lambda_crit_37068(float* _37071_38949, float* _37072_38950, struct_14389 _37073_38951, float* _37074_38952, char* _37075_38953) {
    int  _38956;
    int p_38956;
    int  _38959;
    int p_38959;
    int  _38962;
    int p_38962;
    int  _38965;
    int p_38965;
    int  _38968;
    int p_38968;
    int  _38971;
    int p_38971;
    #line 1 "/home/rafael/Utilities/anydsl/runtime/platforms/intrinsics_cuda.impala"
    _38956 = blockIdx_x();
    p_38956 = _38956;
    l38954: ;
        _38956 = p_38956;
        #line 1 "/home/rafael/Utilities/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _38959 = blockDim_x();
        p_38959 = _38959;
    l38957: ;
        _38959 = p_38959;
        #line 1 "/home/rafael/Utilities/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _38962 = threadIdx_x();
        p_38962 = _38962;
    l38960: ;
        _38962 = p_38962;
        #line 1 "/home/rafael/Utilities/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _38965 = blockIdx_y();
        p_38965 = _38965;
    l38963: ;
        _38965 = p_38965;
        #line 1 "/home/rafael/Utilities/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _38968 = blockDim_y();
        p_38968 = _38968;
    l38966: ;
        _38968 = p_38968;
        #line 1 "/home/rafael/Utilities/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _38971 = threadIdx_y();
        p_38971 = _38971;
    l38969: ;
        _38971 = p_38971;
        #line 170 "dsl.impala"
        int _38975;
        _38975 = _37073_38951.e5;
        #line 88 "gpu_device.impala"
        int _38978;
        _38978 = _38956 * _38959;
        #line 88 "gpu_device.impala"
        int _38979;
        _38979 = _38978 + _38962;
        #line 73 "gpu_device.impala"
        int _38976;
        _38976 = 2 * _38975;
        #line 88 "gpu_device.impala"
        int _38980;
        _38980 = _38979 + _38976;
        #line 73 "gpu_device.impala"
        int offset_38977;
        offset_38977 = 5 + _38976;
        #line 88 "gpu_device.impala"
        int x_38981;
        x_38981 = 5 + _38980;
        #line 91 "gpu_device.impala"
        bool _38982;
        _38982 = offset_38977 < x_38981;
        #line 91 "gpu_device.impala"
        if (_38982) goto l38983; else goto l39091;
    l39091: ;
        #line 93 "gpu_device.impala"
        goto l39088;
    l38983: ;
        #line 91 "gpu_device.impala"
        int _38984;
        _38984 = _38975 + _38976;
        #line 91 "gpu_device.impala"
        int _38985;
        _38985 = 5 + _38984;
        #line 91 "gpu_device.impala"
        bool _38986;
        _38986 = x_38981 < _38985;
        #line 91 "gpu_device.impala"
        if (_38986) goto l38987; else goto l39090;
    l39090: ;
        #line 93 "gpu_device.impala"
        goto l39088;
    l38987: ;
        #line 89 "gpu_device.impala"
        int _38989;
        _38989 = _38965 * _38968;
        #line 89 "gpu_device.impala"
        int y_38990;
        y_38990 = _38989 + _38971;
        #line 91 "gpu_device.impala"
        bool _38991;
        _38991 = 1 < y_38990;
        #line 91 "gpu_device.impala"
        if (_38991) goto l38992; else goto l39089;
    l39089: ;
        #line 93 "gpu_device.impala"
        goto l39088;
    l38992: ;
        #line 72 "gpu_device.impala"
        int _38994;
        _38994 = _37073_38951.e6;
        #line 91 "gpu_device.impala"
        int _38995;
        _38995 = _38994 - 2;
        #line 91 "gpu_device.impala"
        bool _38996;
        _38996 = y_38990 < _38995;
        #line 91 "gpu_device.impala"
        if (_38996) goto l38997; else goto l39087;
    l39087: ;
        #line 93 "gpu_device.impala"
        goto l39088;
    l39088: ;
        #line 93 "gpu_device.impala"
        goto l39016;
    l38997: ;
        #line 170 "dsl.impala"
        int _39010;
        _39010 = y_38990 * _38975;
        #line 170 "dsl.impala"
        int _39011;
        _39011 = _39010 + _38980;
        #line 36 "/home/rafael/Utilities/anydsl/runtime/src/runtime.impala"
        struct_14391 _39009;
        _39009.e0 = 1;
        _39009.e1 = _37075_38953;
        #line 170 "dsl.impala"
        int _39012;
        _39012 = 5 + _39011;
        #line 13 "gpu_device.impala"
        write_38998(_39009, _39012, 0.000000e+00f);
    l39014: ;
        #line 175 "dsl.impala"
        int _39041;
        _39041 = 4 + _39011;
        #line 9 "gpu_device.impala"
        float* i_39061;
        i_39061 = _37071_38949 + 3;
        #line 9 "gpu_device.impala"
        float* i_39053;
        i_39053 = _37072_38950 + _39012;
        #line 175 "dsl.impala"
        int _39065;
        _39065 = 6 + _39011;
        #line 9 "gpu_device.impala"
        float* i_39050;
        i_39050 = _37071_38949 + 2;
        #line 9 "gpu_device.impala"
        float* i_39037;
        i_39037 = _37071_38949 + 1;
        #line 175 "dsl.impala"
        int _39027;
        _39027 = 3 + _39011;
        #line 9 "gpu_device.impala"
        float* i_39018;
        i_39018 = _37074_38952 + _39012;
        #line 9 "gpu_device.impala"
        float* i_39074;
        i_39074 = _37071_38949 + 4;
        #line 9 "gpu_device.impala"
        float* i_39023;
        i_39023 = _37071_38949 + 0;
        #line 175 "dsl.impala"
        int _39078;
        _39078 = 7 + _39011;
        #line 9 "gpu_device.impala"
        float* i_39066;
        i_39066 = _37072_38950 + _39065;
        #line 9 "gpu_device.impala"
        float* i_39042;
        i_39042 = _37072_38950 + _39041;
        #line 9 "gpu_device.impala"
        float* i_39028;
        i_39028 = _37072_38950 + _39027;
        #line 10 "gpu_device.impala"
        float _39019;
        _39019 = *i_39018;
        #line 9 "gpu_device.impala"
        float* i_39079;
        i_39079 = _37072_38950 + _39078;
        #line 10 "gpu_device.impala"
        float _39031;
        _39031 = _39019;
        #line 10 "gpu_device.impala"
        float _39024;
        _39024 = *i_39023;
        #line 10 "gpu_device.impala"
        float _39032;
        _39032 = _39024;
        #line 10 "gpu_device.impala"
        float _39029;
        _39029 = *i_39028;
        #line 10 "gpu_device.impala"
        float _39033;
        _39033 = _39029;
        #line 175 "dsl.impala"
        float _39034;
        _39034 = _39032 * _39033;
        #line 174 "dsl.impala"
        float _39035;
        _39035 = _39031 + _39034;
        #line 14 "gpu_device.impala"
        *i_39018 = _39035;
        #line 10 "gpu_device.impala"
        float _39038;
        _39038 = *i_39037;
        #line 10 "gpu_device.impala"
        float _39045;
        _39045 = _39038;
        #line 10 "gpu_device.impala"
        float _39043;
        _39043 = *i_39042;
        #line 10 "gpu_device.impala"
        float _39046;
        _39046 = _39043;
        #line 175 "dsl.impala"
        float _39047;
        _39047 = _39045 * _39046;
        #line 174 "dsl.impala"
        float _39048;
        _39048 = _39035 + _39047;
        #line 14 "gpu_device.impala"
        *i_39018 = _39048;
        #line 10 "gpu_device.impala"
        float _39051;
        _39051 = *i_39050;
        #line 10 "gpu_device.impala"
        float _39056;
        _39056 = _39051;
        #line 10 "gpu_device.impala"
        float _39054;
        _39054 = *i_39053;
        #line 10 "gpu_device.impala"
        float _39057;
        _39057 = _39054;
        #line 175 "dsl.impala"
        float _39058;
        _39058 = _39056 * _39057;
        #line 174 "dsl.impala"
        float _39059;
        _39059 = _39048 + _39058;
        #line 14 "gpu_device.impala"
        *i_39018 = _39059;
        #line 10 "gpu_device.impala"
        float _39062;
        _39062 = *i_39061;
        #line 10 "gpu_device.impala"
        float _39069;
        _39069 = _39062;
        #line 10 "gpu_device.impala"
        float _39067;
        _39067 = *i_39066;
        #line 10 "gpu_device.impala"
        float _39070;
        _39070 = _39067;
        #line 175 "dsl.impala"
        float _39071;
        _39071 = _39069 * _39070;
        #line 174 "dsl.impala"
        float _39072;
        _39072 = _39059 + _39071;
        #line 14 "gpu_device.impala"
        *i_39018 = _39072;
        #line 10 "gpu_device.impala"
        float _39075;
        _39075 = *i_39074;
        #line 10 "gpu_device.impala"
        float _39082;
        _39082 = _39075;
        #line 10 "gpu_device.impala"
        float _39080;
        _39080 = *i_39079;
        #line 10 "gpu_device.impala"
        float _39083;
        _39083 = _39080;
        #line 175 "dsl.impala"
        float _39084;
        _39084 = _39082 * _39083;
        #line 174 "dsl.impala"
        float _39085;
        _39085 = _39072 + _39084;
        #line 14 "gpu_device.impala"
        *i_39018 = _39085;
        #line 93 "gpu_device.impala"
        goto l39016;
    l39016: ;
        return ;
}

__global__ void lambda_crit_37448(char* _37451_39232, char* _37452_39233, struct_14389 _37453_39234, char* _37454_39235, char* _37455_39236) {
    int  _39239;
    int p_39239;
    int  _39242;
    int p_39242;
    int  _39245;
    int p_39245;
    int  _39248;
    int p_39248;
    int  _39251;
    int p_39251;
    int  _39254;
    int p_39254;
    float  read_39295;
    float pread_39295;
    float  read_39303;
    float pread_39303;
    float  read_39309;
    float pread_39309;
    float  read_39313;
    float pread_39313;
    float  read_39321;
    float pread_39321;
    float  read_39327;
    float pread_39327;
    float  read_39333;
    float pread_39333;
    float  read_39338;
    float pread_39338;
    float  read_39344;
    float pread_39344;
    float  read_39348;
    float pread_39348;
    float  read_39355;
    float pread_39355;
    float  read_39360;
    float pread_39360;
    float  read_39369;
    float pread_39369;
    float  read_39372;
    float pread_39372;
    float  read_39378;
    float pread_39378;
    float  read_39382;
    float pread_39382;
    float  read_39389;
    float pread_39389;
    float  read_39392;
    float pread_39392;
    float  read_39398;
    float pread_39398;
    float  read_39401;
    float pread_39401;
    float  read_39407;
    float pread_39407;
    float  read_39411;
    float pread_39411;
    float  read_39418;
    float pread_39418;
    float  read_39421;
    float pread_39421;
    bool  _39432;
    bool p_39432;
    bool  _39434;
    bool p_39434;
    float  read_39437;
    float pread_39437;
    float  read_39444;
    float pread_39444;
    float  read_39450;
    float pread_39450;
    float  read_39453;
    float pread_39453;
    float  read_39456;
    float pread_39456;
    float  read_39459;
    float pread_39459;
    #line 1 "/home/rafael/Utilities/anydsl/runtime/platforms/intrinsics_cuda.impala"
    _39239 = blockIdx_x();
    p_39239 = _39239;
    l39237: ;
        _39239 = p_39239;
        #line 1 "/home/rafael/Utilities/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _39242 = blockDim_x();
        p_39242 = _39242;
    l39240: ;
        _39242 = p_39242;
        #line 1 "/home/rafael/Utilities/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _39245 = threadIdx_x();
        p_39245 = _39245;
    l39243: ;
        _39245 = p_39245;
        #line 1 "/home/rafael/Utilities/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _39248 = blockIdx_y();
        p_39248 = _39248;
    l39246: ;
        _39248 = p_39248;
        #line 1 "/home/rafael/Utilities/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _39251 = blockDim_y();
        p_39251 = _39251;
    l39249: ;
        _39251 = p_39251;
        #line 1 "/home/rafael/Utilities/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _39254 = threadIdx_y();
        p_39254 = _39254;
    l39252: ;
        _39254 = p_39254;
        #line 135 "gpu_device.impala"
        int _39258;
        _39258 = _39239 * _39242;
        #line 242 "dsl.impala"
        int _39255;
        _39255 = _37453_39234.e5;
        #line 321 "dsl.impala"
        int _39256;
        _39256 = 2 * _39255;
        #line 321 "dsl.impala"
        int _39257;
        _39257 = 5 + _39256;
        #line 135 "gpu_device.impala"
        int _39259;
        _39259 = _39258 + _39245;
        #line 135 "gpu_device.impala"
        int _39260;
        _39260 = _39259 + _39256;
        #line 135 "gpu_device.impala"
        int x_39261;
        x_39261 = 5 + _39260;
        #line 138 "gpu_device.impala"
        bool _39262;
        _39262 = _39257 < x_39261;
        #line 138 "gpu_device.impala"
        if (_39262) goto l39263; else goto l39477;
    l39477: ;
        #line 140 "gpu_device.impala"
        goto l39474;
    l39263: ;
        #line 138 "gpu_device.impala"
        int _39264;
        _39264 = _39255 + _39256;
        #line 138 "gpu_device.impala"
        int _39265;
        _39265 = 5 + _39264;
        #line 138 "gpu_device.impala"
        bool _39266;
        _39266 = x_39261 < _39265;
        #line 138 "gpu_device.impala"
        if (_39266) goto l39267; else goto l39476;
    l39476: ;
        #line 140 "gpu_device.impala"
        goto l39474;
    l39267: ;
        #line 136 "gpu_device.impala"
        int _39268;
        _39268 = _39248 * _39251;
        #line 136 "gpu_device.impala"
        int y_39269;
        y_39269 = _39268 + _39254;
        #line 138 "gpu_device.impala"
        bool _39270;
        _39270 = 1 < y_39269;
        #line 138 "gpu_device.impala"
        if (_39270) goto l39271; else goto l39475;
    l39475: ;
        #line 140 "gpu_device.impala"
        goto l39474;
    l39271: ;
        #line 322 "dsl.impala"
        int _39272;
        _39272 = _37453_39234.e6;
        #line 322 "dsl.impala"
        int _39273;
        _39273 = _39272 - 2;
        #line 138 "gpu_device.impala"
        bool _39274;
        _39274 = y_39269 < _39273;
        #line 138 "gpu_device.impala"
        if (_39274) goto l39275; else goto l39473;
    l39473: ;
        #line 140 "gpu_device.impala"
        goto l39474;
    l39474: ;
        #line 140 "gpu_device.impala"
        goto l39466;
    l39275: ;
        #line 36 "/home/rafael/Utilities/anydsl/runtime/src/runtime.impala"
        struct_14391 _39276;
        _39276.e0 = 1;
        _39276.e1 = _37454_39235;
        #line 242 "dsl.impala"
        int _39277;
        _39277 = y_39269 * _39255;
        #line 242 "dsl.impala"
        int _39278;
        _39278 = _39277 + _39260;
        #line 242 "dsl.impala"
        int _39279;
        _39279 = 5 + _39278;
        #line 13 "gpu_device.impala"
        write_38998(_39276, _39279, 0.000000e+00f);
    l39280: ;
        #line 9 "gpu_device.impala"
        read_39295 = read_39282(_39276, _39279);
        pread_39295 = read_39295;
    l39293: ;
        read_39295 = pread_39295;
        #line 247 "dsl.impala"
        int _39299;
        _39299 = x_39261 - 1;
        #line 36 "/home/rafael/Utilities/anydsl/runtime/src/runtime.impala"
        struct_14391 _39296;
        _39296.e0 = 1;
        _39296.e1 = _37455_39236;
        #line 247 "dsl.impala"
        int _39297;
        _39297 = y_39269 - 1;
        #line 247 "dsl.impala"
        int _39298;
        _39298 = _39297 * _39255;
        #line 247 "dsl.impala"
        int _39300;
        _39300 = _39298 + _39299;
        #line 9 "gpu_device.impala"
        read_39303 = read_39282(_39296, _39300);
        pread_39303 = read_39303;
    l39301: ;
        read_39303 = pread_39303;
        #line 246 "dsl.impala"
        float _39304;
        _39304 = read_39295 - read_39303;
        #line 13 "gpu_device.impala"
        write_38998(_39276, _39279, _39304);
    l39305: ;
        #line 9 "gpu_device.impala"
        read_39309 = read_39282(_39276, _39279);
        pread_39309 = read_39309;
    l39307: ;
        read_39309 = pread_39309;
        #line 253 "dsl.impala"
        int _39310;
        _39310 = _39277 + _39299;
        #line 9 "gpu_device.impala"
        read_39313 = read_39282(_39296, _39310);
        pread_39313 = read_39313;
    l39311: ;
        read_39313 = pread_39313;
        #line 253 "dsl.impala"
        float _39315;
        _39315 = 2.000000e+00f * read_39313;
        #line 252 "dsl.impala"
        float _39316;
        _39316 = read_39309 - _39315;
        #line 13 "gpu_device.impala"
        write_38998(_39276, _39279, _39316);
    l39317: ;
        #line 9 "gpu_device.impala"
        read_39321 = read_39282(_39276, _39279);
        pread_39321 = read_39321;
    l39319: ;
        read_39321 = pread_39321;
        #line 259 "dsl.impala"
        int _39322;
        _39322 = 1 + y_39269;
        #line 259 "dsl.impala"
        int _39323;
        _39323 = _39322 * _39255;
        #line 259 "dsl.impala"
        int _39324;
        _39324 = _39323 + _39299;
        #line 9 "gpu_device.impala"
        read_39327 = read_39282(_39296, _39324);
        pread_39327 = read_39327;
    l39325: ;
        read_39327 = pread_39327;
        #line 258 "dsl.impala"
        float _39328;
        _39328 = read_39321 - read_39327;
        #line 13 "gpu_device.impala"
        write_38998(_39276, _39279, _39328);
    l39329: ;
        #line 9 "gpu_device.impala"
        read_39333 = read_39282(_39276, _39279);
        pread_39333 = read_39333;
    l39331: ;
        read_39333 = pread_39333;
        #line 265 "dsl.impala"
        int _39334;
        _39334 = _39298 + _39260;
        #line 265 "dsl.impala"
        int _39335;
        _39335 = 6 + _39334;
        #line 9 "gpu_device.impala"
        read_39338 = read_39282(_39296, _39335);
        pread_39338 = read_39338;
    l39336: ;
        read_39338 = pread_39338;
        #line 264 "dsl.impala"
        float _39339;
        _39339 = read_39333 + read_39338;
        #line 13 "gpu_device.impala"
        write_38998(_39276, _39279, _39339);
    l39340: ;
        #line 9 "gpu_device.impala"
        read_39344 = read_39282(_39276, _39279);
        pread_39344 = read_39344;
    l39342: ;
        read_39344 = pread_39344;
        #line 271 "dsl.impala"
        int _39345;
        _39345 = 6 + _39278;
        #line 9 "gpu_device.impala"
        read_39348 = read_39282(_39296, _39345);
        pread_39348 = read_39348;
    l39346: ;
        read_39348 = pread_39348;
        #line 271 "dsl.impala"
        float _39349;
        _39349 = 2.000000e+00f * read_39348;
        #line 270 "dsl.impala"
        float _39350;
        _39350 = read_39344 + _39349;
        #line 13 "gpu_device.impala"
        write_38998(_39276, _39279, _39350);
    l39351: ;
        #line 9 "gpu_device.impala"
        read_39355 = read_39282(_39276, _39279);
        pread_39355 = read_39355;
    l39353: ;
        read_39355 = pread_39355;
        #line 277 "dsl.impala"
        int _39356;
        _39356 = _39323 + _39260;
        #line 277 "dsl.impala"
        int _39357;
        _39357 = 6 + _39356;
        #line 9 "gpu_device.impala"
        read_39360 = read_39282(_39296, _39357);
        pread_39360 = read_39360;
    l39358: ;
        read_39360 = pread_39360;
        #line 276 "dsl.impala"
        float _39361;
        _39361 = read_39355 + read_39360;
        #line 13 "gpu_device.impala"
        write_38998(_39276, _39279, _39361);
    l39362: ;
        #line 36 "/home/rafael/Utilities/anydsl/runtime/src/runtime.impala"
        struct_14391 _39364;
        _39364.e0 = 1;
        _39364.e1 = _37452_39233;
        #line 13 "gpu_device.impala"
        write_38998(_39364, _39279, 0.000000e+00f);
    l39365: ;
        #line 9 "gpu_device.impala"
        read_39369 = read_39282(_39364, _39279);
        pread_39369 = read_39369;
    l39367: ;
        read_39369 = pread_39369;
        #line 9 "gpu_device.impala"
        read_39372 = read_39282(_39296, _39300);
        pread_39372 = read_39372;
    l39370: ;
        read_39372 = pread_39372;
        #line 285 "dsl.impala"
        float _39373;
        _39373 = read_39369 - read_39372;
        #line 13 "gpu_device.impala"
        write_38998(_39364, _39279, _39373);
    l39374: ;
        #line 9 "gpu_device.impala"
        read_39378 = read_39282(_39364, _39279);
        pread_39378 = read_39378;
    l39376: ;
        read_39378 = pread_39378;
        #line 265 "dsl.impala"
        int _39379;
        _39379 = 5 + _39334;
        #line 9 "gpu_device.impala"
        read_39382 = read_39282(_39296, _39379);
        pread_39382 = read_39382;
    l39380: ;
        read_39382 = pread_39382;
        #line 292 "dsl.impala"
        float _39383;
        _39383 = 2.000000e+00f * read_39382;
        #line 291 "dsl.impala"
        float _39384;
        _39384 = read_39378 - _39383;
        #line 13 "gpu_device.impala"
        write_38998(_39364, _39279, _39384);
    l39385: ;
        #line 9 "gpu_device.impala"
        read_39389 = read_39282(_39364, _39279);
        pread_39389 = read_39389;
    l39387: ;
        read_39389 = pread_39389;
        #line 9 "gpu_device.impala"
        read_39392 = read_39282(_39296, _39335);
        pread_39392 = read_39392;
    l39390: ;
        read_39392 = pread_39392;
        #line 297 "dsl.impala"
        float _39393;
        _39393 = read_39389 - read_39392;
        #line 13 "gpu_device.impala"
        write_38998(_39364, _39279, _39393);
    l39394: ;
        #line 9 "gpu_device.impala"
        read_39398 = read_39282(_39364, _39279);
        pread_39398 = read_39398;
    l39396: ;
        read_39398 = pread_39398;
        #line 9 "gpu_device.impala"
        read_39401 = read_39282(_39296, _39324);
        pread_39401 = read_39401;
    l39399: ;
        read_39401 = pread_39401;
        #line 303 "dsl.impala"
        float _39402;
        _39402 = read_39398 + read_39401;
        #line 13 "gpu_device.impala"
        write_38998(_39364, _39279, _39402);
    l39403: ;
        #line 9 "gpu_device.impala"
        read_39407 = read_39282(_39364, _39279);
        pread_39407 = read_39407;
    l39405: ;
        read_39407 = pread_39407;
        #line 277 "dsl.impala"
        int _39408;
        _39408 = 5 + _39356;
        #line 9 "gpu_device.impala"
        read_39411 = read_39282(_39296, _39408);
        pread_39411 = read_39411;
    l39409: ;
        read_39411 = pread_39411;
        #line 310 "dsl.impala"
        float _39412;
        _39412 = 2.000000e+00f * read_39411;
        #line 309 "dsl.impala"
        float _39413;
        _39413 = read_39407 + _39412;
        #line 13 "gpu_device.impala"
        write_38998(_39364, _39279, _39413);
    l39414: ;
        #line 9 "gpu_device.impala"
        read_39418 = read_39282(_39364, _39279);
        pread_39418 = read_39418;
    l39416: ;
        read_39418 = pread_39418;
        #line 9 "gpu_device.impala"
        read_39421 = read_39282(_39296, _39357);
        pread_39421 = read_39421;
    l39419: ;
        read_39421 = pread_39421;
        #line 315 "dsl.impala"
        float _39422;
        _39422 = read_39418 + read_39421;
        #line 13 "gpu_device.impala"
        write_38998(_39364, _39279, _39422);
    l39423: ;
        #line 138 "gpu_device.impala"
        if (_39262) goto l39425; else goto l39472;
    l39472: ;
        #line 322 "dsl.impala"
        goto l39471;
    l39425: ;
        #line 321 "dsl.impala"
        int _39426;
        _39426 = 3 * _39255;
        #line 321 "dsl.impala"
        int _39427;
        _39427 = 4 + _39426;
        #line 321 "dsl.impala"
        bool _39428;
        _39428 = x_39261 < _39427;
        #line 321 "dsl.impala"
        if (_39428) goto l39429; else goto l39470;
    l39470: ;
        #line 322 "dsl.impala"
        goto l39471;
    l39471: ;
        #line 322 "dsl.impala"
        p_39434 = false;
        goto l39433;
    l39429: ;
        #line 138 "gpu_device.impala"
        if (_39270) goto l39430; else goto l39468;
    l39468: ;
        #line 322 "dsl.impala"
        p_39432 = false;
        goto l39431;
    l39430: ;
        #line 322 "dsl.impala"
        p_39432 = _39274;
        goto l39431;
    l39431: ;
        _39432 = p_39432;
        #line 322 "dsl.impala"
        p_39434 = _39432;
        goto l39433;
    l39433: ;
        _39434 = p_39434;
        #line 9 "gpu_device.impala"
        read_39437 = read_39282(_39276, _39279);
        pread_39437 = read_39437;
    l39435: ;
        read_39437 = pread_39437;
        #line 320 "dsl.impala"
        float condition_39438;
        condition_39438 = (float)_39434;
        #line 327 "dsl.impala"
        float _39439;
        _39439 = read_39437 * condition_39438;
        #line 13 "gpu_device.impala"
        write_38998(_39276, _39279, _39439);
    l39440: ;
        #line 9 "gpu_device.impala"
        read_39444 = read_39282(_39364, _39279);
        pread_39444 = read_39444;
    l39442: ;
        read_39444 = pread_39444;
        #line 332 "dsl.impala"
        float _39445;
        _39445 = read_39444 * condition_39438;
        #line 13 "gpu_device.impala"
        write_38998(_39364, _39279, _39445);
    l39446: ;
        #line 9 "gpu_device.impala"
        read_39450 = read_39282(_39276, _39279);
        pread_39450 = read_39450;
    l39448: ;
        read_39450 = pread_39450;
        #line 9 "gpu_device.impala"
        read_39453 = read_39282(_39276, _39279);
        pread_39453 = read_39453;
    l39451: ;
        read_39453 = pread_39453;
        #line 9 "gpu_device.impala"
        read_39456 = read_39282(_39364, _39279);
        pread_39456 = read_39456;
    l39454: ;
        read_39456 = pread_39456;
        #line 9 "gpu_device.impala"
        read_39459 = read_39282(_39364, _39279);
        pread_39459 = read_39459;
    l39457: ;
        read_39459 = pread_39459;
        #line 337 "dsl.impala"
        float _39461;
        _39461 = read_39450 * read_39453;
        #line 338 "dsl.impala"
        float _39462;
        _39462 = read_39456 * read_39459;
        #line 36 "/home/rafael/Utilities/anydsl/runtime/src/runtime.impala"
        struct_14391 _39460;
        _39460.e0 = 1;
        _39460.e1 = _37451_39232;
        #line 337 "dsl.impala"
        float _39463;
        _39463 = _39461 + _39462;
        #line 13 "gpu_device.impala"
        write_38998(_39460, _39279, _39463);
    l39464: ;
        #line 140 "gpu_device.impala"
        goto l39466;
    l39466: ;
        return ;
}

__global__ void lambda_36802() {
    int  _38915;
    int p_38915;
    int  _38921;
    int p_38921;
    int  _38927;
    int p_38927;
    int  _38933;
    int p_38933;
    int  _38939;
    int p_38939;
    int  _38945;
    int p_38945;
    #line 1 "/home/rafael/Utilities/anydsl/runtime/platforms/intrinsics_cuda.impala"
    _38915 = blockIdx_x();
    p_38915 = _38915;
    l38913: ;
        _38915 = p_38915;
        #line 1 "/home/rafael/Utilities/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _38921 = blockDim_x();
        p_38921 = _38921;
    l38919: ;
        _38921 = p_38921;
        #line 1 "/home/rafael/Utilities/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _38927 = threadIdx_x();
        p_38927 = _38927;
    l38925: ;
        _38927 = p_38927;
        #line 1 "/home/rafael/Utilities/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _38933 = blockIdx_y();
        p_38933 = _38933;
    l38931: ;
        _38933 = p_38933;
        #line 1 "/home/rafael/Utilities/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _38939 = blockDim_y();
        p_38939 = _38939;
    l38937: ;
        _38939 = p_38939;
        #line 1 "/home/rafael/Utilities/anydsl/runtime/platforms/intrinsics_cuda.impala"
        _38945 = threadIdx_y();
        p_38945 = _38945;
    l38943: ;
        _38945 = p_38945;
        return ;
}

__device__ int abs_39538(int a) {
  return a * (a >= 0) - a * (a < 0);
}

__device__ void write_38998(struct_14391, int, float);
__device__ void write_38998(struct_14391 buf_39876, int i_39877, float v_39878) {
    #line 14 "gpu_device.impala"
    char* _39881;
    _39881 = buf_39876.e1;
    *((float *) _39881 + i_39877) = v_39878;
}

__device__ float read_39282(struct_14391 buf_39423, int i_39424) {
    #line 10 "gpu_device.impala"
    char* _39426;
    _39426 = buf_39423.e1;
    return *((float *) _39426 + i_39424);
}

}
