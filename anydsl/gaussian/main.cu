
#include <hip/hip_runtime.h>
extern "C" {
typedef struct array_1590 {
    float e[3];
} array_1590;
typedef struct array_1591 {
    array_1590 e[3];
} array_1591;

__device__ inline int threadIdx_x() { return threadIdx.x; }
__device__ inline int threadIdx_y() { return threadIdx.y; }
__device__ inline int threadIdx_z() { return threadIdx.z; }
__device__ inline int blockIdx_x() { return blockIdx.x; }
__device__ inline int blockIdx_y() { return blockIdx.y; }
__device__ inline int blockIdx_z() { return blockIdx.z; }
__device__ inline int blockDim_x() { return blockDim.x; }
__device__ inline int blockDim_y() { return blockDim.y; }
__device__ inline int blockDim_z() { return blockDim.z; }
__device__ inline int gridDim_x() { return gridDim.x; }
__device__ inline int gridDim_y() { return gridDim.y; }
__device__ inline int gridDim_z() { return gridDim.z; }
__global__ void lambda_crit_1951(int*, int, int, array_1591, int, int);
__device__ int acc_bidx_2355();
__device__ int acc_bdimx_2367();
__device__ int acc_tidx_2379();
__device__ int acc_bidy_2391();
__device__ int acc_bdimy_2403();
__device__ int acc_tidy_2415();

__global__ void lambda_crit_1951(int* _1954_2349, int _1955_2350, int _1956_2351, array_1591 _1957_2352, int _1958_2353, int _1959_2354) {
    int  acc_bidx_2366;
    int pacc_bidx_2366;
    int  acc_bdimx_2378;
    int pacc_bdimx_2378;
    int  acc_tidx_2390;
    int pacc_tidx_2390;
    int  acc_bidy_2402;
    int pacc_bidy_2402;
    int  acc_bdimy_2414;
    int pacc_bdimy_2414;
    int  acc_tidy_2426;
    int pacc_tidy_2426;
    int  min_2428;
    int pmin_2428;
    int*  data_2430;
    int* pdata_2430;
    float  sum_2431;
    float psum_2431;
    int  min_2435;
    int pmin_2435;
    float  sum_2437;
    float psum_2437;
    float  sum_2457;
    float psum_2457;
    acc_bidx_2366 = acc_bidx_2355();
    pacc_bidx_2366 = acc_bidx_2366;
    l2364: ;
        acc_bidx_2366 = pacc_bidx_2366;
        acc_bdimx_2378 = acc_bdimx_2367();
        pacc_bdimx_2378 = acc_bdimx_2378;
    l2376: ;
        acc_bdimx_2378 = pacc_bdimx_2378;
        acc_tidx_2390 = acc_tidx_2379();
        pacc_tidx_2390 = acc_tidx_2390;
    l2388: ;
        acc_tidx_2390 = pacc_tidx_2390;
        acc_bidy_2402 = acc_bidy_2391();
        pacc_bidy_2402 = acc_bidy_2402;
    l2400: ;
        acc_bidy_2402 = pacc_bidy_2402;
        acc_bdimy_2414 = acc_bdimy_2403();
        pacc_bdimy_2414 = acc_bdimy_2414;
    l2412: ;
        acc_bdimy_2414 = pacc_bdimy_2414;
        acc_tidy_2426 = acc_tidy_2415();
        pacc_tidy_2426 = acc_tidy_2426;
    l2424: ;
        acc_tidy_2426 = pacc_tidy_2426;
        int _2441;
        _2441 = acc_bidx_2366 * acc_bdimx_2378;
        int x_2442;
        x_2442 = _2441 + acc_tidx_2390;
        int _2448;
        _2448 = acc_bidy_2402 * acc_bdimy_2414;
        int y_2449;
        y_2449 = _2448 + acc_tidy_2426;
        pmin_2428 = _1959_2354;
        pdata_2430 = _1954_2349;
        psum_2431 = 0.000000e+00f;
        goto l2427;
    l2427: ;
        min_2428 = pmin_2428;
        data_2430 = pdata_2430;
        sum_2431 = psum_2431;
        bool _2432;
        _2432 = min_2428 < _1956_2351;
        if (_2432) goto l2433; else goto l2482;
    l2482: ;
        int _2483;
        _2483 = y_2449 * _1955_2350;
        int _2484;
        _2484 = _2483 + x_2442;
        int* _2485;
        _2485 = data_2430 + _2484;
        int _2486;
        _2486 = (int)sum_2431;
        *_2485 = _2486;
        return ;
    l2433: ;
        int _2443;
        _2443 = x_2442 + min_2428;
        bool _2444;
        _2444 = -1 < _2443;
        bool _2446;
        _2446 = _2443 < _1955_2350;
        int _2466;
        _2466 = min_2428 + _1956_2351;
        array_1590 _2467;
        _2467 = _1957_2352.e[_2466];
        pmin_2435 = _1959_2354;
        psum_2437 = sum_2431;
        goto l2434;
    l2434: ;
        min_2435 = pmin_2435;
        sum_2437 = psum_2437;
        bool _2438;
        _2438 = min_2435 < _1956_2351;
        if (_2438) goto l2439; else goto l2480;
    l2480: ;
        int _2481;
        _2481 = 1 + min_2428;
        pmin_2428 = _2481;
        pdata_2430 = data_2430;
        psum_2431 = sum_2437;
        goto l2427;
    l2439: ;
        if (_2444) goto l2445; else goto l2479;
    l2479: ;
        goto l2476;
    l2445: ;
        if (_2446) goto l2447; else goto l2478;
    l2478: ;
        goto l2476;
    l2447: ;
        int _2450;
        _2450 = y_2449 + min_2435;
        bool _2451;
        _2451 = -1 < _2450;
        if (_2451) goto l2452; else goto l2477;
    l2477: ;
        goto l2476;
    l2452: ;
        bool _2453;
        _2453 = _2450 < _1958_2353;
        if (_2453) goto l2454; else goto l2475;
    l2475: ;
        goto l2476;
    l2476: ;
        psum_2457 = sum_2437;
        goto l2455;
    l2454: ;
        int _2460;
        _2460 = _2450 * _1955_2350;
        int _2461;
        _2461 = _2460 + _2443;
        int* _2462;
        _2462 = data_2430 + _2461;
        int _2463;
        _2463 = *_2462;
        int _2468;
        _2468 = min_2435 + _1956_2351;
        float _2469;
        _2469 = _2467.e[_2468];
        int _2471;
        _2471 = _2463;
        float _2472;
        _2472 = (float)_2471;
        float _2473;
        _2473 = _2469 * _2472;
        float _2474;
        _2474 = sum_2437 + _2473;
        psum_2457 = _2474;
        goto l2455;
    l2455: ;
        sum_2457 = psum_2457;
        int _2459;
        _2459 = 1 + min_2435;
        pmin_2435 = _2459;
        psum_2437 = sum_2457;
        goto l2434;
}

__device__ int acc_bidx_2355() {
    int  blockIdx_x_2363;
    int pblockIdx_x_2363;
    blockIdx_x_2363 = blockIdx_x();
    pblockIdx_x_2363 = blockIdx_x_2363;
    l2361: ;
        blockIdx_x_2363 = pblockIdx_x_2363;
        return blockIdx_x_2363;
}

__device__ int acc_bdimx_2367() {
    int  blockDim_x_2375;
    int pblockDim_x_2375;
    blockDim_x_2375 = blockDim_x();
    pblockDim_x_2375 = blockDim_x_2375;
    l2373: ;
        blockDim_x_2375 = pblockDim_x_2375;
        return blockDim_x_2375;
}

__device__ int acc_tidx_2379() {
    int  threadIdx_x_2387;
    int pthreadIdx_x_2387;
    threadIdx_x_2387 = threadIdx_x();
    pthreadIdx_x_2387 = threadIdx_x_2387;
    l2385: ;
        threadIdx_x_2387 = pthreadIdx_x_2387;
        return threadIdx_x_2387;
}

__device__ int acc_bidy_2391() {
    int  blockIdx_y_2399;
    int pblockIdx_y_2399;
    blockIdx_y_2399 = blockIdx_y();
    pblockIdx_y_2399 = blockIdx_y_2399;
    l2397: ;
        blockIdx_y_2399 = pblockIdx_y_2399;
        return blockIdx_y_2399;
}

__device__ int acc_bdimy_2403() {
    int  blockDim_y_2411;
    int pblockDim_y_2411;
    blockDim_y_2411 = blockDim_y();
    pblockDim_y_2411 = blockDim_y_2411;
    l2409: ;
        blockDim_y_2411 = pblockDim_y_2411;
        return blockDim_y_2411;
}

__device__ int acc_tidy_2415() {
    int  threadIdx_y_2423;
    int pthreadIdx_y_2423;
    threadIdx_y_2423 = threadIdx_y();
    pthreadIdx_y_2423 = threadIdx_y_2423;
    l2421: ;
        threadIdx_y_2423 = pthreadIdx_y_2423;
        return threadIdx_y_2423;
}

}
