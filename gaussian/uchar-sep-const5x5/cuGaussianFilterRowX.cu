#include "hip/hip_runtime.h"
#ifndef _CUGAUSSIANFILTERROWX_CU_
#define _CUGAUSSIANFILTERROWX_CU_

#include "hipacc_types.hpp"
#include "hipacc_math_functions.hpp"

texture<uchar, hipTextureType1D, hipReadModeElementType> _texinputX;
const textureReference *_texinputXRef;

extern "C" {
__global__ __launch_bounds__ (64*1) void cuGaussianFilterRowXKernel(float * __restrict__ iter, int iter_width, int iter_height, int iter_stride, int input_width, int input_height, int input_stride, int bh_start_left, int bh_start_right, int bh_start_bottom, int bh_fall_back) {
    const int gid_x = blockDim.x * blockIdx.x + threadIdx.x;
    const int gid_y = blockDim.y * blockIdx.y * 8 + threadIdx.y;
    if (bh_fall_back)
        goto BH_FB;
    if (blockIdx.y >= bh_start_bottom)
        goto BH_B;
    if (blockIdx.x >= bh_start_right)
        goto BH_R;
    if (blockIdx.x < bh_start_left)
        goto BH_L;
    goto BH_NO;
  BH_FB:
    {
        if (gid_x < iter_width) {
            if (gid_y < iter_height) {
                float _tmp0 = 0.F;
                {
                    int _gid_x1 = gid_x + -2;
                    int _gid_y1 = gid_y + 0;
                    if (_gid_x1 >= input_width)
                        _gid_x1 = input_width - 1;
                    if (_gid_x1 < 0)
                        _gid_x1 = 0;
                    _tmp0 += 0.0707660019F * tex1Dfetch(_texinputX, (_gid_y1) * input_stride + _gid_x1);
                }
                {
                    int _gid_x2 = gid_x + -1;
                    int _gid_y2 = gid_y + 0;
                    if (_gid_x2 >= input_width)
                        _gid_x2 = input_width - 1;
                    if (_gid_x2 < 0)
                        _gid_x2 = 0;
                    _tmp0 += 0.244460002F * tex1Dfetch(_texinputX, (_gid_y2) * input_stride + _gid_x2);
                }
                {
                    int _gid_x3 = gid_x + 0;
                    int _gid_y3 = gid_y + 0;
                    if (_gid_x3 >= input_width)
                        _gid_x3 = input_width - 1;
                    if (_gid_x3 < 0)
                        _gid_x3 = 0;
                    _tmp0 += 0.369545996F * tex1Dfetch(_texinputX, (_gid_y3) * input_stride + _gid_x3);
                }
                {
                    int _gid_x4 = gid_x + 1;
                    int _gid_y4 = gid_y + 0;
                    if (_gid_x4 >= input_width)
                        _gid_x4 = input_width - 1;
                    if (_gid_x4 < 0)
                        _gid_x4 = 0;
                    _tmp0 += 0.244460002F * tex1Dfetch(_texinputX, (_gid_y4) * input_stride + _gid_x4);
                }
                {
                    int _gid_x5 = gid_x + 2;
                    int _gid_y5 = gid_y + 0;
                    if (_gid_x5 >= input_width)
                        _gid_x5 = input_width - 1;
                    if (_gid_x5 < 0)
                        _gid_x5 = 0;
                    _tmp0 += 0.0707660019F * tex1Dfetch(_texinputX, (_gid_y5) * input_stride + _gid_x5);
                }
                iter[(gid_y) * iter_stride + gid_x] = (float)(_tmp0);
            }
        }
        if (gid_x < iter_width) {
            if (gid_y + 1 * (int)blockDim.y < iter_height) {
                float _tmp6 = 0.F;
                {
                    int _gid_x7 = gid_x + -2;
                    int _gid_y7 = gid_y + 1 * (int)blockDim.y + 0;
                    if (_gid_x7 >= input_width)
                        _gid_x7 = input_width - 1;
                    if (_gid_x7 < 0)
                        _gid_x7 = 0;
                    _tmp6 += 0.0707660019F * tex1Dfetch(_texinputX, (_gid_y7) * input_stride + _gid_x7);
                }
                {
                    int _gid_x8 = gid_x + -1;
                    int _gid_y8 = gid_y + 1 * (int)blockDim.y + 0;
                    if (_gid_x8 >= input_width)
                        _gid_x8 = input_width - 1;
                    if (_gid_x8 < 0)
                        _gid_x8 = 0;
                    _tmp6 += 0.244460002F * tex1Dfetch(_texinputX, (_gid_y8) * input_stride + _gid_x8);
                }
                {
                    int _gid_x9 = gid_x + 0;
                    int _gid_y9 = gid_y + 1 * (int)blockDim.y + 0;
                    if (_gid_x9 >= input_width)
                        _gid_x9 = input_width - 1;
                    if (_gid_x9 < 0)
                        _gid_x9 = 0;
                    _tmp6 += 0.369545996F * tex1Dfetch(_texinputX, (_gid_y9) * input_stride + _gid_x9);
                }
                {
                    int _gid_x10 = gid_x + 1;
                    int _gid_y10 = gid_y + 1 * (int)blockDim.y + 0;
                    if (_gid_x10 >= input_width)
                        _gid_x10 = input_width - 1;
                    if (_gid_x10 < 0)
                        _gid_x10 = 0;
                    _tmp6 += 0.244460002F * tex1Dfetch(_texinputX, (_gid_y10) * input_stride + _gid_x10);
                }
                {
                    int _gid_x11 = gid_x + 2;
                    int _gid_y11 = gid_y + 1 * (int)blockDim.y + 0;
                    if (_gid_x11 >= input_width)
                        _gid_x11 = input_width - 1;
                    if (_gid_x11 < 0)
                        _gid_x11 = 0;
                    _tmp6 += 0.0707660019F * tex1Dfetch(_texinputX, (_gid_y11) * input_stride + _gid_x11);
                }
                iter[(gid_y + 1 * (int)blockDim.y) * iter_stride + gid_x] = (float)(_tmp6);
            }
        }
        if (gid_x < iter_width) {
            if (gid_y + 2 * (int)blockDim.y < iter_height) {
                float _tmp12 = 0.F;
                {
                    int _gid_x13 = gid_x + -2;
                    int _gid_y13 = gid_y + 2 * (int)blockDim.y + 0;
                    if (_gid_x13 >= input_width)
                        _gid_x13 = input_width - 1;
                    if (_gid_x13 < 0)
                        _gid_x13 = 0;
                    _tmp12 += 0.0707660019F * tex1Dfetch(_texinputX, (_gid_y13) * input_stride + _gid_x13);
                }
                {
                    int _gid_x14 = gid_x + -1;
                    int _gid_y14 = gid_y + 2 * (int)blockDim.y + 0;
                    if (_gid_x14 >= input_width)
                        _gid_x14 = input_width - 1;
                    if (_gid_x14 < 0)
                        _gid_x14 = 0;
                    _tmp12 += 0.244460002F * tex1Dfetch(_texinputX, (_gid_y14) * input_stride + _gid_x14);
                }
                {
                    int _gid_x15 = gid_x + 0;
                    int _gid_y15 = gid_y + 2 * (int)blockDim.y + 0;
                    if (_gid_x15 >= input_width)
                        _gid_x15 = input_width - 1;
                    if (_gid_x15 < 0)
                        _gid_x15 = 0;
                    _tmp12 += 0.369545996F * tex1Dfetch(_texinputX, (_gid_y15) * input_stride + _gid_x15);
                }
                {
                    int _gid_x16 = gid_x + 1;
                    int _gid_y16 = gid_y + 2 * (int)blockDim.y + 0;
                    if (_gid_x16 >= input_width)
                        _gid_x16 = input_width - 1;
                    if (_gid_x16 < 0)
                        _gid_x16 = 0;
                    _tmp12 += 0.244460002F * tex1Dfetch(_texinputX, (_gid_y16) * input_stride + _gid_x16);
                }
                {
                    int _gid_x17 = gid_x + 2;
                    int _gid_y17 = gid_y + 2 * (int)blockDim.y + 0;
                    if (_gid_x17 >= input_width)
                        _gid_x17 = input_width - 1;
                    if (_gid_x17 < 0)
                        _gid_x17 = 0;
                    _tmp12 += 0.0707660019F * tex1Dfetch(_texinputX, (_gid_y17) * input_stride + _gid_x17);
                }
                iter[(gid_y + 2 * (int)blockDim.y) * iter_stride + gid_x] = (float)(_tmp12);
            }
        }
        if (gid_x < iter_width) {
            if (gid_y + 3 * (int)blockDim.y < iter_height) {
                float _tmp18 = 0.F;
                {
                    int _gid_x19 = gid_x + -2;
                    int _gid_y19 = gid_y + 3 * (int)blockDim.y + 0;
                    if (_gid_x19 >= input_width)
                        _gid_x19 = input_width - 1;
                    if (_gid_x19 < 0)
                        _gid_x19 = 0;
                    _tmp18 += 0.0707660019F * tex1Dfetch(_texinputX, (_gid_y19) * input_stride + _gid_x19);
                }
                {
                    int _gid_x20 = gid_x + -1;
                    int _gid_y20 = gid_y + 3 * (int)blockDim.y + 0;
                    if (_gid_x20 >= input_width)
                        _gid_x20 = input_width - 1;
                    if (_gid_x20 < 0)
                        _gid_x20 = 0;
                    _tmp18 += 0.244460002F * tex1Dfetch(_texinputX, (_gid_y20) * input_stride + _gid_x20);
                }
                {
                    int _gid_x21 = gid_x + 0;
                    int _gid_y21 = gid_y + 3 * (int)blockDim.y + 0;
                    if (_gid_x21 >= input_width)
                        _gid_x21 = input_width - 1;
                    if (_gid_x21 < 0)
                        _gid_x21 = 0;
                    _tmp18 += 0.369545996F * tex1Dfetch(_texinputX, (_gid_y21) * input_stride + _gid_x21);
                }
                {
                    int _gid_x22 = gid_x + 1;
                    int _gid_y22 = gid_y + 3 * (int)blockDim.y + 0;
                    if (_gid_x22 >= input_width)
                        _gid_x22 = input_width - 1;
                    if (_gid_x22 < 0)
                        _gid_x22 = 0;
                    _tmp18 += 0.244460002F * tex1Dfetch(_texinputX, (_gid_y22) * input_stride + _gid_x22);
                }
                {
                    int _gid_x23 = gid_x + 2;
                    int _gid_y23 = gid_y + 3 * (int)blockDim.y + 0;
                    if (_gid_x23 >= input_width)
                        _gid_x23 = input_width - 1;
                    if (_gid_x23 < 0)
                        _gid_x23 = 0;
                    _tmp18 += 0.0707660019F * tex1Dfetch(_texinputX, (_gid_y23) * input_stride + _gid_x23);
                }
                iter[(gid_y + 3 * (int)blockDim.y) * iter_stride + gid_x] = (float)(_tmp18);
            }
        }
        if (gid_x < iter_width) {
            if (gid_y + 4 * (int)blockDim.y < iter_height) {
                float _tmp24 = 0.F;
                {
                    int _gid_x25 = gid_x + -2;
                    int _gid_y25 = gid_y + 4 * (int)blockDim.y + 0;
                    if (_gid_x25 >= input_width)
                        _gid_x25 = input_width - 1;
                    if (_gid_x25 < 0)
                        _gid_x25 = 0;
                    _tmp24 += 0.0707660019F * tex1Dfetch(_texinputX, (_gid_y25) * input_stride + _gid_x25);
                }
                {
                    int _gid_x26 = gid_x + -1;
                    int _gid_y26 = gid_y + 4 * (int)blockDim.y + 0;
                    if (_gid_x26 >= input_width)
                        _gid_x26 = input_width - 1;
                    if (_gid_x26 < 0)
                        _gid_x26 = 0;
                    _tmp24 += 0.244460002F * tex1Dfetch(_texinputX, (_gid_y26) * input_stride + _gid_x26);
                }
                {
                    int _gid_x27 = gid_x + 0;
                    int _gid_y27 = gid_y + 4 * (int)blockDim.y + 0;
                    if (_gid_x27 >= input_width)
                        _gid_x27 = input_width - 1;
                    if (_gid_x27 < 0)
                        _gid_x27 = 0;
                    _tmp24 += 0.369545996F * tex1Dfetch(_texinputX, (_gid_y27) * input_stride + _gid_x27);
                }
                {
                    int _gid_x28 = gid_x + 1;
                    int _gid_y28 = gid_y + 4 * (int)blockDim.y + 0;
                    if (_gid_x28 >= input_width)
                        _gid_x28 = input_width - 1;
                    if (_gid_x28 < 0)
                        _gid_x28 = 0;
                    _tmp24 += 0.244460002F * tex1Dfetch(_texinputX, (_gid_y28) * input_stride + _gid_x28);
                }
                {
                    int _gid_x29 = gid_x + 2;
                    int _gid_y29 = gid_y + 4 * (int)blockDim.y + 0;
                    if (_gid_x29 >= input_width)
                        _gid_x29 = input_width - 1;
                    if (_gid_x29 < 0)
                        _gid_x29 = 0;
                    _tmp24 += 0.0707660019F * tex1Dfetch(_texinputX, (_gid_y29) * input_stride + _gid_x29);
                }
                iter[(gid_y + 4 * (int)blockDim.y) * iter_stride + gid_x] = (float)(_tmp24);
            }
        }
        if (gid_x < iter_width) {
            if (gid_y + 5 * (int)blockDim.y < iter_height) {
                float _tmp30 = 0.F;
                {
                    int _gid_x31 = gid_x + -2;
                    int _gid_y31 = gid_y + 5 * (int)blockDim.y + 0;
                    if (_gid_x31 >= input_width)
                        _gid_x31 = input_width - 1;
                    if (_gid_x31 < 0)
                        _gid_x31 = 0;
                    _tmp30 += 0.0707660019F * tex1Dfetch(_texinputX, (_gid_y31) * input_stride + _gid_x31);
                }
                {
                    int _gid_x32 = gid_x + -1;
                    int _gid_y32 = gid_y + 5 * (int)blockDim.y + 0;
                    if (_gid_x32 >= input_width)
                        _gid_x32 = input_width - 1;
                    if (_gid_x32 < 0)
                        _gid_x32 = 0;
                    _tmp30 += 0.244460002F * tex1Dfetch(_texinputX, (_gid_y32) * input_stride + _gid_x32);
                }
                {
                    int _gid_x33 = gid_x + 0;
                    int _gid_y33 = gid_y + 5 * (int)blockDim.y + 0;
                    if (_gid_x33 >= input_width)
                        _gid_x33 = input_width - 1;
                    if (_gid_x33 < 0)
                        _gid_x33 = 0;
                    _tmp30 += 0.369545996F * tex1Dfetch(_texinputX, (_gid_y33) * input_stride + _gid_x33);
                }
                {
                    int _gid_x34 = gid_x + 1;
                    int _gid_y34 = gid_y + 5 * (int)blockDim.y + 0;
                    if (_gid_x34 >= input_width)
                        _gid_x34 = input_width - 1;
                    if (_gid_x34 < 0)
                        _gid_x34 = 0;
                    _tmp30 += 0.244460002F * tex1Dfetch(_texinputX, (_gid_y34) * input_stride + _gid_x34);
                }
                {
                    int _gid_x35 = gid_x + 2;
                    int _gid_y35 = gid_y + 5 * (int)blockDim.y + 0;
                    if (_gid_x35 >= input_width)
                        _gid_x35 = input_width - 1;
                    if (_gid_x35 < 0)
                        _gid_x35 = 0;
                    _tmp30 += 0.0707660019F * tex1Dfetch(_texinputX, (_gid_y35) * input_stride + _gid_x35);
                }
                iter[(gid_y + 5 * (int)blockDim.y) * iter_stride + gid_x] = (float)(_tmp30);
            }
        }
        if (gid_x < iter_width) {
            if (gid_y + 6 * (int)blockDim.y < iter_height) {
                float _tmp36 = 0.F;
                {
                    int _gid_x37 = gid_x + -2;
                    int _gid_y37 = gid_y + 6 * (int)blockDim.y + 0;
                    if (_gid_x37 >= input_width)
                        _gid_x37 = input_width - 1;
                    if (_gid_x37 < 0)
                        _gid_x37 = 0;
                    _tmp36 += 0.0707660019F * tex1Dfetch(_texinputX, (_gid_y37) * input_stride + _gid_x37);
                }
                {
                    int _gid_x38 = gid_x + -1;
                    int _gid_y38 = gid_y + 6 * (int)blockDim.y + 0;
                    if (_gid_x38 >= input_width)
                        _gid_x38 = input_width - 1;
                    if (_gid_x38 < 0)
                        _gid_x38 = 0;
                    _tmp36 += 0.244460002F * tex1Dfetch(_texinputX, (_gid_y38) * input_stride + _gid_x38);
                }
                {
                    int _gid_x39 = gid_x + 0;
                    int _gid_y39 = gid_y + 6 * (int)blockDim.y + 0;
                    if (_gid_x39 >= input_width)
                        _gid_x39 = input_width - 1;
                    if (_gid_x39 < 0)
                        _gid_x39 = 0;
                    _tmp36 += 0.369545996F * tex1Dfetch(_texinputX, (_gid_y39) * input_stride + _gid_x39);
                }
                {
                    int _gid_x40 = gid_x + 1;
                    int _gid_y40 = gid_y + 6 * (int)blockDim.y + 0;
                    if (_gid_x40 >= input_width)
                        _gid_x40 = input_width - 1;
                    if (_gid_x40 < 0)
                        _gid_x40 = 0;
                    _tmp36 += 0.244460002F * tex1Dfetch(_texinputX, (_gid_y40) * input_stride + _gid_x40);
                }
                {
                    int _gid_x41 = gid_x + 2;
                    int _gid_y41 = gid_y + 6 * (int)blockDim.y + 0;
                    if (_gid_x41 >= input_width)
                        _gid_x41 = input_width - 1;
                    if (_gid_x41 < 0)
                        _gid_x41 = 0;
                    _tmp36 += 0.0707660019F * tex1Dfetch(_texinputX, (_gid_y41) * input_stride + _gid_x41);
                }
                iter[(gid_y + 6 * (int)blockDim.y) * iter_stride + gid_x] = (float)(_tmp36);
            }
        }
        if (gid_x < iter_width) {
            if (gid_y + 7 * (int)blockDim.y < iter_height) {
                float _tmp42 = 0.F;
                {
                    int _gid_x43 = gid_x + -2;
                    int _gid_y43 = gid_y + 7 * (int)blockDim.y + 0;
                    if (_gid_x43 >= input_width)
                        _gid_x43 = input_width - 1;
                    if (_gid_x43 < 0)
                        _gid_x43 = 0;
                    _tmp42 += 0.0707660019F * tex1Dfetch(_texinputX, (_gid_y43) * input_stride + _gid_x43);
                }
                {
                    int _gid_x44 = gid_x + -1;
                    int _gid_y44 = gid_y + 7 * (int)blockDim.y + 0;
                    if (_gid_x44 >= input_width)
                        _gid_x44 = input_width - 1;
                    if (_gid_x44 < 0)
                        _gid_x44 = 0;
                    _tmp42 += 0.244460002F * tex1Dfetch(_texinputX, (_gid_y44) * input_stride + _gid_x44);
                }
                {
                    int _gid_x45 = gid_x + 0;
                    int _gid_y45 = gid_y + 7 * (int)blockDim.y + 0;
                    if (_gid_x45 >= input_width)
                        _gid_x45 = input_width - 1;
                    if (_gid_x45 < 0)
                        _gid_x45 = 0;
                    _tmp42 += 0.369545996F * tex1Dfetch(_texinputX, (_gid_y45) * input_stride + _gid_x45);
                }
                {
                    int _gid_x46 = gid_x + 1;
                    int _gid_y46 = gid_y + 7 * (int)blockDim.y + 0;
                    if (_gid_x46 >= input_width)
                        _gid_x46 = input_width - 1;
                    if (_gid_x46 < 0)
                        _gid_x46 = 0;
                    _tmp42 += 0.244460002F * tex1Dfetch(_texinputX, (_gid_y46) * input_stride + _gid_x46);
                }
                {
                    int _gid_x47 = gid_x + 2;
                    int _gid_y47 = gid_y + 7 * (int)blockDim.y + 0;
                    if (_gid_x47 >= input_width)
                        _gid_x47 = input_width - 1;
                    if (_gid_x47 < 0)
                        _gid_x47 = 0;
                    _tmp42 += 0.0707660019F * tex1Dfetch(_texinputX, (_gid_y47) * input_stride + _gid_x47);
                }
                iter[(gid_y + 7 * (int)blockDim.y) * iter_stride + gid_x] = (float)(_tmp42);
            }
        }
    }
    goto BH_EXIT;
  BH_B:
    {
        if (gid_y < iter_height) {
            float _tmp48 = 0.F;
            {
                int _gid_x49 = gid_x + -2;
                int _gid_y49 = gid_y + 0;
                if (_gid_y49 >= input_height)
                    _gid_y49 = input_height - 1;
                _tmp48 += 0.0707660019F * tex1Dfetch(_texinputX, (_gid_y49) * input_stride + _gid_x49);
            }
            {
                int _gid_x50 = gid_x + -1;
                int _gid_y50 = gid_y + 0;
                if (_gid_y50 >= input_height)
                    _gid_y50 = input_height - 1;
                _tmp48 += 0.244460002F * tex1Dfetch(_texinputX, (_gid_y50) * input_stride + _gid_x50);
            }
            {
                int _gid_x51 = gid_x + 0;
                int _gid_y51 = gid_y + 0;
                if (_gid_y51 >= input_height)
                    _gid_y51 = input_height - 1;
                _tmp48 += 0.369545996F * tex1Dfetch(_texinputX, (_gid_y51) * input_stride + _gid_x51);
            }
            {
                int _gid_x52 = gid_x + 1;
                int _gid_y52 = gid_y + 0;
                if (_gid_y52 >= input_height)
                    _gid_y52 = input_height - 1;
                _tmp48 += 0.244460002F * tex1Dfetch(_texinputX, (_gid_y52) * input_stride + _gid_x52);
            }
            {
                int _gid_x53 = gid_x + 2;
                int _gid_y53 = gid_y + 0;
                if (_gid_y53 >= input_height)
                    _gid_y53 = input_height - 1;
                _tmp48 += 0.0707660019F * tex1Dfetch(_texinputX, (_gid_y53) * input_stride + _gid_x53);
            }
            iter[(gid_y) * iter_stride + gid_x] = (float)(_tmp48);
        }
        if (gid_y + 1 * (int)blockDim.y < iter_height) {
            float _tmp54 = 0.F;
            {
                int _gid_x55 = gid_x + -2;
                int _gid_y55 = gid_y + 1 * (int)blockDim.y + 0;
                if (_gid_y55 >= input_height)
                    _gid_y55 = input_height - 1;
                _tmp54 += 0.0707660019F * tex1Dfetch(_texinputX, (_gid_y55) * input_stride + _gid_x55);
            }
            {
                int _gid_x56 = gid_x + -1;
                int _gid_y56 = gid_y + 1 * (int)blockDim.y + 0;
                if (_gid_y56 >= input_height)
                    _gid_y56 = input_height - 1;
                _tmp54 += 0.244460002F * tex1Dfetch(_texinputX, (_gid_y56) * input_stride + _gid_x56);
            }
            {
                int _gid_x57 = gid_x + 0;
                int _gid_y57 = gid_y + 1 * (int)blockDim.y + 0;
                if (_gid_y57 >= input_height)
                    _gid_y57 = input_height - 1;
                _tmp54 += 0.369545996F * tex1Dfetch(_texinputX, (_gid_y57) * input_stride + _gid_x57);
            }
            {
                int _gid_x58 = gid_x + 1;
                int _gid_y58 = gid_y + 1 * (int)blockDim.y + 0;
                if (_gid_y58 >= input_height)
                    _gid_y58 = input_height - 1;
                _tmp54 += 0.244460002F * tex1Dfetch(_texinputX, (_gid_y58) * input_stride + _gid_x58);
            }
            {
                int _gid_x59 = gid_x + 2;
                int _gid_y59 = gid_y + 1 * (int)blockDim.y + 0;
                if (_gid_y59 >= input_height)
                    _gid_y59 = input_height - 1;
                _tmp54 += 0.0707660019F * tex1Dfetch(_texinputX, (_gid_y59) * input_stride + _gid_x59);
            }
            iter[(gid_y + 1 * (int)blockDim.y) * iter_stride + gid_x] = (float)(_tmp54);
        }
        if (gid_y + 2 * (int)blockDim.y < iter_height) {
            float _tmp60 = 0.F;
            {
                int _gid_x61 = gid_x + -2;
                int _gid_y61 = gid_y + 2 * (int)blockDim.y + 0;
                if (_gid_y61 >= input_height)
                    _gid_y61 = input_height - 1;
                _tmp60 += 0.0707660019F * tex1Dfetch(_texinputX, (_gid_y61) * input_stride + _gid_x61);
            }
            {
                int _gid_x62 = gid_x + -1;
                int _gid_y62 = gid_y + 2 * (int)blockDim.y + 0;
                if (_gid_y62 >= input_height)
                    _gid_y62 = input_height - 1;
                _tmp60 += 0.244460002F * tex1Dfetch(_texinputX, (_gid_y62) * input_stride + _gid_x62);
            }
            {
                int _gid_x63 = gid_x + 0;
                int _gid_y63 = gid_y + 2 * (int)blockDim.y + 0;
                if (_gid_y63 >= input_height)
                    _gid_y63 = input_height - 1;
                _tmp60 += 0.369545996F * tex1Dfetch(_texinputX, (_gid_y63) * input_stride + _gid_x63);
            }
            {
                int _gid_x64 = gid_x + 1;
                int _gid_y64 = gid_y + 2 * (int)blockDim.y + 0;
                if (_gid_y64 >= input_height)
                    _gid_y64 = input_height - 1;
                _tmp60 += 0.244460002F * tex1Dfetch(_texinputX, (_gid_y64) * input_stride + _gid_x64);
            }
            {
                int _gid_x65 = gid_x + 2;
                int _gid_y65 = gid_y + 2 * (int)blockDim.y + 0;
                if (_gid_y65 >= input_height)
                    _gid_y65 = input_height - 1;
                _tmp60 += 0.0707660019F * tex1Dfetch(_texinputX, (_gid_y65) * input_stride + _gid_x65);
            }
            iter[(gid_y + 2 * (int)blockDim.y) * iter_stride + gid_x] = (float)(_tmp60);
        }
        if (gid_y + 3 * (int)blockDim.y < iter_height) {
            float _tmp66 = 0.F;
            {
                int _gid_x67 = gid_x + -2;
                int _gid_y67 = gid_y + 3 * (int)blockDim.y + 0;
                if (_gid_y67 >= input_height)
                    _gid_y67 = input_height - 1;
                _tmp66 += 0.0707660019F * tex1Dfetch(_texinputX, (_gid_y67) * input_stride + _gid_x67);
            }
            {
                int _gid_x68 = gid_x + -1;
                int _gid_y68 = gid_y + 3 * (int)blockDim.y + 0;
                if (_gid_y68 >= input_height)
                    _gid_y68 = input_height - 1;
                _tmp66 += 0.244460002F * tex1Dfetch(_texinputX, (_gid_y68) * input_stride + _gid_x68);
            }
            {
                int _gid_x69 = gid_x + 0;
                int _gid_y69 = gid_y + 3 * (int)blockDim.y + 0;
                if (_gid_y69 >= input_height)
                    _gid_y69 = input_height - 1;
                _tmp66 += 0.369545996F * tex1Dfetch(_texinputX, (_gid_y69) * input_stride + _gid_x69);
            }
            {
                int _gid_x70 = gid_x + 1;
                int _gid_y70 = gid_y + 3 * (int)blockDim.y + 0;
                if (_gid_y70 >= input_height)
                    _gid_y70 = input_height - 1;
                _tmp66 += 0.244460002F * tex1Dfetch(_texinputX, (_gid_y70) * input_stride + _gid_x70);
            }
            {
                int _gid_x71 = gid_x + 2;
                int _gid_y71 = gid_y + 3 * (int)blockDim.y + 0;
                if (_gid_y71 >= input_height)
                    _gid_y71 = input_height - 1;
                _tmp66 += 0.0707660019F * tex1Dfetch(_texinputX, (_gid_y71) * input_stride + _gid_x71);
            }
            iter[(gid_y + 3 * (int)blockDim.y) * iter_stride + gid_x] = (float)(_tmp66);
        }
        if (gid_y + 4 * (int)blockDim.y < iter_height) {
            float _tmp72 = 0.F;
            {
                int _gid_x73 = gid_x + -2;
                int _gid_y73 = gid_y + 4 * (int)blockDim.y + 0;
                if (_gid_y73 >= input_height)
                    _gid_y73 = input_height - 1;
                _tmp72 += 0.0707660019F * tex1Dfetch(_texinputX, (_gid_y73) * input_stride + _gid_x73);
            }
            {
                int _gid_x74 = gid_x + -1;
                int _gid_y74 = gid_y + 4 * (int)blockDim.y + 0;
                if (_gid_y74 >= input_height)
                    _gid_y74 = input_height - 1;
                _tmp72 += 0.244460002F * tex1Dfetch(_texinputX, (_gid_y74) * input_stride + _gid_x74);
            }
            {
                int _gid_x75 = gid_x + 0;
                int _gid_y75 = gid_y + 4 * (int)blockDim.y + 0;
                if (_gid_y75 >= input_height)
                    _gid_y75 = input_height - 1;
                _tmp72 += 0.369545996F * tex1Dfetch(_texinputX, (_gid_y75) * input_stride + _gid_x75);
            }
            {
                int _gid_x76 = gid_x + 1;
                int _gid_y76 = gid_y + 4 * (int)blockDim.y + 0;
                if (_gid_y76 >= input_height)
                    _gid_y76 = input_height - 1;
                _tmp72 += 0.244460002F * tex1Dfetch(_texinputX, (_gid_y76) * input_stride + _gid_x76);
            }
            {
                int _gid_x77 = gid_x + 2;
                int _gid_y77 = gid_y + 4 * (int)blockDim.y + 0;
                if (_gid_y77 >= input_height)
                    _gid_y77 = input_height - 1;
                _tmp72 += 0.0707660019F * tex1Dfetch(_texinputX, (_gid_y77) * input_stride + _gid_x77);
            }
            iter[(gid_y + 4 * (int)blockDim.y) * iter_stride + gid_x] = (float)(_tmp72);
        }
        if (gid_y + 5 * (int)blockDim.y < iter_height) {
            float _tmp78 = 0.F;
            {
                int _gid_x79 = gid_x + -2;
                int _gid_y79 = gid_y + 5 * (int)blockDim.y + 0;
                if (_gid_y79 >= input_height)
                    _gid_y79 = input_height - 1;
                _tmp78 += 0.0707660019F * tex1Dfetch(_texinputX, (_gid_y79) * input_stride + _gid_x79);
            }
            {
                int _gid_x80 = gid_x + -1;
                int _gid_y80 = gid_y + 5 * (int)blockDim.y + 0;
                if (_gid_y80 >= input_height)
                    _gid_y80 = input_height - 1;
                _tmp78 += 0.244460002F * tex1Dfetch(_texinputX, (_gid_y80) * input_stride + _gid_x80);
            }
            {
                int _gid_x81 = gid_x + 0;
                int _gid_y81 = gid_y + 5 * (int)blockDim.y + 0;
                if (_gid_y81 >= input_height)
                    _gid_y81 = input_height - 1;
                _tmp78 += 0.369545996F * tex1Dfetch(_texinputX, (_gid_y81) * input_stride + _gid_x81);
            }
            {
                int _gid_x82 = gid_x + 1;
                int _gid_y82 = gid_y + 5 * (int)blockDim.y + 0;
                if (_gid_y82 >= input_height)
                    _gid_y82 = input_height - 1;
                _tmp78 += 0.244460002F * tex1Dfetch(_texinputX, (_gid_y82) * input_stride + _gid_x82);
            }
            {
                int _gid_x83 = gid_x + 2;
                int _gid_y83 = gid_y + 5 * (int)blockDim.y + 0;
                if (_gid_y83 >= input_height)
                    _gid_y83 = input_height - 1;
                _tmp78 += 0.0707660019F * tex1Dfetch(_texinputX, (_gid_y83) * input_stride + _gid_x83);
            }
            iter[(gid_y + 5 * (int)blockDim.y) * iter_stride + gid_x] = (float)(_tmp78);
        }
        if (gid_y + 6 * (int)blockDim.y < iter_height) {
            float _tmp84 = 0.F;
            {
                int _gid_x85 = gid_x + -2;
                int _gid_y85 = gid_y + 6 * (int)blockDim.y + 0;
                if (_gid_y85 >= input_height)
                    _gid_y85 = input_height - 1;
                _tmp84 += 0.0707660019F * tex1Dfetch(_texinputX, (_gid_y85) * input_stride + _gid_x85);
            }
            {
                int _gid_x86 = gid_x + -1;
                int _gid_y86 = gid_y + 6 * (int)blockDim.y + 0;
                if (_gid_y86 >= input_height)
                    _gid_y86 = input_height - 1;
                _tmp84 += 0.244460002F * tex1Dfetch(_texinputX, (_gid_y86) * input_stride + _gid_x86);
            }
            {
                int _gid_x87 = gid_x + 0;
                int _gid_y87 = gid_y + 6 * (int)blockDim.y + 0;
                if (_gid_y87 >= input_height)
                    _gid_y87 = input_height - 1;
                _tmp84 += 0.369545996F * tex1Dfetch(_texinputX, (_gid_y87) * input_stride + _gid_x87);
            }
            {
                int _gid_x88 = gid_x + 1;
                int _gid_y88 = gid_y + 6 * (int)blockDim.y + 0;
                if (_gid_y88 >= input_height)
                    _gid_y88 = input_height - 1;
                _tmp84 += 0.244460002F * tex1Dfetch(_texinputX, (_gid_y88) * input_stride + _gid_x88);
            }
            {
                int _gid_x89 = gid_x + 2;
                int _gid_y89 = gid_y + 6 * (int)blockDim.y + 0;
                if (_gid_y89 >= input_height)
                    _gid_y89 = input_height - 1;
                _tmp84 += 0.0707660019F * tex1Dfetch(_texinputX, (_gid_y89) * input_stride + _gid_x89);
            }
            iter[(gid_y + 6 * (int)blockDim.y) * iter_stride + gid_x] = (float)(_tmp84);
        }
        if (gid_y + 7 * (int)blockDim.y < iter_height) {
            float _tmp90 = 0.F;
            {
                int _gid_x91 = gid_x + -2;
                int _gid_y91 = gid_y + 7 * (int)blockDim.y + 0;
                if (_gid_y91 >= input_height)
                    _gid_y91 = input_height - 1;
                _tmp90 += 0.0707660019F * tex1Dfetch(_texinputX, (_gid_y91) * input_stride + _gid_x91);
            }
            {
                int _gid_x92 = gid_x + -1;
                int _gid_y92 = gid_y + 7 * (int)blockDim.y + 0;
                if (_gid_y92 >= input_height)
                    _gid_y92 = input_height - 1;
                _tmp90 += 0.244460002F * tex1Dfetch(_texinputX, (_gid_y92) * input_stride + _gid_x92);
            }
            {
                int _gid_x93 = gid_x + 0;
                int _gid_y93 = gid_y + 7 * (int)blockDim.y + 0;
                if (_gid_y93 >= input_height)
                    _gid_y93 = input_height - 1;
                _tmp90 += 0.369545996F * tex1Dfetch(_texinputX, (_gid_y93) * input_stride + _gid_x93);
            }
            {
                int _gid_x94 = gid_x + 1;
                int _gid_y94 = gid_y + 7 * (int)blockDim.y + 0;
                if (_gid_y94 >= input_height)
                    _gid_y94 = input_height - 1;
                _tmp90 += 0.244460002F * tex1Dfetch(_texinputX, (_gid_y94) * input_stride + _gid_x94);
            }
            {
                int _gid_x95 = gid_x + 2;
                int _gid_y95 = gid_y + 7 * (int)blockDim.y + 0;
                if (_gid_y95 >= input_height)
                    _gid_y95 = input_height - 1;
                _tmp90 += 0.0707660019F * tex1Dfetch(_texinputX, (_gid_y95) * input_stride + _gid_x95);
            }
            iter[(gid_y + 7 * (int)blockDim.y) * iter_stride + gid_x] = (float)(_tmp90);
        }
    }
    goto BH_EXIT;
  BH_R:
    {
        if (gid_x < iter_width) {
            if (gid_y < iter_height) {
                float _tmp96 = 0.F;
                {
                    int _gid_x97 = gid_x + -2;
                    int _gid_y97 = gid_y + 0;
                    if (_gid_x97 >= input_width)
                        _gid_x97 = input_width - 1;
                    _tmp96 += 0.0707660019F * tex1Dfetch(_texinputX, (_gid_y97) * input_stride + _gid_x97);
                }
                {
                    int _gid_x98 = gid_x + -1;
                    int _gid_y98 = gid_y + 0;
                    if (_gid_x98 >= input_width)
                        _gid_x98 = input_width - 1;
                    _tmp96 += 0.244460002F * tex1Dfetch(_texinputX, (_gid_y98) * input_stride + _gid_x98);
                }
                {
                    int _gid_x99 = gid_x + 0;
                    int _gid_y99 = gid_y + 0;
                    if (_gid_x99 >= input_width)
                        _gid_x99 = input_width - 1;
                    _tmp96 += 0.369545996F * tex1Dfetch(_texinputX, (_gid_y99) * input_stride + _gid_x99);
                }
                {
                    int _gid_x100 = gid_x + 1;
                    int _gid_y100 = gid_y + 0;
                    if (_gid_x100 >= input_width)
                        _gid_x100 = input_width - 1;
                    _tmp96 += 0.244460002F * tex1Dfetch(_texinputX, (_gid_y100) * input_stride + _gid_x100);
                }
                {
                    int _gid_x101 = gid_x + 2;
                    int _gid_y101 = gid_y + 0;
                    if (_gid_x101 >= input_width)
                        _gid_x101 = input_width - 1;
                    _tmp96 += 0.0707660019F * tex1Dfetch(_texinputX, (_gid_y101) * input_stride + _gid_x101);
                }
                iter[(gid_y) * iter_stride + gid_x] = (float)(_tmp96);
            }
        }
        if (gid_x < iter_width) {
            if (gid_y + 1 * (int)blockDim.y < iter_height) {
                float _tmp102 = 0.F;
                {
                    int _gid_x103 = gid_x + -2;
                    int _gid_y103 = gid_y + 1 * (int)blockDim.y + 0;
                    if (_gid_x103 >= input_width)
                        _gid_x103 = input_width - 1;
                    _tmp102 += 0.0707660019F * tex1Dfetch(_texinputX, (_gid_y103) * input_stride + _gid_x103);
                }
                {
                    int _gid_x104 = gid_x + -1;
                    int _gid_y104 = gid_y + 1 * (int)blockDim.y + 0;
                    if (_gid_x104 >= input_width)
                        _gid_x104 = input_width - 1;
                    _tmp102 += 0.244460002F * tex1Dfetch(_texinputX, (_gid_y104) * input_stride + _gid_x104);
                }
                {
                    int _gid_x105 = gid_x + 0;
                    int _gid_y105 = gid_y + 1 * (int)blockDim.y + 0;
                    if (_gid_x105 >= input_width)
                        _gid_x105 = input_width - 1;
                    _tmp102 += 0.369545996F * tex1Dfetch(_texinputX, (_gid_y105) * input_stride + _gid_x105);
                }
                {
                    int _gid_x106 = gid_x + 1;
                    int _gid_y106 = gid_y + 1 * (int)blockDim.y + 0;
                    if (_gid_x106 >= input_width)
                        _gid_x106 = input_width - 1;
                    _tmp102 += 0.244460002F * tex1Dfetch(_texinputX, (_gid_y106) * input_stride + _gid_x106);
                }
                {
                    int _gid_x107 = gid_x + 2;
                    int _gid_y107 = gid_y + 1 * (int)blockDim.y + 0;
                    if (_gid_x107 >= input_width)
                        _gid_x107 = input_width - 1;
                    _tmp102 += 0.0707660019F * tex1Dfetch(_texinputX, (_gid_y107) * input_stride + _gid_x107);
                }
                iter[(gid_y + 1 * (int)blockDim.y) * iter_stride + gid_x] = (float)(_tmp102);
            }
        }
        if (gid_x < iter_width) {
            if (gid_y + 2 * (int)blockDim.y < iter_height) {
                float _tmp108 = 0.F;
                {
                    int _gid_x109 = gid_x + -2;
                    int _gid_y109 = gid_y + 2 * (int)blockDim.y + 0;
                    if (_gid_x109 >= input_width)
                        _gid_x109 = input_width - 1;
                    _tmp108 += 0.0707660019F * tex1Dfetch(_texinputX, (_gid_y109) * input_stride + _gid_x109);
                }
                {
                    int _gid_x110 = gid_x + -1;
                    int _gid_y110 = gid_y + 2 * (int)blockDim.y + 0;
                    if (_gid_x110 >= input_width)
                        _gid_x110 = input_width - 1;
                    _tmp108 += 0.244460002F * tex1Dfetch(_texinputX, (_gid_y110) * input_stride + _gid_x110);
                }
                {
                    int _gid_x111 = gid_x + 0;
                    int _gid_y111 = gid_y + 2 * (int)blockDim.y + 0;
                    if (_gid_x111 >= input_width)
                        _gid_x111 = input_width - 1;
                    _tmp108 += 0.369545996F * tex1Dfetch(_texinputX, (_gid_y111) * input_stride + _gid_x111);
                }
                {
                    int _gid_x112 = gid_x + 1;
                    int _gid_y112 = gid_y + 2 * (int)blockDim.y + 0;
                    if (_gid_x112 >= input_width)
                        _gid_x112 = input_width - 1;
                    _tmp108 += 0.244460002F * tex1Dfetch(_texinputX, (_gid_y112) * input_stride + _gid_x112);
                }
                {
                    int _gid_x113 = gid_x + 2;
                    int _gid_y113 = gid_y + 2 * (int)blockDim.y + 0;
                    if (_gid_x113 >= input_width)
                        _gid_x113 = input_width - 1;
                    _tmp108 += 0.0707660019F * tex1Dfetch(_texinputX, (_gid_y113) * input_stride + _gid_x113);
                }
                iter[(gid_y + 2 * (int)blockDim.y) * iter_stride + gid_x] = (float)(_tmp108);
            }
        }
        if (gid_x < iter_width) {
            if (gid_y + 3 * (int)blockDim.y < iter_height) {
                float _tmp114 = 0.F;
                {
                    int _gid_x115 = gid_x + -2;
                    int _gid_y115 = gid_y + 3 * (int)blockDim.y + 0;
                    if (_gid_x115 >= input_width)
                        _gid_x115 = input_width - 1;
                    _tmp114 += 0.0707660019F * tex1Dfetch(_texinputX, (_gid_y115) * input_stride + _gid_x115);
                }
                {
                    int _gid_x116 = gid_x + -1;
                    int _gid_y116 = gid_y + 3 * (int)blockDim.y + 0;
                    if (_gid_x116 >= input_width)
                        _gid_x116 = input_width - 1;
                    _tmp114 += 0.244460002F * tex1Dfetch(_texinputX, (_gid_y116) * input_stride + _gid_x116);
                }
                {
                    int _gid_x117 = gid_x + 0;
                    int _gid_y117 = gid_y + 3 * (int)blockDim.y + 0;
                    if (_gid_x117 >= input_width)
                        _gid_x117 = input_width - 1;
                    _tmp114 += 0.369545996F * tex1Dfetch(_texinputX, (_gid_y117) * input_stride + _gid_x117);
                }
                {
                    int _gid_x118 = gid_x + 1;
                    int _gid_y118 = gid_y + 3 * (int)blockDim.y + 0;
                    if (_gid_x118 >= input_width)
                        _gid_x118 = input_width - 1;
                    _tmp114 += 0.244460002F * tex1Dfetch(_texinputX, (_gid_y118) * input_stride + _gid_x118);
                }
                {
                    int _gid_x119 = gid_x + 2;
                    int _gid_y119 = gid_y + 3 * (int)blockDim.y + 0;
                    if (_gid_x119 >= input_width)
                        _gid_x119 = input_width - 1;
                    _tmp114 += 0.0707660019F * tex1Dfetch(_texinputX, (_gid_y119) * input_stride + _gid_x119);
                }
                iter[(gid_y + 3 * (int)blockDim.y) * iter_stride + gid_x] = (float)(_tmp114);
            }
        }
        if (gid_x < iter_width) {
            if (gid_y + 4 * (int)blockDim.y < iter_height) {
                float _tmp120 = 0.F;
                {
                    int _gid_x121 = gid_x + -2;
                    int _gid_y121 = gid_y + 4 * (int)blockDim.y + 0;
                    if (_gid_x121 >= input_width)
                        _gid_x121 = input_width - 1;
                    _tmp120 += 0.0707660019F * tex1Dfetch(_texinputX, (_gid_y121) * input_stride + _gid_x121);
                }
                {
                    int _gid_x122 = gid_x + -1;
                    int _gid_y122 = gid_y + 4 * (int)blockDim.y + 0;
                    if (_gid_x122 >= input_width)
                        _gid_x122 = input_width - 1;
                    _tmp120 += 0.244460002F * tex1Dfetch(_texinputX, (_gid_y122) * input_stride + _gid_x122);
                }
                {
                    int _gid_x123 = gid_x + 0;
                    int _gid_y123 = gid_y + 4 * (int)blockDim.y + 0;
                    if (_gid_x123 >= input_width)
                        _gid_x123 = input_width - 1;
                    _tmp120 += 0.369545996F * tex1Dfetch(_texinputX, (_gid_y123) * input_stride + _gid_x123);
                }
                {
                    int _gid_x124 = gid_x + 1;
                    int _gid_y124 = gid_y + 4 * (int)blockDim.y + 0;
                    if (_gid_x124 >= input_width)
                        _gid_x124 = input_width - 1;
                    _tmp120 += 0.244460002F * tex1Dfetch(_texinputX, (_gid_y124) * input_stride + _gid_x124);
                }
                {
                    int _gid_x125 = gid_x + 2;
                    int _gid_y125 = gid_y + 4 * (int)blockDim.y + 0;
                    if (_gid_x125 >= input_width)
                        _gid_x125 = input_width - 1;
                    _tmp120 += 0.0707660019F * tex1Dfetch(_texinputX, (_gid_y125) * input_stride + _gid_x125);
                }
                iter[(gid_y + 4 * (int)blockDim.y) * iter_stride + gid_x] = (float)(_tmp120);
            }
        }
        if (gid_x < iter_width) {
            if (gid_y + 5 * (int)blockDim.y < iter_height) {
                float _tmp126 = 0.F;
                {
                    int _gid_x127 = gid_x + -2;
                    int _gid_y127 = gid_y + 5 * (int)blockDim.y + 0;
                    if (_gid_x127 >= input_width)
                        _gid_x127 = input_width - 1;
                    _tmp126 += 0.0707660019F * tex1Dfetch(_texinputX, (_gid_y127) * input_stride + _gid_x127);
                }
                {
                    int _gid_x128 = gid_x + -1;
                    int _gid_y128 = gid_y + 5 * (int)blockDim.y + 0;
                    if (_gid_x128 >= input_width)
                        _gid_x128 = input_width - 1;
                    _tmp126 += 0.244460002F * tex1Dfetch(_texinputX, (_gid_y128) * input_stride + _gid_x128);
                }
                {
                    int _gid_x129 = gid_x + 0;
                    int _gid_y129 = gid_y + 5 * (int)blockDim.y + 0;
                    if (_gid_x129 >= input_width)
                        _gid_x129 = input_width - 1;
                    _tmp126 += 0.369545996F * tex1Dfetch(_texinputX, (_gid_y129) * input_stride + _gid_x129);
                }
                {
                    int _gid_x130 = gid_x + 1;
                    int _gid_y130 = gid_y + 5 * (int)blockDim.y + 0;
                    if (_gid_x130 >= input_width)
                        _gid_x130 = input_width - 1;
                    _tmp126 += 0.244460002F * tex1Dfetch(_texinputX, (_gid_y130) * input_stride + _gid_x130);
                }
                {
                    int _gid_x131 = gid_x + 2;
                    int _gid_y131 = gid_y + 5 * (int)blockDim.y + 0;
                    if (_gid_x131 >= input_width)
                        _gid_x131 = input_width - 1;
                    _tmp126 += 0.0707660019F * tex1Dfetch(_texinputX, (_gid_y131) * input_stride + _gid_x131);
                }
                iter[(gid_y + 5 * (int)blockDim.y) * iter_stride + gid_x] = (float)(_tmp126);
            }
        }
        if (gid_x < iter_width) {
            if (gid_y + 6 * (int)blockDim.y < iter_height) {
                float _tmp132 = 0.F;
                {
                    int _gid_x133 = gid_x + -2;
                    int _gid_y133 = gid_y + 6 * (int)blockDim.y + 0;
                    if (_gid_x133 >= input_width)
                        _gid_x133 = input_width - 1;
                    _tmp132 += 0.0707660019F * tex1Dfetch(_texinputX, (_gid_y133) * input_stride + _gid_x133);
                }
                {
                    int _gid_x134 = gid_x + -1;
                    int _gid_y134 = gid_y + 6 * (int)blockDim.y + 0;
                    if (_gid_x134 >= input_width)
                        _gid_x134 = input_width - 1;
                    _tmp132 += 0.244460002F * tex1Dfetch(_texinputX, (_gid_y134) * input_stride + _gid_x134);
                }
                {
                    int _gid_x135 = gid_x + 0;
                    int _gid_y135 = gid_y + 6 * (int)blockDim.y + 0;
                    if (_gid_x135 >= input_width)
                        _gid_x135 = input_width - 1;
                    _tmp132 += 0.369545996F * tex1Dfetch(_texinputX, (_gid_y135) * input_stride + _gid_x135);
                }
                {
                    int _gid_x136 = gid_x + 1;
                    int _gid_y136 = gid_y + 6 * (int)blockDim.y + 0;
                    if (_gid_x136 >= input_width)
                        _gid_x136 = input_width - 1;
                    _tmp132 += 0.244460002F * tex1Dfetch(_texinputX, (_gid_y136) * input_stride + _gid_x136);
                }
                {
                    int _gid_x137 = gid_x + 2;
                    int _gid_y137 = gid_y + 6 * (int)blockDim.y + 0;
                    if (_gid_x137 >= input_width)
                        _gid_x137 = input_width - 1;
                    _tmp132 += 0.0707660019F * tex1Dfetch(_texinputX, (_gid_y137) * input_stride + _gid_x137);
                }
                iter[(gid_y + 6 * (int)blockDim.y) * iter_stride + gid_x] = (float)(_tmp132);
            }
        }
        if (gid_x < iter_width) {
            if (gid_y + 7 * (int)blockDim.y < iter_height) {
                float _tmp138 = 0.F;
                {
                    int _gid_x139 = gid_x + -2;
                    int _gid_y139 = gid_y + 7 * (int)blockDim.y + 0;
                    if (_gid_x139 >= input_width)
                        _gid_x139 = input_width - 1;
                    _tmp138 += 0.0707660019F * tex1Dfetch(_texinputX, (_gid_y139) * input_stride + _gid_x139);
                }
                {
                    int _gid_x140 = gid_x + -1;
                    int _gid_y140 = gid_y + 7 * (int)blockDim.y + 0;
                    if (_gid_x140 >= input_width)
                        _gid_x140 = input_width - 1;
                    _tmp138 += 0.244460002F * tex1Dfetch(_texinputX, (_gid_y140) * input_stride + _gid_x140);
                }
                {
                    int _gid_x141 = gid_x + 0;
                    int _gid_y141 = gid_y + 7 * (int)blockDim.y + 0;
                    if (_gid_x141 >= input_width)
                        _gid_x141 = input_width - 1;
                    _tmp138 += 0.369545996F * tex1Dfetch(_texinputX, (_gid_y141) * input_stride + _gid_x141);
                }
                {
                    int _gid_x142 = gid_x + 1;
                    int _gid_y142 = gid_y + 7 * (int)blockDim.y + 0;
                    if (_gid_x142 >= input_width)
                        _gid_x142 = input_width - 1;
                    _tmp138 += 0.244460002F * tex1Dfetch(_texinputX, (_gid_y142) * input_stride + _gid_x142);
                }
                {
                    int _gid_x143 = gid_x + 2;
                    int _gid_y143 = gid_y + 7 * (int)blockDim.y + 0;
                    if (_gid_x143 >= input_width)
                        _gid_x143 = input_width - 1;
                    _tmp138 += 0.0707660019F * tex1Dfetch(_texinputX, (_gid_y143) * input_stride + _gid_x143);
                }
                iter[(gid_y + 7 * (int)blockDim.y) * iter_stride + gid_x] = (float)(_tmp138);
            }
        }
    }
    goto BH_EXIT;
  BH_L:
    {
        if (gid_y < iter_height) {
            float _tmp144 = 0.F;
            {
                int _gid_x145 = gid_x + -2;
                int _gid_y145 = gid_y + 0;
                if (_gid_x145 < 0)
                    _gid_x145 = 0;
                _tmp144 += 0.0707660019F * tex1Dfetch(_texinputX, (_gid_y145) * input_stride + _gid_x145);
            }
            {
                int _gid_x146 = gid_x + -1;
                int _gid_y146 = gid_y + 0;
                if (_gid_x146 < 0)
                    _gid_x146 = 0;
                _tmp144 += 0.244460002F * tex1Dfetch(_texinputX, (_gid_y146) * input_stride + _gid_x146);
            }
            {
                int _gid_x147 = gid_x + 0;
                int _gid_y147 = gid_y + 0;
                if (_gid_x147 < 0)
                    _gid_x147 = 0;
                _tmp144 += 0.369545996F * tex1Dfetch(_texinputX, (_gid_y147) * input_stride + _gid_x147);
            }
            {
                int _gid_x148 = gid_x + 1;
                int _gid_y148 = gid_y + 0;
                if (_gid_x148 < 0)
                    _gid_x148 = 0;
                _tmp144 += 0.244460002F * tex1Dfetch(_texinputX, (_gid_y148) * input_stride + _gid_x148);
            }
            {
                int _gid_x149 = gid_x + 2;
                int _gid_y149 = gid_y + 0;
                if (_gid_x149 < 0)
                    _gid_x149 = 0;
                _tmp144 += 0.0707660019F * tex1Dfetch(_texinputX, (_gid_y149) * input_stride + _gid_x149);
            }
            iter[(gid_y) * iter_stride + gid_x] = (float)(_tmp144);
        }
        if (gid_y + 1 * (int)blockDim.y < iter_height) {
            float _tmp150 = 0.F;
            {
                int _gid_x151 = gid_x + -2;
                int _gid_y151 = gid_y + 1 * (int)blockDim.y + 0;
                if (_gid_x151 < 0)
                    _gid_x151 = 0;
                _tmp150 += 0.0707660019F * tex1Dfetch(_texinputX, (_gid_y151) * input_stride + _gid_x151);
            }
            {
                int _gid_x152 = gid_x + -1;
                int _gid_y152 = gid_y + 1 * (int)blockDim.y + 0;
                if (_gid_x152 < 0)
                    _gid_x152 = 0;
                _tmp150 += 0.244460002F * tex1Dfetch(_texinputX, (_gid_y152) * input_stride + _gid_x152);
            }
            {
                int _gid_x153 = gid_x + 0;
                int _gid_y153 = gid_y + 1 * (int)blockDim.y + 0;
                if (_gid_x153 < 0)
                    _gid_x153 = 0;
                _tmp150 += 0.369545996F * tex1Dfetch(_texinputX, (_gid_y153) * input_stride + _gid_x153);
            }
            {
                int _gid_x154 = gid_x + 1;
                int _gid_y154 = gid_y + 1 * (int)blockDim.y + 0;
                if (_gid_x154 < 0)
                    _gid_x154 = 0;
                _tmp150 += 0.244460002F * tex1Dfetch(_texinputX, (_gid_y154) * input_stride + _gid_x154);
            }
            {
                int _gid_x155 = gid_x + 2;
                int _gid_y155 = gid_y + 1 * (int)blockDim.y + 0;
                if (_gid_x155 < 0)
                    _gid_x155 = 0;
                _tmp150 += 0.0707660019F * tex1Dfetch(_texinputX, (_gid_y155) * input_stride + _gid_x155);
            }
            iter[(gid_y + 1 * (int)blockDim.y) * iter_stride + gid_x] = (float)(_tmp150);
        }
        if (gid_y + 2 * (int)blockDim.y < iter_height) {
            float _tmp156 = 0.F;
            {
                int _gid_x157 = gid_x + -2;
                int _gid_y157 = gid_y + 2 * (int)blockDim.y + 0;
                if (_gid_x157 < 0)
                    _gid_x157 = 0;
                _tmp156 += 0.0707660019F * tex1Dfetch(_texinputX, (_gid_y157) * input_stride + _gid_x157);
            }
            {
                int _gid_x158 = gid_x + -1;
                int _gid_y158 = gid_y + 2 * (int)blockDim.y + 0;
                if (_gid_x158 < 0)
                    _gid_x158 = 0;
                _tmp156 += 0.244460002F * tex1Dfetch(_texinputX, (_gid_y158) * input_stride + _gid_x158);
            }
            {
                int _gid_x159 = gid_x + 0;
                int _gid_y159 = gid_y + 2 * (int)blockDim.y + 0;
                if (_gid_x159 < 0)
                    _gid_x159 = 0;
                _tmp156 += 0.369545996F * tex1Dfetch(_texinputX, (_gid_y159) * input_stride + _gid_x159);
            }
            {
                int _gid_x160 = gid_x + 1;
                int _gid_y160 = gid_y + 2 * (int)blockDim.y + 0;
                if (_gid_x160 < 0)
                    _gid_x160 = 0;
                _tmp156 += 0.244460002F * tex1Dfetch(_texinputX, (_gid_y160) * input_stride + _gid_x160);
            }
            {
                int _gid_x161 = gid_x + 2;
                int _gid_y161 = gid_y + 2 * (int)blockDim.y + 0;
                if (_gid_x161 < 0)
                    _gid_x161 = 0;
                _tmp156 += 0.0707660019F * tex1Dfetch(_texinputX, (_gid_y161) * input_stride + _gid_x161);
            }
            iter[(gid_y + 2 * (int)blockDim.y) * iter_stride + gid_x] = (float)(_tmp156);
        }
        if (gid_y + 3 * (int)blockDim.y < iter_height) {
            float _tmp162 = 0.F;
            {
                int _gid_x163 = gid_x + -2;
                int _gid_y163 = gid_y + 3 * (int)blockDim.y + 0;
                if (_gid_x163 < 0)
                    _gid_x163 = 0;
                _tmp162 += 0.0707660019F * tex1Dfetch(_texinputX, (_gid_y163) * input_stride + _gid_x163);
            }
            {
                int _gid_x164 = gid_x + -1;
                int _gid_y164 = gid_y + 3 * (int)blockDim.y + 0;
                if (_gid_x164 < 0)
                    _gid_x164 = 0;
                _tmp162 += 0.244460002F * tex1Dfetch(_texinputX, (_gid_y164) * input_stride + _gid_x164);
            }
            {
                int _gid_x165 = gid_x + 0;
                int _gid_y165 = gid_y + 3 * (int)blockDim.y + 0;
                if (_gid_x165 < 0)
                    _gid_x165 = 0;
                _tmp162 += 0.369545996F * tex1Dfetch(_texinputX, (_gid_y165) * input_stride + _gid_x165);
            }
            {
                int _gid_x166 = gid_x + 1;
                int _gid_y166 = gid_y + 3 * (int)blockDim.y + 0;
                if (_gid_x166 < 0)
                    _gid_x166 = 0;
                _tmp162 += 0.244460002F * tex1Dfetch(_texinputX, (_gid_y166) * input_stride + _gid_x166);
            }
            {
                int _gid_x167 = gid_x + 2;
                int _gid_y167 = gid_y + 3 * (int)blockDim.y + 0;
                if (_gid_x167 < 0)
                    _gid_x167 = 0;
                _tmp162 += 0.0707660019F * tex1Dfetch(_texinputX, (_gid_y167) * input_stride + _gid_x167);
            }
            iter[(gid_y + 3 * (int)blockDim.y) * iter_stride + gid_x] = (float)(_tmp162);
        }
        if (gid_y + 4 * (int)blockDim.y < iter_height) {
            float _tmp168 = 0.F;
            {
                int _gid_x169 = gid_x + -2;
                int _gid_y169 = gid_y + 4 * (int)blockDim.y + 0;
                if (_gid_x169 < 0)
                    _gid_x169 = 0;
                _tmp168 += 0.0707660019F * tex1Dfetch(_texinputX, (_gid_y169) * input_stride + _gid_x169);
            }
            {
                int _gid_x170 = gid_x + -1;
                int _gid_y170 = gid_y + 4 * (int)blockDim.y + 0;
                if (_gid_x170 < 0)
                    _gid_x170 = 0;
                _tmp168 += 0.244460002F * tex1Dfetch(_texinputX, (_gid_y170) * input_stride + _gid_x170);
            }
            {
                int _gid_x171 = gid_x + 0;
                int _gid_y171 = gid_y + 4 * (int)blockDim.y + 0;
                if (_gid_x171 < 0)
                    _gid_x171 = 0;
                _tmp168 += 0.369545996F * tex1Dfetch(_texinputX, (_gid_y171) * input_stride + _gid_x171);
            }
            {
                int _gid_x172 = gid_x + 1;
                int _gid_y172 = gid_y + 4 * (int)blockDim.y + 0;
                if (_gid_x172 < 0)
                    _gid_x172 = 0;
                _tmp168 += 0.244460002F * tex1Dfetch(_texinputX, (_gid_y172) * input_stride + _gid_x172);
            }
            {
                int _gid_x173 = gid_x + 2;
                int _gid_y173 = gid_y + 4 * (int)blockDim.y + 0;
                if (_gid_x173 < 0)
                    _gid_x173 = 0;
                _tmp168 += 0.0707660019F * tex1Dfetch(_texinputX, (_gid_y173) * input_stride + _gid_x173);
            }
            iter[(gid_y + 4 * (int)blockDim.y) * iter_stride + gid_x] = (float)(_tmp168);
        }
        if (gid_y + 5 * (int)blockDim.y < iter_height) {
            float _tmp174 = 0.F;
            {
                int _gid_x175 = gid_x + -2;
                int _gid_y175 = gid_y + 5 * (int)blockDim.y + 0;
                if (_gid_x175 < 0)
                    _gid_x175 = 0;
                _tmp174 += 0.0707660019F * tex1Dfetch(_texinputX, (_gid_y175) * input_stride + _gid_x175);
            }
            {
                int _gid_x176 = gid_x + -1;
                int _gid_y176 = gid_y + 5 * (int)blockDim.y + 0;
                if (_gid_x176 < 0)
                    _gid_x176 = 0;
                _tmp174 += 0.244460002F * tex1Dfetch(_texinputX, (_gid_y176) * input_stride + _gid_x176);
            }
            {
                int _gid_x177 = gid_x + 0;
                int _gid_y177 = gid_y + 5 * (int)blockDim.y + 0;
                if (_gid_x177 < 0)
                    _gid_x177 = 0;
                _tmp174 += 0.369545996F * tex1Dfetch(_texinputX, (_gid_y177) * input_stride + _gid_x177);
            }
            {
                int _gid_x178 = gid_x + 1;
                int _gid_y178 = gid_y + 5 * (int)blockDim.y + 0;
                if (_gid_x178 < 0)
                    _gid_x178 = 0;
                _tmp174 += 0.244460002F * tex1Dfetch(_texinputX, (_gid_y178) * input_stride + _gid_x178);
            }
            {
                int _gid_x179 = gid_x + 2;
                int _gid_y179 = gid_y + 5 * (int)blockDim.y + 0;
                if (_gid_x179 < 0)
                    _gid_x179 = 0;
                _tmp174 += 0.0707660019F * tex1Dfetch(_texinputX, (_gid_y179) * input_stride + _gid_x179);
            }
            iter[(gid_y + 5 * (int)blockDim.y) * iter_stride + gid_x] = (float)(_tmp174);
        }
        if (gid_y + 6 * (int)blockDim.y < iter_height) {
            float _tmp180 = 0.F;
            {
                int _gid_x181 = gid_x + -2;
                int _gid_y181 = gid_y + 6 * (int)blockDim.y + 0;
                if (_gid_x181 < 0)
                    _gid_x181 = 0;
                _tmp180 += 0.0707660019F * tex1Dfetch(_texinputX, (_gid_y181) * input_stride + _gid_x181);
            }
            {
                int _gid_x182 = gid_x + -1;
                int _gid_y182 = gid_y + 6 * (int)blockDim.y + 0;
                if (_gid_x182 < 0)
                    _gid_x182 = 0;
                _tmp180 += 0.244460002F * tex1Dfetch(_texinputX, (_gid_y182) * input_stride + _gid_x182);
            }
            {
                int _gid_x183 = gid_x + 0;
                int _gid_y183 = gid_y + 6 * (int)blockDim.y + 0;
                if (_gid_x183 < 0)
                    _gid_x183 = 0;
                _tmp180 += 0.369545996F * tex1Dfetch(_texinputX, (_gid_y183) * input_stride + _gid_x183);
            }
            {
                int _gid_x184 = gid_x + 1;
                int _gid_y184 = gid_y + 6 * (int)blockDim.y + 0;
                if (_gid_x184 < 0)
                    _gid_x184 = 0;
                _tmp180 += 0.244460002F * tex1Dfetch(_texinputX, (_gid_y184) * input_stride + _gid_x184);
            }
            {
                int _gid_x185 = gid_x + 2;
                int _gid_y185 = gid_y + 6 * (int)blockDim.y + 0;
                if (_gid_x185 < 0)
                    _gid_x185 = 0;
                _tmp180 += 0.0707660019F * tex1Dfetch(_texinputX, (_gid_y185) * input_stride + _gid_x185);
            }
            iter[(gid_y + 6 * (int)blockDim.y) * iter_stride + gid_x] = (float)(_tmp180);
        }
        if (gid_y + 7 * (int)blockDim.y < iter_height) {
            float _tmp186 = 0.F;
            {
                int _gid_x187 = gid_x + -2;
                int _gid_y187 = gid_y + 7 * (int)blockDim.y + 0;
                if (_gid_x187 < 0)
                    _gid_x187 = 0;
                _tmp186 += 0.0707660019F * tex1Dfetch(_texinputX, (_gid_y187) * input_stride + _gid_x187);
            }
            {
                int _gid_x188 = gid_x + -1;
                int _gid_y188 = gid_y + 7 * (int)blockDim.y + 0;
                if (_gid_x188 < 0)
                    _gid_x188 = 0;
                _tmp186 += 0.244460002F * tex1Dfetch(_texinputX, (_gid_y188) * input_stride + _gid_x188);
            }
            {
                int _gid_x189 = gid_x + 0;
                int _gid_y189 = gid_y + 7 * (int)blockDim.y + 0;
                if (_gid_x189 < 0)
                    _gid_x189 = 0;
                _tmp186 += 0.369545996F * tex1Dfetch(_texinputX, (_gid_y189) * input_stride + _gid_x189);
            }
            {
                int _gid_x190 = gid_x + 1;
                int _gid_y190 = gid_y + 7 * (int)blockDim.y + 0;
                if (_gid_x190 < 0)
                    _gid_x190 = 0;
                _tmp186 += 0.244460002F * tex1Dfetch(_texinputX, (_gid_y190) * input_stride + _gid_x190);
            }
            {
                int _gid_x191 = gid_x + 2;
                int _gid_y191 = gid_y + 7 * (int)blockDim.y + 0;
                if (_gid_x191 < 0)
                    _gid_x191 = 0;
                _tmp186 += 0.0707660019F * tex1Dfetch(_texinputX, (_gid_y191) * input_stride + _gid_x191);
            }
            iter[(gid_y + 7 * (int)blockDim.y) * iter_stride + gid_x] = (float)(_tmp186);
        }
    }
    goto BH_EXIT;
  BH_NO:
    {
        {
            float _tmp192 = 0.F;
            {
                _tmp192 += 0.0707660019F * tex1Dfetch(_texinputX, (gid_y + 0) * input_stride + gid_x + -2);
            }
            {
                _tmp192 += 0.244460002F * tex1Dfetch(_texinputX, (gid_y + 0) * input_stride + gid_x + -1);
            }
            {
                _tmp192 += 0.369545996F * tex1Dfetch(_texinputX, (gid_y + 0) * input_stride + gid_x + 0);
            }
            {
                _tmp192 += 0.244460002F * tex1Dfetch(_texinputX, (gid_y + 0) * input_stride + gid_x + 1);
            }
            {
                _tmp192 += 0.0707660019F * tex1Dfetch(_texinputX, (gid_y + 0) * input_stride + gid_x + 2);
            }
            iter[(gid_y) * iter_stride + gid_x] = (float)(_tmp192);
        }
        {
            float _tmp193 = 0.F;
            {
                _tmp193 += 0.0707660019F * tex1Dfetch(_texinputX, (gid_y + 1 * (int)blockDim.y + 0) * input_stride + gid_x + -2);
            }
            {
                _tmp193 += 0.244460002F * tex1Dfetch(_texinputX, (gid_y + 1 * (int)blockDim.y + 0) * input_stride + gid_x + -1);
            }
            {
                _tmp193 += 0.369545996F * tex1Dfetch(_texinputX, (gid_y + 1 * (int)blockDim.y + 0) * input_stride + gid_x + 0);
            }
            {
                _tmp193 += 0.244460002F * tex1Dfetch(_texinputX, (gid_y + 1 * (int)blockDim.y + 0) * input_stride + gid_x + 1);
            }
            {
                _tmp193 += 0.0707660019F * tex1Dfetch(_texinputX, (gid_y + 1 * (int)blockDim.y + 0) * input_stride + gid_x + 2);
            }
            iter[(gid_y + 1 * (int)blockDim.y) * iter_stride + gid_x] = (float)(_tmp193);
        }
        {
            float _tmp194 = 0.F;
            {
                _tmp194 += 0.0707660019F * tex1Dfetch(_texinputX, (gid_y + 2 * (int)blockDim.y + 0) * input_stride + gid_x + -2);
            }
            {
                _tmp194 += 0.244460002F * tex1Dfetch(_texinputX, (gid_y + 2 * (int)blockDim.y + 0) * input_stride + gid_x + -1);
            }
            {
                _tmp194 += 0.369545996F * tex1Dfetch(_texinputX, (gid_y + 2 * (int)blockDim.y + 0) * input_stride + gid_x + 0);
            }
            {
                _tmp194 += 0.244460002F * tex1Dfetch(_texinputX, (gid_y + 2 * (int)blockDim.y + 0) * input_stride + gid_x + 1);
            }
            {
                _tmp194 += 0.0707660019F * tex1Dfetch(_texinputX, (gid_y + 2 * (int)blockDim.y + 0) * input_stride + gid_x + 2);
            }
            iter[(gid_y + 2 * (int)blockDim.y) * iter_stride + gid_x] = (float)(_tmp194);
        }
        {
            float _tmp195 = 0.F;
            {
                _tmp195 += 0.0707660019F * tex1Dfetch(_texinputX, (gid_y + 3 * (int)blockDim.y + 0) * input_stride + gid_x + -2);
            }
            {
                _tmp195 += 0.244460002F * tex1Dfetch(_texinputX, (gid_y + 3 * (int)blockDim.y + 0) * input_stride + gid_x + -1);
            }
            {
                _tmp195 += 0.369545996F * tex1Dfetch(_texinputX, (gid_y + 3 * (int)blockDim.y + 0) * input_stride + gid_x + 0);
            }
            {
                _tmp195 += 0.244460002F * tex1Dfetch(_texinputX, (gid_y + 3 * (int)blockDim.y + 0) * input_stride + gid_x + 1);
            }
            {
                _tmp195 += 0.0707660019F * tex1Dfetch(_texinputX, (gid_y + 3 * (int)blockDim.y + 0) * input_stride + gid_x + 2);
            }
            iter[(gid_y + 3 * (int)blockDim.y) * iter_stride + gid_x] = (float)(_tmp195);
        }
        {
            float _tmp196 = 0.F;
            {
                _tmp196 += 0.0707660019F * tex1Dfetch(_texinputX, (gid_y + 4 * (int)blockDim.y + 0) * input_stride + gid_x + -2);
            }
            {
                _tmp196 += 0.244460002F * tex1Dfetch(_texinputX, (gid_y + 4 * (int)blockDim.y + 0) * input_stride + gid_x + -1);
            }
            {
                _tmp196 += 0.369545996F * tex1Dfetch(_texinputX, (gid_y + 4 * (int)blockDim.y + 0) * input_stride + gid_x + 0);
            }
            {
                _tmp196 += 0.244460002F * tex1Dfetch(_texinputX, (gid_y + 4 * (int)blockDim.y + 0) * input_stride + gid_x + 1);
            }
            {
                _tmp196 += 0.0707660019F * tex1Dfetch(_texinputX, (gid_y + 4 * (int)blockDim.y + 0) * input_stride + gid_x + 2);
            }
            iter[(gid_y + 4 * (int)blockDim.y) * iter_stride + gid_x] = (float)(_tmp196);
        }
        {
            float _tmp197 = 0.F;
            {
                _tmp197 += 0.0707660019F * tex1Dfetch(_texinputX, (gid_y + 5 * (int)blockDim.y + 0) * input_stride + gid_x + -2);
            }
            {
                _tmp197 += 0.244460002F * tex1Dfetch(_texinputX, (gid_y + 5 * (int)blockDim.y + 0) * input_stride + gid_x + -1);
            }
            {
                _tmp197 += 0.369545996F * tex1Dfetch(_texinputX, (gid_y + 5 * (int)blockDim.y + 0) * input_stride + gid_x + 0);
            }
            {
                _tmp197 += 0.244460002F * tex1Dfetch(_texinputX, (gid_y + 5 * (int)blockDim.y + 0) * input_stride + gid_x + 1);
            }
            {
                _tmp197 += 0.0707660019F * tex1Dfetch(_texinputX, (gid_y + 5 * (int)blockDim.y + 0) * input_stride + gid_x + 2);
            }
            iter[(gid_y + 5 * (int)blockDim.y) * iter_stride + gid_x] = (float)(_tmp197);
        }
        {
            float _tmp198 = 0.F;
            {
                _tmp198 += 0.0707660019F * tex1Dfetch(_texinputX, (gid_y + 6 * (int)blockDim.y + 0) * input_stride + gid_x + -2);
            }
            {
                _tmp198 += 0.244460002F * tex1Dfetch(_texinputX, (gid_y + 6 * (int)blockDim.y + 0) * input_stride + gid_x + -1);
            }
            {
                _tmp198 += 0.369545996F * tex1Dfetch(_texinputX, (gid_y + 6 * (int)blockDim.y + 0) * input_stride + gid_x + 0);
            }
            {
                _tmp198 += 0.244460002F * tex1Dfetch(_texinputX, (gid_y + 6 * (int)blockDim.y + 0) * input_stride + gid_x + 1);
            }
            {
                _tmp198 += 0.0707660019F * tex1Dfetch(_texinputX, (gid_y + 6 * (int)blockDim.y + 0) * input_stride + gid_x + 2);
            }
            iter[(gid_y + 6 * (int)blockDim.y) * iter_stride + gid_x] = (float)(_tmp198);
        }
        {
            float _tmp199 = 0.F;
            {
                _tmp199 += 0.0707660019F * tex1Dfetch(_texinputX, (gid_y + 7 * (int)blockDim.y + 0) * input_stride + gid_x + -2);
            }
            {
                _tmp199 += 0.244460002F * tex1Dfetch(_texinputX, (gid_y + 7 * (int)blockDim.y + 0) * input_stride + gid_x + -1);
            }
            {
                _tmp199 += 0.369545996F * tex1Dfetch(_texinputX, (gid_y + 7 * (int)blockDim.y + 0) * input_stride + gid_x + 0);
            }
            {
                _tmp199 += 0.244460002F * tex1Dfetch(_texinputX, (gid_y + 7 * (int)blockDim.y + 0) * input_stride + gid_x + 1);
            }
            {
                _tmp199 += 0.0707660019F * tex1Dfetch(_texinputX, (gid_y + 7 * (int)blockDim.y + 0) * input_stride + gid_x + 2);
            }
            iter[(gid_y + 7 * (int)blockDim.y) * iter_stride + gid_x] = (float)(_tmp199);
        }
    }
    goto BH_EXIT;
  BH_EXIT:
    ;
}
}

#endif //_CUGAUSSIANFILTERROWX_CU_

