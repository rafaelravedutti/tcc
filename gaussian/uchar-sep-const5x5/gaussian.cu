#include "hip/hip_runtime.h"
#include "hipacc_cu.hpp"

#include "cuGaussianFilterRowX.cu"
#include "cuGaussianFilterColumnY.cu"
#include <algorithm>
#include <cmath>
#include <cstdlib>
#include <iostream>
#include <vector>
#include <sys/time.h>
#include <opencv2/opencv.hpp>
//---

#ifndef BH_MODE
#  define BH_MODE           CLAMP
#endif

#define PIXEL_CAST(a)       (pixel_t)(a)
#define TMP_PIXEL_CAST(a)   (tmp_pixel_t)(a)

#define pixel_t             uchar
#define tmp_pixel_t         float

#define USE_LAMBDA


// get time in milliseconds
double time_ms () {
    struct timeval tv;
    gettimeofday (&tv, NULL);

    return ((double)(tv.tv_sec) * 1e+3 + (double)(tv.tv_usec) * 1e-3);
}

// Gaussian blur filter in Hipacc
#ifdef NO_SEP

class GaussianFilter : public Kernel<pixel_t> {
    private:
        Accessor<pixel_t> &input;
        Mask<float> &mask;
        const int size_x, size_y;

    public:
        GaussianFilter(IterationSpace<pixel_t> &iter, Accessor<pixel_t>
                &input, Mask<float> &mask, const int size_x, const int size_y) :
            Kernel(iter),
            input(input),
            mask(mask),
            size_x(size_x),
            size_y(size_y)
        { add_accessor(&input); }

        #ifdef USE_LAMBDA

        void kernel() {
            output() = PIXEL_CAST(convolve(mask, Reduce::SUM, [&] () -> float {
                    return mask() * input(mask);
                    }) + 0.5f);
        }

        #else

        void kernel() {
            const int anchor_x = size_x >> 1;
            const int anchor_y = size_y >> 1;
            float sum = 0.5f;

            for (int yf = -anchor_y; yf<=anchor_y; ++yf) {
                for (int xf = -anchor_x; xf<=anchor_x; ++xf) {
                    sum += mask(xf, yf) * input(xf, yf);
                }
            }

            output() = PIXEL_CAST(sum);
        }

        #endif
};

#else



#endif

int main(int argc, const char *argv[]) {
    hipaccInitCUDA();
    
    const int size_x = SIZE_X;
    const int size_y = SIZE_Y;
    const double sigma1 = ((size_x-1)*0.5 - 1)*0.3 + 0.8;
    const double sigma2 = ((size_y-1)*0.5 - 1)*0.3 + 0.8;

    // filter coefficients
    #ifdef CONST_MASK

    // only filter kernel sizes 3x3, 5x5, and 7x7 implemented
    if (size_x != size_y || !(size_x == 3 || size_x == 5 || size_x == 7)) {
        std::cerr << "Wrong filter kernel size. Currently supported values: 3x3, 5x5, and 7x7!" << std::endl;
        exit(EXIT_FAILURE);
    }

    // convolution filter mask
    const float filter_x[1][SIZE_X] = {
        #if SIZE_X == 3
        { 0.238994f, 0.522011f, 0.238994f }
        #endif
        #if SIZE_X == 5
        { 0.070766f, 0.244460f, 0.369546f, 0.244460f, 0.070766f }
        #endif
        #if SIZE_X == 7
        { 0.028995f, 0.103818f, 0.223173f, 0.288026f, 0.223173f, 0.103818f, 0.028995f }
        #endif
    };

    const float filter_y[SIZE_Y][1] = {
        #if SIZE_Y == 3
        { 0.238994f }, { 0.522011f }, { 0.238994f }
        #endif
        #if SIZE_Y == 5
        { 0.070766f }, { 0.244460f }, { 0.369546f }, { 0.244460f }, { 0.070766f }
        #endif
        #if SIZE_Y == 7
        { 0.028995f }, { 0.103818f }, { 0.223173f }, { 0.288026f }, { 0.223173f }, { 0.103818f }, { 0.028995f }
        #endif
    };

    const float filter_xy[SIZE_Y][SIZE_X] = {
        #if SIZE_X == 3
        { 0.057118f, 0.124758f, 0.057118f },
        { 0.124758f, 0.272496f, 0.124758f },
        { 0.057118f, 0.124758f, 0.057118f }
        #endif
        #if SIZE_X == 5
        { 0.005008f, 0.017300f, 0.026151f, 0.017300f, 0.005008f },
        { 0.017300f, 0.059761f, 0.090339f, 0.059761f, 0.017300f },
        { 0.026151f, 0.090339f, 0.136565f, 0.090339f, 0.026151f },
        { 0.017300f, 0.059761f, 0.090339f, 0.059761f, 0.017300f },
        { 0.005008f, 0.017300f, 0.026151f, 0.017300f, 0.005008f }
        #endif
        #if SIZE_X == 7
        { 0.000841, 0.003010, 0.006471, 0.008351, 0.006471, 0.003010, 0.000841 },
        { 0.003010, 0.010778, 0.023169, 0.029902, 0.023169, 0.010778, 0.003010 },
        { 0.006471, 0.023169, 0.049806, 0.064280, 0.049806, 0.023169, 0.006471 },
        { 0.008351, 0.029902, 0.064280, 0.082959, 0.064280, 0.029902, 0.008351 },
        { 0.006471, 0.023169, 0.049806, 0.064280, 0.049806, 0.023169, 0.006471 },
        { 0.003010, 0.010778, 0.023169, 0.029902, 0.023169, 0.010778, 0.003010 },
        { 0.000841, 0.003010, 0.006471, 0.008351, 0.006471, 0.003010, 0.000841 }
        #endif
    };

    #else

    float filter_x[1][SIZE_X];
    float filter_y[SIZE_Y][1];
    float filter_xy[SIZE_Y][SIZE_X];

    double scale2X = -0.5/(sigma1*sigma1);
    double scale2Y = -0.5/(sigma2*sigma2);
    double sum_x = 0;
    double sum_y = 0;

    for (int i=0; i < size_x; ++i) {
        double x = i - (size_x-1)*0.5;
        double t = std::exp(scale2X*x*x);

        filter_x[0][i] = (float)t;
        sum_x += filter_x[0][i];
    }
    for (int i=0; i < size_y; ++i) {
        double x = i - (size_y-1)*0.5;
        double t = std::exp(scale2Y*x*x);

        filter_y[i][0] = (float)t;
        sum_y += filter_y[i][0];
    }

    sum_x = 1./sum_x;
    sum_y = 1./sum_y;
    for (int i=0; i < size_x; ++i) {
        filter_x[0][i] = (float)(filter_x[0][i]*sum_x);
    }
    for (int i=0; i < size_y; ++i) {
        filter_y[i][0] = (float)(filter_y[i][0]*sum_y);
    }

    for (int y=0; y < size_y; ++y) {
        for (int x=0; x < size_x; ++x) {
            filter_xy[y][x] = filter_x[0][x]*filter_y[y][0];
        }
    }

    #endif

    cv::Mat frame, frame_colored;

    frame_colored = cv::imread(argv[1], CV_LOAD_IMAGE_COLOR);
    cvtColor(frame_colored, frame, CV_BGR2GRAY);

    const int width = frame.cols;
    const int height = frame.rows;
    float timing = 0;

    // input and output image of width x height pixels
    HipaccImage input = hipaccCreateMemory<uchar>(NULL, width, height, 256);
    HipaccImage output = hipaccCreateMemory<uchar>(NULL, width, height, 256);
    HipaccImage tmp = hipaccCreateMemory<float>(NULL, width, height, 256);

    hipaccWriteMemory(input, frame.data);

    // filter mask
    
    
    

    HipaccAccessor iter_out(output);
    HipaccAccessor iter_tmp(tmp);

    #ifdef NO_SEP

    BoundaryCondition<pixel_t> cond(input, mask, Boundary::BH_MODE);
    Accessor<pixel_t> acc(cond);
    GaussianFilter XY(iter_out, acc, mask, size_x, size_y);

    XY.execute();
    timing = hipacc_last_kernel_timing();

    #else

    
    HipaccAccessor acc(input);
    

    
    HipaccAccessor acc_tmp(tmp);
    

    hipacc_launch_info X_info0(2, 0, iter_tmp, 8, 1);
    dim3 block0(64, 1);
    dim3 grid0(hipaccCalcGridFromBlock(X_info0, block0));

    hipaccPrepareKernelLaunch(X_info0, block0);
    hipaccConfigureCall(grid0, block0);

    size_t offset0 = 0;
    hipGetTextureReference(&_texinputXRef, HIP_SYMBOL(&_texinputX));
    hipaccBindTexture<uchar>(Linear1D, _texinputXRef, acc.img);
    hipaccSetupArgument(&iter_tmp.img.mem, sizeof(float *), offset0);
    hipaccSetupArgument(&iter_tmp.width, sizeof(const int), offset0);
    hipaccSetupArgument(&iter_tmp.height, sizeof(const int), offset0);
    hipaccSetupArgument(&iter_tmp.img.stride, sizeof(const int), offset0);
    hipaccSetupArgument(&acc.width, sizeof(const int), offset0);
    hipaccSetupArgument(&acc.height, sizeof(const int), offset0);
    hipaccSetupArgument(&acc.img.stride, sizeof(const int), offset0);
    hipaccSetupArgument(&X_info0.bh_start_left, sizeof(const int), offset0);
    hipaccSetupArgument(&X_info0.bh_start_right, sizeof(const int), offset0);
    hipaccSetupArgument(&X_info0.bh_start_bottom, sizeof(const int), offset0);
    hipaccSetupArgument(&X_info0.bh_fall_back, sizeof(const int), offset0);
    
    hipaccLaunchKernel((const void *)&cuGaussianFilterRowXKernel, "cuGaussianFilterRowXKernel", grid0, block0);
    timing = hipacc_last_kernel_timing();
    hipacc_launch_info Y_info0(0, 2, iter_out, 8, 1);
    dim3 block1(32, 2);
    dim3 grid1(hipaccCalcGridFromBlock(Y_info0, block1));

    hipaccPrepareKernelLaunch(Y_info0, block1);
    hipaccConfigureCall(grid1, block1);

    size_t offset1 = 0;
    hipGetTextureReference(&_texinputYRef, HIP_SYMBOL(&_texinputY));
    hipaccBindTexture<float>(Linear1D, _texinputYRef, acc_tmp.img);
    hipaccSetupArgument(&iter_out.img.mem, sizeof(uchar *), offset1);
    hipaccSetupArgument(&iter_out.width, sizeof(const int), offset1);
    hipaccSetupArgument(&iter_out.height, sizeof(const int), offset1);
    hipaccSetupArgument(&iter_out.img.stride, sizeof(const int), offset1);
    hipaccSetupArgument(&acc_tmp.width, sizeof(const int), offset1);
    hipaccSetupArgument(&acc_tmp.height, sizeof(const int), offset1);
    hipaccSetupArgument(&acc_tmp.img.stride, sizeof(const int), offset1);
    hipaccSetupArgument(&Y_info0.bh_start_right, sizeof(const int), offset1);
    hipaccSetupArgument(&Y_info0.bh_start_top, sizeof(const int), offset1);
    hipaccSetupArgument(&Y_info0.bh_start_bottom, sizeof(const int), offset1);
    hipaccSetupArgument(&Y_info0.bh_fall_back, sizeof(const int), offset1);
    
    hipaccLaunchKernel((const void *)&cuGaussianFilterColumnYKernel, "cuGaussianFilterColumnYKernel", grid1, block1);
    timing += hipacc_last_kernel_timing();

    #endif

    std::cerr << "Timing: " << timing << " ms, " << (width*height/timing)/1000 << " Mpixel/s" << std::endl;

    // OpenCV display image
    std::string outputfn;
    std::vector<int> compression_params;

    frame.data = hipaccReadMemory<uchar>(output);
    outputfn = argv[1];
    outputfn = outputfn.substr(0, outputfn.find_last_of(".")) + "-filtered.jpg";
    compression_params.push_back(CV_IMWRITE_JPEG_QUALITY);
    compression_params.push_back(100);

    try {
      cv::imwrite(outputfn, frame, compression_params);
    } catch(std::runtime_error &ex) {
      fprintf(stderr, "JPEG compression exception: %s\n", ex.what());
      return 1;
    }

    cv::namedWindow("Result", cv::WINDOW_NORMAL);
    cv::imshow("Result", frame);
    cv::waitKey(0);

    fprintf(stdout, "Done!\n");
    hipaccReleaseMemory<uchar>(input);
    hipaccReleaseMemory<uchar>(output);
    hipaccReleaseMemory<float>(tmp);
    return 0;
}
