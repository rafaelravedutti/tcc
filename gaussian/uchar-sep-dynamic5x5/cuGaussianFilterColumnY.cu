#include "hip/hip_runtime.h"
#ifndef _CUGAUSSIANFILTERCOLUMNY_CU_
#define _CUGAUSSIANFILTERCOLUMNY_CU_

#include "hipacc_types.hpp"
#include "hipacc_math_functions.hpp"

texture<float, hipTextureType1D, hipReadModeElementType> _texinputY;
const textureReference *_texinputYRef;
__device__ __constant__ float _constmask_yY[5][1];


extern "C" {
__global__ __launch_bounds__ (32*2) void cuGaussianFilterColumnYKernel(uchar * __restrict__ iter, int iter_width, int iter_height, int iter_stride, int input_width, int input_height, int input_stride, int bh_start_right, int bh_start_top, int bh_start_bottom, int bh_fall_back) {
    const int gid_x = blockDim.x * blockIdx.x + threadIdx.x;
    const int gid_y = blockDim.y * blockIdx.y * 8 + threadIdx.y;
    if (bh_fall_back)
        goto BH_FB;
    if (blockIdx.y < bh_start_top)
        goto BH_T;
    if (blockIdx.y >= bh_start_bottom)
        goto BH_B;
    if (blockIdx.x >= bh_start_right)
        goto BH_R;
    goto BH_NO;
  BH_FB:
    {
        if (gid_x < iter_width) {
            if (gid_y < iter_height) {
                float _tmp0 = 0.F;
                {
                    int _gid_x1 = gid_x + 0;
                    int _gid_y1 = gid_y + -2;
                    if (_gid_y1 >= input_height)
                        _gid_y1 = input_height - 1;
                    if (_gid_y1 < 0)
                        _gid_y1 = 0;
                    _tmp0 += _constmask_yY[0][0] * tex1Dfetch(_texinputY, (_gid_y1) * input_stride + _gid_x1);
                }
                {
                    int _gid_x2 = gid_x + 0;
                    int _gid_y2 = gid_y + -1;
                    if (_gid_y2 >= input_height)
                        _gid_y2 = input_height - 1;
                    if (_gid_y2 < 0)
                        _gid_y2 = 0;
                    _tmp0 += _constmask_yY[1][0] * tex1Dfetch(_texinputY, (_gid_y2) * input_stride + _gid_x2);
                }
                {
                    int _gid_x3 = gid_x + 0;
                    int _gid_y3 = gid_y + 0;
                    if (_gid_y3 >= input_height)
                        _gid_y3 = input_height - 1;
                    if (_gid_y3 < 0)
                        _gid_y3 = 0;
                    _tmp0 += _constmask_yY[2][0] * tex1Dfetch(_texinputY, (_gid_y3) * input_stride + _gid_x3);
                }
                {
                    int _gid_x4 = gid_x + 0;
                    int _gid_y4 = gid_y + 1;
                    if (_gid_y4 >= input_height)
                        _gid_y4 = input_height - 1;
                    if (_gid_y4 < 0)
                        _gid_y4 = 0;
                    _tmp0 += _constmask_yY[3][0] * tex1Dfetch(_texinputY, (_gid_y4) * input_stride + _gid_x4);
                }
                {
                    int _gid_x5 = gid_x + 0;
                    int _gid_y5 = gid_y + 2;
                    if (_gid_y5 >= input_height)
                        _gid_y5 = input_height - 1;
                    if (_gid_y5 < 0)
                        _gid_y5 = 0;
                    _tmp0 += _constmask_yY[4][0] * tex1Dfetch(_texinputY, (_gid_y5) * input_stride + _gid_x5);
                }
                iter[(gid_y) * iter_stride + gid_x] = (uchar)(_tmp0 + 0.5F);
            }
        }
        if (gid_x < iter_width) {
            if (gid_y + 1 * (int)blockDim.y < iter_height) {
                float _tmp6 = 0.F;
                {
                    int _gid_x7 = gid_x + 0;
                    int _gid_y7 = gid_y + 1 * (int)blockDim.y + -2;
                    if (_gid_y7 >= input_height)
                        _gid_y7 = input_height - 1;
                    if (_gid_y7 < 0)
                        _gid_y7 = 0;
                    _tmp6 += _constmask_yY[0][0] * tex1Dfetch(_texinputY, (_gid_y7) * input_stride + _gid_x7);
                }
                {
                    int _gid_x8 = gid_x + 0;
                    int _gid_y8 = gid_y + 1 * (int)blockDim.y + -1;
                    if (_gid_y8 >= input_height)
                        _gid_y8 = input_height - 1;
                    if (_gid_y8 < 0)
                        _gid_y8 = 0;
                    _tmp6 += _constmask_yY[1][0] * tex1Dfetch(_texinputY, (_gid_y8) * input_stride + _gid_x8);
                }
                {
                    int _gid_x9 = gid_x + 0;
                    int _gid_y9 = gid_y + 1 * (int)blockDim.y + 0;
                    if (_gid_y9 >= input_height)
                        _gid_y9 = input_height - 1;
                    if (_gid_y9 < 0)
                        _gid_y9 = 0;
                    _tmp6 += _constmask_yY[2][0] * tex1Dfetch(_texinputY, (_gid_y9) * input_stride + _gid_x9);
                }
                {
                    int _gid_x10 = gid_x + 0;
                    int _gid_y10 = gid_y + 1 * (int)blockDim.y + 1;
                    if (_gid_y10 >= input_height)
                        _gid_y10 = input_height - 1;
                    if (_gid_y10 < 0)
                        _gid_y10 = 0;
                    _tmp6 += _constmask_yY[3][0] * tex1Dfetch(_texinputY, (_gid_y10) * input_stride + _gid_x10);
                }
                {
                    int _gid_x11 = gid_x + 0;
                    int _gid_y11 = gid_y + 1 * (int)blockDim.y + 2;
                    if (_gid_y11 >= input_height)
                        _gid_y11 = input_height - 1;
                    if (_gid_y11 < 0)
                        _gid_y11 = 0;
                    _tmp6 += _constmask_yY[4][0] * tex1Dfetch(_texinputY, (_gid_y11) * input_stride + _gid_x11);
                }
                iter[(gid_y + 1 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp6 + 0.5F);
            }
        }
        if (gid_x < iter_width) {
            if (gid_y + 2 * (int)blockDim.y < iter_height) {
                float _tmp12 = 0.F;
                {
                    int _gid_x13 = gid_x + 0;
                    int _gid_y13 = gid_y + 2 * (int)blockDim.y + -2;
                    if (_gid_y13 >= input_height)
                        _gid_y13 = input_height - 1;
                    if (_gid_y13 < 0)
                        _gid_y13 = 0;
                    _tmp12 += _constmask_yY[0][0] * tex1Dfetch(_texinputY, (_gid_y13) * input_stride + _gid_x13);
                }
                {
                    int _gid_x14 = gid_x + 0;
                    int _gid_y14 = gid_y + 2 * (int)blockDim.y + -1;
                    if (_gid_y14 >= input_height)
                        _gid_y14 = input_height - 1;
                    if (_gid_y14 < 0)
                        _gid_y14 = 0;
                    _tmp12 += _constmask_yY[1][0] * tex1Dfetch(_texinputY, (_gid_y14) * input_stride + _gid_x14);
                }
                {
                    int _gid_x15 = gid_x + 0;
                    int _gid_y15 = gid_y + 2 * (int)blockDim.y + 0;
                    if (_gid_y15 >= input_height)
                        _gid_y15 = input_height - 1;
                    if (_gid_y15 < 0)
                        _gid_y15 = 0;
                    _tmp12 += _constmask_yY[2][0] * tex1Dfetch(_texinputY, (_gid_y15) * input_stride + _gid_x15);
                }
                {
                    int _gid_x16 = gid_x + 0;
                    int _gid_y16 = gid_y + 2 * (int)blockDim.y + 1;
                    if (_gid_y16 >= input_height)
                        _gid_y16 = input_height - 1;
                    if (_gid_y16 < 0)
                        _gid_y16 = 0;
                    _tmp12 += _constmask_yY[3][0] * tex1Dfetch(_texinputY, (_gid_y16) * input_stride + _gid_x16);
                }
                {
                    int _gid_x17 = gid_x + 0;
                    int _gid_y17 = gid_y + 2 * (int)blockDim.y + 2;
                    if (_gid_y17 >= input_height)
                        _gid_y17 = input_height - 1;
                    if (_gid_y17 < 0)
                        _gid_y17 = 0;
                    _tmp12 += _constmask_yY[4][0] * tex1Dfetch(_texinputY, (_gid_y17) * input_stride + _gid_x17);
                }
                iter[(gid_y + 2 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp12 + 0.5F);
            }
        }
        if (gid_x < iter_width) {
            if (gid_y + 3 * (int)blockDim.y < iter_height) {
                float _tmp18 = 0.F;
                {
                    int _gid_x19 = gid_x + 0;
                    int _gid_y19 = gid_y + 3 * (int)blockDim.y + -2;
                    if (_gid_y19 >= input_height)
                        _gid_y19 = input_height - 1;
                    if (_gid_y19 < 0)
                        _gid_y19 = 0;
                    _tmp18 += _constmask_yY[0][0] * tex1Dfetch(_texinputY, (_gid_y19) * input_stride + _gid_x19);
                }
                {
                    int _gid_x20 = gid_x + 0;
                    int _gid_y20 = gid_y + 3 * (int)blockDim.y + -1;
                    if (_gid_y20 >= input_height)
                        _gid_y20 = input_height - 1;
                    if (_gid_y20 < 0)
                        _gid_y20 = 0;
                    _tmp18 += _constmask_yY[1][0] * tex1Dfetch(_texinputY, (_gid_y20) * input_stride + _gid_x20);
                }
                {
                    int _gid_x21 = gid_x + 0;
                    int _gid_y21 = gid_y + 3 * (int)blockDim.y + 0;
                    if (_gid_y21 >= input_height)
                        _gid_y21 = input_height - 1;
                    if (_gid_y21 < 0)
                        _gid_y21 = 0;
                    _tmp18 += _constmask_yY[2][0] * tex1Dfetch(_texinputY, (_gid_y21) * input_stride + _gid_x21);
                }
                {
                    int _gid_x22 = gid_x + 0;
                    int _gid_y22 = gid_y + 3 * (int)blockDim.y + 1;
                    if (_gid_y22 >= input_height)
                        _gid_y22 = input_height - 1;
                    if (_gid_y22 < 0)
                        _gid_y22 = 0;
                    _tmp18 += _constmask_yY[3][0] * tex1Dfetch(_texinputY, (_gid_y22) * input_stride + _gid_x22);
                }
                {
                    int _gid_x23 = gid_x + 0;
                    int _gid_y23 = gid_y + 3 * (int)blockDim.y + 2;
                    if (_gid_y23 >= input_height)
                        _gid_y23 = input_height - 1;
                    if (_gid_y23 < 0)
                        _gid_y23 = 0;
                    _tmp18 += _constmask_yY[4][0] * tex1Dfetch(_texinputY, (_gid_y23) * input_stride + _gid_x23);
                }
                iter[(gid_y + 3 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp18 + 0.5F);
            }
        }
        if (gid_x < iter_width) {
            if (gid_y + 4 * (int)blockDim.y < iter_height) {
                float _tmp24 = 0.F;
                {
                    int _gid_x25 = gid_x + 0;
                    int _gid_y25 = gid_y + 4 * (int)blockDim.y + -2;
                    if (_gid_y25 >= input_height)
                        _gid_y25 = input_height - 1;
                    if (_gid_y25 < 0)
                        _gid_y25 = 0;
                    _tmp24 += _constmask_yY[0][0] * tex1Dfetch(_texinputY, (_gid_y25) * input_stride + _gid_x25);
                }
                {
                    int _gid_x26 = gid_x + 0;
                    int _gid_y26 = gid_y + 4 * (int)blockDim.y + -1;
                    if (_gid_y26 >= input_height)
                        _gid_y26 = input_height - 1;
                    if (_gid_y26 < 0)
                        _gid_y26 = 0;
                    _tmp24 += _constmask_yY[1][0] * tex1Dfetch(_texinputY, (_gid_y26) * input_stride + _gid_x26);
                }
                {
                    int _gid_x27 = gid_x + 0;
                    int _gid_y27 = gid_y + 4 * (int)blockDim.y + 0;
                    if (_gid_y27 >= input_height)
                        _gid_y27 = input_height - 1;
                    if (_gid_y27 < 0)
                        _gid_y27 = 0;
                    _tmp24 += _constmask_yY[2][0] * tex1Dfetch(_texinputY, (_gid_y27) * input_stride + _gid_x27);
                }
                {
                    int _gid_x28 = gid_x + 0;
                    int _gid_y28 = gid_y + 4 * (int)blockDim.y + 1;
                    if (_gid_y28 >= input_height)
                        _gid_y28 = input_height - 1;
                    if (_gid_y28 < 0)
                        _gid_y28 = 0;
                    _tmp24 += _constmask_yY[3][0] * tex1Dfetch(_texinputY, (_gid_y28) * input_stride + _gid_x28);
                }
                {
                    int _gid_x29 = gid_x + 0;
                    int _gid_y29 = gid_y + 4 * (int)blockDim.y + 2;
                    if (_gid_y29 >= input_height)
                        _gid_y29 = input_height - 1;
                    if (_gid_y29 < 0)
                        _gid_y29 = 0;
                    _tmp24 += _constmask_yY[4][0] * tex1Dfetch(_texinputY, (_gid_y29) * input_stride + _gid_x29);
                }
                iter[(gid_y + 4 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp24 + 0.5F);
            }
        }
        if (gid_x < iter_width) {
            if (gid_y + 5 * (int)blockDim.y < iter_height) {
                float _tmp30 = 0.F;
                {
                    int _gid_x31 = gid_x + 0;
                    int _gid_y31 = gid_y + 5 * (int)blockDim.y + -2;
                    if (_gid_y31 >= input_height)
                        _gid_y31 = input_height - 1;
                    if (_gid_y31 < 0)
                        _gid_y31 = 0;
                    _tmp30 += _constmask_yY[0][0] * tex1Dfetch(_texinputY, (_gid_y31) * input_stride + _gid_x31);
                }
                {
                    int _gid_x32 = gid_x + 0;
                    int _gid_y32 = gid_y + 5 * (int)blockDim.y + -1;
                    if (_gid_y32 >= input_height)
                        _gid_y32 = input_height - 1;
                    if (_gid_y32 < 0)
                        _gid_y32 = 0;
                    _tmp30 += _constmask_yY[1][0] * tex1Dfetch(_texinputY, (_gid_y32) * input_stride + _gid_x32);
                }
                {
                    int _gid_x33 = gid_x + 0;
                    int _gid_y33 = gid_y + 5 * (int)blockDim.y + 0;
                    if (_gid_y33 >= input_height)
                        _gid_y33 = input_height - 1;
                    if (_gid_y33 < 0)
                        _gid_y33 = 0;
                    _tmp30 += _constmask_yY[2][0] * tex1Dfetch(_texinputY, (_gid_y33) * input_stride + _gid_x33);
                }
                {
                    int _gid_x34 = gid_x + 0;
                    int _gid_y34 = gid_y + 5 * (int)blockDim.y + 1;
                    if (_gid_y34 >= input_height)
                        _gid_y34 = input_height - 1;
                    if (_gid_y34 < 0)
                        _gid_y34 = 0;
                    _tmp30 += _constmask_yY[3][0] * tex1Dfetch(_texinputY, (_gid_y34) * input_stride + _gid_x34);
                }
                {
                    int _gid_x35 = gid_x + 0;
                    int _gid_y35 = gid_y + 5 * (int)blockDim.y + 2;
                    if (_gid_y35 >= input_height)
                        _gid_y35 = input_height - 1;
                    if (_gid_y35 < 0)
                        _gid_y35 = 0;
                    _tmp30 += _constmask_yY[4][0] * tex1Dfetch(_texinputY, (_gid_y35) * input_stride + _gid_x35);
                }
                iter[(gid_y + 5 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp30 + 0.5F);
            }
        }
        if (gid_x < iter_width) {
            if (gid_y + 6 * (int)blockDim.y < iter_height) {
                float _tmp36 = 0.F;
                {
                    int _gid_x37 = gid_x + 0;
                    int _gid_y37 = gid_y + 6 * (int)blockDim.y + -2;
                    if (_gid_y37 >= input_height)
                        _gid_y37 = input_height - 1;
                    if (_gid_y37 < 0)
                        _gid_y37 = 0;
                    _tmp36 += _constmask_yY[0][0] * tex1Dfetch(_texinputY, (_gid_y37) * input_stride + _gid_x37);
                }
                {
                    int _gid_x38 = gid_x + 0;
                    int _gid_y38 = gid_y + 6 * (int)blockDim.y + -1;
                    if (_gid_y38 >= input_height)
                        _gid_y38 = input_height - 1;
                    if (_gid_y38 < 0)
                        _gid_y38 = 0;
                    _tmp36 += _constmask_yY[1][0] * tex1Dfetch(_texinputY, (_gid_y38) * input_stride + _gid_x38);
                }
                {
                    int _gid_x39 = gid_x + 0;
                    int _gid_y39 = gid_y + 6 * (int)blockDim.y + 0;
                    if (_gid_y39 >= input_height)
                        _gid_y39 = input_height - 1;
                    if (_gid_y39 < 0)
                        _gid_y39 = 0;
                    _tmp36 += _constmask_yY[2][0] * tex1Dfetch(_texinputY, (_gid_y39) * input_stride + _gid_x39);
                }
                {
                    int _gid_x40 = gid_x + 0;
                    int _gid_y40 = gid_y + 6 * (int)blockDim.y + 1;
                    if (_gid_y40 >= input_height)
                        _gid_y40 = input_height - 1;
                    if (_gid_y40 < 0)
                        _gid_y40 = 0;
                    _tmp36 += _constmask_yY[3][0] * tex1Dfetch(_texinputY, (_gid_y40) * input_stride + _gid_x40);
                }
                {
                    int _gid_x41 = gid_x + 0;
                    int _gid_y41 = gid_y + 6 * (int)blockDim.y + 2;
                    if (_gid_y41 >= input_height)
                        _gid_y41 = input_height - 1;
                    if (_gid_y41 < 0)
                        _gid_y41 = 0;
                    _tmp36 += _constmask_yY[4][0] * tex1Dfetch(_texinputY, (_gid_y41) * input_stride + _gid_x41);
                }
                iter[(gid_y + 6 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp36 + 0.5F);
            }
        }
        if (gid_x < iter_width) {
            if (gid_y + 7 * (int)blockDim.y < iter_height) {
                float _tmp42 = 0.F;
                {
                    int _gid_x43 = gid_x + 0;
                    int _gid_y43 = gid_y + 7 * (int)blockDim.y + -2;
                    if (_gid_y43 >= input_height)
                        _gid_y43 = input_height - 1;
                    if (_gid_y43 < 0)
                        _gid_y43 = 0;
                    _tmp42 += _constmask_yY[0][0] * tex1Dfetch(_texinputY, (_gid_y43) * input_stride + _gid_x43);
                }
                {
                    int _gid_x44 = gid_x + 0;
                    int _gid_y44 = gid_y + 7 * (int)blockDim.y + -1;
                    if (_gid_y44 >= input_height)
                        _gid_y44 = input_height - 1;
                    if (_gid_y44 < 0)
                        _gid_y44 = 0;
                    _tmp42 += _constmask_yY[1][0] * tex1Dfetch(_texinputY, (_gid_y44) * input_stride + _gid_x44);
                }
                {
                    int _gid_x45 = gid_x + 0;
                    int _gid_y45 = gid_y + 7 * (int)blockDim.y + 0;
                    if (_gid_y45 >= input_height)
                        _gid_y45 = input_height - 1;
                    if (_gid_y45 < 0)
                        _gid_y45 = 0;
                    _tmp42 += _constmask_yY[2][0] * tex1Dfetch(_texinputY, (_gid_y45) * input_stride + _gid_x45);
                }
                {
                    int _gid_x46 = gid_x + 0;
                    int _gid_y46 = gid_y + 7 * (int)blockDim.y + 1;
                    if (_gid_y46 >= input_height)
                        _gid_y46 = input_height - 1;
                    if (_gid_y46 < 0)
                        _gid_y46 = 0;
                    _tmp42 += _constmask_yY[3][0] * tex1Dfetch(_texinputY, (_gid_y46) * input_stride + _gid_x46);
                }
                {
                    int _gid_x47 = gid_x + 0;
                    int _gid_y47 = gid_y + 7 * (int)blockDim.y + 2;
                    if (_gid_y47 >= input_height)
                        _gid_y47 = input_height - 1;
                    if (_gid_y47 < 0)
                        _gid_y47 = 0;
                    _tmp42 += _constmask_yY[4][0] * tex1Dfetch(_texinputY, (_gid_y47) * input_stride + _gid_x47);
                }
                iter[(gid_y + 7 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp42 + 0.5F);
            }
        }
    }
    goto BH_EXIT;
  BH_T:
    {
        if (gid_x < iter_width) {
            {
                float _tmp48 = 0.F;
                {
                    int _gid_x49 = gid_x + 0;
                    int _gid_y49 = gid_y + -2;
                    if (_gid_y49 < 0)
                        _gid_y49 = 0;
                    _tmp48 += _constmask_yY[0][0] * tex1Dfetch(_texinputY, (_gid_y49) * input_stride + _gid_x49);
                }
                {
                    int _gid_x50 = gid_x + 0;
                    int _gid_y50 = gid_y + -1;
                    if (_gid_y50 < 0)
                        _gid_y50 = 0;
                    _tmp48 += _constmask_yY[1][0] * tex1Dfetch(_texinputY, (_gid_y50) * input_stride + _gid_x50);
                }
                {
                    int _gid_x51 = gid_x + 0;
                    int _gid_y51 = gid_y + 0;
                    if (_gid_y51 < 0)
                        _gid_y51 = 0;
                    _tmp48 += _constmask_yY[2][0] * tex1Dfetch(_texinputY, (_gid_y51) * input_stride + _gid_x51);
                }
                {
                    int _gid_x52 = gid_x + 0;
                    int _gid_y52 = gid_y + 1;
                    if (_gid_y52 < 0)
                        _gid_y52 = 0;
                    _tmp48 += _constmask_yY[3][0] * tex1Dfetch(_texinputY, (_gid_y52) * input_stride + _gid_x52);
                }
                {
                    int _gid_x53 = gid_x + 0;
                    int _gid_y53 = gid_y + 2;
                    if (_gid_y53 < 0)
                        _gid_y53 = 0;
                    _tmp48 += _constmask_yY[4][0] * tex1Dfetch(_texinputY, (_gid_y53) * input_stride + _gid_x53);
                }
                iter[(gid_y) * iter_stride + gid_x] = (uchar)(_tmp48 + 0.5F);
            }
        }
        if (gid_x < iter_width) {
            {
                float _tmp54 = 0.F;
                {
                    int _gid_x55 = gid_x + 0;
                    int _gid_y55 = gid_y + 1 * (int)blockDim.y + -2;
                    if (_gid_y55 < 0)
                        _gid_y55 = 0;
                    _tmp54 += _constmask_yY[0][0] * tex1Dfetch(_texinputY, (_gid_y55) * input_stride + _gid_x55);
                }
                {
                    int _gid_x56 = gid_x + 0;
                    int _gid_y56 = gid_y + 1 * (int)blockDim.y + -1;
                    if (_gid_y56 < 0)
                        _gid_y56 = 0;
                    _tmp54 += _constmask_yY[1][0] * tex1Dfetch(_texinputY, (_gid_y56) * input_stride + _gid_x56);
                }
                {
                    int _gid_x57 = gid_x + 0;
                    int _gid_y57 = gid_y + 1 * (int)blockDim.y + 0;
                    if (_gid_y57 < 0)
                        _gid_y57 = 0;
                    _tmp54 += _constmask_yY[2][0] * tex1Dfetch(_texinputY, (_gid_y57) * input_stride + _gid_x57);
                }
                {
                    int _gid_x58 = gid_x + 0;
                    int _gid_y58 = gid_y + 1 * (int)blockDim.y + 1;
                    if (_gid_y58 < 0)
                        _gid_y58 = 0;
                    _tmp54 += _constmask_yY[3][0] * tex1Dfetch(_texinputY, (_gid_y58) * input_stride + _gid_x58);
                }
                {
                    int _gid_x59 = gid_x + 0;
                    int _gid_y59 = gid_y + 1 * (int)blockDim.y + 2;
                    if (_gid_y59 < 0)
                        _gid_y59 = 0;
                    _tmp54 += _constmask_yY[4][0] * tex1Dfetch(_texinputY, (_gid_y59) * input_stride + _gid_x59);
                }
                iter[(gid_y + 1 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp54 + 0.5F);
            }
        }
        if (gid_x < iter_width) {
            {
                float _tmp60 = 0.F;
                {
                    int _gid_x61 = gid_x + 0;
                    int _gid_y61 = gid_y + 2 * (int)blockDim.y + -2;
                    if (_gid_y61 < 0)
                        _gid_y61 = 0;
                    _tmp60 += _constmask_yY[0][0] * tex1Dfetch(_texinputY, (_gid_y61) * input_stride + _gid_x61);
                }
                {
                    int _gid_x62 = gid_x + 0;
                    int _gid_y62 = gid_y + 2 * (int)blockDim.y + -1;
                    if (_gid_y62 < 0)
                        _gid_y62 = 0;
                    _tmp60 += _constmask_yY[1][0] * tex1Dfetch(_texinputY, (_gid_y62) * input_stride + _gid_x62);
                }
                {
                    int _gid_x63 = gid_x + 0;
                    int _gid_y63 = gid_y + 2 * (int)blockDim.y + 0;
                    if (_gid_y63 < 0)
                        _gid_y63 = 0;
                    _tmp60 += _constmask_yY[2][0] * tex1Dfetch(_texinputY, (_gid_y63) * input_stride + _gid_x63);
                }
                {
                    int _gid_x64 = gid_x + 0;
                    int _gid_y64 = gid_y + 2 * (int)blockDim.y + 1;
                    if (_gid_y64 < 0)
                        _gid_y64 = 0;
                    _tmp60 += _constmask_yY[3][0] * tex1Dfetch(_texinputY, (_gid_y64) * input_stride + _gid_x64);
                }
                {
                    int _gid_x65 = gid_x + 0;
                    int _gid_y65 = gid_y + 2 * (int)blockDim.y + 2;
                    if (_gid_y65 < 0)
                        _gid_y65 = 0;
                    _tmp60 += _constmask_yY[4][0] * tex1Dfetch(_texinputY, (_gid_y65) * input_stride + _gid_x65);
                }
                iter[(gid_y + 2 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp60 + 0.5F);
            }
        }
        if (gid_x < iter_width) {
            {
                float _tmp66 = 0.F;
                {
                    int _gid_x67 = gid_x + 0;
                    int _gid_y67 = gid_y + 3 * (int)blockDim.y + -2;
                    if (_gid_y67 < 0)
                        _gid_y67 = 0;
                    _tmp66 += _constmask_yY[0][0] * tex1Dfetch(_texinputY, (_gid_y67) * input_stride + _gid_x67);
                }
                {
                    int _gid_x68 = gid_x + 0;
                    int _gid_y68 = gid_y + 3 * (int)blockDim.y + -1;
                    if (_gid_y68 < 0)
                        _gid_y68 = 0;
                    _tmp66 += _constmask_yY[1][0] * tex1Dfetch(_texinputY, (_gid_y68) * input_stride + _gid_x68);
                }
                {
                    int _gid_x69 = gid_x + 0;
                    int _gid_y69 = gid_y + 3 * (int)blockDim.y + 0;
                    if (_gid_y69 < 0)
                        _gid_y69 = 0;
                    _tmp66 += _constmask_yY[2][0] * tex1Dfetch(_texinputY, (_gid_y69) * input_stride + _gid_x69);
                }
                {
                    int _gid_x70 = gid_x + 0;
                    int _gid_y70 = gid_y + 3 * (int)blockDim.y + 1;
                    if (_gid_y70 < 0)
                        _gid_y70 = 0;
                    _tmp66 += _constmask_yY[3][0] * tex1Dfetch(_texinputY, (_gid_y70) * input_stride + _gid_x70);
                }
                {
                    int _gid_x71 = gid_x + 0;
                    int _gid_y71 = gid_y + 3 * (int)blockDim.y + 2;
                    if (_gid_y71 < 0)
                        _gid_y71 = 0;
                    _tmp66 += _constmask_yY[4][0] * tex1Dfetch(_texinputY, (_gid_y71) * input_stride + _gid_x71);
                }
                iter[(gid_y + 3 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp66 + 0.5F);
            }
        }
        if (gid_x < iter_width) {
            {
                float _tmp72 = 0.F;
                {
                    int _gid_x73 = gid_x + 0;
                    int _gid_y73 = gid_y + 4 * (int)blockDim.y + -2;
                    if (_gid_y73 < 0)
                        _gid_y73 = 0;
                    _tmp72 += _constmask_yY[0][0] * tex1Dfetch(_texinputY, (_gid_y73) * input_stride + _gid_x73);
                }
                {
                    int _gid_x74 = gid_x + 0;
                    int _gid_y74 = gid_y + 4 * (int)blockDim.y + -1;
                    if (_gid_y74 < 0)
                        _gid_y74 = 0;
                    _tmp72 += _constmask_yY[1][0] * tex1Dfetch(_texinputY, (_gid_y74) * input_stride + _gid_x74);
                }
                {
                    int _gid_x75 = gid_x + 0;
                    int _gid_y75 = gid_y + 4 * (int)blockDim.y + 0;
                    if (_gid_y75 < 0)
                        _gid_y75 = 0;
                    _tmp72 += _constmask_yY[2][0] * tex1Dfetch(_texinputY, (_gid_y75) * input_stride + _gid_x75);
                }
                {
                    int _gid_x76 = gid_x + 0;
                    int _gid_y76 = gid_y + 4 * (int)blockDim.y + 1;
                    if (_gid_y76 < 0)
                        _gid_y76 = 0;
                    _tmp72 += _constmask_yY[3][0] * tex1Dfetch(_texinputY, (_gid_y76) * input_stride + _gid_x76);
                }
                {
                    int _gid_x77 = gid_x + 0;
                    int _gid_y77 = gid_y + 4 * (int)blockDim.y + 2;
                    if (_gid_y77 < 0)
                        _gid_y77 = 0;
                    _tmp72 += _constmask_yY[4][0] * tex1Dfetch(_texinputY, (_gid_y77) * input_stride + _gid_x77);
                }
                iter[(gid_y + 4 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp72 + 0.5F);
            }
        }
        if (gid_x < iter_width) {
            {
                float _tmp78 = 0.F;
                {
                    int _gid_x79 = gid_x + 0;
                    int _gid_y79 = gid_y + 5 * (int)blockDim.y + -2;
                    if (_gid_y79 < 0)
                        _gid_y79 = 0;
                    _tmp78 += _constmask_yY[0][0] * tex1Dfetch(_texinputY, (_gid_y79) * input_stride + _gid_x79);
                }
                {
                    int _gid_x80 = gid_x + 0;
                    int _gid_y80 = gid_y + 5 * (int)blockDim.y + -1;
                    if (_gid_y80 < 0)
                        _gid_y80 = 0;
                    _tmp78 += _constmask_yY[1][0] * tex1Dfetch(_texinputY, (_gid_y80) * input_stride + _gid_x80);
                }
                {
                    int _gid_x81 = gid_x + 0;
                    int _gid_y81 = gid_y + 5 * (int)blockDim.y + 0;
                    if (_gid_y81 < 0)
                        _gid_y81 = 0;
                    _tmp78 += _constmask_yY[2][0] * tex1Dfetch(_texinputY, (_gid_y81) * input_stride + _gid_x81);
                }
                {
                    int _gid_x82 = gid_x + 0;
                    int _gid_y82 = gid_y + 5 * (int)blockDim.y + 1;
                    if (_gid_y82 < 0)
                        _gid_y82 = 0;
                    _tmp78 += _constmask_yY[3][0] * tex1Dfetch(_texinputY, (_gid_y82) * input_stride + _gid_x82);
                }
                {
                    int _gid_x83 = gid_x + 0;
                    int _gid_y83 = gid_y + 5 * (int)blockDim.y + 2;
                    if (_gid_y83 < 0)
                        _gid_y83 = 0;
                    _tmp78 += _constmask_yY[4][0] * tex1Dfetch(_texinputY, (_gid_y83) * input_stride + _gid_x83);
                }
                iter[(gid_y + 5 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp78 + 0.5F);
            }
        }
        if (gid_x < iter_width) {
            {
                float _tmp84 = 0.F;
                {
                    int _gid_x85 = gid_x + 0;
                    int _gid_y85 = gid_y + 6 * (int)blockDim.y + -2;
                    if (_gid_y85 < 0)
                        _gid_y85 = 0;
                    _tmp84 += _constmask_yY[0][0] * tex1Dfetch(_texinputY, (_gid_y85) * input_stride + _gid_x85);
                }
                {
                    int _gid_x86 = gid_x + 0;
                    int _gid_y86 = gid_y + 6 * (int)blockDim.y + -1;
                    if (_gid_y86 < 0)
                        _gid_y86 = 0;
                    _tmp84 += _constmask_yY[1][0] * tex1Dfetch(_texinputY, (_gid_y86) * input_stride + _gid_x86);
                }
                {
                    int _gid_x87 = gid_x + 0;
                    int _gid_y87 = gid_y + 6 * (int)blockDim.y + 0;
                    if (_gid_y87 < 0)
                        _gid_y87 = 0;
                    _tmp84 += _constmask_yY[2][0] * tex1Dfetch(_texinputY, (_gid_y87) * input_stride + _gid_x87);
                }
                {
                    int _gid_x88 = gid_x + 0;
                    int _gid_y88 = gid_y + 6 * (int)blockDim.y + 1;
                    if (_gid_y88 < 0)
                        _gid_y88 = 0;
                    _tmp84 += _constmask_yY[3][0] * tex1Dfetch(_texinputY, (_gid_y88) * input_stride + _gid_x88);
                }
                {
                    int _gid_x89 = gid_x + 0;
                    int _gid_y89 = gid_y + 6 * (int)blockDim.y + 2;
                    if (_gid_y89 < 0)
                        _gid_y89 = 0;
                    _tmp84 += _constmask_yY[4][0] * tex1Dfetch(_texinputY, (_gid_y89) * input_stride + _gid_x89);
                }
                iter[(gid_y + 6 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp84 + 0.5F);
            }
        }
        if (gid_x < iter_width) {
            {
                float _tmp90 = 0.F;
                {
                    int _gid_x91 = gid_x + 0;
                    int _gid_y91 = gid_y + 7 * (int)blockDim.y + -2;
                    if (_gid_y91 < 0)
                        _gid_y91 = 0;
                    _tmp90 += _constmask_yY[0][0] * tex1Dfetch(_texinputY, (_gid_y91) * input_stride + _gid_x91);
                }
                {
                    int _gid_x92 = gid_x + 0;
                    int _gid_y92 = gid_y + 7 * (int)blockDim.y + -1;
                    if (_gid_y92 < 0)
                        _gid_y92 = 0;
                    _tmp90 += _constmask_yY[1][0] * tex1Dfetch(_texinputY, (_gid_y92) * input_stride + _gid_x92);
                }
                {
                    int _gid_x93 = gid_x + 0;
                    int _gid_y93 = gid_y + 7 * (int)blockDim.y + 0;
                    if (_gid_y93 < 0)
                        _gid_y93 = 0;
                    _tmp90 += _constmask_yY[2][0] * tex1Dfetch(_texinputY, (_gid_y93) * input_stride + _gid_x93);
                }
                {
                    int _gid_x94 = gid_x + 0;
                    int _gid_y94 = gid_y + 7 * (int)blockDim.y + 1;
                    if (_gid_y94 < 0)
                        _gid_y94 = 0;
                    _tmp90 += _constmask_yY[3][0] * tex1Dfetch(_texinputY, (_gid_y94) * input_stride + _gid_x94);
                }
                {
                    int _gid_x95 = gid_x + 0;
                    int _gid_y95 = gid_y + 7 * (int)blockDim.y + 2;
                    if (_gid_y95 < 0)
                        _gid_y95 = 0;
                    _tmp90 += _constmask_yY[4][0] * tex1Dfetch(_texinputY, (_gid_y95) * input_stride + _gid_x95);
                }
                iter[(gid_y + 7 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp90 + 0.5F);
            }
        }
    }
    goto BH_EXIT;
  BH_B:
    {
        if (gid_x < iter_width) {
            if (gid_y < iter_height) {
                float _tmp96 = 0.F;
                {
                    int _gid_x97 = gid_x + 0;
                    int _gid_y97 = gid_y + -2;
                    if (_gid_y97 >= input_height)
                        _gid_y97 = input_height - 1;
                    _tmp96 += _constmask_yY[0][0] * tex1Dfetch(_texinputY, (_gid_y97) * input_stride + _gid_x97);
                }
                {
                    int _gid_x98 = gid_x + 0;
                    int _gid_y98 = gid_y + -1;
                    if (_gid_y98 >= input_height)
                        _gid_y98 = input_height - 1;
                    _tmp96 += _constmask_yY[1][0] * tex1Dfetch(_texinputY, (_gid_y98) * input_stride + _gid_x98);
                }
                {
                    int _gid_x99 = gid_x + 0;
                    int _gid_y99 = gid_y + 0;
                    if (_gid_y99 >= input_height)
                        _gid_y99 = input_height - 1;
                    _tmp96 += _constmask_yY[2][0] * tex1Dfetch(_texinputY, (_gid_y99) * input_stride + _gid_x99);
                }
                {
                    int _gid_x100 = gid_x + 0;
                    int _gid_y100 = gid_y + 1;
                    if (_gid_y100 >= input_height)
                        _gid_y100 = input_height - 1;
                    _tmp96 += _constmask_yY[3][0] * tex1Dfetch(_texinputY, (_gid_y100) * input_stride + _gid_x100);
                }
                {
                    int _gid_x101 = gid_x + 0;
                    int _gid_y101 = gid_y + 2;
                    if (_gid_y101 >= input_height)
                        _gid_y101 = input_height - 1;
                    _tmp96 += _constmask_yY[4][0] * tex1Dfetch(_texinputY, (_gid_y101) * input_stride + _gid_x101);
                }
                iter[(gid_y) * iter_stride + gid_x] = (uchar)(_tmp96 + 0.5F);
            }
        }
        if (gid_x < iter_width) {
            if (gid_y + 1 * (int)blockDim.y < iter_height) {
                float _tmp102 = 0.F;
                {
                    int _gid_x103 = gid_x + 0;
                    int _gid_y103 = gid_y + 1 * (int)blockDim.y + -2;
                    if (_gid_y103 >= input_height)
                        _gid_y103 = input_height - 1;
                    _tmp102 += _constmask_yY[0][0] * tex1Dfetch(_texinputY, (_gid_y103) * input_stride + _gid_x103);
                }
                {
                    int _gid_x104 = gid_x + 0;
                    int _gid_y104 = gid_y + 1 * (int)blockDim.y + -1;
                    if (_gid_y104 >= input_height)
                        _gid_y104 = input_height - 1;
                    _tmp102 += _constmask_yY[1][0] * tex1Dfetch(_texinputY, (_gid_y104) * input_stride + _gid_x104);
                }
                {
                    int _gid_x105 = gid_x + 0;
                    int _gid_y105 = gid_y + 1 * (int)blockDim.y + 0;
                    if (_gid_y105 >= input_height)
                        _gid_y105 = input_height - 1;
                    _tmp102 += _constmask_yY[2][0] * tex1Dfetch(_texinputY, (_gid_y105) * input_stride + _gid_x105);
                }
                {
                    int _gid_x106 = gid_x + 0;
                    int _gid_y106 = gid_y + 1 * (int)blockDim.y + 1;
                    if (_gid_y106 >= input_height)
                        _gid_y106 = input_height - 1;
                    _tmp102 += _constmask_yY[3][0] * tex1Dfetch(_texinputY, (_gid_y106) * input_stride + _gid_x106);
                }
                {
                    int _gid_x107 = gid_x + 0;
                    int _gid_y107 = gid_y + 1 * (int)blockDim.y + 2;
                    if (_gid_y107 >= input_height)
                        _gid_y107 = input_height - 1;
                    _tmp102 += _constmask_yY[4][0] * tex1Dfetch(_texinputY, (_gid_y107) * input_stride + _gid_x107);
                }
                iter[(gid_y + 1 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp102 + 0.5F);
            }
        }
        if (gid_x < iter_width) {
            if (gid_y + 2 * (int)blockDim.y < iter_height) {
                float _tmp108 = 0.F;
                {
                    int _gid_x109 = gid_x + 0;
                    int _gid_y109 = gid_y + 2 * (int)blockDim.y + -2;
                    if (_gid_y109 >= input_height)
                        _gid_y109 = input_height - 1;
                    _tmp108 += _constmask_yY[0][0] * tex1Dfetch(_texinputY, (_gid_y109) * input_stride + _gid_x109);
                }
                {
                    int _gid_x110 = gid_x + 0;
                    int _gid_y110 = gid_y + 2 * (int)blockDim.y + -1;
                    if (_gid_y110 >= input_height)
                        _gid_y110 = input_height - 1;
                    _tmp108 += _constmask_yY[1][0] * tex1Dfetch(_texinputY, (_gid_y110) * input_stride + _gid_x110);
                }
                {
                    int _gid_x111 = gid_x + 0;
                    int _gid_y111 = gid_y + 2 * (int)blockDim.y + 0;
                    if (_gid_y111 >= input_height)
                        _gid_y111 = input_height - 1;
                    _tmp108 += _constmask_yY[2][0] * tex1Dfetch(_texinputY, (_gid_y111) * input_stride + _gid_x111);
                }
                {
                    int _gid_x112 = gid_x + 0;
                    int _gid_y112 = gid_y + 2 * (int)blockDim.y + 1;
                    if (_gid_y112 >= input_height)
                        _gid_y112 = input_height - 1;
                    _tmp108 += _constmask_yY[3][0] * tex1Dfetch(_texinputY, (_gid_y112) * input_stride + _gid_x112);
                }
                {
                    int _gid_x113 = gid_x + 0;
                    int _gid_y113 = gid_y + 2 * (int)blockDim.y + 2;
                    if (_gid_y113 >= input_height)
                        _gid_y113 = input_height - 1;
                    _tmp108 += _constmask_yY[4][0] * tex1Dfetch(_texinputY, (_gid_y113) * input_stride + _gid_x113);
                }
                iter[(gid_y + 2 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp108 + 0.5F);
            }
        }
        if (gid_x < iter_width) {
            if (gid_y + 3 * (int)blockDim.y < iter_height) {
                float _tmp114 = 0.F;
                {
                    int _gid_x115 = gid_x + 0;
                    int _gid_y115 = gid_y + 3 * (int)blockDim.y + -2;
                    if (_gid_y115 >= input_height)
                        _gid_y115 = input_height - 1;
                    _tmp114 += _constmask_yY[0][0] * tex1Dfetch(_texinputY, (_gid_y115) * input_stride + _gid_x115);
                }
                {
                    int _gid_x116 = gid_x + 0;
                    int _gid_y116 = gid_y + 3 * (int)blockDim.y + -1;
                    if (_gid_y116 >= input_height)
                        _gid_y116 = input_height - 1;
                    _tmp114 += _constmask_yY[1][0] * tex1Dfetch(_texinputY, (_gid_y116) * input_stride + _gid_x116);
                }
                {
                    int _gid_x117 = gid_x + 0;
                    int _gid_y117 = gid_y + 3 * (int)blockDim.y + 0;
                    if (_gid_y117 >= input_height)
                        _gid_y117 = input_height - 1;
                    _tmp114 += _constmask_yY[2][0] * tex1Dfetch(_texinputY, (_gid_y117) * input_stride + _gid_x117);
                }
                {
                    int _gid_x118 = gid_x + 0;
                    int _gid_y118 = gid_y + 3 * (int)blockDim.y + 1;
                    if (_gid_y118 >= input_height)
                        _gid_y118 = input_height - 1;
                    _tmp114 += _constmask_yY[3][0] * tex1Dfetch(_texinputY, (_gid_y118) * input_stride + _gid_x118);
                }
                {
                    int _gid_x119 = gid_x + 0;
                    int _gid_y119 = gid_y + 3 * (int)blockDim.y + 2;
                    if (_gid_y119 >= input_height)
                        _gid_y119 = input_height - 1;
                    _tmp114 += _constmask_yY[4][0] * tex1Dfetch(_texinputY, (_gid_y119) * input_stride + _gid_x119);
                }
                iter[(gid_y + 3 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp114 + 0.5F);
            }
        }
        if (gid_x < iter_width) {
            if (gid_y + 4 * (int)blockDim.y < iter_height) {
                float _tmp120 = 0.F;
                {
                    int _gid_x121 = gid_x + 0;
                    int _gid_y121 = gid_y + 4 * (int)blockDim.y + -2;
                    if (_gid_y121 >= input_height)
                        _gid_y121 = input_height - 1;
                    _tmp120 += _constmask_yY[0][0] * tex1Dfetch(_texinputY, (_gid_y121) * input_stride + _gid_x121);
                }
                {
                    int _gid_x122 = gid_x + 0;
                    int _gid_y122 = gid_y + 4 * (int)blockDim.y + -1;
                    if (_gid_y122 >= input_height)
                        _gid_y122 = input_height - 1;
                    _tmp120 += _constmask_yY[1][0] * tex1Dfetch(_texinputY, (_gid_y122) * input_stride + _gid_x122);
                }
                {
                    int _gid_x123 = gid_x + 0;
                    int _gid_y123 = gid_y + 4 * (int)blockDim.y + 0;
                    if (_gid_y123 >= input_height)
                        _gid_y123 = input_height - 1;
                    _tmp120 += _constmask_yY[2][0] * tex1Dfetch(_texinputY, (_gid_y123) * input_stride + _gid_x123);
                }
                {
                    int _gid_x124 = gid_x + 0;
                    int _gid_y124 = gid_y + 4 * (int)blockDim.y + 1;
                    if (_gid_y124 >= input_height)
                        _gid_y124 = input_height - 1;
                    _tmp120 += _constmask_yY[3][0] * tex1Dfetch(_texinputY, (_gid_y124) * input_stride + _gid_x124);
                }
                {
                    int _gid_x125 = gid_x + 0;
                    int _gid_y125 = gid_y + 4 * (int)blockDim.y + 2;
                    if (_gid_y125 >= input_height)
                        _gid_y125 = input_height - 1;
                    _tmp120 += _constmask_yY[4][0] * tex1Dfetch(_texinputY, (_gid_y125) * input_stride + _gid_x125);
                }
                iter[(gid_y + 4 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp120 + 0.5F);
            }
        }
        if (gid_x < iter_width) {
            if (gid_y + 5 * (int)blockDim.y < iter_height) {
                float _tmp126 = 0.F;
                {
                    int _gid_x127 = gid_x + 0;
                    int _gid_y127 = gid_y + 5 * (int)blockDim.y + -2;
                    if (_gid_y127 >= input_height)
                        _gid_y127 = input_height - 1;
                    _tmp126 += _constmask_yY[0][0] * tex1Dfetch(_texinputY, (_gid_y127) * input_stride + _gid_x127);
                }
                {
                    int _gid_x128 = gid_x + 0;
                    int _gid_y128 = gid_y + 5 * (int)blockDim.y + -1;
                    if (_gid_y128 >= input_height)
                        _gid_y128 = input_height - 1;
                    _tmp126 += _constmask_yY[1][0] * tex1Dfetch(_texinputY, (_gid_y128) * input_stride + _gid_x128);
                }
                {
                    int _gid_x129 = gid_x + 0;
                    int _gid_y129 = gid_y + 5 * (int)blockDim.y + 0;
                    if (_gid_y129 >= input_height)
                        _gid_y129 = input_height - 1;
                    _tmp126 += _constmask_yY[2][0] * tex1Dfetch(_texinputY, (_gid_y129) * input_stride + _gid_x129);
                }
                {
                    int _gid_x130 = gid_x + 0;
                    int _gid_y130 = gid_y + 5 * (int)blockDim.y + 1;
                    if (_gid_y130 >= input_height)
                        _gid_y130 = input_height - 1;
                    _tmp126 += _constmask_yY[3][0] * tex1Dfetch(_texinputY, (_gid_y130) * input_stride + _gid_x130);
                }
                {
                    int _gid_x131 = gid_x + 0;
                    int _gid_y131 = gid_y + 5 * (int)blockDim.y + 2;
                    if (_gid_y131 >= input_height)
                        _gid_y131 = input_height - 1;
                    _tmp126 += _constmask_yY[4][0] * tex1Dfetch(_texinputY, (_gid_y131) * input_stride + _gid_x131);
                }
                iter[(gid_y + 5 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp126 + 0.5F);
            }
        }
        if (gid_x < iter_width) {
            if (gid_y + 6 * (int)blockDim.y < iter_height) {
                float _tmp132 = 0.F;
                {
                    int _gid_x133 = gid_x + 0;
                    int _gid_y133 = gid_y + 6 * (int)blockDim.y + -2;
                    if (_gid_y133 >= input_height)
                        _gid_y133 = input_height - 1;
                    _tmp132 += _constmask_yY[0][0] * tex1Dfetch(_texinputY, (_gid_y133) * input_stride + _gid_x133);
                }
                {
                    int _gid_x134 = gid_x + 0;
                    int _gid_y134 = gid_y + 6 * (int)blockDim.y + -1;
                    if (_gid_y134 >= input_height)
                        _gid_y134 = input_height - 1;
                    _tmp132 += _constmask_yY[1][0] * tex1Dfetch(_texinputY, (_gid_y134) * input_stride + _gid_x134);
                }
                {
                    int _gid_x135 = gid_x + 0;
                    int _gid_y135 = gid_y + 6 * (int)blockDim.y + 0;
                    if (_gid_y135 >= input_height)
                        _gid_y135 = input_height - 1;
                    _tmp132 += _constmask_yY[2][0] * tex1Dfetch(_texinputY, (_gid_y135) * input_stride + _gid_x135);
                }
                {
                    int _gid_x136 = gid_x + 0;
                    int _gid_y136 = gid_y + 6 * (int)blockDim.y + 1;
                    if (_gid_y136 >= input_height)
                        _gid_y136 = input_height - 1;
                    _tmp132 += _constmask_yY[3][0] * tex1Dfetch(_texinputY, (_gid_y136) * input_stride + _gid_x136);
                }
                {
                    int _gid_x137 = gid_x + 0;
                    int _gid_y137 = gid_y + 6 * (int)blockDim.y + 2;
                    if (_gid_y137 >= input_height)
                        _gid_y137 = input_height - 1;
                    _tmp132 += _constmask_yY[4][0] * tex1Dfetch(_texinputY, (_gid_y137) * input_stride + _gid_x137);
                }
                iter[(gid_y + 6 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp132 + 0.5F);
            }
        }
        if (gid_x < iter_width) {
            if (gid_y + 7 * (int)blockDim.y < iter_height) {
                float _tmp138 = 0.F;
                {
                    int _gid_x139 = gid_x + 0;
                    int _gid_y139 = gid_y + 7 * (int)blockDim.y + -2;
                    if (_gid_y139 >= input_height)
                        _gid_y139 = input_height - 1;
                    _tmp138 += _constmask_yY[0][0] * tex1Dfetch(_texinputY, (_gid_y139) * input_stride + _gid_x139);
                }
                {
                    int _gid_x140 = gid_x + 0;
                    int _gid_y140 = gid_y + 7 * (int)blockDim.y + -1;
                    if (_gid_y140 >= input_height)
                        _gid_y140 = input_height - 1;
                    _tmp138 += _constmask_yY[1][0] * tex1Dfetch(_texinputY, (_gid_y140) * input_stride + _gid_x140);
                }
                {
                    int _gid_x141 = gid_x + 0;
                    int _gid_y141 = gid_y + 7 * (int)blockDim.y + 0;
                    if (_gid_y141 >= input_height)
                        _gid_y141 = input_height - 1;
                    _tmp138 += _constmask_yY[2][0] * tex1Dfetch(_texinputY, (_gid_y141) * input_stride + _gid_x141);
                }
                {
                    int _gid_x142 = gid_x + 0;
                    int _gid_y142 = gid_y + 7 * (int)blockDim.y + 1;
                    if (_gid_y142 >= input_height)
                        _gid_y142 = input_height - 1;
                    _tmp138 += _constmask_yY[3][0] * tex1Dfetch(_texinputY, (_gid_y142) * input_stride + _gid_x142);
                }
                {
                    int _gid_x143 = gid_x + 0;
                    int _gid_y143 = gid_y + 7 * (int)blockDim.y + 2;
                    if (_gid_y143 >= input_height)
                        _gid_y143 = input_height - 1;
                    _tmp138 += _constmask_yY[4][0] * tex1Dfetch(_texinputY, (_gid_y143) * input_stride + _gid_x143);
                }
                iter[(gid_y + 7 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp138 + 0.5F);
            }
        }
    }
    goto BH_EXIT;
  BH_R:
    {
        if (gid_x < iter_width) {
            {
                float _tmp144 = 0.F;
                {
                    int _gid_x145 = gid_x + 0;
                    int _gid_y145 = gid_y + -2;
                    if (_gid_x145 >= input_width)
                        _gid_x145 = input_width - 1;
                    _tmp144 += _constmask_yY[0][0] * tex1Dfetch(_texinputY, (_gid_y145) * input_stride + _gid_x145);
                }
                {
                    int _gid_x146 = gid_x + 0;
                    int _gid_y146 = gid_y + -1;
                    if (_gid_x146 >= input_width)
                        _gid_x146 = input_width - 1;
                    _tmp144 += _constmask_yY[1][0] * tex1Dfetch(_texinputY, (_gid_y146) * input_stride + _gid_x146);
                }
                {
                    int _gid_x147 = gid_x + 0;
                    int _gid_y147 = gid_y + 0;
                    if (_gid_x147 >= input_width)
                        _gid_x147 = input_width - 1;
                    _tmp144 += _constmask_yY[2][0] * tex1Dfetch(_texinputY, (_gid_y147) * input_stride + _gid_x147);
                }
                {
                    int _gid_x148 = gid_x + 0;
                    int _gid_y148 = gid_y + 1;
                    if (_gid_x148 >= input_width)
                        _gid_x148 = input_width - 1;
                    _tmp144 += _constmask_yY[3][0] * tex1Dfetch(_texinputY, (_gid_y148) * input_stride + _gid_x148);
                }
                {
                    int _gid_x149 = gid_x + 0;
                    int _gid_y149 = gid_y + 2;
                    if (_gid_x149 >= input_width)
                        _gid_x149 = input_width - 1;
                    _tmp144 += _constmask_yY[4][0] * tex1Dfetch(_texinputY, (_gid_y149) * input_stride + _gid_x149);
                }
                iter[(gid_y) * iter_stride + gid_x] = (uchar)(_tmp144 + 0.5F);
            }
        }
        if (gid_x < iter_width) {
            {
                float _tmp150 = 0.F;
                {
                    int _gid_x151 = gid_x + 0;
                    int _gid_y151 = gid_y + 1 * (int)blockDim.y + -2;
                    if (_gid_x151 >= input_width)
                        _gid_x151 = input_width - 1;
                    _tmp150 += _constmask_yY[0][0] * tex1Dfetch(_texinputY, (_gid_y151) * input_stride + _gid_x151);
                }
                {
                    int _gid_x152 = gid_x + 0;
                    int _gid_y152 = gid_y + 1 * (int)blockDim.y + -1;
                    if (_gid_x152 >= input_width)
                        _gid_x152 = input_width - 1;
                    _tmp150 += _constmask_yY[1][0] * tex1Dfetch(_texinputY, (_gid_y152) * input_stride + _gid_x152);
                }
                {
                    int _gid_x153 = gid_x + 0;
                    int _gid_y153 = gid_y + 1 * (int)blockDim.y + 0;
                    if (_gid_x153 >= input_width)
                        _gid_x153 = input_width - 1;
                    _tmp150 += _constmask_yY[2][0] * tex1Dfetch(_texinputY, (_gid_y153) * input_stride + _gid_x153);
                }
                {
                    int _gid_x154 = gid_x + 0;
                    int _gid_y154 = gid_y + 1 * (int)blockDim.y + 1;
                    if (_gid_x154 >= input_width)
                        _gid_x154 = input_width - 1;
                    _tmp150 += _constmask_yY[3][0] * tex1Dfetch(_texinputY, (_gid_y154) * input_stride + _gid_x154);
                }
                {
                    int _gid_x155 = gid_x + 0;
                    int _gid_y155 = gid_y + 1 * (int)blockDim.y + 2;
                    if (_gid_x155 >= input_width)
                        _gid_x155 = input_width - 1;
                    _tmp150 += _constmask_yY[4][0] * tex1Dfetch(_texinputY, (_gid_y155) * input_stride + _gid_x155);
                }
                iter[(gid_y + 1 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp150 + 0.5F);
            }
        }
        if (gid_x < iter_width) {
            {
                float _tmp156 = 0.F;
                {
                    int _gid_x157 = gid_x + 0;
                    int _gid_y157 = gid_y + 2 * (int)blockDim.y + -2;
                    if (_gid_x157 >= input_width)
                        _gid_x157 = input_width - 1;
                    _tmp156 += _constmask_yY[0][0] * tex1Dfetch(_texinputY, (_gid_y157) * input_stride + _gid_x157);
                }
                {
                    int _gid_x158 = gid_x + 0;
                    int _gid_y158 = gid_y + 2 * (int)blockDim.y + -1;
                    if (_gid_x158 >= input_width)
                        _gid_x158 = input_width - 1;
                    _tmp156 += _constmask_yY[1][0] * tex1Dfetch(_texinputY, (_gid_y158) * input_stride + _gid_x158);
                }
                {
                    int _gid_x159 = gid_x + 0;
                    int _gid_y159 = gid_y + 2 * (int)blockDim.y + 0;
                    if (_gid_x159 >= input_width)
                        _gid_x159 = input_width - 1;
                    _tmp156 += _constmask_yY[2][0] * tex1Dfetch(_texinputY, (_gid_y159) * input_stride + _gid_x159);
                }
                {
                    int _gid_x160 = gid_x + 0;
                    int _gid_y160 = gid_y + 2 * (int)blockDim.y + 1;
                    if (_gid_x160 >= input_width)
                        _gid_x160 = input_width - 1;
                    _tmp156 += _constmask_yY[3][0] * tex1Dfetch(_texinputY, (_gid_y160) * input_stride + _gid_x160);
                }
                {
                    int _gid_x161 = gid_x + 0;
                    int _gid_y161 = gid_y + 2 * (int)blockDim.y + 2;
                    if (_gid_x161 >= input_width)
                        _gid_x161 = input_width - 1;
                    _tmp156 += _constmask_yY[4][0] * tex1Dfetch(_texinputY, (_gid_y161) * input_stride + _gid_x161);
                }
                iter[(gid_y + 2 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp156 + 0.5F);
            }
        }
        if (gid_x < iter_width) {
            {
                float _tmp162 = 0.F;
                {
                    int _gid_x163 = gid_x + 0;
                    int _gid_y163 = gid_y + 3 * (int)blockDim.y + -2;
                    if (_gid_x163 >= input_width)
                        _gid_x163 = input_width - 1;
                    _tmp162 += _constmask_yY[0][0] * tex1Dfetch(_texinputY, (_gid_y163) * input_stride + _gid_x163);
                }
                {
                    int _gid_x164 = gid_x + 0;
                    int _gid_y164 = gid_y + 3 * (int)blockDim.y + -1;
                    if (_gid_x164 >= input_width)
                        _gid_x164 = input_width - 1;
                    _tmp162 += _constmask_yY[1][0] * tex1Dfetch(_texinputY, (_gid_y164) * input_stride + _gid_x164);
                }
                {
                    int _gid_x165 = gid_x + 0;
                    int _gid_y165 = gid_y + 3 * (int)blockDim.y + 0;
                    if (_gid_x165 >= input_width)
                        _gid_x165 = input_width - 1;
                    _tmp162 += _constmask_yY[2][0] * tex1Dfetch(_texinputY, (_gid_y165) * input_stride + _gid_x165);
                }
                {
                    int _gid_x166 = gid_x + 0;
                    int _gid_y166 = gid_y + 3 * (int)blockDim.y + 1;
                    if (_gid_x166 >= input_width)
                        _gid_x166 = input_width - 1;
                    _tmp162 += _constmask_yY[3][0] * tex1Dfetch(_texinputY, (_gid_y166) * input_stride + _gid_x166);
                }
                {
                    int _gid_x167 = gid_x + 0;
                    int _gid_y167 = gid_y + 3 * (int)blockDim.y + 2;
                    if (_gid_x167 >= input_width)
                        _gid_x167 = input_width - 1;
                    _tmp162 += _constmask_yY[4][0] * tex1Dfetch(_texinputY, (_gid_y167) * input_stride + _gid_x167);
                }
                iter[(gid_y + 3 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp162 + 0.5F);
            }
        }
        if (gid_x < iter_width) {
            {
                float _tmp168 = 0.F;
                {
                    int _gid_x169 = gid_x + 0;
                    int _gid_y169 = gid_y + 4 * (int)blockDim.y + -2;
                    if (_gid_x169 >= input_width)
                        _gid_x169 = input_width - 1;
                    _tmp168 += _constmask_yY[0][0] * tex1Dfetch(_texinputY, (_gid_y169) * input_stride + _gid_x169);
                }
                {
                    int _gid_x170 = gid_x + 0;
                    int _gid_y170 = gid_y + 4 * (int)blockDim.y + -1;
                    if (_gid_x170 >= input_width)
                        _gid_x170 = input_width - 1;
                    _tmp168 += _constmask_yY[1][0] * tex1Dfetch(_texinputY, (_gid_y170) * input_stride + _gid_x170);
                }
                {
                    int _gid_x171 = gid_x + 0;
                    int _gid_y171 = gid_y + 4 * (int)blockDim.y + 0;
                    if (_gid_x171 >= input_width)
                        _gid_x171 = input_width - 1;
                    _tmp168 += _constmask_yY[2][0] * tex1Dfetch(_texinputY, (_gid_y171) * input_stride + _gid_x171);
                }
                {
                    int _gid_x172 = gid_x + 0;
                    int _gid_y172 = gid_y + 4 * (int)blockDim.y + 1;
                    if (_gid_x172 >= input_width)
                        _gid_x172 = input_width - 1;
                    _tmp168 += _constmask_yY[3][0] * tex1Dfetch(_texinputY, (_gid_y172) * input_stride + _gid_x172);
                }
                {
                    int _gid_x173 = gid_x + 0;
                    int _gid_y173 = gid_y + 4 * (int)blockDim.y + 2;
                    if (_gid_x173 >= input_width)
                        _gid_x173 = input_width - 1;
                    _tmp168 += _constmask_yY[4][0] * tex1Dfetch(_texinputY, (_gid_y173) * input_stride + _gid_x173);
                }
                iter[(gid_y + 4 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp168 + 0.5F);
            }
        }
        if (gid_x < iter_width) {
            {
                float _tmp174 = 0.F;
                {
                    int _gid_x175 = gid_x + 0;
                    int _gid_y175 = gid_y + 5 * (int)blockDim.y + -2;
                    if (_gid_x175 >= input_width)
                        _gid_x175 = input_width - 1;
                    _tmp174 += _constmask_yY[0][0] * tex1Dfetch(_texinputY, (_gid_y175) * input_stride + _gid_x175);
                }
                {
                    int _gid_x176 = gid_x + 0;
                    int _gid_y176 = gid_y + 5 * (int)blockDim.y + -1;
                    if (_gid_x176 >= input_width)
                        _gid_x176 = input_width - 1;
                    _tmp174 += _constmask_yY[1][0] * tex1Dfetch(_texinputY, (_gid_y176) * input_stride + _gid_x176);
                }
                {
                    int _gid_x177 = gid_x + 0;
                    int _gid_y177 = gid_y + 5 * (int)blockDim.y + 0;
                    if (_gid_x177 >= input_width)
                        _gid_x177 = input_width - 1;
                    _tmp174 += _constmask_yY[2][0] * tex1Dfetch(_texinputY, (_gid_y177) * input_stride + _gid_x177);
                }
                {
                    int _gid_x178 = gid_x + 0;
                    int _gid_y178 = gid_y + 5 * (int)blockDim.y + 1;
                    if (_gid_x178 >= input_width)
                        _gid_x178 = input_width - 1;
                    _tmp174 += _constmask_yY[3][0] * tex1Dfetch(_texinputY, (_gid_y178) * input_stride + _gid_x178);
                }
                {
                    int _gid_x179 = gid_x + 0;
                    int _gid_y179 = gid_y + 5 * (int)blockDim.y + 2;
                    if (_gid_x179 >= input_width)
                        _gid_x179 = input_width - 1;
                    _tmp174 += _constmask_yY[4][0] * tex1Dfetch(_texinputY, (_gid_y179) * input_stride + _gid_x179);
                }
                iter[(gid_y + 5 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp174 + 0.5F);
            }
        }
        if (gid_x < iter_width) {
            {
                float _tmp180 = 0.F;
                {
                    int _gid_x181 = gid_x + 0;
                    int _gid_y181 = gid_y + 6 * (int)blockDim.y + -2;
                    if (_gid_x181 >= input_width)
                        _gid_x181 = input_width - 1;
                    _tmp180 += _constmask_yY[0][0] * tex1Dfetch(_texinputY, (_gid_y181) * input_stride + _gid_x181);
                }
                {
                    int _gid_x182 = gid_x + 0;
                    int _gid_y182 = gid_y + 6 * (int)blockDim.y + -1;
                    if (_gid_x182 >= input_width)
                        _gid_x182 = input_width - 1;
                    _tmp180 += _constmask_yY[1][0] * tex1Dfetch(_texinputY, (_gid_y182) * input_stride + _gid_x182);
                }
                {
                    int _gid_x183 = gid_x + 0;
                    int _gid_y183 = gid_y + 6 * (int)blockDim.y + 0;
                    if (_gid_x183 >= input_width)
                        _gid_x183 = input_width - 1;
                    _tmp180 += _constmask_yY[2][0] * tex1Dfetch(_texinputY, (_gid_y183) * input_stride + _gid_x183);
                }
                {
                    int _gid_x184 = gid_x + 0;
                    int _gid_y184 = gid_y + 6 * (int)blockDim.y + 1;
                    if (_gid_x184 >= input_width)
                        _gid_x184 = input_width - 1;
                    _tmp180 += _constmask_yY[3][0] * tex1Dfetch(_texinputY, (_gid_y184) * input_stride + _gid_x184);
                }
                {
                    int _gid_x185 = gid_x + 0;
                    int _gid_y185 = gid_y + 6 * (int)blockDim.y + 2;
                    if (_gid_x185 >= input_width)
                        _gid_x185 = input_width - 1;
                    _tmp180 += _constmask_yY[4][0] * tex1Dfetch(_texinputY, (_gid_y185) * input_stride + _gid_x185);
                }
                iter[(gid_y + 6 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp180 + 0.5F);
            }
        }
        if (gid_x < iter_width) {
            {
                float _tmp186 = 0.F;
                {
                    int _gid_x187 = gid_x + 0;
                    int _gid_y187 = gid_y + 7 * (int)blockDim.y + -2;
                    if (_gid_x187 >= input_width)
                        _gid_x187 = input_width - 1;
                    _tmp186 += _constmask_yY[0][0] * tex1Dfetch(_texinputY, (_gid_y187) * input_stride + _gid_x187);
                }
                {
                    int _gid_x188 = gid_x + 0;
                    int _gid_y188 = gid_y + 7 * (int)blockDim.y + -1;
                    if (_gid_x188 >= input_width)
                        _gid_x188 = input_width - 1;
                    _tmp186 += _constmask_yY[1][0] * tex1Dfetch(_texinputY, (_gid_y188) * input_stride + _gid_x188);
                }
                {
                    int _gid_x189 = gid_x + 0;
                    int _gid_y189 = gid_y + 7 * (int)blockDim.y + 0;
                    if (_gid_x189 >= input_width)
                        _gid_x189 = input_width - 1;
                    _tmp186 += _constmask_yY[2][0] * tex1Dfetch(_texinputY, (_gid_y189) * input_stride + _gid_x189);
                }
                {
                    int _gid_x190 = gid_x + 0;
                    int _gid_y190 = gid_y + 7 * (int)blockDim.y + 1;
                    if (_gid_x190 >= input_width)
                        _gid_x190 = input_width - 1;
                    _tmp186 += _constmask_yY[3][0] * tex1Dfetch(_texinputY, (_gid_y190) * input_stride + _gid_x190);
                }
                {
                    int _gid_x191 = gid_x + 0;
                    int _gid_y191 = gid_y + 7 * (int)blockDim.y + 2;
                    if (_gid_x191 >= input_width)
                        _gid_x191 = input_width - 1;
                    _tmp186 += _constmask_yY[4][0] * tex1Dfetch(_texinputY, (_gid_y191) * input_stride + _gid_x191);
                }
                iter[(gid_y + 7 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp186 + 0.5F);
            }
        }
    }
    goto BH_EXIT;
  BH_NO:
    {
        {
            float _tmp192 = 0.F;
            {
                _tmp192 += _constmask_yY[0][0] * tex1Dfetch(_texinputY, (gid_y + -2) * input_stride + gid_x + 0);
            }
            {
                _tmp192 += _constmask_yY[1][0] * tex1Dfetch(_texinputY, (gid_y + -1) * input_stride + gid_x + 0);
            }
            {
                _tmp192 += _constmask_yY[2][0] * tex1Dfetch(_texinputY, (gid_y + 0) * input_stride + gid_x + 0);
            }
            {
                _tmp192 += _constmask_yY[3][0] * tex1Dfetch(_texinputY, (gid_y + 1) * input_stride + gid_x + 0);
            }
            {
                _tmp192 += _constmask_yY[4][0] * tex1Dfetch(_texinputY, (gid_y + 2) * input_stride + gid_x + 0);
            }
            iter[(gid_y) * iter_stride + gid_x] = (uchar)(_tmp192 + 0.5F);
        }
        {
            float _tmp193 = 0.F;
            {
                _tmp193 += _constmask_yY[0][0] * tex1Dfetch(_texinputY, (gid_y + 1 * (int)blockDim.y + -2) * input_stride + gid_x + 0);
            }
            {
                _tmp193 += _constmask_yY[1][0] * tex1Dfetch(_texinputY, (gid_y + 1 * (int)blockDim.y + -1) * input_stride + gid_x + 0);
            }
            {
                _tmp193 += _constmask_yY[2][0] * tex1Dfetch(_texinputY, (gid_y + 1 * (int)blockDim.y + 0) * input_stride + gid_x + 0);
            }
            {
                _tmp193 += _constmask_yY[3][0] * tex1Dfetch(_texinputY, (gid_y + 1 * (int)blockDim.y + 1) * input_stride + gid_x + 0);
            }
            {
                _tmp193 += _constmask_yY[4][0] * tex1Dfetch(_texinputY, (gid_y + 1 * (int)blockDim.y + 2) * input_stride + gid_x + 0);
            }
            iter[(gid_y + 1 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp193 + 0.5F);
        }
        {
            float _tmp194 = 0.F;
            {
                _tmp194 += _constmask_yY[0][0] * tex1Dfetch(_texinputY, (gid_y + 2 * (int)blockDim.y + -2) * input_stride + gid_x + 0);
            }
            {
                _tmp194 += _constmask_yY[1][0] * tex1Dfetch(_texinputY, (gid_y + 2 * (int)blockDim.y + -1) * input_stride + gid_x + 0);
            }
            {
                _tmp194 += _constmask_yY[2][0] * tex1Dfetch(_texinputY, (gid_y + 2 * (int)blockDim.y + 0) * input_stride + gid_x + 0);
            }
            {
                _tmp194 += _constmask_yY[3][0] * tex1Dfetch(_texinputY, (gid_y + 2 * (int)blockDim.y + 1) * input_stride + gid_x + 0);
            }
            {
                _tmp194 += _constmask_yY[4][0] * tex1Dfetch(_texinputY, (gid_y + 2 * (int)blockDim.y + 2) * input_stride + gid_x + 0);
            }
            iter[(gid_y + 2 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp194 + 0.5F);
        }
        {
            float _tmp195 = 0.F;
            {
                _tmp195 += _constmask_yY[0][0] * tex1Dfetch(_texinputY, (gid_y + 3 * (int)blockDim.y + -2) * input_stride + gid_x + 0);
            }
            {
                _tmp195 += _constmask_yY[1][0] * tex1Dfetch(_texinputY, (gid_y + 3 * (int)blockDim.y + -1) * input_stride + gid_x + 0);
            }
            {
                _tmp195 += _constmask_yY[2][0] * tex1Dfetch(_texinputY, (gid_y + 3 * (int)blockDim.y + 0) * input_stride + gid_x + 0);
            }
            {
                _tmp195 += _constmask_yY[3][0] * tex1Dfetch(_texinputY, (gid_y + 3 * (int)blockDim.y + 1) * input_stride + gid_x + 0);
            }
            {
                _tmp195 += _constmask_yY[4][0] * tex1Dfetch(_texinputY, (gid_y + 3 * (int)blockDim.y + 2) * input_stride + gid_x + 0);
            }
            iter[(gid_y + 3 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp195 + 0.5F);
        }
        {
            float _tmp196 = 0.F;
            {
                _tmp196 += _constmask_yY[0][0] * tex1Dfetch(_texinputY, (gid_y + 4 * (int)blockDim.y + -2) * input_stride + gid_x + 0);
            }
            {
                _tmp196 += _constmask_yY[1][0] * tex1Dfetch(_texinputY, (gid_y + 4 * (int)blockDim.y + -1) * input_stride + gid_x + 0);
            }
            {
                _tmp196 += _constmask_yY[2][0] * tex1Dfetch(_texinputY, (gid_y + 4 * (int)blockDim.y + 0) * input_stride + gid_x + 0);
            }
            {
                _tmp196 += _constmask_yY[3][0] * tex1Dfetch(_texinputY, (gid_y + 4 * (int)blockDim.y + 1) * input_stride + gid_x + 0);
            }
            {
                _tmp196 += _constmask_yY[4][0] * tex1Dfetch(_texinputY, (gid_y + 4 * (int)blockDim.y + 2) * input_stride + gid_x + 0);
            }
            iter[(gid_y + 4 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp196 + 0.5F);
        }
        {
            float _tmp197 = 0.F;
            {
                _tmp197 += _constmask_yY[0][0] * tex1Dfetch(_texinputY, (gid_y + 5 * (int)blockDim.y + -2) * input_stride + gid_x + 0);
            }
            {
                _tmp197 += _constmask_yY[1][0] * tex1Dfetch(_texinputY, (gid_y + 5 * (int)blockDim.y + -1) * input_stride + gid_x + 0);
            }
            {
                _tmp197 += _constmask_yY[2][0] * tex1Dfetch(_texinputY, (gid_y + 5 * (int)blockDim.y + 0) * input_stride + gid_x + 0);
            }
            {
                _tmp197 += _constmask_yY[3][0] * tex1Dfetch(_texinputY, (gid_y + 5 * (int)blockDim.y + 1) * input_stride + gid_x + 0);
            }
            {
                _tmp197 += _constmask_yY[4][0] * tex1Dfetch(_texinputY, (gid_y + 5 * (int)blockDim.y + 2) * input_stride + gid_x + 0);
            }
            iter[(gid_y + 5 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp197 + 0.5F);
        }
        {
            float _tmp198 = 0.F;
            {
                _tmp198 += _constmask_yY[0][0] * tex1Dfetch(_texinputY, (gid_y + 6 * (int)blockDim.y + -2) * input_stride + gid_x + 0);
            }
            {
                _tmp198 += _constmask_yY[1][0] * tex1Dfetch(_texinputY, (gid_y + 6 * (int)blockDim.y + -1) * input_stride + gid_x + 0);
            }
            {
                _tmp198 += _constmask_yY[2][0] * tex1Dfetch(_texinputY, (gid_y + 6 * (int)blockDim.y + 0) * input_stride + gid_x + 0);
            }
            {
                _tmp198 += _constmask_yY[3][0] * tex1Dfetch(_texinputY, (gid_y + 6 * (int)blockDim.y + 1) * input_stride + gid_x + 0);
            }
            {
                _tmp198 += _constmask_yY[4][0] * tex1Dfetch(_texinputY, (gid_y + 6 * (int)blockDim.y + 2) * input_stride + gid_x + 0);
            }
            iter[(gid_y + 6 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp198 + 0.5F);
        }
        {
            float _tmp199 = 0.F;
            {
                _tmp199 += _constmask_yY[0][0] * tex1Dfetch(_texinputY, (gid_y + 7 * (int)blockDim.y + -2) * input_stride + gid_x + 0);
            }
            {
                _tmp199 += _constmask_yY[1][0] * tex1Dfetch(_texinputY, (gid_y + 7 * (int)blockDim.y + -1) * input_stride + gid_x + 0);
            }
            {
                _tmp199 += _constmask_yY[2][0] * tex1Dfetch(_texinputY, (gid_y + 7 * (int)blockDim.y + 0) * input_stride + gid_x + 0);
            }
            {
                _tmp199 += _constmask_yY[3][0] * tex1Dfetch(_texinputY, (gid_y + 7 * (int)blockDim.y + 1) * input_stride + gid_x + 0);
            }
            {
                _tmp199 += _constmask_yY[4][0] * tex1Dfetch(_texinputY, (gid_y + 7 * (int)blockDim.y + 2) * input_stride + gid_x + 0);
            }
            iter[(gid_y + 7 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp199 + 0.5F);
        }
    }
    goto BH_EXIT;
  BH_EXIT:
    ;
}
}

#endif //_CUGAUSSIANFILTERCOLUMNY_CU_

