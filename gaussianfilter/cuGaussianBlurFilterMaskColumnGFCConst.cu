#include "hip/hip_runtime.h"
#ifndef _CUGAUSSIANBLURFILTERMASKCOLUMNGFCCONST_CU_
#define _CUGAUSSIANBLURFILTERMASKCOLUMNGFCCONST_CU_

#include "hipacc_types.hpp"
#include "hipacc_math_functions.hpp"


extern "C" {
__global__ __launch_bounds__ (128*1) void cuGaussianBlurFilterMaskColumnGFCConstKernel(uchar * __restrict__ iter, int iter_width, int iter_stride, const float * __restrict__ input, int input_width, int input_height, int input_stride, int bh_start_right, int bh_start_top, int bh_start_bottom, int bh_fall_back) {
    const int gid_x = blockDim.x * blockIdx.x + threadIdx.x;
    const int gid_y = blockDim.y * blockIdx.y + threadIdx.y;
    if (bh_fall_back)
        goto BH_FB;
    if (blockIdx.y < bh_start_top)
        goto BH_T;
    if (blockIdx.y >= bh_start_bottom)
        goto BH_B;
    if (blockIdx.x >= bh_start_right)
        goto BH_R;
    goto BH_NO;
  BH_FB:
    {
        if (gid_x < iter_width) {
            {
                float _tmp0 = 0.F;
                {
                    int _gid_x1 = gid_x + 0;
                    int _gid_y1 = gid_y + -2;
                    float _tmp1 = 1.F;
                    if (_gid_y1 >= 0 && _gid_y1 < input_height)
                        _tmp1 = input[(_gid_y1) * input_stride + _gid_x1];
                    _tmp0 += 0.0707660019F * _tmp1;
                }
                {
                    int _gid_x2 = gid_x + 0;
                    int _gid_y2 = gid_y + -1;
                    float _tmp2 = 1.F;
                    if (_gid_y2 >= 0 && _gid_y2 < input_height)
                        _tmp2 = input[(_gid_y2) * input_stride + _gid_x2];
                    _tmp0 += 0.244460002F * _tmp2;
                }
                {
                    int _gid_x3 = gid_x + 0;
                    int _gid_y3 = gid_y + 0;
                    float _tmp3 = 1.F;
                    if (_gid_y3 >= 0 && _gid_y3 < input_height)
                        _tmp3 = input[(_gid_y3) * input_stride + _gid_x3];
                    _tmp0 += 0.369545996F * _tmp3;
                }
                {
                    int _gid_x4 = gid_x + 0;
                    int _gid_y4 = gid_y + 1;
                    float _tmp4 = 1.F;
                    if (_gid_y4 >= 0 && _gid_y4 < input_height)
                        _tmp4 = input[(_gid_y4) * input_stride + _gid_x4];
                    _tmp0 += 0.244460002F * _tmp4;
                }
                {
                    int _gid_x5 = gid_x + 0;
                    int _gid_y5 = gid_y + 2;
                    float _tmp5 = 1.F;
                    if (_gid_y5 >= 0 && _gid_y5 < input_height)
                        _tmp5 = input[(_gid_y5) * input_stride + _gid_x5];
                    _tmp0 += 0.0707660019F * _tmp5;
                }
                iter[(gid_y) * iter_stride + gid_x] = (uchar)(_tmp0 + 0.5F);
            }
        }
    }
    goto BH_EXIT;
  BH_T:
    {
        if (gid_x < iter_width) {
            {
                float _tmp6 = 0.F;
                {
                    int _gid_x7 = gid_x + 0;
                    int _gid_y7 = gid_y + -2;
                    float _tmp7 = 1.F;
                    if (_gid_y7 >= 0)
                        _tmp7 = input[(_gid_y7) * input_stride + _gid_x7];
                    _tmp6 += 0.0707660019F * _tmp7;
                }
                {
                    int _gid_x8 = gid_x + 0;
                    int _gid_y8 = gid_y + -1;
                    float _tmp8 = 1.F;
                    if (_gid_y8 >= 0)
                        _tmp8 = input[(_gid_y8) * input_stride + _gid_x8];
                    _tmp6 += 0.244460002F * _tmp8;
                }
                {
                    int _gid_x9 = gid_x + 0;
                    int _gid_y9 = gid_y + 0;
                    float _tmp9 = 1.F;
                    if (_gid_y9 >= 0)
                        _tmp9 = input[(_gid_y9) * input_stride + _gid_x9];
                    _tmp6 += 0.369545996F * _tmp9;
                }
                {
                    int _gid_x10 = gid_x + 0;
                    int _gid_y10 = gid_y + 1;
                    float _tmp10 = 1.F;
                    if (_gid_y10 >= 0)
                        _tmp10 = input[(_gid_y10) * input_stride + _gid_x10];
                    _tmp6 += 0.244460002F * _tmp10;
                }
                {
                    int _gid_x11 = gid_x + 0;
                    int _gid_y11 = gid_y + 2;
                    float _tmp11 = 1.F;
                    if (_gid_y11 >= 0)
                        _tmp11 = input[(_gid_y11) * input_stride + _gid_x11];
                    _tmp6 += 0.0707660019F * _tmp11;
                }
                iter[(gid_y) * iter_stride + gid_x] = (uchar)(_tmp6 + 0.5F);
            }
        }
    }
    goto BH_EXIT;
  BH_B:
    {
        if (gid_x < iter_width) {
            {
                float _tmp12 = 0.F;
                {
                    int _gid_x13 = gid_x + 0;
                    int _gid_y13 = gid_y + -2;
                    float _tmp13 = 1.F;
                    if (_gid_y13 < input_height)
                        _tmp13 = input[(_gid_y13) * input_stride + _gid_x13];
                    _tmp12 += 0.0707660019F * _tmp13;
                }
                {
                    int _gid_x14 = gid_x + 0;
                    int _gid_y14 = gid_y + -1;
                    float _tmp14 = 1.F;
                    if (_gid_y14 < input_height)
                        _tmp14 = input[(_gid_y14) * input_stride + _gid_x14];
                    _tmp12 += 0.244460002F * _tmp14;
                }
                {
                    int _gid_x15 = gid_x + 0;
                    int _gid_y15 = gid_y + 0;
                    float _tmp15 = 1.F;
                    if (_gid_y15 < input_height)
                        _tmp15 = input[(_gid_y15) * input_stride + _gid_x15];
                    _tmp12 += 0.369545996F * _tmp15;
                }
                {
                    int _gid_x16 = gid_x + 0;
                    int _gid_y16 = gid_y + 1;
                    float _tmp16 = 1.F;
                    if (_gid_y16 < input_height)
                        _tmp16 = input[(_gid_y16) * input_stride + _gid_x16];
                    _tmp12 += 0.244460002F * _tmp16;
                }
                {
                    int _gid_x17 = gid_x + 0;
                    int _gid_y17 = gid_y + 2;
                    float _tmp17 = 1.F;
                    if (_gid_y17 < input_height)
                        _tmp17 = input[(_gid_y17) * input_stride + _gid_x17];
                    _tmp12 += 0.0707660019F * _tmp17;
                }
                iter[(gid_y) * iter_stride + gid_x] = (uchar)(_tmp12 + 0.5F);
            }
        }
    }
    goto BH_EXIT;
  BH_R:
    {
        if (gid_x < iter_width) {
            {
                float _tmp18 = 0.F;
                {
                    int _gid_x19 = gid_x + 0;
                    int _gid_y19 = gid_y + -2;
                    float _tmp19 = 1.F;
                    if (_gid_x19 < input_width)
                        _tmp19 = input[(_gid_y19) * input_stride + _gid_x19];
                    _tmp18 += 0.0707660019F * _tmp19;
                }
                {
                    int _gid_x20 = gid_x + 0;
                    int _gid_y20 = gid_y + -1;
                    float _tmp20 = 1.F;
                    if (_gid_x20 < input_width)
                        _tmp20 = input[(_gid_y20) * input_stride + _gid_x20];
                    _tmp18 += 0.244460002F * _tmp20;
                }
                {
                    int _gid_x21 = gid_x + 0;
                    int _gid_y21 = gid_y + 0;
                    float _tmp21 = 1.F;
                    if (_gid_x21 < input_width)
                        _tmp21 = input[(_gid_y21) * input_stride + _gid_x21];
                    _tmp18 += 0.369545996F * _tmp21;
                }
                {
                    int _gid_x22 = gid_x + 0;
                    int _gid_y22 = gid_y + 1;
                    float _tmp22 = 1.F;
                    if (_gid_x22 < input_width)
                        _tmp22 = input[(_gid_y22) * input_stride + _gid_x22];
                    _tmp18 += 0.244460002F * _tmp22;
                }
                {
                    int _gid_x23 = gid_x + 0;
                    int _gid_y23 = gid_y + 2;
                    float _tmp23 = 1.F;
                    if (_gid_x23 < input_width)
                        _tmp23 = input[(_gid_y23) * input_stride + _gid_x23];
                    _tmp18 += 0.0707660019F * _tmp23;
                }
                iter[(gid_y) * iter_stride + gid_x] = (uchar)(_tmp18 + 0.5F);
            }
        }
    }
    goto BH_EXIT;
  BH_NO:
    {
        {
            float _tmp24 = 0.F;
            {
                _tmp24 += 0.0707660019F * input[(gid_y + -2) * input_stride + gid_x + 0];
            }
            {
                _tmp24 += 0.244460002F * input[(gid_y + -1) * input_stride + gid_x + 0];
            }
            {
                _tmp24 += 0.369545996F * input[(gid_y + 0) * input_stride + gid_x + 0];
            }
            {
                _tmp24 += 0.244460002F * input[(gid_y + 1) * input_stride + gid_x + 0];
            }
            {
                _tmp24 += 0.0707660019F * input[(gid_y + 2) * input_stride + gid_x + 0];
            }
            iter[(gid_y) * iter_stride + gid_x] = (uchar)(_tmp24 + 0.5F);
        }
    }
    goto BH_EXIT;
  BH_EXIT:
    ;
}
}

#endif //_CUGAUSSIANBLURFILTERMASKCOLUMNGFCCONST_CU_

