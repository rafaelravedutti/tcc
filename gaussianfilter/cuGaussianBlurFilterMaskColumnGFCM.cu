#include "hip/hip_runtime.h"
#ifndef _CUGAUSSIANBLURFILTERMASKCOLUMNGFCM_CU_
#define _CUGAUSSIANBLURFILTERMASKCOLUMNGFCM_CU_

#include "hipacc_types.hpp"
#include "hipacc_math_functions.hpp"


extern "C" {
__global__ __launch_bounds__ (128*1) void cuGaussianBlurFilterMaskColumnGFCMKernel(uchar * __restrict__ iter, int iter_width, int iter_stride, const float * __restrict__ input, int input_width, int input_height, int input_stride, int bh_start_right, int bh_start_top, int bh_start_bottom, int bh_fall_back) {
    const int gid_x = blockDim.x * blockIdx.x + threadIdx.x;
    const int gid_y = blockDim.y * blockIdx.y + threadIdx.y;
    if (bh_fall_back)
        goto BH_FB;
    if (blockIdx.y < bh_start_top)
        goto BH_T;
    if (blockIdx.y >= bh_start_bottom)
        goto BH_B;
    if (blockIdx.x >= bh_start_right)
        goto BH_R;
    goto BH_NO;
  BH_FB:
    {
        if (gid_x < iter_width) {
            {
                float _tmp0 = 0.F;
                {
                    int _gid_x1 = gid_x + 0;
                    int _gid_y1 = gid_y + -2;
                    if (_gid_y1 >= input_height)
                        _gid_y1 = input_height - (_gid_y1 + 1 - (input_height));
                    if (_gid_y1 < 0)
                        _gid_y1 = 0 + (0 - _gid_y1 - 1);
                    _tmp0 += 0.0707660019F * input[(_gid_y1) * input_stride + _gid_x1];
                }
                {
                    int _gid_x2 = gid_x + 0;
                    int _gid_y2 = gid_y + -1;
                    if (_gid_y2 >= input_height)
                        _gid_y2 = input_height - (_gid_y2 + 1 - (input_height));
                    if (_gid_y2 < 0)
                        _gid_y2 = 0 + (0 - _gid_y2 - 1);
                    _tmp0 += 0.244460002F * input[(_gid_y2) * input_stride + _gid_x2];
                }
                {
                    int _gid_x3 = gid_x + 0;
                    int _gid_y3 = gid_y + 0;
                    if (_gid_y3 >= input_height)
                        _gid_y3 = input_height - (_gid_y3 + 1 - (input_height));
                    if (_gid_y3 < 0)
                        _gid_y3 = 0 + (0 - _gid_y3 - 1);
                    _tmp0 += 0.369545996F * input[(_gid_y3) * input_stride + _gid_x3];
                }
                {
                    int _gid_x4 = gid_x + 0;
                    int _gid_y4 = gid_y + 1;
                    if (_gid_y4 >= input_height)
                        _gid_y4 = input_height - (_gid_y4 + 1 - (input_height));
                    if (_gid_y4 < 0)
                        _gid_y4 = 0 + (0 - _gid_y4 - 1);
                    _tmp0 += 0.244460002F * input[(_gid_y4) * input_stride + _gid_x4];
                }
                {
                    int _gid_x5 = gid_x + 0;
                    int _gid_y5 = gid_y + 2;
                    if (_gid_y5 >= input_height)
                        _gid_y5 = input_height - (_gid_y5 + 1 - (input_height));
                    if (_gid_y5 < 0)
                        _gid_y5 = 0 + (0 - _gid_y5 - 1);
                    _tmp0 += 0.0707660019F * input[(_gid_y5) * input_stride + _gid_x5];
                }
                iter[(gid_y) * iter_stride + gid_x] = (uchar)(_tmp0 + 0.5F);
            }
        }
    }
    goto BH_EXIT;
  BH_T:
    {
        if (gid_x < iter_width) {
            {
                float _tmp6 = 0.F;
                {
                    int _gid_x7 = gid_x + 0;
                    int _gid_y7 = gid_y + -2;
                    if (_gid_y7 < 0)
                        _gid_y7 = 0 + (0 - _gid_y7 - 1);
                    _tmp6 += 0.0707660019F * input[(_gid_y7) * input_stride + _gid_x7];
                }
                {
                    int _gid_x8 = gid_x + 0;
                    int _gid_y8 = gid_y + -1;
                    if (_gid_y8 < 0)
                        _gid_y8 = 0 + (0 - _gid_y8 - 1);
                    _tmp6 += 0.244460002F * input[(_gid_y8) * input_stride + _gid_x8];
                }
                {
                    int _gid_x9 = gid_x + 0;
                    int _gid_y9 = gid_y + 0;
                    if (_gid_y9 < 0)
                        _gid_y9 = 0 + (0 - _gid_y9 - 1);
                    _tmp6 += 0.369545996F * input[(_gid_y9) * input_stride + _gid_x9];
                }
                {
                    int _gid_x10 = gid_x + 0;
                    int _gid_y10 = gid_y + 1;
                    if (_gid_y10 < 0)
                        _gid_y10 = 0 + (0 - _gid_y10 - 1);
                    _tmp6 += 0.244460002F * input[(_gid_y10) * input_stride + _gid_x10];
                }
                {
                    int _gid_x11 = gid_x + 0;
                    int _gid_y11 = gid_y + 2;
                    if (_gid_y11 < 0)
                        _gid_y11 = 0 + (0 - _gid_y11 - 1);
                    _tmp6 += 0.0707660019F * input[(_gid_y11) * input_stride + _gid_x11];
                }
                iter[(gid_y) * iter_stride + gid_x] = (uchar)(_tmp6 + 0.5F);
            }
        }
    }
    goto BH_EXIT;
  BH_B:
    {
        if (gid_x < iter_width) {
            {
                float _tmp12 = 0.F;
                {
                    int _gid_x13 = gid_x + 0;
                    int _gid_y13 = gid_y + -2;
                    if (_gid_y13 >= input_height)
                        _gid_y13 = input_height - (_gid_y13 + 1 - (input_height));
                    _tmp12 += 0.0707660019F * input[(_gid_y13) * input_stride + _gid_x13];
                }
                {
                    int _gid_x14 = gid_x + 0;
                    int _gid_y14 = gid_y + -1;
                    if (_gid_y14 >= input_height)
                        _gid_y14 = input_height - (_gid_y14 + 1 - (input_height));
                    _tmp12 += 0.244460002F * input[(_gid_y14) * input_stride + _gid_x14];
                }
                {
                    int _gid_x15 = gid_x + 0;
                    int _gid_y15 = gid_y + 0;
                    if (_gid_y15 >= input_height)
                        _gid_y15 = input_height - (_gid_y15 + 1 - (input_height));
                    _tmp12 += 0.369545996F * input[(_gid_y15) * input_stride + _gid_x15];
                }
                {
                    int _gid_x16 = gid_x + 0;
                    int _gid_y16 = gid_y + 1;
                    if (_gid_y16 >= input_height)
                        _gid_y16 = input_height - (_gid_y16 + 1 - (input_height));
                    _tmp12 += 0.244460002F * input[(_gid_y16) * input_stride + _gid_x16];
                }
                {
                    int _gid_x17 = gid_x + 0;
                    int _gid_y17 = gid_y + 2;
                    if (_gid_y17 >= input_height)
                        _gid_y17 = input_height - (_gid_y17 + 1 - (input_height));
                    _tmp12 += 0.0707660019F * input[(_gid_y17) * input_stride + _gid_x17];
                }
                iter[(gid_y) * iter_stride + gid_x] = (uchar)(_tmp12 + 0.5F);
            }
        }
    }
    goto BH_EXIT;
  BH_R:
    {
        if (gid_x < iter_width) {
            {
                float _tmp18 = 0.F;
                {
                    int _gid_x19 = gid_x + 0;
                    int _gid_y19 = gid_y + -2;
                    if (_gid_x19 >= input_width)
                        _gid_x19 = input_width - (_gid_x19 + 1 - (input_width));
                    _tmp18 += 0.0707660019F * input[(_gid_y19) * input_stride + _gid_x19];
                }
                {
                    int _gid_x20 = gid_x + 0;
                    int _gid_y20 = gid_y + -1;
                    if (_gid_x20 >= input_width)
                        _gid_x20 = input_width - (_gid_x20 + 1 - (input_width));
                    _tmp18 += 0.244460002F * input[(_gid_y20) * input_stride + _gid_x20];
                }
                {
                    int _gid_x21 = gid_x + 0;
                    int _gid_y21 = gid_y + 0;
                    if (_gid_x21 >= input_width)
                        _gid_x21 = input_width - (_gid_x21 + 1 - (input_width));
                    _tmp18 += 0.369545996F * input[(_gid_y21) * input_stride + _gid_x21];
                }
                {
                    int _gid_x22 = gid_x + 0;
                    int _gid_y22 = gid_y + 1;
                    if (_gid_x22 >= input_width)
                        _gid_x22 = input_width - (_gid_x22 + 1 - (input_width));
                    _tmp18 += 0.244460002F * input[(_gid_y22) * input_stride + _gid_x22];
                }
                {
                    int _gid_x23 = gid_x + 0;
                    int _gid_y23 = gid_y + 2;
                    if (_gid_x23 >= input_width)
                        _gid_x23 = input_width - (_gid_x23 + 1 - (input_width));
                    _tmp18 += 0.0707660019F * input[(_gid_y23) * input_stride + _gid_x23];
                }
                iter[(gid_y) * iter_stride + gid_x] = (uchar)(_tmp18 + 0.5F);
            }
        }
    }
    goto BH_EXIT;
  BH_NO:
    {
        {
            float _tmp24 = 0.F;
            {
                _tmp24 += 0.0707660019F * input[(gid_y + -2) * input_stride + gid_x + 0];
            }
            {
                _tmp24 += 0.244460002F * input[(gid_y + -1) * input_stride + gid_x + 0];
            }
            {
                _tmp24 += 0.369545996F * input[(gid_y + 0) * input_stride + gid_x + 0];
            }
            {
                _tmp24 += 0.244460002F * input[(gid_y + 1) * input_stride + gid_x + 0];
            }
            {
                _tmp24 += 0.0707660019F * input[(gid_y + 2) * input_stride + gid_x + 0];
            }
            iter[(gid_y) * iter_stride + gid_x] = (uchar)(_tmp24 + 0.5F);
        }
    }
    goto BH_EXIT;
  BH_EXIT:
    ;
}
}

#endif //_CUGAUSSIANBLURFILTERMASKCOLUMNGFCM_CU_

