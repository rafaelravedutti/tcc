#include "hip/hip_runtime.h"
#ifndef _CUGAUSSIANBLURFILTERMASKROWGFRCONST_CU_
#define _CUGAUSSIANBLURFILTERMASKROWGFRCONST_CU_

#include "hipacc_types.hpp"
#include "hipacc_math_functions.hpp"


extern "C" {
__global__ __launch_bounds__ (128*1) void cuGaussianBlurFilterMaskRowGFRConstKernel(float * __restrict__ iter, int iter_width, int iter_stride, const uchar * __restrict__ input, int input_width, int input_height, int input_stride, int bh_start_left, int bh_start_right, int bh_fall_back) {
    const int gid_x = blockDim.x * blockIdx.x + threadIdx.x;
    const int gid_y = blockDim.y * blockIdx.y + threadIdx.y;
    if (bh_fall_back)
        goto BH_FB;
    if (blockIdx.x >= bh_start_right)
        goto BH_R;
    if (blockIdx.x < bh_start_left)
        goto BH_L;
    goto BH_NO;
  BH_FB:
    {
        if (gid_x < iter_width) {
            {
                float _tmp0 = 0.F;
                {
                    int _gid_x1 = gid_x + -2;
                    int _gid_y1 = gid_y + 0;
                    uchar _tmp1 = '1';
                    if (_gid_x1 >= 0 && _gid_x1 < input_width)
                        _tmp1 = input[(_gid_y1) * input_stride + _gid_x1];
                    _tmp0 += 0.0707660019F * _tmp1;
                }
                {
                    int _gid_x2 = gid_x + -1;
                    int _gid_y2 = gid_y + 0;
                    uchar _tmp2 = '1';
                    if (_gid_x2 >= 0 && _gid_x2 < input_width)
                        _tmp2 = input[(_gid_y2) * input_stride + _gid_x2];
                    _tmp0 += 0.244460002F * _tmp2;
                }
                {
                    int _gid_x3 = gid_x + 0;
                    int _gid_y3 = gid_y + 0;
                    uchar _tmp3 = '1';
                    if (_gid_x3 >= 0 && _gid_x3 < input_width)
                        _tmp3 = input[(_gid_y3) * input_stride + _gid_x3];
                    _tmp0 += 0.369545996F * _tmp3;
                }
                {
                    int _gid_x4 = gid_x + 1;
                    int _gid_y4 = gid_y + 0;
                    uchar _tmp4 = '1';
                    if (_gid_x4 >= 0 && _gid_x4 < input_width)
                        _tmp4 = input[(_gid_y4) * input_stride + _gid_x4];
                    _tmp0 += 0.244460002F * _tmp4;
                }
                {
                    int _gid_x5 = gid_x + 2;
                    int _gid_y5 = gid_y + 0;
                    uchar _tmp5 = '1';
                    if (_gid_x5 >= 0 && _gid_x5 < input_width)
                        _tmp5 = input[(_gid_y5) * input_stride + _gid_x5];
                    _tmp0 += 0.0707660019F * _tmp5;
                }
                iter[(gid_y) * iter_stride + gid_x] = _tmp0;
            }
        }
    }
    goto BH_EXIT;
  BH_R:
    {
        if (gid_x < iter_width) {
            {
                float _tmp6 = 0.F;
                {
                    int _gid_x7 = gid_x + -2;
                    int _gid_y7 = gid_y + 0;
                    uchar _tmp7 = '1';
                    if (_gid_x7 < input_width)
                        _tmp7 = input[(_gid_y7) * input_stride + _gid_x7];
                    _tmp6 += 0.0707660019F * _tmp7;
                }
                {
                    int _gid_x8 = gid_x + -1;
                    int _gid_y8 = gid_y + 0;
                    uchar _tmp8 = '1';
                    if (_gid_x8 < input_width)
                        _tmp8 = input[(_gid_y8) * input_stride + _gid_x8];
                    _tmp6 += 0.244460002F * _tmp8;
                }
                {
                    int _gid_x9 = gid_x + 0;
                    int _gid_y9 = gid_y + 0;
                    uchar _tmp9 = '1';
                    if (_gid_x9 < input_width)
                        _tmp9 = input[(_gid_y9) * input_stride + _gid_x9];
                    _tmp6 += 0.369545996F * _tmp9;
                }
                {
                    int _gid_x10 = gid_x + 1;
                    int _gid_y10 = gid_y + 0;
                    uchar _tmp10 = '1';
                    if (_gid_x10 < input_width)
                        _tmp10 = input[(_gid_y10) * input_stride + _gid_x10];
                    _tmp6 += 0.244460002F * _tmp10;
                }
                {
                    int _gid_x11 = gid_x + 2;
                    int _gid_y11 = gid_y + 0;
                    uchar _tmp11 = '1';
                    if (_gid_x11 < input_width)
                        _tmp11 = input[(_gid_y11) * input_stride + _gid_x11];
                    _tmp6 += 0.0707660019F * _tmp11;
                }
                iter[(gid_y) * iter_stride + gid_x] = _tmp6;
            }
        }
    }
    goto BH_EXIT;
  BH_L:
    {
        {
            float _tmp12 = 0.F;
            {
                int _gid_x13 = gid_x + -2;
                int _gid_y13 = gid_y + 0;
                uchar _tmp13 = '1';
                if (_gid_x13 >= 0)
                    _tmp13 = input[(_gid_y13) * input_stride + _gid_x13];
                _tmp12 += 0.0707660019F * _tmp13;
            }
            {
                int _gid_x14 = gid_x + -1;
                int _gid_y14 = gid_y + 0;
                uchar _tmp14 = '1';
                if (_gid_x14 >= 0)
                    _tmp14 = input[(_gid_y14) * input_stride + _gid_x14];
                _tmp12 += 0.244460002F * _tmp14;
            }
            {
                int _gid_x15 = gid_x + 0;
                int _gid_y15 = gid_y + 0;
                uchar _tmp15 = '1';
                if (_gid_x15 >= 0)
                    _tmp15 = input[(_gid_y15) * input_stride + _gid_x15];
                _tmp12 += 0.369545996F * _tmp15;
            }
            {
                int _gid_x16 = gid_x + 1;
                int _gid_y16 = gid_y + 0;
                uchar _tmp16 = '1';
                if (_gid_x16 >= 0)
                    _tmp16 = input[(_gid_y16) * input_stride + _gid_x16];
                _tmp12 += 0.244460002F * _tmp16;
            }
            {
                int _gid_x17 = gid_x + 2;
                int _gid_y17 = gid_y + 0;
                uchar _tmp17 = '1';
                if (_gid_x17 >= 0)
                    _tmp17 = input[(_gid_y17) * input_stride + _gid_x17];
                _tmp12 += 0.0707660019F * _tmp17;
            }
            iter[(gid_y) * iter_stride + gid_x] = _tmp12;
        }
    }
    goto BH_EXIT;
  BH_NO:
    {
        {
            float _tmp18 = 0.F;
            {
                _tmp18 += 0.0707660019F * input[(gid_y + 0) * input_stride + gid_x + -2];
            }
            {
                _tmp18 += 0.244460002F * input[(gid_y + 0) * input_stride + gid_x + -1];
            }
            {
                _tmp18 += 0.369545996F * input[(gid_y + 0) * input_stride + gid_x + 0];
            }
            {
                _tmp18 += 0.244460002F * input[(gid_y + 0) * input_stride + gid_x + 1];
            }
            {
                _tmp18 += 0.0707660019F * input[(gid_y + 0) * input_stride + gid_x + 2];
            }
            iter[(gid_y) * iter_stride + gid_x] = _tmp18;
        }
    }
    goto BH_EXIT;
  BH_EXIT:
    ;
}
}

#endif //_CUGAUSSIANBLURFILTERMASKROWGFRCONST_CU_

