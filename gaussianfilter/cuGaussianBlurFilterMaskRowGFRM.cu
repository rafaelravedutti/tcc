#include "hip/hip_runtime.h"
#ifndef _CUGAUSSIANBLURFILTERMASKROWGFRM_CU_
#define _CUGAUSSIANBLURFILTERMASKROWGFRM_CU_

#include "hipacc_types.hpp"
#include "hipacc_math_functions.hpp"


extern "C" {
__global__ __launch_bounds__ (128*1) void cuGaussianBlurFilterMaskRowGFRMKernel(float * __restrict__ iter, int iter_width, int iter_stride, const uchar * __restrict__ input, int input_width, int input_height, int input_stride, int bh_start_left, int bh_start_right, int bh_fall_back) {
    const int gid_x = blockDim.x * blockIdx.x + threadIdx.x;
    const int gid_y = blockDim.y * blockIdx.y + threadIdx.y;
    if (bh_fall_back)
        goto BH_FB;
    if (blockIdx.x >= bh_start_right)
        goto BH_R;
    if (blockIdx.x < bh_start_left)
        goto BH_L;
    goto BH_NO;
  BH_FB:
    {
        if (gid_x < iter_width) {
            {
                float _tmp0 = 0.F;
                {
                    int _gid_x1 = gid_x + -2;
                    int _gid_y1 = gid_y + 0;
                    if (_gid_x1 >= input_width)
                        _gid_x1 = input_width - (_gid_x1 + 1 - (input_width));
                    if (_gid_x1 < 0)
                        _gid_x1 = 0 + (0 - _gid_x1 - 1);
                    _tmp0 += 0.0707660019F * input[(_gid_y1) * input_stride + _gid_x1];
                }
                {
                    int _gid_x2 = gid_x + -1;
                    int _gid_y2 = gid_y + 0;
                    if (_gid_x2 >= input_width)
                        _gid_x2 = input_width - (_gid_x2 + 1 - (input_width));
                    if (_gid_x2 < 0)
                        _gid_x2 = 0 + (0 - _gid_x2 - 1);
                    _tmp0 += 0.244460002F * input[(_gid_y2) * input_stride + _gid_x2];
                }
                {
                    int _gid_x3 = gid_x + 0;
                    int _gid_y3 = gid_y + 0;
                    if (_gid_x3 >= input_width)
                        _gid_x3 = input_width - (_gid_x3 + 1 - (input_width));
                    if (_gid_x3 < 0)
                        _gid_x3 = 0 + (0 - _gid_x3 - 1);
                    _tmp0 += 0.369545996F * input[(_gid_y3) * input_stride + _gid_x3];
                }
                {
                    int _gid_x4 = gid_x + 1;
                    int _gid_y4 = gid_y + 0;
                    if (_gid_x4 >= input_width)
                        _gid_x4 = input_width - (_gid_x4 + 1 - (input_width));
                    if (_gid_x4 < 0)
                        _gid_x4 = 0 + (0 - _gid_x4 - 1);
                    _tmp0 += 0.244460002F * input[(_gid_y4) * input_stride + _gid_x4];
                }
                {
                    int _gid_x5 = gid_x + 2;
                    int _gid_y5 = gid_y + 0;
                    if (_gid_x5 >= input_width)
                        _gid_x5 = input_width - (_gid_x5 + 1 - (input_width));
                    if (_gid_x5 < 0)
                        _gid_x5 = 0 + (0 - _gid_x5 - 1);
                    _tmp0 += 0.0707660019F * input[(_gid_y5) * input_stride + _gid_x5];
                }
                iter[(gid_y) * iter_stride + gid_x] = _tmp0;
            }
        }
    }
    goto BH_EXIT;
  BH_R:
    {
        if (gid_x < iter_width) {
            {
                float _tmp6 = 0.F;
                {
                    int _gid_x7 = gid_x + -2;
                    int _gid_y7 = gid_y + 0;
                    if (_gid_x7 >= input_width)
                        _gid_x7 = input_width - (_gid_x7 + 1 - (input_width));
                    _tmp6 += 0.0707660019F * input[(_gid_y7) * input_stride + _gid_x7];
                }
                {
                    int _gid_x8 = gid_x + -1;
                    int _gid_y8 = gid_y + 0;
                    if (_gid_x8 >= input_width)
                        _gid_x8 = input_width - (_gid_x8 + 1 - (input_width));
                    _tmp6 += 0.244460002F * input[(_gid_y8) * input_stride + _gid_x8];
                }
                {
                    int _gid_x9 = gid_x + 0;
                    int _gid_y9 = gid_y + 0;
                    if (_gid_x9 >= input_width)
                        _gid_x9 = input_width - (_gid_x9 + 1 - (input_width));
                    _tmp6 += 0.369545996F * input[(_gid_y9) * input_stride + _gid_x9];
                }
                {
                    int _gid_x10 = gid_x + 1;
                    int _gid_y10 = gid_y + 0;
                    if (_gid_x10 >= input_width)
                        _gid_x10 = input_width - (_gid_x10 + 1 - (input_width));
                    _tmp6 += 0.244460002F * input[(_gid_y10) * input_stride + _gid_x10];
                }
                {
                    int _gid_x11 = gid_x + 2;
                    int _gid_y11 = gid_y + 0;
                    if (_gid_x11 >= input_width)
                        _gid_x11 = input_width - (_gid_x11 + 1 - (input_width));
                    _tmp6 += 0.0707660019F * input[(_gid_y11) * input_stride + _gid_x11];
                }
                iter[(gid_y) * iter_stride + gid_x] = _tmp6;
            }
        }
    }
    goto BH_EXIT;
  BH_L:
    {
        {
            float _tmp12 = 0.F;
            {
                int _gid_x13 = gid_x + -2;
                int _gid_y13 = gid_y + 0;
                if (_gid_x13 < 0)
                    _gid_x13 = 0 + (0 - _gid_x13 - 1);
                _tmp12 += 0.0707660019F * input[(_gid_y13) * input_stride + _gid_x13];
            }
            {
                int _gid_x14 = gid_x + -1;
                int _gid_y14 = gid_y + 0;
                if (_gid_x14 < 0)
                    _gid_x14 = 0 + (0 - _gid_x14 - 1);
                _tmp12 += 0.244460002F * input[(_gid_y14) * input_stride + _gid_x14];
            }
            {
                int _gid_x15 = gid_x + 0;
                int _gid_y15 = gid_y + 0;
                if (_gid_x15 < 0)
                    _gid_x15 = 0 + (0 - _gid_x15 - 1);
                _tmp12 += 0.369545996F * input[(_gid_y15) * input_stride + _gid_x15];
            }
            {
                int _gid_x16 = gid_x + 1;
                int _gid_y16 = gid_y + 0;
                if (_gid_x16 < 0)
                    _gid_x16 = 0 + (0 - _gid_x16 - 1);
                _tmp12 += 0.244460002F * input[(_gid_y16) * input_stride + _gid_x16];
            }
            {
                int _gid_x17 = gid_x + 2;
                int _gid_y17 = gid_y + 0;
                if (_gid_x17 < 0)
                    _gid_x17 = 0 + (0 - _gid_x17 - 1);
                _tmp12 += 0.0707660019F * input[(_gid_y17) * input_stride + _gid_x17];
            }
            iter[(gid_y) * iter_stride + gid_x] = _tmp12;
        }
    }
    goto BH_EXIT;
  BH_NO:
    {
        {
            float _tmp18 = 0.F;
            {
                _tmp18 += 0.0707660019F * input[(gid_y + 0) * input_stride + gid_x + -2];
            }
            {
                _tmp18 += 0.244460002F * input[(gid_y + 0) * input_stride + gid_x + -1];
            }
            {
                _tmp18 += 0.369545996F * input[(gid_y + 0) * input_stride + gid_x + 0];
            }
            {
                _tmp18 += 0.244460002F * input[(gid_y + 0) * input_stride + gid_x + 1];
            }
            {
                _tmp18 += 0.0707660019F * input[(gid_y + 0) * input_stride + gid_x + 2];
            }
            iter[(gid_y) * iter_stride + gid_x] = _tmp18;
        }
    }
    goto BH_EXIT;
  BH_EXIT:
    ;
}
}

#endif //_CUGAUSSIANBLURFILTERMASKROWGFRM_CU_

