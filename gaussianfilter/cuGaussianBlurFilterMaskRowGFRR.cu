#include "hip/hip_runtime.h"
#ifndef _CUGAUSSIANBLURFILTERMASKROWGFRR_CU_
#define _CUGAUSSIANBLURFILTERMASKROWGFRR_CU_

#include "hipacc_types.hpp"
#include "hipacc_math_functions.hpp"


extern "C" {
__global__ __launch_bounds__ (128*1) void cuGaussianBlurFilterMaskRowGFRRKernel(float * __restrict__ iter, int iter_width, int iter_stride, const uchar * __restrict__ input, int input_width, int input_height, int input_stride, int bh_start_left, int bh_start_right, int bh_fall_back) {
    const int gid_x = blockDim.x * blockIdx.x + threadIdx.x;
    const int gid_y = blockDim.y * blockIdx.y + threadIdx.y;
    if (bh_fall_back)
        goto BH_FB;
    if (blockIdx.x >= bh_start_right)
        goto BH_R;
    if (blockIdx.x < bh_start_left)
        goto BH_L;
    goto BH_NO;
  BH_FB:
    {
        if (gid_x < iter_width) {
            {
                float _tmp0 = 0.F;
                {
                    int _gid_x1 = gid_x + -2;
                    int _gid_y1 = gid_y + 0;
                    while (_gid_x1 >= input_width)
                        _gid_x1 = _gid_x1 - input_width;
                    while (_gid_x1 < 0)
                        _gid_x1 = _gid_x1 + input_width;
                    _tmp0 += 0.0707660019F * input[(_gid_y1) * input_stride + _gid_x1];
                }
                {
                    int _gid_x2 = gid_x + -1;
                    int _gid_y2 = gid_y + 0;
                    while (_gid_x2 >= input_width)
                        _gid_x2 = _gid_x2 - input_width;
                    while (_gid_x2 < 0)
                        _gid_x2 = _gid_x2 + input_width;
                    _tmp0 += 0.244460002F * input[(_gid_y2) * input_stride + _gid_x2];
                }
                {
                    int _gid_x3 = gid_x + 0;
                    int _gid_y3 = gid_y + 0;
                    while (_gid_x3 >= input_width)
                        _gid_x3 = _gid_x3 - input_width;
                    while (_gid_x3 < 0)
                        _gid_x3 = _gid_x3 + input_width;
                    _tmp0 += 0.369545996F * input[(_gid_y3) * input_stride + _gid_x3];
                }
                {
                    int _gid_x4 = gid_x + 1;
                    int _gid_y4 = gid_y + 0;
                    while (_gid_x4 >= input_width)
                        _gid_x4 = _gid_x4 - input_width;
                    while (_gid_x4 < 0)
                        _gid_x4 = _gid_x4 + input_width;
                    _tmp0 += 0.244460002F * input[(_gid_y4) * input_stride + _gid_x4];
                }
                {
                    int _gid_x5 = gid_x + 2;
                    int _gid_y5 = gid_y + 0;
                    while (_gid_x5 >= input_width)
                        _gid_x5 = _gid_x5 - input_width;
                    while (_gid_x5 < 0)
                        _gid_x5 = _gid_x5 + input_width;
                    _tmp0 += 0.0707660019F * input[(_gid_y5) * input_stride + _gid_x5];
                }
                iter[(gid_y) * iter_stride + gid_x] = _tmp0;
            }
        }
    }
    goto BH_EXIT;
  BH_R:
    {
        if (gid_x < iter_width) {
            {
                float _tmp6 = 0.F;
                {
                    int _gid_x7 = gid_x + -2;
                    int _gid_y7 = gid_y + 0;
                    while (_gid_x7 >= input_width)
                        _gid_x7 = _gid_x7 - input_width;
                    _tmp6 += 0.0707660019F * input[(_gid_y7) * input_stride + _gid_x7];
                }
                {
                    int _gid_x8 = gid_x + -1;
                    int _gid_y8 = gid_y + 0;
                    while (_gid_x8 >= input_width)
                        _gid_x8 = _gid_x8 - input_width;
                    _tmp6 += 0.244460002F * input[(_gid_y8) * input_stride + _gid_x8];
                }
                {
                    int _gid_x9 = gid_x + 0;
                    int _gid_y9 = gid_y + 0;
                    while (_gid_x9 >= input_width)
                        _gid_x9 = _gid_x9 - input_width;
                    _tmp6 += 0.369545996F * input[(_gid_y9) * input_stride + _gid_x9];
                }
                {
                    int _gid_x10 = gid_x + 1;
                    int _gid_y10 = gid_y + 0;
                    while (_gid_x10 >= input_width)
                        _gid_x10 = _gid_x10 - input_width;
                    _tmp6 += 0.244460002F * input[(_gid_y10) * input_stride + _gid_x10];
                }
                {
                    int _gid_x11 = gid_x + 2;
                    int _gid_y11 = gid_y + 0;
                    while (_gid_x11 >= input_width)
                        _gid_x11 = _gid_x11 - input_width;
                    _tmp6 += 0.0707660019F * input[(_gid_y11) * input_stride + _gid_x11];
                }
                iter[(gid_y) * iter_stride + gid_x] = _tmp6;
            }
        }
    }
    goto BH_EXIT;
  BH_L:
    {
        {
            float _tmp12 = 0.F;
            {
                int _gid_x13 = gid_x + -2;
                int _gid_y13 = gid_y + 0;
                while (_gid_x13 < 0)
                    _gid_x13 = _gid_x13 + input_width;
                _tmp12 += 0.0707660019F * input[(_gid_y13) * input_stride + _gid_x13];
            }
            {
                int _gid_x14 = gid_x + -1;
                int _gid_y14 = gid_y + 0;
                while (_gid_x14 < 0)
                    _gid_x14 = _gid_x14 + input_width;
                _tmp12 += 0.244460002F * input[(_gid_y14) * input_stride + _gid_x14];
            }
            {
                int _gid_x15 = gid_x + 0;
                int _gid_y15 = gid_y + 0;
                while (_gid_x15 < 0)
                    _gid_x15 = _gid_x15 + input_width;
                _tmp12 += 0.369545996F * input[(_gid_y15) * input_stride + _gid_x15];
            }
            {
                int _gid_x16 = gid_x + 1;
                int _gid_y16 = gid_y + 0;
                while (_gid_x16 < 0)
                    _gid_x16 = _gid_x16 + input_width;
                _tmp12 += 0.244460002F * input[(_gid_y16) * input_stride + _gid_x16];
            }
            {
                int _gid_x17 = gid_x + 2;
                int _gid_y17 = gid_y + 0;
                while (_gid_x17 < 0)
                    _gid_x17 = _gid_x17 + input_width;
                _tmp12 += 0.0707660019F * input[(_gid_y17) * input_stride + _gid_x17];
            }
            iter[(gid_y) * iter_stride + gid_x] = _tmp12;
        }
    }
    goto BH_EXIT;
  BH_NO:
    {
        {
            float _tmp18 = 0.F;
            {
                _tmp18 += 0.0707660019F * input[(gid_y + 0) * input_stride + gid_x + -2];
            }
            {
                _tmp18 += 0.244460002F * input[(gid_y + 0) * input_stride + gid_x + -1];
            }
            {
                _tmp18 += 0.369545996F * input[(gid_y + 0) * input_stride + gid_x + 0];
            }
            {
                _tmp18 += 0.244460002F * input[(gid_y + 0) * input_stride + gid_x + 1];
            }
            {
                _tmp18 += 0.0707660019F * input[(gid_y + 0) * input_stride + gid_x + 2];
            }
            iter[(gid_y) * iter_stride + gid_x] = _tmp18;
        }
    }
    goto BH_EXIT;
  BH_EXIT:
    ;
}
}

#endif //_CUGAUSSIANBLURFILTERMASKROWGFRR_CU_

