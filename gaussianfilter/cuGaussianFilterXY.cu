#include "hip/hip_runtime.h"
#ifndef _CUGAUSSIANFILTERXY_CU_
#define _CUGAUSSIANFILTERXY_CU_

#include "hipacc_types.hpp"
#include "hipacc_math_functions.hpp"

texture<uchar, hipTextureType1D, hipReadModeElementType> _texinputXY;
const textureReference *_texinputXYRef;
__device__ __constant__ float _constmaskXY[5][5];


extern "C" {
__global__ __launch_bounds__ (32*2) void cuGaussianFilterXYKernel(uchar * __restrict__ iter, int iter_width, int iter_height, int iter_stride, int input_width, int input_height, int input_stride, int size_x, int size_y, int bh_start_left, int bh_start_right, int bh_start_top, int bh_start_bottom, int bh_fall_back) {
    const int gid_x = blockDim.x * blockIdx.x + threadIdx.x;
    const int gid_y = blockDim.y * blockIdx.y * 8 + threadIdx.y;
    uchar _smeminput[20][97] __attribute__((shared));
    if (bh_fall_back)
        goto BH_FB;
    if (blockIdx.x < bh_start_left && blockIdx.y < bh_start_top)
        goto BH_TL;
    if (blockIdx.x >= bh_start_right && blockIdx.y < bh_start_top)
        goto BH_TR;
    if (blockIdx.y < bh_start_top)
        goto BH_T;
    if (blockIdx.y >= bh_start_bottom && blockIdx.x < bh_start_left)
        goto BH_BL;
    if (blockIdx.y >= bh_start_bottom && blockIdx.x >= bh_start_right)
        goto BH_BR;
    if (blockIdx.y >= bh_start_bottom)
        goto BH_B;
    if (blockIdx.x >= bh_start_right)
        goto BH_R;
    if (blockIdx.x < bh_start_left)
        goto BH_L;
    goto BH_NO;
  BH_FB:
    {
        int _gid_x0 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y0 = gid_y + (-2);
        if (_gid_x0 >= input_width)
            _gid_x0 = input_width - 1;
        if (_gid_y0 >= input_height)
            _gid_y0 = input_height - 1;
        if (_gid_x0 < 0)
            _gid_x0 = 0;
        if (_gid_y0 < 0)
            _gid_y0 = 0;
        _smeminput[(int)threadIdx.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y0) * input_stride + _gid_x0);
        int _gid_x1 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y1 = gid_y + (-2);
        if (_gid_x1 >= input_width)
            _gid_x1 = input_width - 1;
        if (_gid_y1 >= input_height)
            _gid_y1 = input_height - 1;
        if (_gid_x1 < 0)
            _gid_x1 = 0;
        if (_gid_y1 < 0)
            _gid_y1 = 0;
        _smeminput[(int)threadIdx.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y1) * input_stride + _gid_x1);
        int _gid_x2 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y2 = gid_y + (-2);
        if (_gid_x2 >= input_width)
            _gid_x2 = input_width - 1;
        if (_gid_y2 >= input_height)
            _gid_y2 = input_height - 1;
        if (_gid_x2 < 0)
            _gid_x2 = 0;
        if (_gid_y2 < 0)
            _gid_y2 = 0;
        _smeminput[(int)threadIdx.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y2) * input_stride + _gid_x2);
        int _gid_x3 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y3 = gid_y + 1 * (int)blockDim.y + (-2);
        if (_gid_x3 >= input_width)
            _gid_x3 = input_width - 1;
        if (_gid_y3 >= input_height)
            _gid_y3 = input_height - 1;
        if (_gid_x3 < 0)
            _gid_x3 = 0;
        if (_gid_y3 < 0)
            _gid_y3 = 0;
        _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y3) * input_stride + _gid_x3);
        int _gid_x4 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y4 = gid_y + 1 * (int)blockDim.y + (-2);
        if (_gid_x4 >= input_width)
            _gid_x4 = input_width - 1;
        if (_gid_y4 >= input_height)
            _gid_y4 = input_height - 1;
        if (_gid_x4 < 0)
            _gid_x4 = 0;
        if (_gid_y4 < 0)
            _gid_y4 = 0;
        _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y4) * input_stride + _gid_x4);
        int _gid_x5 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y5 = gid_y + 1 * (int)blockDim.y + (-2);
        if (_gid_x5 >= input_width)
            _gid_x5 = input_width - 1;
        if (_gid_y5 >= input_height)
            _gid_y5 = input_height - 1;
        if (_gid_x5 < 0)
            _gid_x5 = 0;
        if (_gid_y5 < 0)
            _gid_y5 = 0;
        _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y5) * input_stride + _gid_x5);
        int _gid_x6 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y6 = gid_y + 2 * (int)blockDim.y + (-2);
        if (_gid_x6 >= input_width)
            _gid_x6 = input_width - 1;
        if (_gid_y6 >= input_height)
            _gid_y6 = input_height - 1;
        if (_gid_x6 < 0)
            _gid_x6 = 0;
        if (_gid_y6 < 0)
            _gid_y6 = 0;
        _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y6) * input_stride + _gid_x6);
        int _gid_x7 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y7 = gid_y + 2 * (int)blockDim.y + (-2);
        if (_gid_x7 >= input_width)
            _gid_x7 = input_width - 1;
        if (_gid_y7 >= input_height)
            _gid_y7 = input_height - 1;
        if (_gid_x7 < 0)
            _gid_x7 = 0;
        if (_gid_y7 < 0)
            _gid_y7 = 0;
        _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y7) * input_stride + _gid_x7);
        int _gid_x8 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y8 = gid_y + 2 * (int)blockDim.y + (-2);
        if (_gid_x8 >= input_width)
            _gid_x8 = input_width - 1;
        if (_gid_y8 >= input_height)
            _gid_y8 = input_height - 1;
        if (_gid_x8 < 0)
            _gid_x8 = 0;
        if (_gid_y8 < 0)
            _gid_y8 = 0;
        _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y8) * input_stride + _gid_x8);
        int _gid_x9 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y9 = gid_y + 3 * (int)blockDim.y + (-2);
        if (_gid_x9 >= input_width)
            _gid_x9 = input_width - 1;
        if (_gid_y9 >= input_height)
            _gid_y9 = input_height - 1;
        if (_gid_x9 < 0)
            _gid_x9 = 0;
        if (_gid_y9 < 0)
            _gid_y9 = 0;
        _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y9) * input_stride + _gid_x9);
        int _gid_x10 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y10 = gid_y + 3 * (int)blockDim.y + (-2);
        if (_gid_x10 >= input_width)
            _gid_x10 = input_width - 1;
        if (_gid_y10 >= input_height)
            _gid_y10 = input_height - 1;
        if (_gid_x10 < 0)
            _gid_x10 = 0;
        if (_gid_y10 < 0)
            _gid_y10 = 0;
        _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y10) * input_stride + _gid_x10);
        int _gid_x11 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y11 = gid_y + 3 * (int)blockDim.y + (-2);
        if (_gid_x11 >= input_width)
            _gid_x11 = input_width - 1;
        if (_gid_y11 >= input_height)
            _gid_y11 = input_height - 1;
        if (_gid_x11 < 0)
            _gid_x11 = 0;
        if (_gid_y11 < 0)
            _gid_y11 = 0;
        _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y11) * input_stride + _gid_x11);
        int _gid_x12 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y12 = gid_y + 4 * (int)blockDim.y + (-2);
        if (_gid_x12 >= input_width)
            _gid_x12 = input_width - 1;
        if (_gid_y12 >= input_height)
            _gid_y12 = input_height - 1;
        if (_gid_x12 < 0)
            _gid_x12 = 0;
        if (_gid_y12 < 0)
            _gid_y12 = 0;
        _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y12) * input_stride + _gid_x12);
        int _gid_x13 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y13 = gid_y + 4 * (int)blockDim.y + (-2);
        if (_gid_x13 >= input_width)
            _gid_x13 = input_width - 1;
        if (_gid_y13 >= input_height)
            _gid_y13 = input_height - 1;
        if (_gid_x13 < 0)
            _gid_x13 = 0;
        if (_gid_y13 < 0)
            _gid_y13 = 0;
        _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y13) * input_stride + _gid_x13);
        int _gid_x14 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y14 = gid_y + 4 * (int)blockDim.y + (-2);
        if (_gid_x14 >= input_width)
            _gid_x14 = input_width - 1;
        if (_gid_y14 >= input_height)
            _gid_y14 = input_height - 1;
        if (_gid_x14 < 0)
            _gid_x14 = 0;
        if (_gid_y14 < 0)
            _gid_y14 = 0;
        _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y14) * input_stride + _gid_x14);
        int _gid_x15 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y15 = gid_y + 5 * (int)blockDim.y + (-2);
        if (_gid_x15 >= input_width)
            _gid_x15 = input_width - 1;
        if (_gid_y15 >= input_height)
            _gid_y15 = input_height - 1;
        if (_gid_x15 < 0)
            _gid_x15 = 0;
        if (_gid_y15 < 0)
            _gid_y15 = 0;
        _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y15) * input_stride + _gid_x15);
        int _gid_x16 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y16 = gid_y + 5 * (int)blockDim.y + (-2);
        if (_gid_x16 >= input_width)
            _gid_x16 = input_width - 1;
        if (_gid_y16 >= input_height)
            _gid_y16 = input_height - 1;
        if (_gid_x16 < 0)
            _gid_x16 = 0;
        if (_gid_y16 < 0)
            _gid_y16 = 0;
        _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y16) * input_stride + _gid_x16);
        int _gid_x17 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y17 = gid_y + 5 * (int)blockDim.y + (-2);
        if (_gid_x17 >= input_width)
            _gid_x17 = input_width - 1;
        if (_gid_y17 >= input_height)
            _gid_y17 = input_height - 1;
        if (_gid_x17 < 0)
            _gid_x17 = 0;
        if (_gid_y17 < 0)
            _gid_y17 = 0;
        _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y17) * input_stride + _gid_x17);
        int _gid_x18 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y18 = gid_y + 6 * (int)blockDim.y + (-2);
        if (_gid_x18 >= input_width)
            _gid_x18 = input_width - 1;
        if (_gid_y18 >= input_height)
            _gid_y18 = input_height - 1;
        if (_gid_x18 < 0)
            _gid_x18 = 0;
        if (_gid_y18 < 0)
            _gid_y18 = 0;
        _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y18) * input_stride + _gid_x18);
        int _gid_x19 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y19 = gid_y + 6 * (int)blockDim.y + (-2);
        if (_gid_x19 >= input_width)
            _gid_x19 = input_width - 1;
        if (_gid_y19 >= input_height)
            _gid_y19 = input_height - 1;
        if (_gid_x19 < 0)
            _gid_x19 = 0;
        if (_gid_y19 < 0)
            _gid_y19 = 0;
        _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y19) * input_stride + _gid_x19);
        int _gid_x20 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y20 = gid_y + 6 * (int)blockDim.y + (-2);
        if (_gid_x20 >= input_width)
            _gid_x20 = input_width - 1;
        if (_gid_y20 >= input_height)
            _gid_y20 = input_height - 1;
        if (_gid_x20 < 0)
            _gid_x20 = 0;
        if (_gid_y20 < 0)
            _gid_y20 = 0;
        _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y20) * input_stride + _gid_x20);
        int _gid_x21 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y21 = gid_y + 7 * (int)blockDim.y + (-2);
        if (_gid_x21 >= input_width)
            _gid_x21 = input_width - 1;
        if (_gid_y21 >= input_height)
            _gid_y21 = input_height - 1;
        if (_gid_x21 < 0)
            _gid_x21 = 0;
        if (_gid_y21 < 0)
            _gid_y21 = 0;
        _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y21) * input_stride + _gid_x21);
        int _gid_x22 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y22 = gid_y + 7 * (int)blockDim.y + (-2);
        if (_gid_x22 >= input_width)
            _gid_x22 = input_width - 1;
        if (_gid_y22 >= input_height)
            _gid_y22 = input_height - 1;
        if (_gid_x22 < 0)
            _gid_x22 = 0;
        if (_gid_y22 < 0)
            _gid_y22 = 0;
        _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y22) * input_stride + _gid_x22);
        int _gid_x23 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y23 = gid_y + 7 * (int)blockDim.y + (-2);
        if (_gid_x23 >= input_width)
            _gid_x23 = input_width - 1;
        if (_gid_y23 >= input_height)
            _gid_y23 = input_height - 1;
        if (_gid_x23 < 0)
            _gid_x23 = 0;
        if (_gid_y23 < 0)
            _gid_y23 = 0;
        _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y23) * input_stride + _gid_x23);
        int _gid_x24 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y24 = gid_y + 8 * (int)blockDim.y + (-2);
        if (_gid_x24 >= input_width)
            _gid_x24 = input_width - 1;
        if (_gid_y24 >= input_height)
            _gid_y24 = input_height - 1;
        if (_gid_x24 < 0)
            _gid_x24 = 0;
        if (_gid_y24 < 0)
            _gid_y24 = 0;
        _smeminput[(int)threadIdx.y + 8 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y24) * input_stride + _gid_x24);
        int _gid_x25 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y25 = gid_y + 8 * (int)blockDim.y + (-2);
        if (_gid_x25 >= input_width)
            _gid_x25 = input_width - 1;
        if (_gid_y25 >= input_height)
            _gid_y25 = input_height - 1;
        if (_gid_x25 < 0)
            _gid_x25 = 0;
        if (_gid_y25 < 0)
            _gid_y25 = 0;
        _smeminput[(int)threadIdx.y + 8 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y25) * input_stride + _gid_x25);
        int _gid_x26 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y26 = gid_y + 8 * (int)blockDim.y + (-2);
        if (_gid_x26 >= input_width)
            _gid_x26 = input_width - 1;
        if (_gid_y26 >= input_height)
            _gid_y26 = input_height - 1;
        if (_gid_x26 < 0)
            _gid_x26 = 0;
        if (_gid_y26 < 0)
            _gid_y26 = 0;
        _smeminput[(int)threadIdx.y + 8 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y26) * input_stride + _gid_x26);
        int _gid_x27 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y27 = gid_y + 9 * (int)blockDim.y + (-2);
        if (_gid_x27 >= input_width)
            _gid_x27 = input_width - 1;
        if (_gid_y27 >= input_height)
            _gid_y27 = input_height - 1;
        if (_gid_x27 < 0)
            _gid_x27 = 0;
        if (_gid_y27 < 0)
            _gid_y27 = 0;
        _smeminput[(int)threadIdx.y + 9 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y27) * input_stride + _gid_x27);
        int _gid_x28 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y28 = gid_y + 9 * (int)blockDim.y + (-2);
        if (_gid_x28 >= input_width)
            _gid_x28 = input_width - 1;
        if (_gid_y28 >= input_height)
            _gid_y28 = input_height - 1;
        if (_gid_x28 < 0)
            _gid_x28 = 0;
        if (_gid_y28 < 0)
            _gid_y28 = 0;
        _smeminput[(int)threadIdx.y + 9 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y28) * input_stride + _gid_x28);
        int _gid_x29 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y29 = gid_y + 9 * (int)blockDim.y + (-2);
        if (_gid_x29 >= input_width)
            _gid_x29 = input_width - 1;
        if (_gid_y29 >= input_height)
            _gid_y29 = input_height - 1;
        if (_gid_x29 < 0)
            _gid_x29 = 0;
        if (_gid_y29 < 0)
            _gid_y29 = 0;
        _smeminput[(int)threadIdx.y + 9 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y29) * input_stride + _gid_x29);
        __syncthreads();
        if (gid_x < iter_width) {
            if (gid_y < iter_height) {
                const int anchor_x = size_x >> 1;
                const int anchor_y = size_y >> 1;
                float sum = 0.5F;
                for (int yf = -anchor_y; yf <= anchor_y; ++yf) {
                    for (int xf = -anchor_x; xf <= anchor_x; ++xf) {
                        sum += _constmaskXY[yf + 2][xf + 2] * _smeminput[(int)threadIdx.y + yf + 2][(int)threadIdx.x + xf + 32];
                    }
                }
                iter[(gid_y) * iter_stride + gid_x] = (uchar)sum;
            }
        }
        if (gid_x < iter_width) {
            if (gid_y + 1 * (int)blockDim.y < iter_height) {
                const int anchor_x = size_x >> 1;
                const int anchor_y = size_y >> 1;
                float sum = 0.5F;
                for (int yf = -anchor_y; yf <= anchor_y; ++yf) {
                    for (int xf = -anchor_x; xf <= anchor_x; ++xf) {
                        sum += _constmaskXY[yf + 2][xf + 2] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + yf + 2][(int)threadIdx.x + xf + 32];
                    }
                }
                iter[(gid_y + 1 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)sum;
            }
        }
        if (gid_x < iter_width) {
            if (gid_y + 2 * (int)blockDim.y < iter_height) {
                const int anchor_x = size_x >> 1;
                const int anchor_y = size_y >> 1;
                float sum = 0.5F;
                for (int yf = -anchor_y; yf <= anchor_y; ++yf) {
                    for (int xf = -anchor_x; xf <= anchor_x; ++xf) {
                        sum += _constmaskXY[yf + 2][xf + 2] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + yf + 2][(int)threadIdx.x + xf + 32];
                    }
                }
                iter[(gid_y + 2 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)sum;
            }
        }
        if (gid_x < iter_width) {
            if (gid_y + 3 * (int)blockDim.y < iter_height) {
                const int anchor_x = size_x >> 1;
                const int anchor_y = size_y >> 1;
                float sum = 0.5F;
                for (int yf = -anchor_y; yf <= anchor_y; ++yf) {
                    for (int xf = -anchor_x; xf <= anchor_x; ++xf) {
                        sum += _constmaskXY[yf + 2][xf + 2] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + yf + 2][(int)threadIdx.x + xf + 32];
                    }
                }
                iter[(gid_y + 3 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)sum;
            }
        }
        if (gid_x < iter_width) {
            if (gid_y + 4 * (int)blockDim.y < iter_height) {
                const int anchor_x = size_x >> 1;
                const int anchor_y = size_y >> 1;
                float sum = 0.5F;
                for (int yf = -anchor_y; yf <= anchor_y; ++yf) {
                    for (int xf = -anchor_x; xf <= anchor_x; ++xf) {
                        sum += _constmaskXY[yf + 2][xf + 2] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + yf + 2][(int)threadIdx.x + xf + 32];
                    }
                }
                iter[(gid_y + 4 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)sum;
            }
        }
        if (gid_x < iter_width) {
            if (gid_y + 5 * (int)blockDim.y < iter_height) {
                const int anchor_x = size_x >> 1;
                const int anchor_y = size_y >> 1;
                float sum = 0.5F;
                for (int yf = -anchor_y; yf <= anchor_y; ++yf) {
                    for (int xf = -anchor_x; xf <= anchor_x; ++xf) {
                        sum += _constmaskXY[yf + 2][xf + 2] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + yf + 2][(int)threadIdx.x + xf + 32];
                    }
                }
                iter[(gid_y + 5 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)sum;
            }
        }
        if (gid_x < iter_width) {
            if (gid_y + 6 * (int)blockDim.y < iter_height) {
                const int anchor_x = size_x >> 1;
                const int anchor_y = size_y >> 1;
                float sum = 0.5F;
                for (int yf = -anchor_y; yf <= anchor_y; ++yf) {
                    for (int xf = -anchor_x; xf <= anchor_x; ++xf) {
                        sum += _constmaskXY[yf + 2][xf + 2] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + yf + 2][(int)threadIdx.x + xf + 32];
                    }
                }
                iter[(gid_y + 6 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)sum;
            }
        }
        if (gid_x < iter_width) {
            if (gid_y + 7 * (int)blockDim.y < iter_height) {
                const int anchor_x = size_x >> 1;
                const int anchor_y = size_y >> 1;
                float sum = 0.5F;
                for (int yf = -anchor_y; yf <= anchor_y; ++yf) {
                    for (int xf = -anchor_x; xf <= anchor_x; ++xf) {
                        sum += _constmaskXY[yf + 2][xf + 2] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + yf + 2][(int)threadIdx.x + xf + 32];
                    }
                }
                iter[(gid_y + 7 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)sum;
            }
        }
    }
    goto BH_EXIT;
  BH_TL:
    {
        int _gid_x30 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y30 = gid_y + (-2);
        if (_gid_x30 < 0)
            _gid_x30 = 0;
        if (_gid_y30 < 0)
            _gid_y30 = 0;
        _smeminput[(int)threadIdx.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y30) * input_stride + _gid_x30);
        int _gid_x31 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y31 = gid_y + (-2);
        if (_gid_x31 < 0)
            _gid_x31 = 0;
        if (_gid_y31 < 0)
            _gid_y31 = 0;
        _smeminput[(int)threadIdx.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y31) * input_stride + _gid_x31);
        int _gid_x32 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y32 = gid_y + (-2);
        if (_gid_x32 < 0)
            _gid_x32 = 0;
        if (_gid_y32 < 0)
            _gid_y32 = 0;
        _smeminput[(int)threadIdx.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y32) * input_stride + _gid_x32);
        int _gid_x33 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y33 = gid_y + 1 * (int)blockDim.y + (-2);
        if (_gid_x33 < 0)
            _gid_x33 = 0;
        if (_gid_y33 < 0)
            _gid_y33 = 0;
        _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y33) * input_stride + _gid_x33);
        int _gid_x34 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y34 = gid_y + 1 * (int)blockDim.y + (-2);
        if (_gid_x34 < 0)
            _gid_x34 = 0;
        if (_gid_y34 < 0)
            _gid_y34 = 0;
        _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y34) * input_stride + _gid_x34);
        int _gid_x35 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y35 = gid_y + 1 * (int)blockDim.y + (-2);
        if (_gid_x35 < 0)
            _gid_x35 = 0;
        if (_gid_y35 < 0)
            _gid_y35 = 0;
        _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y35) * input_stride + _gid_x35);
        int _gid_x36 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y36 = gid_y + 2 * (int)blockDim.y + (-2);
        if (_gid_x36 < 0)
            _gid_x36 = 0;
        if (_gid_y36 < 0)
            _gid_y36 = 0;
        _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y36) * input_stride + _gid_x36);
        int _gid_x37 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y37 = gid_y + 2 * (int)blockDim.y + (-2);
        if (_gid_x37 < 0)
            _gid_x37 = 0;
        if (_gid_y37 < 0)
            _gid_y37 = 0;
        _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y37) * input_stride + _gid_x37);
        int _gid_x38 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y38 = gid_y + 2 * (int)blockDim.y + (-2);
        if (_gid_x38 < 0)
            _gid_x38 = 0;
        if (_gid_y38 < 0)
            _gid_y38 = 0;
        _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y38) * input_stride + _gid_x38);
        int _gid_x39 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y39 = gid_y + 3 * (int)blockDim.y + (-2);
        if (_gid_x39 < 0)
            _gid_x39 = 0;
        if (_gid_y39 < 0)
            _gid_y39 = 0;
        _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y39) * input_stride + _gid_x39);
        int _gid_x40 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y40 = gid_y + 3 * (int)blockDim.y + (-2);
        if (_gid_x40 < 0)
            _gid_x40 = 0;
        if (_gid_y40 < 0)
            _gid_y40 = 0;
        _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y40) * input_stride + _gid_x40);
        int _gid_x41 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y41 = gid_y + 3 * (int)blockDim.y + (-2);
        if (_gid_x41 < 0)
            _gid_x41 = 0;
        if (_gid_y41 < 0)
            _gid_y41 = 0;
        _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y41) * input_stride + _gid_x41);
        int _gid_x42 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y42 = gid_y + 4 * (int)blockDim.y + (-2);
        if (_gid_x42 < 0)
            _gid_x42 = 0;
        if (_gid_y42 < 0)
            _gid_y42 = 0;
        _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y42) * input_stride + _gid_x42);
        int _gid_x43 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y43 = gid_y + 4 * (int)blockDim.y + (-2);
        if (_gid_x43 < 0)
            _gid_x43 = 0;
        if (_gid_y43 < 0)
            _gid_y43 = 0;
        _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y43) * input_stride + _gid_x43);
        int _gid_x44 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y44 = gid_y + 4 * (int)blockDim.y + (-2);
        if (_gid_x44 < 0)
            _gid_x44 = 0;
        if (_gid_y44 < 0)
            _gid_y44 = 0;
        _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y44) * input_stride + _gid_x44);
        int _gid_x45 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y45 = gid_y + 5 * (int)blockDim.y + (-2);
        if (_gid_x45 < 0)
            _gid_x45 = 0;
        if (_gid_y45 < 0)
            _gid_y45 = 0;
        _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y45) * input_stride + _gid_x45);
        int _gid_x46 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y46 = gid_y + 5 * (int)blockDim.y + (-2);
        if (_gid_x46 < 0)
            _gid_x46 = 0;
        if (_gid_y46 < 0)
            _gid_y46 = 0;
        _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y46) * input_stride + _gid_x46);
        int _gid_x47 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y47 = gid_y + 5 * (int)blockDim.y + (-2);
        if (_gid_x47 < 0)
            _gid_x47 = 0;
        if (_gid_y47 < 0)
            _gid_y47 = 0;
        _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y47) * input_stride + _gid_x47);
        int _gid_x48 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y48 = gid_y + 6 * (int)blockDim.y + (-2);
        if (_gid_x48 < 0)
            _gid_x48 = 0;
        if (_gid_y48 < 0)
            _gid_y48 = 0;
        _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y48) * input_stride + _gid_x48);
        int _gid_x49 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y49 = gid_y + 6 * (int)blockDim.y + (-2);
        if (_gid_x49 < 0)
            _gid_x49 = 0;
        if (_gid_y49 < 0)
            _gid_y49 = 0;
        _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y49) * input_stride + _gid_x49);
        int _gid_x50 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y50 = gid_y + 6 * (int)blockDim.y + (-2);
        if (_gid_x50 < 0)
            _gid_x50 = 0;
        if (_gid_y50 < 0)
            _gid_y50 = 0;
        _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y50) * input_stride + _gid_x50);
        int _gid_x51 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y51 = gid_y + 7 * (int)blockDim.y + (-2);
        if (_gid_x51 < 0)
            _gid_x51 = 0;
        if (_gid_y51 < 0)
            _gid_y51 = 0;
        _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y51) * input_stride + _gid_x51);
        int _gid_x52 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y52 = gid_y + 7 * (int)blockDim.y + (-2);
        if (_gid_x52 < 0)
            _gid_x52 = 0;
        if (_gid_y52 < 0)
            _gid_y52 = 0;
        _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y52) * input_stride + _gid_x52);
        int _gid_x53 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y53 = gid_y + 7 * (int)blockDim.y + (-2);
        if (_gid_x53 < 0)
            _gid_x53 = 0;
        if (_gid_y53 < 0)
            _gid_y53 = 0;
        _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y53) * input_stride + _gid_x53);
        int _gid_x54 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y54 = gid_y + 8 * (int)blockDim.y + (-2);
        if (_gid_x54 < 0)
            _gid_x54 = 0;
        if (_gid_y54 < 0)
            _gid_y54 = 0;
        _smeminput[(int)threadIdx.y + 8 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y54) * input_stride + _gid_x54);
        int _gid_x55 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y55 = gid_y + 8 * (int)blockDim.y + (-2);
        if (_gid_x55 < 0)
            _gid_x55 = 0;
        if (_gid_y55 < 0)
            _gid_y55 = 0;
        _smeminput[(int)threadIdx.y + 8 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y55) * input_stride + _gid_x55);
        int _gid_x56 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y56 = gid_y + 8 * (int)blockDim.y + (-2);
        if (_gid_x56 < 0)
            _gid_x56 = 0;
        if (_gid_y56 < 0)
            _gid_y56 = 0;
        _smeminput[(int)threadIdx.y + 8 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y56) * input_stride + _gid_x56);
        int _gid_x57 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y57 = gid_y + 9 * (int)blockDim.y + (-2);
        if (_gid_x57 < 0)
            _gid_x57 = 0;
        if (_gid_y57 < 0)
            _gid_y57 = 0;
        _smeminput[(int)threadIdx.y + 9 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y57) * input_stride + _gid_x57);
        int _gid_x58 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y58 = gid_y + 9 * (int)blockDim.y + (-2);
        if (_gid_x58 < 0)
            _gid_x58 = 0;
        if (_gid_y58 < 0)
            _gid_y58 = 0;
        _smeminput[(int)threadIdx.y + 9 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y58) * input_stride + _gid_x58);
        int _gid_x59 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y59 = gid_y + 9 * (int)blockDim.y + (-2);
        if (_gid_x59 < 0)
            _gid_x59 = 0;
        if (_gid_y59 < 0)
            _gid_y59 = 0;
        _smeminput[(int)threadIdx.y + 9 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y59) * input_stride + _gid_x59);
        __syncthreads();
        {
            const int anchor_x = size_x >> 1;
            const int anchor_y = size_y >> 1;
            float sum = 0.5F;
            for (int yf = -anchor_y; yf <= anchor_y; ++yf) {
                for (int xf = -anchor_x; xf <= anchor_x; ++xf) {
                    sum += _constmaskXY[yf + 2][xf + 2] * _smeminput[(int)threadIdx.y + yf + 2][(int)threadIdx.x + xf + 32];
                }
            }
            iter[(gid_y) * iter_stride + gid_x] = (uchar)sum;
        }
        {
            const int anchor_x = size_x >> 1;
            const int anchor_y = size_y >> 1;
            float sum = 0.5F;
            for (int yf = -anchor_y; yf <= anchor_y; ++yf) {
                for (int xf = -anchor_x; xf <= anchor_x; ++xf) {
                    sum += _constmaskXY[yf + 2][xf + 2] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + yf + 2][(int)threadIdx.x + xf + 32];
                }
            }
            iter[(gid_y + 1 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)sum;
        }
        {
            const int anchor_x = size_x >> 1;
            const int anchor_y = size_y >> 1;
            float sum = 0.5F;
            for (int yf = -anchor_y; yf <= anchor_y; ++yf) {
                for (int xf = -anchor_x; xf <= anchor_x; ++xf) {
                    sum += _constmaskXY[yf + 2][xf + 2] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + yf + 2][(int)threadIdx.x + xf + 32];
                }
            }
            iter[(gid_y + 2 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)sum;
        }
        {
            const int anchor_x = size_x >> 1;
            const int anchor_y = size_y >> 1;
            float sum = 0.5F;
            for (int yf = -anchor_y; yf <= anchor_y; ++yf) {
                for (int xf = -anchor_x; xf <= anchor_x; ++xf) {
                    sum += _constmaskXY[yf + 2][xf + 2] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + yf + 2][(int)threadIdx.x + xf + 32];
                }
            }
            iter[(gid_y + 3 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)sum;
        }
        {
            const int anchor_x = size_x >> 1;
            const int anchor_y = size_y >> 1;
            float sum = 0.5F;
            for (int yf = -anchor_y; yf <= anchor_y; ++yf) {
                for (int xf = -anchor_x; xf <= anchor_x; ++xf) {
                    sum += _constmaskXY[yf + 2][xf + 2] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + yf + 2][(int)threadIdx.x + xf + 32];
                }
            }
            iter[(gid_y + 4 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)sum;
        }
        {
            const int anchor_x = size_x >> 1;
            const int anchor_y = size_y >> 1;
            float sum = 0.5F;
            for (int yf = -anchor_y; yf <= anchor_y; ++yf) {
                for (int xf = -anchor_x; xf <= anchor_x; ++xf) {
                    sum += _constmaskXY[yf + 2][xf + 2] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + yf + 2][(int)threadIdx.x + xf + 32];
                }
            }
            iter[(gid_y + 5 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)sum;
        }
        {
            const int anchor_x = size_x >> 1;
            const int anchor_y = size_y >> 1;
            float sum = 0.5F;
            for (int yf = -anchor_y; yf <= anchor_y; ++yf) {
                for (int xf = -anchor_x; xf <= anchor_x; ++xf) {
                    sum += _constmaskXY[yf + 2][xf + 2] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + yf + 2][(int)threadIdx.x + xf + 32];
                }
            }
            iter[(gid_y + 6 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)sum;
        }
        {
            const int anchor_x = size_x >> 1;
            const int anchor_y = size_y >> 1;
            float sum = 0.5F;
            for (int yf = -anchor_y; yf <= anchor_y; ++yf) {
                for (int xf = -anchor_x; xf <= anchor_x; ++xf) {
                    sum += _constmaskXY[yf + 2][xf + 2] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + yf + 2][(int)threadIdx.x + xf + 32];
                }
            }
            iter[(gid_y + 7 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)sum;
        }
    }
    goto BH_EXIT;
  BH_TR:
    {
        int _gid_x60 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y60 = gid_y + (-2);
        if (_gid_x60 >= input_width)
            _gid_x60 = input_width - 1;
        if (_gid_y60 < 0)
            _gid_y60 = 0;
        _smeminput[(int)threadIdx.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y60) * input_stride + _gid_x60);
        int _gid_x61 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y61 = gid_y + (-2);
        if (_gid_x61 >= input_width)
            _gid_x61 = input_width - 1;
        if (_gid_y61 < 0)
            _gid_y61 = 0;
        _smeminput[(int)threadIdx.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y61) * input_stride + _gid_x61);
        int _gid_x62 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y62 = gid_y + (-2);
        if (_gid_x62 >= input_width)
            _gid_x62 = input_width - 1;
        if (_gid_y62 < 0)
            _gid_y62 = 0;
        _smeminput[(int)threadIdx.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y62) * input_stride + _gid_x62);
        int _gid_x63 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y63 = gid_y + 1 * (int)blockDim.y + (-2);
        if (_gid_x63 >= input_width)
            _gid_x63 = input_width - 1;
        if (_gid_y63 < 0)
            _gid_y63 = 0;
        _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y63) * input_stride + _gid_x63);
        int _gid_x64 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y64 = gid_y + 1 * (int)blockDim.y + (-2);
        if (_gid_x64 >= input_width)
            _gid_x64 = input_width - 1;
        if (_gid_y64 < 0)
            _gid_y64 = 0;
        _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y64) * input_stride + _gid_x64);
        int _gid_x65 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y65 = gid_y + 1 * (int)blockDim.y + (-2);
        if (_gid_x65 >= input_width)
            _gid_x65 = input_width - 1;
        if (_gid_y65 < 0)
            _gid_y65 = 0;
        _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y65) * input_stride + _gid_x65);
        int _gid_x66 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y66 = gid_y + 2 * (int)blockDim.y + (-2);
        if (_gid_x66 >= input_width)
            _gid_x66 = input_width - 1;
        if (_gid_y66 < 0)
            _gid_y66 = 0;
        _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y66) * input_stride + _gid_x66);
        int _gid_x67 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y67 = gid_y + 2 * (int)blockDim.y + (-2);
        if (_gid_x67 >= input_width)
            _gid_x67 = input_width - 1;
        if (_gid_y67 < 0)
            _gid_y67 = 0;
        _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y67) * input_stride + _gid_x67);
        int _gid_x68 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y68 = gid_y + 2 * (int)blockDim.y + (-2);
        if (_gid_x68 >= input_width)
            _gid_x68 = input_width - 1;
        if (_gid_y68 < 0)
            _gid_y68 = 0;
        _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y68) * input_stride + _gid_x68);
        int _gid_x69 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y69 = gid_y + 3 * (int)blockDim.y + (-2);
        if (_gid_x69 >= input_width)
            _gid_x69 = input_width - 1;
        if (_gid_y69 < 0)
            _gid_y69 = 0;
        _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y69) * input_stride + _gid_x69);
        int _gid_x70 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y70 = gid_y + 3 * (int)blockDim.y + (-2);
        if (_gid_x70 >= input_width)
            _gid_x70 = input_width - 1;
        if (_gid_y70 < 0)
            _gid_y70 = 0;
        _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y70) * input_stride + _gid_x70);
        int _gid_x71 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y71 = gid_y + 3 * (int)blockDim.y + (-2);
        if (_gid_x71 >= input_width)
            _gid_x71 = input_width - 1;
        if (_gid_y71 < 0)
            _gid_y71 = 0;
        _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y71) * input_stride + _gid_x71);
        int _gid_x72 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y72 = gid_y + 4 * (int)blockDim.y + (-2);
        if (_gid_x72 >= input_width)
            _gid_x72 = input_width - 1;
        if (_gid_y72 < 0)
            _gid_y72 = 0;
        _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y72) * input_stride + _gid_x72);
        int _gid_x73 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y73 = gid_y + 4 * (int)blockDim.y + (-2);
        if (_gid_x73 >= input_width)
            _gid_x73 = input_width - 1;
        if (_gid_y73 < 0)
            _gid_y73 = 0;
        _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y73) * input_stride + _gid_x73);
        int _gid_x74 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y74 = gid_y + 4 * (int)blockDim.y + (-2);
        if (_gid_x74 >= input_width)
            _gid_x74 = input_width - 1;
        if (_gid_y74 < 0)
            _gid_y74 = 0;
        _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y74) * input_stride + _gid_x74);
        int _gid_x75 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y75 = gid_y + 5 * (int)blockDim.y + (-2);
        if (_gid_x75 >= input_width)
            _gid_x75 = input_width - 1;
        if (_gid_y75 < 0)
            _gid_y75 = 0;
        _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y75) * input_stride + _gid_x75);
        int _gid_x76 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y76 = gid_y + 5 * (int)blockDim.y + (-2);
        if (_gid_x76 >= input_width)
            _gid_x76 = input_width - 1;
        if (_gid_y76 < 0)
            _gid_y76 = 0;
        _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y76) * input_stride + _gid_x76);
        int _gid_x77 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y77 = gid_y + 5 * (int)blockDim.y + (-2);
        if (_gid_x77 >= input_width)
            _gid_x77 = input_width - 1;
        if (_gid_y77 < 0)
            _gid_y77 = 0;
        _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y77) * input_stride + _gid_x77);
        int _gid_x78 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y78 = gid_y + 6 * (int)blockDim.y + (-2);
        if (_gid_x78 >= input_width)
            _gid_x78 = input_width - 1;
        if (_gid_y78 < 0)
            _gid_y78 = 0;
        _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y78) * input_stride + _gid_x78);
        int _gid_x79 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y79 = gid_y + 6 * (int)blockDim.y + (-2);
        if (_gid_x79 >= input_width)
            _gid_x79 = input_width - 1;
        if (_gid_y79 < 0)
            _gid_y79 = 0;
        _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y79) * input_stride + _gid_x79);
        int _gid_x80 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y80 = gid_y + 6 * (int)blockDim.y + (-2);
        if (_gid_x80 >= input_width)
            _gid_x80 = input_width - 1;
        if (_gid_y80 < 0)
            _gid_y80 = 0;
        _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y80) * input_stride + _gid_x80);
        int _gid_x81 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y81 = gid_y + 7 * (int)blockDim.y + (-2);
        if (_gid_x81 >= input_width)
            _gid_x81 = input_width - 1;
        if (_gid_y81 < 0)
            _gid_y81 = 0;
        _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y81) * input_stride + _gid_x81);
        int _gid_x82 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y82 = gid_y + 7 * (int)blockDim.y + (-2);
        if (_gid_x82 >= input_width)
            _gid_x82 = input_width - 1;
        if (_gid_y82 < 0)
            _gid_y82 = 0;
        _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y82) * input_stride + _gid_x82);
        int _gid_x83 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y83 = gid_y + 7 * (int)blockDim.y + (-2);
        if (_gid_x83 >= input_width)
            _gid_x83 = input_width - 1;
        if (_gid_y83 < 0)
            _gid_y83 = 0;
        _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y83) * input_stride + _gid_x83);
        int _gid_x84 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y84 = gid_y + 8 * (int)blockDim.y + (-2);
        if (_gid_x84 >= input_width)
            _gid_x84 = input_width - 1;
        if (_gid_y84 < 0)
            _gid_y84 = 0;
        _smeminput[(int)threadIdx.y + 8 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y84) * input_stride + _gid_x84);
        int _gid_x85 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y85 = gid_y + 8 * (int)blockDim.y + (-2);
        if (_gid_x85 >= input_width)
            _gid_x85 = input_width - 1;
        if (_gid_y85 < 0)
            _gid_y85 = 0;
        _smeminput[(int)threadIdx.y + 8 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y85) * input_stride + _gid_x85);
        int _gid_x86 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y86 = gid_y + 8 * (int)blockDim.y + (-2);
        if (_gid_x86 >= input_width)
            _gid_x86 = input_width - 1;
        if (_gid_y86 < 0)
            _gid_y86 = 0;
        _smeminput[(int)threadIdx.y + 8 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y86) * input_stride + _gid_x86);
        int _gid_x87 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y87 = gid_y + 9 * (int)blockDim.y + (-2);
        if (_gid_x87 >= input_width)
            _gid_x87 = input_width - 1;
        if (_gid_y87 < 0)
            _gid_y87 = 0;
        _smeminput[(int)threadIdx.y + 9 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y87) * input_stride + _gid_x87);
        int _gid_x88 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y88 = gid_y + 9 * (int)blockDim.y + (-2);
        if (_gid_x88 >= input_width)
            _gid_x88 = input_width - 1;
        if (_gid_y88 < 0)
            _gid_y88 = 0;
        _smeminput[(int)threadIdx.y + 9 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y88) * input_stride + _gid_x88);
        int _gid_x89 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y89 = gid_y + 9 * (int)blockDim.y + (-2);
        if (_gid_x89 >= input_width)
            _gid_x89 = input_width - 1;
        if (_gid_y89 < 0)
            _gid_y89 = 0;
        _smeminput[(int)threadIdx.y + 9 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y89) * input_stride + _gid_x89);
        __syncthreads();
        if (gid_x < iter_width) {
            {
                const int anchor_x = size_x >> 1;
                const int anchor_y = size_y >> 1;
                float sum = 0.5F;
                for (int yf = -anchor_y; yf <= anchor_y; ++yf) {
                    for (int xf = -anchor_x; xf <= anchor_x; ++xf) {
                        sum += _constmaskXY[yf + 2][xf + 2] * _smeminput[(int)threadIdx.y + yf + 2][(int)threadIdx.x + xf + 32];
                    }
                }
                iter[(gid_y) * iter_stride + gid_x] = (uchar)sum;
            }
        }
        if (gid_x < iter_width) {
            {
                const int anchor_x = size_x >> 1;
                const int anchor_y = size_y >> 1;
                float sum = 0.5F;
                for (int yf = -anchor_y; yf <= anchor_y; ++yf) {
                    for (int xf = -anchor_x; xf <= anchor_x; ++xf) {
                        sum += _constmaskXY[yf + 2][xf + 2] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + yf + 2][(int)threadIdx.x + xf + 32];
                    }
                }
                iter[(gid_y + 1 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)sum;
            }
        }
        if (gid_x < iter_width) {
            {
                const int anchor_x = size_x >> 1;
                const int anchor_y = size_y >> 1;
                float sum = 0.5F;
                for (int yf = -anchor_y; yf <= anchor_y; ++yf) {
                    for (int xf = -anchor_x; xf <= anchor_x; ++xf) {
                        sum += _constmaskXY[yf + 2][xf + 2] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + yf + 2][(int)threadIdx.x + xf + 32];
                    }
                }
                iter[(gid_y + 2 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)sum;
            }
        }
        if (gid_x < iter_width) {
            {
                const int anchor_x = size_x >> 1;
                const int anchor_y = size_y >> 1;
                float sum = 0.5F;
                for (int yf = -anchor_y; yf <= anchor_y; ++yf) {
                    for (int xf = -anchor_x; xf <= anchor_x; ++xf) {
                        sum += _constmaskXY[yf + 2][xf + 2] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + yf + 2][(int)threadIdx.x + xf + 32];
                    }
                }
                iter[(gid_y + 3 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)sum;
            }
        }
        if (gid_x < iter_width) {
            {
                const int anchor_x = size_x >> 1;
                const int anchor_y = size_y >> 1;
                float sum = 0.5F;
                for (int yf = -anchor_y; yf <= anchor_y; ++yf) {
                    for (int xf = -anchor_x; xf <= anchor_x; ++xf) {
                        sum += _constmaskXY[yf + 2][xf + 2] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + yf + 2][(int)threadIdx.x + xf + 32];
                    }
                }
                iter[(gid_y + 4 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)sum;
            }
        }
        if (gid_x < iter_width) {
            {
                const int anchor_x = size_x >> 1;
                const int anchor_y = size_y >> 1;
                float sum = 0.5F;
                for (int yf = -anchor_y; yf <= anchor_y; ++yf) {
                    for (int xf = -anchor_x; xf <= anchor_x; ++xf) {
                        sum += _constmaskXY[yf + 2][xf + 2] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + yf + 2][(int)threadIdx.x + xf + 32];
                    }
                }
                iter[(gid_y + 5 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)sum;
            }
        }
        if (gid_x < iter_width) {
            {
                const int anchor_x = size_x >> 1;
                const int anchor_y = size_y >> 1;
                float sum = 0.5F;
                for (int yf = -anchor_y; yf <= anchor_y; ++yf) {
                    for (int xf = -anchor_x; xf <= anchor_x; ++xf) {
                        sum += _constmaskXY[yf + 2][xf + 2] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + yf + 2][(int)threadIdx.x + xf + 32];
                    }
                }
                iter[(gid_y + 6 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)sum;
            }
        }
        if (gid_x < iter_width) {
            {
                const int anchor_x = size_x >> 1;
                const int anchor_y = size_y >> 1;
                float sum = 0.5F;
                for (int yf = -anchor_y; yf <= anchor_y; ++yf) {
                    for (int xf = -anchor_x; xf <= anchor_x; ++xf) {
                        sum += _constmaskXY[yf + 2][xf + 2] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + yf + 2][(int)threadIdx.x + xf + 32];
                    }
                }
                iter[(gid_y + 7 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)sum;
            }
        }
    }
    goto BH_EXIT;
  BH_T:
    {
        int _gid_x90 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y90 = gid_y + (-2);
        if (_gid_y90 < 0)
            _gid_y90 = 0;
        _smeminput[(int)threadIdx.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y90) * input_stride + _gid_x90);
        int _gid_x91 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y91 = gid_y + (-2);
        if (_gid_y91 < 0)
            _gid_y91 = 0;
        _smeminput[(int)threadIdx.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y91) * input_stride + _gid_x91);
        int _gid_x92 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y92 = gid_y + (-2);
        if (_gid_y92 < 0)
            _gid_y92 = 0;
        _smeminput[(int)threadIdx.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y92) * input_stride + _gid_x92);
        int _gid_x93 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y93 = gid_y + 1 * (int)blockDim.y + (-2);
        if (_gid_y93 < 0)
            _gid_y93 = 0;
        _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y93) * input_stride + _gid_x93);
        int _gid_x94 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y94 = gid_y + 1 * (int)blockDim.y + (-2);
        if (_gid_y94 < 0)
            _gid_y94 = 0;
        _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y94) * input_stride + _gid_x94);
        int _gid_x95 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y95 = gid_y + 1 * (int)blockDim.y + (-2);
        if (_gid_y95 < 0)
            _gid_y95 = 0;
        _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y95) * input_stride + _gid_x95);
        int _gid_x96 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y96 = gid_y + 2 * (int)blockDim.y + (-2);
        if (_gid_y96 < 0)
            _gid_y96 = 0;
        _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y96) * input_stride + _gid_x96);
        int _gid_x97 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y97 = gid_y + 2 * (int)blockDim.y + (-2);
        if (_gid_y97 < 0)
            _gid_y97 = 0;
        _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y97) * input_stride + _gid_x97);
        int _gid_x98 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y98 = gid_y + 2 * (int)blockDim.y + (-2);
        if (_gid_y98 < 0)
            _gid_y98 = 0;
        _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y98) * input_stride + _gid_x98);
        int _gid_x99 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y99 = gid_y + 3 * (int)blockDim.y + (-2);
        if (_gid_y99 < 0)
            _gid_y99 = 0;
        _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y99) * input_stride + _gid_x99);
        int _gid_x100 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y100 = gid_y + 3 * (int)blockDim.y + (-2);
        if (_gid_y100 < 0)
            _gid_y100 = 0;
        _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y100) * input_stride + _gid_x100);
        int _gid_x101 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y101 = gid_y + 3 * (int)blockDim.y + (-2);
        if (_gid_y101 < 0)
            _gid_y101 = 0;
        _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y101) * input_stride + _gid_x101);
        int _gid_x102 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y102 = gid_y + 4 * (int)blockDim.y + (-2);
        if (_gid_y102 < 0)
            _gid_y102 = 0;
        _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y102) * input_stride + _gid_x102);
        int _gid_x103 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y103 = gid_y + 4 * (int)blockDim.y + (-2);
        if (_gid_y103 < 0)
            _gid_y103 = 0;
        _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y103) * input_stride + _gid_x103);
        int _gid_x104 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y104 = gid_y + 4 * (int)blockDim.y + (-2);
        if (_gid_y104 < 0)
            _gid_y104 = 0;
        _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y104) * input_stride + _gid_x104);
        int _gid_x105 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y105 = gid_y + 5 * (int)blockDim.y + (-2);
        if (_gid_y105 < 0)
            _gid_y105 = 0;
        _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y105) * input_stride + _gid_x105);
        int _gid_x106 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y106 = gid_y + 5 * (int)blockDim.y + (-2);
        if (_gid_y106 < 0)
            _gid_y106 = 0;
        _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y106) * input_stride + _gid_x106);
        int _gid_x107 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y107 = gid_y + 5 * (int)blockDim.y + (-2);
        if (_gid_y107 < 0)
            _gid_y107 = 0;
        _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y107) * input_stride + _gid_x107);
        int _gid_x108 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y108 = gid_y + 6 * (int)blockDim.y + (-2);
        if (_gid_y108 < 0)
            _gid_y108 = 0;
        _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y108) * input_stride + _gid_x108);
        int _gid_x109 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y109 = gid_y + 6 * (int)blockDim.y + (-2);
        if (_gid_y109 < 0)
            _gid_y109 = 0;
        _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y109) * input_stride + _gid_x109);
        int _gid_x110 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y110 = gid_y + 6 * (int)blockDim.y + (-2);
        if (_gid_y110 < 0)
            _gid_y110 = 0;
        _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y110) * input_stride + _gid_x110);
        int _gid_x111 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y111 = gid_y + 7 * (int)blockDim.y + (-2);
        if (_gid_y111 < 0)
            _gid_y111 = 0;
        _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y111) * input_stride + _gid_x111);
        int _gid_x112 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y112 = gid_y + 7 * (int)blockDim.y + (-2);
        if (_gid_y112 < 0)
            _gid_y112 = 0;
        _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y112) * input_stride + _gid_x112);
        int _gid_x113 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y113 = gid_y + 7 * (int)blockDim.y + (-2);
        if (_gid_y113 < 0)
            _gid_y113 = 0;
        _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y113) * input_stride + _gid_x113);
        int _gid_x114 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y114 = gid_y + 8 * (int)blockDim.y + (-2);
        if (_gid_y114 < 0)
            _gid_y114 = 0;
        _smeminput[(int)threadIdx.y + 8 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y114) * input_stride + _gid_x114);
        int _gid_x115 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y115 = gid_y + 8 * (int)blockDim.y + (-2);
        if (_gid_y115 < 0)
            _gid_y115 = 0;
        _smeminput[(int)threadIdx.y + 8 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y115) * input_stride + _gid_x115);
        int _gid_x116 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y116 = gid_y + 8 * (int)blockDim.y + (-2);
        if (_gid_y116 < 0)
            _gid_y116 = 0;
        _smeminput[(int)threadIdx.y + 8 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y116) * input_stride + _gid_x116);
        int _gid_x117 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y117 = gid_y + 9 * (int)blockDim.y + (-2);
        if (_gid_y117 < 0)
            _gid_y117 = 0;
        _smeminput[(int)threadIdx.y + 9 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y117) * input_stride + _gid_x117);
        int _gid_x118 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y118 = gid_y + 9 * (int)blockDim.y + (-2);
        if (_gid_y118 < 0)
            _gid_y118 = 0;
        _smeminput[(int)threadIdx.y + 9 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y118) * input_stride + _gid_x118);
        int _gid_x119 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y119 = gid_y + 9 * (int)blockDim.y + (-2);
        if (_gid_y119 < 0)
            _gid_y119 = 0;
        _smeminput[(int)threadIdx.y + 9 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y119) * input_stride + _gid_x119);
        __syncthreads();
        {
            const int anchor_x = size_x >> 1;
            const int anchor_y = size_y >> 1;
            float sum = 0.5F;
            for (int yf = -anchor_y; yf <= anchor_y; ++yf) {
                for (int xf = -anchor_x; xf <= anchor_x; ++xf) {
                    sum += _constmaskXY[yf + 2][xf + 2] * _smeminput[(int)threadIdx.y + yf + 2][(int)threadIdx.x + xf + 32];
                }
            }
            iter[(gid_y) * iter_stride + gid_x] = (uchar)sum;
        }
        {
            const int anchor_x = size_x >> 1;
            const int anchor_y = size_y >> 1;
            float sum = 0.5F;
            for (int yf = -anchor_y; yf <= anchor_y; ++yf) {
                for (int xf = -anchor_x; xf <= anchor_x; ++xf) {
                    sum += _constmaskXY[yf + 2][xf + 2] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + yf + 2][(int)threadIdx.x + xf + 32];
                }
            }
            iter[(gid_y + 1 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)sum;
        }
        {
            const int anchor_x = size_x >> 1;
            const int anchor_y = size_y >> 1;
            float sum = 0.5F;
            for (int yf = -anchor_y; yf <= anchor_y; ++yf) {
                for (int xf = -anchor_x; xf <= anchor_x; ++xf) {
                    sum += _constmaskXY[yf + 2][xf + 2] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + yf + 2][(int)threadIdx.x + xf + 32];
                }
            }
            iter[(gid_y + 2 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)sum;
        }
        {
            const int anchor_x = size_x >> 1;
            const int anchor_y = size_y >> 1;
            float sum = 0.5F;
            for (int yf = -anchor_y; yf <= anchor_y; ++yf) {
                for (int xf = -anchor_x; xf <= anchor_x; ++xf) {
                    sum += _constmaskXY[yf + 2][xf + 2] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + yf + 2][(int)threadIdx.x + xf + 32];
                }
            }
            iter[(gid_y + 3 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)sum;
        }
        {
            const int anchor_x = size_x >> 1;
            const int anchor_y = size_y >> 1;
            float sum = 0.5F;
            for (int yf = -anchor_y; yf <= anchor_y; ++yf) {
                for (int xf = -anchor_x; xf <= anchor_x; ++xf) {
                    sum += _constmaskXY[yf + 2][xf + 2] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + yf + 2][(int)threadIdx.x + xf + 32];
                }
            }
            iter[(gid_y + 4 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)sum;
        }
        {
            const int anchor_x = size_x >> 1;
            const int anchor_y = size_y >> 1;
            float sum = 0.5F;
            for (int yf = -anchor_y; yf <= anchor_y; ++yf) {
                for (int xf = -anchor_x; xf <= anchor_x; ++xf) {
                    sum += _constmaskXY[yf + 2][xf + 2] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + yf + 2][(int)threadIdx.x + xf + 32];
                }
            }
            iter[(gid_y + 5 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)sum;
        }
        {
            const int anchor_x = size_x >> 1;
            const int anchor_y = size_y >> 1;
            float sum = 0.5F;
            for (int yf = -anchor_y; yf <= anchor_y; ++yf) {
                for (int xf = -anchor_x; xf <= anchor_x; ++xf) {
                    sum += _constmaskXY[yf + 2][xf + 2] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + yf + 2][(int)threadIdx.x + xf + 32];
                }
            }
            iter[(gid_y + 6 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)sum;
        }
        {
            const int anchor_x = size_x >> 1;
            const int anchor_y = size_y >> 1;
            float sum = 0.5F;
            for (int yf = -anchor_y; yf <= anchor_y; ++yf) {
                for (int xf = -anchor_x; xf <= anchor_x; ++xf) {
                    sum += _constmaskXY[yf + 2][xf + 2] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + yf + 2][(int)threadIdx.x + xf + 32];
                }
            }
            iter[(gid_y + 7 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)sum;
        }
    }
    goto BH_EXIT;
  BH_BL:
    {
        int _gid_x120 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y120 = gid_y + (-2);
        if (_gid_y120 >= input_height)
            _gid_y120 = input_height - 1;
        if (_gid_x120 < 0)
            _gid_x120 = 0;
        _smeminput[(int)threadIdx.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y120) * input_stride + _gid_x120);
        int _gid_x121 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y121 = gid_y + (-2);
        if (_gid_y121 >= input_height)
            _gid_y121 = input_height - 1;
        if (_gid_x121 < 0)
            _gid_x121 = 0;
        _smeminput[(int)threadIdx.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y121) * input_stride + _gid_x121);
        int _gid_x122 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y122 = gid_y + (-2);
        if (_gid_y122 >= input_height)
            _gid_y122 = input_height - 1;
        if (_gid_x122 < 0)
            _gid_x122 = 0;
        _smeminput[(int)threadIdx.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y122) * input_stride + _gid_x122);
        int _gid_x123 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y123 = gid_y + 1 * (int)blockDim.y + (-2);
        if (_gid_y123 >= input_height)
            _gid_y123 = input_height - 1;
        if (_gid_x123 < 0)
            _gid_x123 = 0;
        _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y123) * input_stride + _gid_x123);
        int _gid_x124 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y124 = gid_y + 1 * (int)blockDim.y + (-2);
        if (_gid_y124 >= input_height)
            _gid_y124 = input_height - 1;
        if (_gid_x124 < 0)
            _gid_x124 = 0;
        _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y124) * input_stride + _gid_x124);
        int _gid_x125 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y125 = gid_y + 1 * (int)blockDim.y + (-2);
        if (_gid_y125 >= input_height)
            _gid_y125 = input_height - 1;
        if (_gid_x125 < 0)
            _gid_x125 = 0;
        _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y125) * input_stride + _gid_x125);
        int _gid_x126 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y126 = gid_y + 2 * (int)blockDim.y + (-2);
        if (_gid_y126 >= input_height)
            _gid_y126 = input_height - 1;
        if (_gid_x126 < 0)
            _gid_x126 = 0;
        _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y126) * input_stride + _gid_x126);
        int _gid_x127 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y127 = gid_y + 2 * (int)blockDim.y + (-2);
        if (_gid_y127 >= input_height)
            _gid_y127 = input_height - 1;
        if (_gid_x127 < 0)
            _gid_x127 = 0;
        _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y127) * input_stride + _gid_x127);
        int _gid_x128 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y128 = gid_y + 2 * (int)blockDim.y + (-2);
        if (_gid_y128 >= input_height)
            _gid_y128 = input_height - 1;
        if (_gid_x128 < 0)
            _gid_x128 = 0;
        _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y128) * input_stride + _gid_x128);
        int _gid_x129 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y129 = gid_y + 3 * (int)blockDim.y + (-2);
        if (_gid_y129 >= input_height)
            _gid_y129 = input_height - 1;
        if (_gid_x129 < 0)
            _gid_x129 = 0;
        _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y129) * input_stride + _gid_x129);
        int _gid_x130 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y130 = gid_y + 3 * (int)blockDim.y + (-2);
        if (_gid_y130 >= input_height)
            _gid_y130 = input_height - 1;
        if (_gid_x130 < 0)
            _gid_x130 = 0;
        _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y130) * input_stride + _gid_x130);
        int _gid_x131 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y131 = gid_y + 3 * (int)blockDim.y + (-2);
        if (_gid_y131 >= input_height)
            _gid_y131 = input_height - 1;
        if (_gid_x131 < 0)
            _gid_x131 = 0;
        _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y131) * input_stride + _gid_x131);
        int _gid_x132 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y132 = gid_y + 4 * (int)blockDim.y + (-2);
        if (_gid_y132 >= input_height)
            _gid_y132 = input_height - 1;
        if (_gid_x132 < 0)
            _gid_x132 = 0;
        _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y132) * input_stride + _gid_x132);
        int _gid_x133 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y133 = gid_y + 4 * (int)blockDim.y + (-2);
        if (_gid_y133 >= input_height)
            _gid_y133 = input_height - 1;
        if (_gid_x133 < 0)
            _gid_x133 = 0;
        _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y133) * input_stride + _gid_x133);
        int _gid_x134 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y134 = gid_y + 4 * (int)blockDim.y + (-2);
        if (_gid_y134 >= input_height)
            _gid_y134 = input_height - 1;
        if (_gid_x134 < 0)
            _gid_x134 = 0;
        _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y134) * input_stride + _gid_x134);
        int _gid_x135 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y135 = gid_y + 5 * (int)blockDim.y + (-2);
        if (_gid_y135 >= input_height)
            _gid_y135 = input_height - 1;
        if (_gid_x135 < 0)
            _gid_x135 = 0;
        _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y135) * input_stride + _gid_x135);
        int _gid_x136 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y136 = gid_y + 5 * (int)blockDim.y + (-2);
        if (_gid_y136 >= input_height)
            _gid_y136 = input_height - 1;
        if (_gid_x136 < 0)
            _gid_x136 = 0;
        _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y136) * input_stride + _gid_x136);
        int _gid_x137 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y137 = gid_y + 5 * (int)blockDim.y + (-2);
        if (_gid_y137 >= input_height)
            _gid_y137 = input_height - 1;
        if (_gid_x137 < 0)
            _gid_x137 = 0;
        _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y137) * input_stride + _gid_x137);
        int _gid_x138 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y138 = gid_y + 6 * (int)blockDim.y + (-2);
        if (_gid_y138 >= input_height)
            _gid_y138 = input_height - 1;
        if (_gid_x138 < 0)
            _gid_x138 = 0;
        _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y138) * input_stride + _gid_x138);
        int _gid_x139 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y139 = gid_y + 6 * (int)blockDim.y + (-2);
        if (_gid_y139 >= input_height)
            _gid_y139 = input_height - 1;
        if (_gid_x139 < 0)
            _gid_x139 = 0;
        _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y139) * input_stride + _gid_x139);
        int _gid_x140 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y140 = gid_y + 6 * (int)blockDim.y + (-2);
        if (_gid_y140 >= input_height)
            _gid_y140 = input_height - 1;
        if (_gid_x140 < 0)
            _gid_x140 = 0;
        _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y140) * input_stride + _gid_x140);
        int _gid_x141 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y141 = gid_y + 7 * (int)blockDim.y + (-2);
        if (_gid_y141 >= input_height)
            _gid_y141 = input_height - 1;
        if (_gid_x141 < 0)
            _gid_x141 = 0;
        _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y141) * input_stride + _gid_x141);
        int _gid_x142 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y142 = gid_y + 7 * (int)blockDim.y + (-2);
        if (_gid_y142 >= input_height)
            _gid_y142 = input_height - 1;
        if (_gid_x142 < 0)
            _gid_x142 = 0;
        _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y142) * input_stride + _gid_x142);
        int _gid_x143 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y143 = gid_y + 7 * (int)blockDim.y + (-2);
        if (_gid_y143 >= input_height)
            _gid_y143 = input_height - 1;
        if (_gid_x143 < 0)
            _gid_x143 = 0;
        _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y143) * input_stride + _gid_x143);
        int _gid_x144 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y144 = gid_y + 8 * (int)blockDim.y + (-2);
        if (_gid_y144 >= input_height)
            _gid_y144 = input_height - 1;
        if (_gid_x144 < 0)
            _gid_x144 = 0;
        _smeminput[(int)threadIdx.y + 8 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y144) * input_stride + _gid_x144);
        int _gid_x145 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y145 = gid_y + 8 * (int)blockDim.y + (-2);
        if (_gid_y145 >= input_height)
            _gid_y145 = input_height - 1;
        if (_gid_x145 < 0)
            _gid_x145 = 0;
        _smeminput[(int)threadIdx.y + 8 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y145) * input_stride + _gid_x145);
        int _gid_x146 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y146 = gid_y + 8 * (int)blockDim.y + (-2);
        if (_gid_y146 >= input_height)
            _gid_y146 = input_height - 1;
        if (_gid_x146 < 0)
            _gid_x146 = 0;
        _smeminput[(int)threadIdx.y + 8 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y146) * input_stride + _gid_x146);
        int _gid_x147 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y147 = gid_y + 9 * (int)blockDim.y + (-2);
        if (_gid_y147 >= input_height)
            _gid_y147 = input_height - 1;
        if (_gid_x147 < 0)
            _gid_x147 = 0;
        _smeminput[(int)threadIdx.y + 9 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y147) * input_stride + _gid_x147);
        int _gid_x148 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y148 = gid_y + 9 * (int)blockDim.y + (-2);
        if (_gid_y148 >= input_height)
            _gid_y148 = input_height - 1;
        if (_gid_x148 < 0)
            _gid_x148 = 0;
        _smeminput[(int)threadIdx.y + 9 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y148) * input_stride + _gid_x148);
        int _gid_x149 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y149 = gid_y + 9 * (int)blockDim.y + (-2);
        if (_gid_y149 >= input_height)
            _gid_y149 = input_height - 1;
        if (_gid_x149 < 0)
            _gid_x149 = 0;
        _smeminput[(int)threadIdx.y + 9 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y149) * input_stride + _gid_x149);
        __syncthreads();
        if (gid_y < iter_height) {
            const int anchor_x = size_x >> 1;
            const int anchor_y = size_y >> 1;
            float sum = 0.5F;
            for (int yf = -anchor_y; yf <= anchor_y; ++yf) {
                for (int xf = -anchor_x; xf <= anchor_x; ++xf) {
                    sum += _constmaskXY[yf + 2][xf + 2] * _smeminput[(int)threadIdx.y + yf + 2][(int)threadIdx.x + xf + 32];
                }
            }
            iter[(gid_y) * iter_stride + gid_x] = (uchar)sum;
        }
        if (gid_y + 1 * (int)blockDim.y < iter_height) {
            const int anchor_x = size_x >> 1;
            const int anchor_y = size_y >> 1;
            float sum = 0.5F;
            for (int yf = -anchor_y; yf <= anchor_y; ++yf) {
                for (int xf = -anchor_x; xf <= anchor_x; ++xf) {
                    sum += _constmaskXY[yf + 2][xf + 2] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + yf + 2][(int)threadIdx.x + xf + 32];
                }
            }
            iter[(gid_y + 1 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)sum;
        }
        if (gid_y + 2 * (int)blockDim.y < iter_height) {
            const int anchor_x = size_x >> 1;
            const int anchor_y = size_y >> 1;
            float sum = 0.5F;
            for (int yf = -anchor_y; yf <= anchor_y; ++yf) {
                for (int xf = -anchor_x; xf <= anchor_x; ++xf) {
                    sum += _constmaskXY[yf + 2][xf + 2] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + yf + 2][(int)threadIdx.x + xf + 32];
                }
            }
            iter[(gid_y + 2 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)sum;
        }
        if (gid_y + 3 * (int)blockDim.y < iter_height) {
            const int anchor_x = size_x >> 1;
            const int anchor_y = size_y >> 1;
            float sum = 0.5F;
            for (int yf = -anchor_y; yf <= anchor_y; ++yf) {
                for (int xf = -anchor_x; xf <= anchor_x; ++xf) {
                    sum += _constmaskXY[yf + 2][xf + 2] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + yf + 2][(int)threadIdx.x + xf + 32];
                }
            }
            iter[(gid_y + 3 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)sum;
        }
        if (gid_y + 4 * (int)blockDim.y < iter_height) {
            const int anchor_x = size_x >> 1;
            const int anchor_y = size_y >> 1;
            float sum = 0.5F;
            for (int yf = -anchor_y; yf <= anchor_y; ++yf) {
                for (int xf = -anchor_x; xf <= anchor_x; ++xf) {
                    sum += _constmaskXY[yf + 2][xf + 2] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + yf + 2][(int)threadIdx.x + xf + 32];
                }
            }
            iter[(gid_y + 4 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)sum;
        }
        if (gid_y + 5 * (int)blockDim.y < iter_height) {
            const int anchor_x = size_x >> 1;
            const int anchor_y = size_y >> 1;
            float sum = 0.5F;
            for (int yf = -anchor_y; yf <= anchor_y; ++yf) {
                for (int xf = -anchor_x; xf <= anchor_x; ++xf) {
                    sum += _constmaskXY[yf + 2][xf + 2] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + yf + 2][(int)threadIdx.x + xf + 32];
                }
            }
            iter[(gid_y + 5 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)sum;
        }
        if (gid_y + 6 * (int)blockDim.y < iter_height) {
            const int anchor_x = size_x >> 1;
            const int anchor_y = size_y >> 1;
            float sum = 0.5F;
            for (int yf = -anchor_y; yf <= anchor_y; ++yf) {
                for (int xf = -anchor_x; xf <= anchor_x; ++xf) {
                    sum += _constmaskXY[yf + 2][xf + 2] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + yf + 2][(int)threadIdx.x + xf + 32];
                }
            }
            iter[(gid_y + 6 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)sum;
        }
        if (gid_y + 7 * (int)blockDim.y < iter_height) {
            const int anchor_x = size_x >> 1;
            const int anchor_y = size_y >> 1;
            float sum = 0.5F;
            for (int yf = -anchor_y; yf <= anchor_y; ++yf) {
                for (int xf = -anchor_x; xf <= anchor_x; ++xf) {
                    sum += _constmaskXY[yf + 2][xf + 2] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + yf + 2][(int)threadIdx.x + xf + 32];
                }
            }
            iter[(gid_y + 7 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)sum;
        }
    }
    goto BH_EXIT;
  BH_BR:
    {
        int _gid_x150 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y150 = gid_y + (-2);
        if (_gid_x150 >= input_width)
            _gid_x150 = input_width - 1;
        if (_gid_y150 >= input_height)
            _gid_y150 = input_height - 1;
        _smeminput[(int)threadIdx.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y150) * input_stride + _gid_x150);
        int _gid_x151 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y151 = gid_y + (-2);
        if (_gid_x151 >= input_width)
            _gid_x151 = input_width - 1;
        if (_gid_y151 >= input_height)
            _gid_y151 = input_height - 1;
        _smeminput[(int)threadIdx.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y151) * input_stride + _gid_x151);
        int _gid_x152 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y152 = gid_y + (-2);
        if (_gid_x152 >= input_width)
            _gid_x152 = input_width - 1;
        if (_gid_y152 >= input_height)
            _gid_y152 = input_height - 1;
        _smeminput[(int)threadIdx.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y152) * input_stride + _gid_x152);
        int _gid_x153 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y153 = gid_y + 1 * (int)blockDim.y + (-2);
        if (_gid_x153 >= input_width)
            _gid_x153 = input_width - 1;
        if (_gid_y153 >= input_height)
            _gid_y153 = input_height - 1;
        _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y153) * input_stride + _gid_x153);
        int _gid_x154 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y154 = gid_y + 1 * (int)blockDim.y + (-2);
        if (_gid_x154 >= input_width)
            _gid_x154 = input_width - 1;
        if (_gid_y154 >= input_height)
            _gid_y154 = input_height - 1;
        _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y154) * input_stride + _gid_x154);
        int _gid_x155 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y155 = gid_y + 1 * (int)blockDim.y + (-2);
        if (_gid_x155 >= input_width)
            _gid_x155 = input_width - 1;
        if (_gid_y155 >= input_height)
            _gid_y155 = input_height - 1;
        _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y155) * input_stride + _gid_x155);
        int _gid_x156 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y156 = gid_y + 2 * (int)blockDim.y + (-2);
        if (_gid_x156 >= input_width)
            _gid_x156 = input_width - 1;
        if (_gid_y156 >= input_height)
            _gid_y156 = input_height - 1;
        _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y156) * input_stride + _gid_x156);
        int _gid_x157 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y157 = gid_y + 2 * (int)blockDim.y + (-2);
        if (_gid_x157 >= input_width)
            _gid_x157 = input_width - 1;
        if (_gid_y157 >= input_height)
            _gid_y157 = input_height - 1;
        _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y157) * input_stride + _gid_x157);
        int _gid_x158 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y158 = gid_y + 2 * (int)blockDim.y + (-2);
        if (_gid_x158 >= input_width)
            _gid_x158 = input_width - 1;
        if (_gid_y158 >= input_height)
            _gid_y158 = input_height - 1;
        _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y158) * input_stride + _gid_x158);
        int _gid_x159 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y159 = gid_y + 3 * (int)blockDim.y + (-2);
        if (_gid_x159 >= input_width)
            _gid_x159 = input_width - 1;
        if (_gid_y159 >= input_height)
            _gid_y159 = input_height - 1;
        _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y159) * input_stride + _gid_x159);
        int _gid_x160 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y160 = gid_y + 3 * (int)blockDim.y + (-2);
        if (_gid_x160 >= input_width)
            _gid_x160 = input_width - 1;
        if (_gid_y160 >= input_height)
            _gid_y160 = input_height - 1;
        _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y160) * input_stride + _gid_x160);
        int _gid_x161 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y161 = gid_y + 3 * (int)blockDim.y + (-2);
        if (_gid_x161 >= input_width)
            _gid_x161 = input_width - 1;
        if (_gid_y161 >= input_height)
            _gid_y161 = input_height - 1;
        _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y161) * input_stride + _gid_x161);
        int _gid_x162 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y162 = gid_y + 4 * (int)blockDim.y + (-2);
        if (_gid_x162 >= input_width)
            _gid_x162 = input_width - 1;
        if (_gid_y162 >= input_height)
            _gid_y162 = input_height - 1;
        _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y162) * input_stride + _gid_x162);
        int _gid_x163 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y163 = gid_y + 4 * (int)blockDim.y + (-2);
        if (_gid_x163 >= input_width)
            _gid_x163 = input_width - 1;
        if (_gid_y163 >= input_height)
            _gid_y163 = input_height - 1;
        _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y163) * input_stride + _gid_x163);
        int _gid_x164 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y164 = gid_y + 4 * (int)blockDim.y + (-2);
        if (_gid_x164 >= input_width)
            _gid_x164 = input_width - 1;
        if (_gid_y164 >= input_height)
            _gid_y164 = input_height - 1;
        _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y164) * input_stride + _gid_x164);
        int _gid_x165 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y165 = gid_y + 5 * (int)blockDim.y + (-2);
        if (_gid_x165 >= input_width)
            _gid_x165 = input_width - 1;
        if (_gid_y165 >= input_height)
            _gid_y165 = input_height - 1;
        _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y165) * input_stride + _gid_x165);
        int _gid_x166 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y166 = gid_y + 5 * (int)blockDim.y + (-2);
        if (_gid_x166 >= input_width)
            _gid_x166 = input_width - 1;
        if (_gid_y166 >= input_height)
            _gid_y166 = input_height - 1;
        _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y166) * input_stride + _gid_x166);
        int _gid_x167 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y167 = gid_y + 5 * (int)blockDim.y + (-2);
        if (_gid_x167 >= input_width)
            _gid_x167 = input_width - 1;
        if (_gid_y167 >= input_height)
            _gid_y167 = input_height - 1;
        _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y167) * input_stride + _gid_x167);
        int _gid_x168 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y168 = gid_y + 6 * (int)blockDim.y + (-2);
        if (_gid_x168 >= input_width)
            _gid_x168 = input_width - 1;
        if (_gid_y168 >= input_height)
            _gid_y168 = input_height - 1;
        _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y168) * input_stride + _gid_x168);
        int _gid_x169 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y169 = gid_y + 6 * (int)blockDim.y + (-2);
        if (_gid_x169 >= input_width)
            _gid_x169 = input_width - 1;
        if (_gid_y169 >= input_height)
            _gid_y169 = input_height - 1;
        _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y169) * input_stride + _gid_x169);
        int _gid_x170 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y170 = gid_y + 6 * (int)blockDim.y + (-2);
        if (_gid_x170 >= input_width)
            _gid_x170 = input_width - 1;
        if (_gid_y170 >= input_height)
            _gid_y170 = input_height - 1;
        _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y170) * input_stride + _gid_x170);
        int _gid_x171 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y171 = gid_y + 7 * (int)blockDim.y + (-2);
        if (_gid_x171 >= input_width)
            _gid_x171 = input_width - 1;
        if (_gid_y171 >= input_height)
            _gid_y171 = input_height - 1;
        _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y171) * input_stride + _gid_x171);
        int _gid_x172 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y172 = gid_y + 7 * (int)blockDim.y + (-2);
        if (_gid_x172 >= input_width)
            _gid_x172 = input_width - 1;
        if (_gid_y172 >= input_height)
            _gid_y172 = input_height - 1;
        _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y172) * input_stride + _gid_x172);
        int _gid_x173 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y173 = gid_y + 7 * (int)blockDim.y + (-2);
        if (_gid_x173 >= input_width)
            _gid_x173 = input_width - 1;
        if (_gid_y173 >= input_height)
            _gid_y173 = input_height - 1;
        _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y173) * input_stride + _gid_x173);
        int _gid_x174 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y174 = gid_y + 8 * (int)blockDim.y + (-2);
        if (_gid_x174 >= input_width)
            _gid_x174 = input_width - 1;
        if (_gid_y174 >= input_height)
            _gid_y174 = input_height - 1;
        _smeminput[(int)threadIdx.y + 8 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y174) * input_stride + _gid_x174);
        int _gid_x175 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y175 = gid_y + 8 * (int)blockDim.y + (-2);
        if (_gid_x175 >= input_width)
            _gid_x175 = input_width - 1;
        if (_gid_y175 >= input_height)
            _gid_y175 = input_height - 1;
        _smeminput[(int)threadIdx.y + 8 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y175) * input_stride + _gid_x175);
        int _gid_x176 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y176 = gid_y + 8 * (int)blockDim.y + (-2);
        if (_gid_x176 >= input_width)
            _gid_x176 = input_width - 1;
        if (_gid_y176 >= input_height)
            _gid_y176 = input_height - 1;
        _smeminput[(int)threadIdx.y + 8 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y176) * input_stride + _gid_x176);
        int _gid_x177 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y177 = gid_y + 9 * (int)blockDim.y + (-2);
        if (_gid_x177 >= input_width)
            _gid_x177 = input_width - 1;
        if (_gid_y177 >= input_height)
            _gid_y177 = input_height - 1;
        _smeminput[(int)threadIdx.y + 9 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y177) * input_stride + _gid_x177);
        int _gid_x178 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y178 = gid_y + 9 * (int)blockDim.y + (-2);
        if (_gid_x178 >= input_width)
            _gid_x178 = input_width - 1;
        if (_gid_y178 >= input_height)
            _gid_y178 = input_height - 1;
        _smeminput[(int)threadIdx.y + 9 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y178) * input_stride + _gid_x178);
        int _gid_x179 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y179 = gid_y + 9 * (int)blockDim.y + (-2);
        if (_gid_x179 >= input_width)
            _gid_x179 = input_width - 1;
        if (_gid_y179 >= input_height)
            _gid_y179 = input_height - 1;
        _smeminput[(int)threadIdx.y + 9 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y179) * input_stride + _gid_x179);
        __syncthreads();
        if (gid_x < iter_width) {
            if (gid_y < iter_height) {
                const int anchor_x = size_x >> 1;
                const int anchor_y = size_y >> 1;
                float sum = 0.5F;
                for (int yf = -anchor_y; yf <= anchor_y; ++yf) {
                    for (int xf = -anchor_x; xf <= anchor_x; ++xf) {
                        sum += _constmaskXY[yf + 2][xf + 2] * _smeminput[(int)threadIdx.y + yf + 2][(int)threadIdx.x + xf + 32];
                    }
                }
                iter[(gid_y) * iter_stride + gid_x] = (uchar)sum;
            }
        }
        if (gid_x < iter_width) {
            if (gid_y + 1 * (int)blockDim.y < iter_height) {
                const int anchor_x = size_x >> 1;
                const int anchor_y = size_y >> 1;
                float sum = 0.5F;
                for (int yf = -anchor_y; yf <= anchor_y; ++yf) {
                    for (int xf = -anchor_x; xf <= anchor_x; ++xf) {
                        sum += _constmaskXY[yf + 2][xf + 2] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + yf + 2][(int)threadIdx.x + xf + 32];
                    }
                }
                iter[(gid_y + 1 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)sum;
            }
        }
        if (gid_x < iter_width) {
            if (gid_y + 2 * (int)blockDim.y < iter_height) {
                const int anchor_x = size_x >> 1;
                const int anchor_y = size_y >> 1;
                float sum = 0.5F;
                for (int yf = -anchor_y; yf <= anchor_y; ++yf) {
                    for (int xf = -anchor_x; xf <= anchor_x; ++xf) {
                        sum += _constmaskXY[yf + 2][xf + 2] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + yf + 2][(int)threadIdx.x + xf + 32];
                    }
                }
                iter[(gid_y + 2 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)sum;
            }
        }
        if (gid_x < iter_width) {
            if (gid_y + 3 * (int)blockDim.y < iter_height) {
                const int anchor_x = size_x >> 1;
                const int anchor_y = size_y >> 1;
                float sum = 0.5F;
                for (int yf = -anchor_y; yf <= anchor_y; ++yf) {
                    for (int xf = -anchor_x; xf <= anchor_x; ++xf) {
                        sum += _constmaskXY[yf + 2][xf + 2] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + yf + 2][(int)threadIdx.x + xf + 32];
                    }
                }
                iter[(gid_y + 3 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)sum;
            }
        }
        if (gid_x < iter_width) {
            if (gid_y + 4 * (int)blockDim.y < iter_height) {
                const int anchor_x = size_x >> 1;
                const int anchor_y = size_y >> 1;
                float sum = 0.5F;
                for (int yf = -anchor_y; yf <= anchor_y; ++yf) {
                    for (int xf = -anchor_x; xf <= anchor_x; ++xf) {
                        sum += _constmaskXY[yf + 2][xf + 2] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + yf + 2][(int)threadIdx.x + xf + 32];
                    }
                }
                iter[(gid_y + 4 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)sum;
            }
        }
        if (gid_x < iter_width) {
            if (gid_y + 5 * (int)blockDim.y < iter_height) {
                const int anchor_x = size_x >> 1;
                const int anchor_y = size_y >> 1;
                float sum = 0.5F;
                for (int yf = -anchor_y; yf <= anchor_y; ++yf) {
                    for (int xf = -anchor_x; xf <= anchor_x; ++xf) {
                        sum += _constmaskXY[yf + 2][xf + 2] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + yf + 2][(int)threadIdx.x + xf + 32];
                    }
                }
                iter[(gid_y + 5 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)sum;
            }
        }
        if (gid_x < iter_width) {
            if (gid_y + 6 * (int)blockDim.y < iter_height) {
                const int anchor_x = size_x >> 1;
                const int anchor_y = size_y >> 1;
                float sum = 0.5F;
                for (int yf = -anchor_y; yf <= anchor_y; ++yf) {
                    for (int xf = -anchor_x; xf <= anchor_x; ++xf) {
                        sum += _constmaskXY[yf + 2][xf + 2] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + yf + 2][(int)threadIdx.x + xf + 32];
                    }
                }
                iter[(gid_y + 6 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)sum;
            }
        }
        if (gid_x < iter_width) {
            if (gid_y + 7 * (int)blockDim.y < iter_height) {
                const int anchor_x = size_x >> 1;
                const int anchor_y = size_y >> 1;
                float sum = 0.5F;
                for (int yf = -anchor_y; yf <= anchor_y; ++yf) {
                    for (int xf = -anchor_x; xf <= anchor_x; ++xf) {
                        sum += _constmaskXY[yf + 2][xf + 2] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + yf + 2][(int)threadIdx.x + xf + 32];
                    }
                }
                iter[(gid_y + 7 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)sum;
            }
        }
    }
    goto BH_EXIT;
  BH_B:
    {
        int _gid_x180 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y180 = gid_y + (-2);
        if (_gid_y180 >= input_height)
            _gid_y180 = input_height - 1;
        _smeminput[(int)threadIdx.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y180) * input_stride + _gid_x180);
        int _gid_x181 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y181 = gid_y + (-2);
        if (_gid_y181 >= input_height)
            _gid_y181 = input_height - 1;
        _smeminput[(int)threadIdx.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y181) * input_stride + _gid_x181);
        int _gid_x182 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y182 = gid_y + (-2);
        if (_gid_y182 >= input_height)
            _gid_y182 = input_height - 1;
        _smeminput[(int)threadIdx.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y182) * input_stride + _gid_x182);
        int _gid_x183 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y183 = gid_y + 1 * (int)blockDim.y + (-2);
        if (_gid_y183 >= input_height)
            _gid_y183 = input_height - 1;
        _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y183) * input_stride + _gid_x183);
        int _gid_x184 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y184 = gid_y + 1 * (int)blockDim.y + (-2);
        if (_gid_y184 >= input_height)
            _gid_y184 = input_height - 1;
        _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y184) * input_stride + _gid_x184);
        int _gid_x185 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y185 = gid_y + 1 * (int)blockDim.y + (-2);
        if (_gid_y185 >= input_height)
            _gid_y185 = input_height - 1;
        _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y185) * input_stride + _gid_x185);
        int _gid_x186 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y186 = gid_y + 2 * (int)blockDim.y + (-2);
        if (_gid_y186 >= input_height)
            _gid_y186 = input_height - 1;
        _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y186) * input_stride + _gid_x186);
        int _gid_x187 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y187 = gid_y + 2 * (int)blockDim.y + (-2);
        if (_gid_y187 >= input_height)
            _gid_y187 = input_height - 1;
        _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y187) * input_stride + _gid_x187);
        int _gid_x188 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y188 = gid_y + 2 * (int)blockDim.y + (-2);
        if (_gid_y188 >= input_height)
            _gid_y188 = input_height - 1;
        _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y188) * input_stride + _gid_x188);
        int _gid_x189 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y189 = gid_y + 3 * (int)blockDim.y + (-2);
        if (_gid_y189 >= input_height)
            _gid_y189 = input_height - 1;
        _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y189) * input_stride + _gid_x189);
        int _gid_x190 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y190 = gid_y + 3 * (int)blockDim.y + (-2);
        if (_gid_y190 >= input_height)
            _gid_y190 = input_height - 1;
        _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y190) * input_stride + _gid_x190);
        int _gid_x191 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y191 = gid_y + 3 * (int)blockDim.y + (-2);
        if (_gid_y191 >= input_height)
            _gid_y191 = input_height - 1;
        _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y191) * input_stride + _gid_x191);
        int _gid_x192 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y192 = gid_y + 4 * (int)blockDim.y + (-2);
        if (_gid_y192 >= input_height)
            _gid_y192 = input_height - 1;
        _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y192) * input_stride + _gid_x192);
        int _gid_x193 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y193 = gid_y + 4 * (int)blockDim.y + (-2);
        if (_gid_y193 >= input_height)
            _gid_y193 = input_height - 1;
        _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y193) * input_stride + _gid_x193);
        int _gid_x194 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y194 = gid_y + 4 * (int)blockDim.y + (-2);
        if (_gid_y194 >= input_height)
            _gid_y194 = input_height - 1;
        _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y194) * input_stride + _gid_x194);
        int _gid_x195 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y195 = gid_y + 5 * (int)blockDim.y + (-2);
        if (_gid_y195 >= input_height)
            _gid_y195 = input_height - 1;
        _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y195) * input_stride + _gid_x195);
        int _gid_x196 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y196 = gid_y + 5 * (int)blockDim.y + (-2);
        if (_gid_y196 >= input_height)
            _gid_y196 = input_height - 1;
        _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y196) * input_stride + _gid_x196);
        int _gid_x197 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y197 = gid_y + 5 * (int)blockDim.y + (-2);
        if (_gid_y197 >= input_height)
            _gid_y197 = input_height - 1;
        _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y197) * input_stride + _gid_x197);
        int _gid_x198 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y198 = gid_y + 6 * (int)blockDim.y + (-2);
        if (_gid_y198 >= input_height)
            _gid_y198 = input_height - 1;
        _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y198) * input_stride + _gid_x198);
        int _gid_x199 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y199 = gid_y + 6 * (int)blockDim.y + (-2);
        if (_gid_y199 >= input_height)
            _gid_y199 = input_height - 1;
        _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y199) * input_stride + _gid_x199);
        int _gid_x200 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y200 = gid_y + 6 * (int)blockDim.y + (-2);
        if (_gid_y200 >= input_height)
            _gid_y200 = input_height - 1;
        _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y200) * input_stride + _gid_x200);
        int _gid_x201 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y201 = gid_y + 7 * (int)blockDim.y + (-2);
        if (_gid_y201 >= input_height)
            _gid_y201 = input_height - 1;
        _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y201) * input_stride + _gid_x201);
        int _gid_x202 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y202 = gid_y + 7 * (int)blockDim.y + (-2);
        if (_gid_y202 >= input_height)
            _gid_y202 = input_height - 1;
        _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y202) * input_stride + _gid_x202);
        int _gid_x203 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y203 = gid_y + 7 * (int)blockDim.y + (-2);
        if (_gid_y203 >= input_height)
            _gid_y203 = input_height - 1;
        _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y203) * input_stride + _gid_x203);
        int _gid_x204 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y204 = gid_y + 8 * (int)blockDim.y + (-2);
        if (_gid_y204 >= input_height)
            _gid_y204 = input_height - 1;
        _smeminput[(int)threadIdx.y + 8 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y204) * input_stride + _gid_x204);
        int _gid_x205 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y205 = gid_y + 8 * (int)blockDim.y + (-2);
        if (_gid_y205 >= input_height)
            _gid_y205 = input_height - 1;
        _smeminput[(int)threadIdx.y + 8 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y205) * input_stride + _gid_x205);
        int _gid_x206 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y206 = gid_y + 8 * (int)blockDim.y + (-2);
        if (_gid_y206 >= input_height)
            _gid_y206 = input_height - 1;
        _smeminput[(int)threadIdx.y + 8 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y206) * input_stride + _gid_x206);
        int _gid_x207 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y207 = gid_y + 9 * (int)blockDim.y + (-2);
        if (_gid_y207 >= input_height)
            _gid_y207 = input_height - 1;
        _smeminput[(int)threadIdx.y + 9 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y207) * input_stride + _gid_x207);
        int _gid_x208 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y208 = gid_y + 9 * (int)blockDim.y + (-2);
        if (_gid_y208 >= input_height)
            _gid_y208 = input_height - 1;
        _smeminput[(int)threadIdx.y + 9 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y208) * input_stride + _gid_x208);
        int _gid_x209 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y209 = gid_y + 9 * (int)blockDim.y + (-2);
        if (_gid_y209 >= input_height)
            _gid_y209 = input_height - 1;
        _smeminput[(int)threadIdx.y + 9 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y209) * input_stride + _gid_x209);
        __syncthreads();
        if (gid_y < iter_height) {
            const int anchor_x = size_x >> 1;
            const int anchor_y = size_y >> 1;
            float sum = 0.5F;
            for (int yf = -anchor_y; yf <= anchor_y; ++yf) {
                for (int xf = -anchor_x; xf <= anchor_x; ++xf) {
                    sum += _constmaskXY[yf + 2][xf + 2] * _smeminput[(int)threadIdx.y + yf + 2][(int)threadIdx.x + xf + 32];
                }
            }
            iter[(gid_y) * iter_stride + gid_x] = (uchar)sum;
        }
        if (gid_y + 1 * (int)blockDim.y < iter_height) {
            const int anchor_x = size_x >> 1;
            const int anchor_y = size_y >> 1;
            float sum = 0.5F;
            for (int yf = -anchor_y; yf <= anchor_y; ++yf) {
                for (int xf = -anchor_x; xf <= anchor_x; ++xf) {
                    sum += _constmaskXY[yf + 2][xf + 2] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + yf + 2][(int)threadIdx.x + xf + 32];
                }
            }
            iter[(gid_y + 1 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)sum;
        }
        if (gid_y + 2 * (int)blockDim.y < iter_height) {
            const int anchor_x = size_x >> 1;
            const int anchor_y = size_y >> 1;
            float sum = 0.5F;
            for (int yf = -anchor_y; yf <= anchor_y; ++yf) {
                for (int xf = -anchor_x; xf <= anchor_x; ++xf) {
                    sum += _constmaskXY[yf + 2][xf + 2] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + yf + 2][(int)threadIdx.x + xf + 32];
                }
            }
            iter[(gid_y + 2 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)sum;
        }
        if (gid_y + 3 * (int)blockDim.y < iter_height) {
            const int anchor_x = size_x >> 1;
            const int anchor_y = size_y >> 1;
            float sum = 0.5F;
            for (int yf = -anchor_y; yf <= anchor_y; ++yf) {
                for (int xf = -anchor_x; xf <= anchor_x; ++xf) {
                    sum += _constmaskXY[yf + 2][xf + 2] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + yf + 2][(int)threadIdx.x + xf + 32];
                }
            }
            iter[(gid_y + 3 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)sum;
        }
        if (gid_y + 4 * (int)blockDim.y < iter_height) {
            const int anchor_x = size_x >> 1;
            const int anchor_y = size_y >> 1;
            float sum = 0.5F;
            for (int yf = -anchor_y; yf <= anchor_y; ++yf) {
                for (int xf = -anchor_x; xf <= anchor_x; ++xf) {
                    sum += _constmaskXY[yf + 2][xf + 2] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + yf + 2][(int)threadIdx.x + xf + 32];
                }
            }
            iter[(gid_y + 4 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)sum;
        }
        if (gid_y + 5 * (int)blockDim.y < iter_height) {
            const int anchor_x = size_x >> 1;
            const int anchor_y = size_y >> 1;
            float sum = 0.5F;
            for (int yf = -anchor_y; yf <= anchor_y; ++yf) {
                for (int xf = -anchor_x; xf <= anchor_x; ++xf) {
                    sum += _constmaskXY[yf + 2][xf + 2] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + yf + 2][(int)threadIdx.x + xf + 32];
                }
            }
            iter[(gid_y + 5 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)sum;
        }
        if (gid_y + 6 * (int)blockDim.y < iter_height) {
            const int anchor_x = size_x >> 1;
            const int anchor_y = size_y >> 1;
            float sum = 0.5F;
            for (int yf = -anchor_y; yf <= anchor_y; ++yf) {
                for (int xf = -anchor_x; xf <= anchor_x; ++xf) {
                    sum += _constmaskXY[yf + 2][xf + 2] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + yf + 2][(int)threadIdx.x + xf + 32];
                }
            }
            iter[(gid_y + 6 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)sum;
        }
        if (gid_y + 7 * (int)blockDim.y < iter_height) {
            const int anchor_x = size_x >> 1;
            const int anchor_y = size_y >> 1;
            float sum = 0.5F;
            for (int yf = -anchor_y; yf <= anchor_y; ++yf) {
                for (int xf = -anchor_x; xf <= anchor_x; ++xf) {
                    sum += _constmaskXY[yf + 2][xf + 2] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + yf + 2][(int)threadIdx.x + xf + 32];
                }
            }
            iter[(gid_y + 7 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)sum;
        }
    }
    goto BH_EXIT;
  BH_R:
    {
        int _gid_x210 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y210 = gid_y + (-2);
        if (_gid_x210 >= input_width)
            _gid_x210 = input_width - 1;
        _smeminput[(int)threadIdx.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y210) * input_stride + _gid_x210);
        int _gid_x211 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y211 = gid_y + (-2);
        if (_gid_x211 >= input_width)
            _gid_x211 = input_width - 1;
        _smeminput[(int)threadIdx.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y211) * input_stride + _gid_x211);
        int _gid_x212 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y212 = gid_y + (-2);
        if (_gid_x212 >= input_width)
            _gid_x212 = input_width - 1;
        _smeminput[(int)threadIdx.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y212) * input_stride + _gid_x212);
        int _gid_x213 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y213 = gid_y + 1 * (int)blockDim.y + (-2);
        if (_gid_x213 >= input_width)
            _gid_x213 = input_width - 1;
        _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y213) * input_stride + _gid_x213);
        int _gid_x214 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y214 = gid_y + 1 * (int)blockDim.y + (-2);
        if (_gid_x214 >= input_width)
            _gid_x214 = input_width - 1;
        _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y214) * input_stride + _gid_x214);
        int _gid_x215 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y215 = gid_y + 1 * (int)blockDim.y + (-2);
        if (_gid_x215 >= input_width)
            _gid_x215 = input_width - 1;
        _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y215) * input_stride + _gid_x215);
        int _gid_x216 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y216 = gid_y + 2 * (int)blockDim.y + (-2);
        if (_gid_x216 >= input_width)
            _gid_x216 = input_width - 1;
        _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y216) * input_stride + _gid_x216);
        int _gid_x217 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y217 = gid_y + 2 * (int)blockDim.y + (-2);
        if (_gid_x217 >= input_width)
            _gid_x217 = input_width - 1;
        _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y217) * input_stride + _gid_x217);
        int _gid_x218 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y218 = gid_y + 2 * (int)blockDim.y + (-2);
        if (_gid_x218 >= input_width)
            _gid_x218 = input_width - 1;
        _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y218) * input_stride + _gid_x218);
        int _gid_x219 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y219 = gid_y + 3 * (int)blockDim.y + (-2);
        if (_gid_x219 >= input_width)
            _gid_x219 = input_width - 1;
        _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y219) * input_stride + _gid_x219);
        int _gid_x220 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y220 = gid_y + 3 * (int)blockDim.y + (-2);
        if (_gid_x220 >= input_width)
            _gid_x220 = input_width - 1;
        _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y220) * input_stride + _gid_x220);
        int _gid_x221 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y221 = gid_y + 3 * (int)blockDim.y + (-2);
        if (_gid_x221 >= input_width)
            _gid_x221 = input_width - 1;
        _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y221) * input_stride + _gid_x221);
        int _gid_x222 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y222 = gid_y + 4 * (int)blockDim.y + (-2);
        if (_gid_x222 >= input_width)
            _gid_x222 = input_width - 1;
        _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y222) * input_stride + _gid_x222);
        int _gid_x223 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y223 = gid_y + 4 * (int)blockDim.y + (-2);
        if (_gid_x223 >= input_width)
            _gid_x223 = input_width - 1;
        _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y223) * input_stride + _gid_x223);
        int _gid_x224 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y224 = gid_y + 4 * (int)blockDim.y + (-2);
        if (_gid_x224 >= input_width)
            _gid_x224 = input_width - 1;
        _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y224) * input_stride + _gid_x224);
        int _gid_x225 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y225 = gid_y + 5 * (int)blockDim.y + (-2);
        if (_gid_x225 >= input_width)
            _gid_x225 = input_width - 1;
        _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y225) * input_stride + _gid_x225);
        int _gid_x226 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y226 = gid_y + 5 * (int)blockDim.y + (-2);
        if (_gid_x226 >= input_width)
            _gid_x226 = input_width - 1;
        _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y226) * input_stride + _gid_x226);
        int _gid_x227 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y227 = gid_y + 5 * (int)blockDim.y + (-2);
        if (_gid_x227 >= input_width)
            _gid_x227 = input_width - 1;
        _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y227) * input_stride + _gid_x227);
        int _gid_x228 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y228 = gid_y + 6 * (int)blockDim.y + (-2);
        if (_gid_x228 >= input_width)
            _gid_x228 = input_width - 1;
        _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y228) * input_stride + _gid_x228);
        int _gid_x229 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y229 = gid_y + 6 * (int)blockDim.y + (-2);
        if (_gid_x229 >= input_width)
            _gid_x229 = input_width - 1;
        _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y229) * input_stride + _gid_x229);
        int _gid_x230 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y230 = gid_y + 6 * (int)blockDim.y + (-2);
        if (_gid_x230 >= input_width)
            _gid_x230 = input_width - 1;
        _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y230) * input_stride + _gid_x230);
        int _gid_x231 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y231 = gid_y + 7 * (int)blockDim.y + (-2);
        if (_gid_x231 >= input_width)
            _gid_x231 = input_width - 1;
        _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y231) * input_stride + _gid_x231);
        int _gid_x232 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y232 = gid_y + 7 * (int)blockDim.y + (-2);
        if (_gid_x232 >= input_width)
            _gid_x232 = input_width - 1;
        _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y232) * input_stride + _gid_x232);
        int _gid_x233 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y233 = gid_y + 7 * (int)blockDim.y + (-2);
        if (_gid_x233 >= input_width)
            _gid_x233 = input_width - 1;
        _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y233) * input_stride + _gid_x233);
        int _gid_x234 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y234 = gid_y + 8 * (int)blockDim.y + (-2);
        if (_gid_x234 >= input_width)
            _gid_x234 = input_width - 1;
        _smeminput[(int)threadIdx.y + 8 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y234) * input_stride + _gid_x234);
        int _gid_x235 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y235 = gid_y + 8 * (int)blockDim.y + (-2);
        if (_gid_x235 >= input_width)
            _gid_x235 = input_width - 1;
        _smeminput[(int)threadIdx.y + 8 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y235) * input_stride + _gid_x235);
        int _gid_x236 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y236 = gid_y + 8 * (int)blockDim.y + (-2);
        if (_gid_x236 >= input_width)
            _gid_x236 = input_width - 1;
        _smeminput[(int)threadIdx.y + 8 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y236) * input_stride + _gid_x236);
        int _gid_x237 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y237 = gid_y + 9 * (int)blockDim.y + (-2);
        if (_gid_x237 >= input_width)
            _gid_x237 = input_width - 1;
        _smeminput[(int)threadIdx.y + 9 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y237) * input_stride + _gid_x237);
        int _gid_x238 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y238 = gid_y + 9 * (int)blockDim.y + (-2);
        if (_gid_x238 >= input_width)
            _gid_x238 = input_width - 1;
        _smeminput[(int)threadIdx.y + 9 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y238) * input_stride + _gid_x238);
        int _gid_x239 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y239 = gid_y + 9 * (int)blockDim.y + (-2);
        if (_gid_x239 >= input_width)
            _gid_x239 = input_width - 1;
        _smeminput[(int)threadIdx.y + 9 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y239) * input_stride + _gid_x239);
        __syncthreads();
        if (gid_x < iter_width) {
            {
                const int anchor_x = size_x >> 1;
                const int anchor_y = size_y >> 1;
                float sum = 0.5F;
                for (int yf = -anchor_y; yf <= anchor_y; ++yf) {
                    for (int xf = -anchor_x; xf <= anchor_x; ++xf) {
                        sum += _constmaskXY[yf + 2][xf + 2] * _smeminput[(int)threadIdx.y + yf + 2][(int)threadIdx.x + xf + 32];
                    }
                }
                iter[(gid_y) * iter_stride + gid_x] = (uchar)sum;
            }
        }
        if (gid_x < iter_width) {
            {
                const int anchor_x = size_x >> 1;
                const int anchor_y = size_y >> 1;
                float sum = 0.5F;
                for (int yf = -anchor_y; yf <= anchor_y; ++yf) {
                    for (int xf = -anchor_x; xf <= anchor_x; ++xf) {
                        sum += _constmaskXY[yf + 2][xf + 2] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + yf + 2][(int)threadIdx.x + xf + 32];
                    }
                }
                iter[(gid_y + 1 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)sum;
            }
        }
        if (gid_x < iter_width) {
            {
                const int anchor_x = size_x >> 1;
                const int anchor_y = size_y >> 1;
                float sum = 0.5F;
                for (int yf = -anchor_y; yf <= anchor_y; ++yf) {
                    for (int xf = -anchor_x; xf <= anchor_x; ++xf) {
                        sum += _constmaskXY[yf + 2][xf + 2] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + yf + 2][(int)threadIdx.x + xf + 32];
                    }
                }
                iter[(gid_y + 2 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)sum;
            }
        }
        if (gid_x < iter_width) {
            {
                const int anchor_x = size_x >> 1;
                const int anchor_y = size_y >> 1;
                float sum = 0.5F;
                for (int yf = -anchor_y; yf <= anchor_y; ++yf) {
                    for (int xf = -anchor_x; xf <= anchor_x; ++xf) {
                        sum += _constmaskXY[yf + 2][xf + 2] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + yf + 2][(int)threadIdx.x + xf + 32];
                    }
                }
                iter[(gid_y + 3 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)sum;
            }
        }
        if (gid_x < iter_width) {
            {
                const int anchor_x = size_x >> 1;
                const int anchor_y = size_y >> 1;
                float sum = 0.5F;
                for (int yf = -anchor_y; yf <= anchor_y; ++yf) {
                    for (int xf = -anchor_x; xf <= anchor_x; ++xf) {
                        sum += _constmaskXY[yf + 2][xf + 2] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + yf + 2][(int)threadIdx.x + xf + 32];
                    }
                }
                iter[(gid_y + 4 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)sum;
            }
        }
        if (gid_x < iter_width) {
            {
                const int anchor_x = size_x >> 1;
                const int anchor_y = size_y >> 1;
                float sum = 0.5F;
                for (int yf = -anchor_y; yf <= anchor_y; ++yf) {
                    for (int xf = -anchor_x; xf <= anchor_x; ++xf) {
                        sum += _constmaskXY[yf + 2][xf + 2] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + yf + 2][(int)threadIdx.x + xf + 32];
                    }
                }
                iter[(gid_y + 5 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)sum;
            }
        }
        if (gid_x < iter_width) {
            {
                const int anchor_x = size_x >> 1;
                const int anchor_y = size_y >> 1;
                float sum = 0.5F;
                for (int yf = -anchor_y; yf <= anchor_y; ++yf) {
                    for (int xf = -anchor_x; xf <= anchor_x; ++xf) {
                        sum += _constmaskXY[yf + 2][xf + 2] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + yf + 2][(int)threadIdx.x + xf + 32];
                    }
                }
                iter[(gid_y + 6 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)sum;
            }
        }
        if (gid_x < iter_width) {
            {
                const int anchor_x = size_x >> 1;
                const int anchor_y = size_y >> 1;
                float sum = 0.5F;
                for (int yf = -anchor_y; yf <= anchor_y; ++yf) {
                    for (int xf = -anchor_x; xf <= anchor_x; ++xf) {
                        sum += _constmaskXY[yf + 2][xf + 2] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + yf + 2][(int)threadIdx.x + xf + 32];
                    }
                }
                iter[(gid_y + 7 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)sum;
            }
        }
    }
    goto BH_EXIT;
  BH_L:
    {
        int _gid_x240 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y240 = gid_y + (-2);
        if (_gid_x240 < 0)
            _gid_x240 = 0;
        _smeminput[(int)threadIdx.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y240) * input_stride + _gid_x240);
        int _gid_x241 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y241 = gid_y + (-2);
        if (_gid_x241 < 0)
            _gid_x241 = 0;
        _smeminput[(int)threadIdx.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y241) * input_stride + _gid_x241);
        int _gid_x242 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y242 = gid_y + (-2);
        if (_gid_x242 < 0)
            _gid_x242 = 0;
        _smeminput[(int)threadIdx.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y242) * input_stride + _gid_x242);
        int _gid_x243 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y243 = gid_y + 1 * (int)blockDim.y + (-2);
        if (_gid_x243 < 0)
            _gid_x243 = 0;
        _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y243) * input_stride + _gid_x243);
        int _gid_x244 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y244 = gid_y + 1 * (int)blockDim.y + (-2);
        if (_gid_x244 < 0)
            _gid_x244 = 0;
        _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y244) * input_stride + _gid_x244);
        int _gid_x245 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y245 = gid_y + 1 * (int)blockDim.y + (-2);
        if (_gid_x245 < 0)
            _gid_x245 = 0;
        _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y245) * input_stride + _gid_x245);
        int _gid_x246 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y246 = gid_y + 2 * (int)blockDim.y + (-2);
        if (_gid_x246 < 0)
            _gid_x246 = 0;
        _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y246) * input_stride + _gid_x246);
        int _gid_x247 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y247 = gid_y + 2 * (int)blockDim.y + (-2);
        if (_gid_x247 < 0)
            _gid_x247 = 0;
        _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y247) * input_stride + _gid_x247);
        int _gid_x248 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y248 = gid_y + 2 * (int)blockDim.y + (-2);
        if (_gid_x248 < 0)
            _gid_x248 = 0;
        _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y248) * input_stride + _gid_x248);
        int _gid_x249 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y249 = gid_y + 3 * (int)blockDim.y + (-2);
        if (_gid_x249 < 0)
            _gid_x249 = 0;
        _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y249) * input_stride + _gid_x249);
        int _gid_x250 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y250 = gid_y + 3 * (int)blockDim.y + (-2);
        if (_gid_x250 < 0)
            _gid_x250 = 0;
        _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y250) * input_stride + _gid_x250);
        int _gid_x251 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y251 = gid_y + 3 * (int)blockDim.y + (-2);
        if (_gid_x251 < 0)
            _gid_x251 = 0;
        _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y251) * input_stride + _gid_x251);
        int _gid_x252 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y252 = gid_y + 4 * (int)blockDim.y + (-2);
        if (_gid_x252 < 0)
            _gid_x252 = 0;
        _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y252) * input_stride + _gid_x252);
        int _gid_x253 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y253 = gid_y + 4 * (int)blockDim.y + (-2);
        if (_gid_x253 < 0)
            _gid_x253 = 0;
        _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y253) * input_stride + _gid_x253);
        int _gid_x254 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y254 = gid_y + 4 * (int)blockDim.y + (-2);
        if (_gid_x254 < 0)
            _gid_x254 = 0;
        _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y254) * input_stride + _gid_x254);
        int _gid_x255 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y255 = gid_y + 5 * (int)blockDim.y + (-2);
        if (_gid_x255 < 0)
            _gid_x255 = 0;
        _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y255) * input_stride + _gid_x255);
        int _gid_x256 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y256 = gid_y + 5 * (int)blockDim.y + (-2);
        if (_gid_x256 < 0)
            _gid_x256 = 0;
        _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y256) * input_stride + _gid_x256);
        int _gid_x257 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y257 = gid_y + 5 * (int)blockDim.y + (-2);
        if (_gid_x257 < 0)
            _gid_x257 = 0;
        _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y257) * input_stride + _gid_x257);
        int _gid_x258 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y258 = gid_y + 6 * (int)blockDim.y + (-2);
        if (_gid_x258 < 0)
            _gid_x258 = 0;
        _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y258) * input_stride + _gid_x258);
        int _gid_x259 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y259 = gid_y + 6 * (int)blockDim.y + (-2);
        if (_gid_x259 < 0)
            _gid_x259 = 0;
        _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y259) * input_stride + _gid_x259);
        int _gid_x260 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y260 = gid_y + 6 * (int)blockDim.y + (-2);
        if (_gid_x260 < 0)
            _gid_x260 = 0;
        _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y260) * input_stride + _gid_x260);
        int _gid_x261 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y261 = gid_y + 7 * (int)blockDim.y + (-2);
        if (_gid_x261 < 0)
            _gid_x261 = 0;
        _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y261) * input_stride + _gid_x261);
        int _gid_x262 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y262 = gid_y + 7 * (int)blockDim.y + (-2);
        if (_gid_x262 < 0)
            _gid_x262 = 0;
        _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y262) * input_stride + _gid_x262);
        int _gid_x263 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y263 = gid_y + 7 * (int)blockDim.y + (-2);
        if (_gid_x263 < 0)
            _gid_x263 = 0;
        _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y263) * input_stride + _gid_x263);
        int _gid_x264 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y264 = gid_y + 8 * (int)blockDim.y + (-2);
        if (_gid_x264 < 0)
            _gid_x264 = 0;
        _smeminput[(int)threadIdx.y + 8 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y264) * input_stride + _gid_x264);
        int _gid_x265 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y265 = gid_y + 8 * (int)blockDim.y + (-2);
        if (_gid_x265 < 0)
            _gid_x265 = 0;
        _smeminput[(int)threadIdx.y + 8 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y265) * input_stride + _gid_x265);
        int _gid_x266 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y266 = gid_y + 8 * (int)blockDim.y + (-2);
        if (_gid_x266 < 0)
            _gid_x266 = 0;
        _smeminput[(int)threadIdx.y + 8 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y266) * input_stride + _gid_x266);
        int _gid_x267 = gid_x + 0 * (int)blockDim.x - 32;
        int _gid_y267 = gid_y + 9 * (int)blockDim.y + (-2);
        if (_gid_x267 < 0)
            _gid_x267 = 0;
        _smeminput[(int)threadIdx.y + 9 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y267) * input_stride + _gid_x267);
        int _gid_x268 = gid_x + 1 * (int)blockDim.x - 32;
        int _gid_y268 = gid_y + 9 * (int)blockDim.y + (-2);
        if (_gid_x268 < 0)
            _gid_x268 = 0;
        _smeminput[(int)threadIdx.y + 9 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y268) * input_stride + _gid_x268);
        int _gid_x269 = gid_x + 2 * (int)blockDim.x - 32;
        int _gid_y269 = gid_y + 9 * (int)blockDim.y + (-2);
        if (_gid_x269 < 0)
            _gid_x269 = 0;
        _smeminput[(int)threadIdx.y + 9 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (_gid_y269) * input_stride + _gid_x269);
        __syncthreads();
        {
            const int anchor_x = size_x >> 1;
            const int anchor_y = size_y >> 1;
            float sum = 0.5F;
            for (int yf = -anchor_y; yf <= anchor_y; ++yf) {
                for (int xf = -anchor_x; xf <= anchor_x; ++xf) {
                    sum += _constmaskXY[yf + 2][xf + 2] * _smeminput[(int)threadIdx.y + yf + 2][(int)threadIdx.x + xf + 32];
                }
            }
            iter[(gid_y) * iter_stride + gid_x] = (uchar)sum;
        }
        {
            const int anchor_x = size_x >> 1;
            const int anchor_y = size_y >> 1;
            float sum = 0.5F;
            for (int yf = -anchor_y; yf <= anchor_y; ++yf) {
                for (int xf = -anchor_x; xf <= anchor_x; ++xf) {
                    sum += _constmaskXY[yf + 2][xf + 2] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + yf + 2][(int)threadIdx.x + xf + 32];
                }
            }
            iter[(gid_y + 1 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)sum;
        }
        {
            const int anchor_x = size_x >> 1;
            const int anchor_y = size_y >> 1;
            float sum = 0.5F;
            for (int yf = -anchor_y; yf <= anchor_y; ++yf) {
                for (int xf = -anchor_x; xf <= anchor_x; ++xf) {
                    sum += _constmaskXY[yf + 2][xf + 2] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + yf + 2][(int)threadIdx.x + xf + 32];
                }
            }
            iter[(gid_y + 2 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)sum;
        }
        {
            const int anchor_x = size_x >> 1;
            const int anchor_y = size_y >> 1;
            float sum = 0.5F;
            for (int yf = -anchor_y; yf <= anchor_y; ++yf) {
                for (int xf = -anchor_x; xf <= anchor_x; ++xf) {
                    sum += _constmaskXY[yf + 2][xf + 2] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + yf + 2][(int)threadIdx.x + xf + 32];
                }
            }
            iter[(gid_y + 3 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)sum;
        }
        {
            const int anchor_x = size_x >> 1;
            const int anchor_y = size_y >> 1;
            float sum = 0.5F;
            for (int yf = -anchor_y; yf <= anchor_y; ++yf) {
                for (int xf = -anchor_x; xf <= anchor_x; ++xf) {
                    sum += _constmaskXY[yf + 2][xf + 2] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + yf + 2][(int)threadIdx.x + xf + 32];
                }
            }
            iter[(gid_y + 4 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)sum;
        }
        {
            const int anchor_x = size_x >> 1;
            const int anchor_y = size_y >> 1;
            float sum = 0.5F;
            for (int yf = -anchor_y; yf <= anchor_y; ++yf) {
                for (int xf = -anchor_x; xf <= anchor_x; ++xf) {
                    sum += _constmaskXY[yf + 2][xf + 2] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + yf + 2][(int)threadIdx.x + xf + 32];
                }
            }
            iter[(gid_y + 5 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)sum;
        }
        {
            const int anchor_x = size_x >> 1;
            const int anchor_y = size_y >> 1;
            float sum = 0.5F;
            for (int yf = -anchor_y; yf <= anchor_y; ++yf) {
                for (int xf = -anchor_x; xf <= anchor_x; ++xf) {
                    sum += _constmaskXY[yf + 2][xf + 2] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + yf + 2][(int)threadIdx.x + xf + 32];
                }
            }
            iter[(gid_y + 6 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)sum;
        }
        {
            const int anchor_x = size_x >> 1;
            const int anchor_y = size_y >> 1;
            float sum = 0.5F;
            for (int yf = -anchor_y; yf <= anchor_y; ++yf) {
                for (int xf = -anchor_x; xf <= anchor_x; ++xf) {
                    sum += _constmaskXY[yf + 2][xf + 2] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + yf + 2][(int)threadIdx.x + xf + 32];
                }
            }
            iter[(gid_y + 7 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)sum;
        }
    }
    goto BH_EXIT;
  BH_NO:
    {
        _smeminput[(int)threadIdx.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (gid_y + (-2)) * input_stride + gid_x + 0 * (int)blockDim.x - 32);
        _smeminput[(int)threadIdx.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (gid_y + (-2)) * input_stride + gid_x + 1 * (int)blockDim.x - 32);
        _smeminput[(int)threadIdx.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (gid_y + (-2)) * input_stride + gid_x + 2 * (int)blockDim.x - 32);
        _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (gid_y + 1 * (int)blockDim.y + (-2)) * input_stride + gid_x + 0 * (int)blockDim.x - 32);
        _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (gid_y + 1 * (int)blockDim.y + (-2)) * input_stride + gid_x + 1 * (int)blockDim.x - 32);
        _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (gid_y + 1 * (int)blockDim.y + (-2)) * input_stride + gid_x + 2 * (int)blockDim.x - 32);
        _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (gid_y + 2 * (int)blockDim.y + (-2)) * input_stride + gid_x + 0 * (int)blockDim.x - 32);
        _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (gid_y + 2 * (int)blockDim.y + (-2)) * input_stride + gid_x + 1 * (int)blockDim.x - 32);
        _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (gid_y + 2 * (int)blockDim.y + (-2)) * input_stride + gid_x + 2 * (int)blockDim.x - 32);
        _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (gid_y + 3 * (int)blockDim.y + (-2)) * input_stride + gid_x + 0 * (int)blockDim.x - 32);
        _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (gid_y + 3 * (int)blockDim.y + (-2)) * input_stride + gid_x + 1 * (int)blockDim.x - 32);
        _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (gid_y + 3 * (int)blockDim.y + (-2)) * input_stride + gid_x + 2 * (int)blockDim.x - 32);
        _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (gid_y + 4 * (int)blockDim.y + (-2)) * input_stride + gid_x + 0 * (int)blockDim.x - 32);
        _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (gid_y + 4 * (int)blockDim.y + (-2)) * input_stride + gid_x + 1 * (int)blockDim.x - 32);
        _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (gid_y + 4 * (int)blockDim.y + (-2)) * input_stride + gid_x + 2 * (int)blockDim.x - 32);
        _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (gid_y + 5 * (int)blockDim.y + (-2)) * input_stride + gid_x + 0 * (int)blockDim.x - 32);
        _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (gid_y + 5 * (int)blockDim.y + (-2)) * input_stride + gid_x + 1 * (int)blockDim.x - 32);
        _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (gid_y + 5 * (int)blockDim.y + (-2)) * input_stride + gid_x + 2 * (int)blockDim.x - 32);
        _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (gid_y + 6 * (int)blockDim.y + (-2)) * input_stride + gid_x + 0 * (int)blockDim.x - 32);
        _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (gid_y + 6 * (int)blockDim.y + (-2)) * input_stride + gid_x + 1 * (int)blockDim.x - 32);
        _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (gid_y + 6 * (int)blockDim.y + (-2)) * input_stride + gid_x + 2 * (int)blockDim.x - 32);
        _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (gid_y + 7 * (int)blockDim.y + (-2)) * input_stride + gid_x + 0 * (int)blockDim.x - 32);
        _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (gid_y + 7 * (int)blockDim.y + (-2)) * input_stride + gid_x + 1 * (int)blockDim.x - 32);
        _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (gid_y + 7 * (int)blockDim.y + (-2)) * input_stride + gid_x + 2 * (int)blockDim.x - 32);
        _smeminput[(int)threadIdx.y + 8 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (gid_y + 8 * (int)blockDim.y + (-2)) * input_stride + gid_x + 0 * (int)blockDim.x - 32);
        _smeminput[(int)threadIdx.y + 8 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (gid_y + 8 * (int)blockDim.y + (-2)) * input_stride + gid_x + 1 * (int)blockDim.x - 32);
        _smeminput[(int)threadIdx.y + 8 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (gid_y + 8 * (int)blockDim.y + (-2)) * input_stride + gid_x + 2 * (int)blockDim.x - 32);
        _smeminput[(int)threadIdx.y + 9 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (gid_y + 9 * (int)blockDim.y + (-2)) * input_stride + gid_x + 0 * (int)blockDim.x - 32);
        _smeminput[(int)threadIdx.y + 9 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (gid_y + 9 * (int)blockDim.y + (-2)) * input_stride + gid_x + 1 * (int)blockDim.x - 32);
        _smeminput[(int)threadIdx.y + 9 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputXY, (gid_y + 9 * (int)blockDim.y + (-2)) * input_stride + gid_x + 2 * (int)blockDim.x - 32);
        __syncthreads();
        {
            const int anchor_x = size_x >> 1;
            const int anchor_y = size_y >> 1;
            float sum = 0.5F;
            for (int yf = -anchor_y; yf <= anchor_y; ++yf) {
                for (int xf = -anchor_x; xf <= anchor_x; ++xf) {
                    sum += _constmaskXY[yf + 2][xf + 2] * _smeminput[(int)threadIdx.y + yf + 2][(int)threadIdx.x + xf + 32];
                }
            }
            iter[(gid_y) * iter_stride + gid_x] = (uchar)sum;
        }
        {
            const int anchor_x = size_x >> 1;
            const int anchor_y = size_y >> 1;
            float sum = 0.5F;
            for (int yf = -anchor_y; yf <= anchor_y; ++yf) {
                for (int xf = -anchor_x; xf <= anchor_x; ++xf) {
                    sum += _constmaskXY[yf + 2][xf + 2] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + yf + 2][(int)threadIdx.x + xf + 32];
                }
            }
            iter[(gid_y + 1 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)sum;
        }
        {
            const int anchor_x = size_x >> 1;
            const int anchor_y = size_y >> 1;
            float sum = 0.5F;
            for (int yf = -anchor_y; yf <= anchor_y; ++yf) {
                for (int xf = -anchor_x; xf <= anchor_x; ++xf) {
                    sum += _constmaskXY[yf + 2][xf + 2] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + yf + 2][(int)threadIdx.x + xf + 32];
                }
            }
            iter[(gid_y + 2 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)sum;
        }
        {
            const int anchor_x = size_x >> 1;
            const int anchor_y = size_y >> 1;
            float sum = 0.5F;
            for (int yf = -anchor_y; yf <= anchor_y; ++yf) {
                for (int xf = -anchor_x; xf <= anchor_x; ++xf) {
                    sum += _constmaskXY[yf + 2][xf + 2] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + yf + 2][(int)threadIdx.x + xf + 32];
                }
            }
            iter[(gid_y + 3 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)sum;
        }
        {
            const int anchor_x = size_x >> 1;
            const int anchor_y = size_y >> 1;
            float sum = 0.5F;
            for (int yf = -anchor_y; yf <= anchor_y; ++yf) {
                for (int xf = -anchor_x; xf <= anchor_x; ++xf) {
                    sum += _constmaskXY[yf + 2][xf + 2] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + yf + 2][(int)threadIdx.x + xf + 32];
                }
            }
            iter[(gid_y + 4 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)sum;
        }
        {
            const int anchor_x = size_x >> 1;
            const int anchor_y = size_y >> 1;
            float sum = 0.5F;
            for (int yf = -anchor_y; yf <= anchor_y; ++yf) {
                for (int xf = -anchor_x; xf <= anchor_x; ++xf) {
                    sum += _constmaskXY[yf + 2][xf + 2] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + yf + 2][(int)threadIdx.x + xf + 32];
                }
            }
            iter[(gid_y + 5 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)sum;
        }
        {
            const int anchor_x = size_x >> 1;
            const int anchor_y = size_y >> 1;
            float sum = 0.5F;
            for (int yf = -anchor_y; yf <= anchor_y; ++yf) {
                for (int xf = -anchor_x; xf <= anchor_x; ++xf) {
                    sum += _constmaskXY[yf + 2][xf + 2] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + yf + 2][(int)threadIdx.x + xf + 32];
                }
            }
            iter[(gid_y + 6 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)sum;
        }
        {
            const int anchor_x = size_x >> 1;
            const int anchor_y = size_y >> 1;
            float sum = 0.5F;
            for (int yf = -anchor_y; yf <= anchor_y; ++yf) {
                for (int xf = -anchor_x; xf <= anchor_x; ++xf) {
                    sum += _constmaskXY[yf + 2][xf + 2] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + yf + 2][(int)threadIdx.x + xf + 32];
                }
            }
            iter[(gid_y + 7 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)sum;
        }
    }
    goto BH_EXIT;
  BH_EXIT:
    ;
}
}

#endif //_CUGAUSSIANFILTERXY_CU_

