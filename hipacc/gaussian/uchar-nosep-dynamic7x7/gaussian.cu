#include "hip/hip_runtime.h"
#include "hipacc_cu.hpp"

#include "cuGaussianFilterXY.cu"
#include <algorithm>
#include <cmath>
#include <cstdlib>
#include <iostream>
#include <vector>
#include <sys/time.h>
#include <opencv2/opencv.hpp>
//---

#ifndef BH_MODE
#  define BH_MODE           CLAMP
#endif

#define PIXEL_CAST(a)       (pixel_t)(a)
#define TMP_PIXEL_CAST(a)   (tmp_pixel_t)(a)

#define pixel_t             uchar
#define tmp_pixel_t         float

#define USE_LAMBDA


// get time in milliseconds
double time_ms () {
    struct timeval tv;
    gettimeofday (&tv, NULL);

    return ((double)(tv.tv_sec) * 1e+3 + (double)(tv.tv_usec) * 1e-3);
}

// Gaussian blur filter in Hipacc
#ifdef NO_SEP


#else

class GaussianFilterRow : public Kernel<tmp_pixel_t> {
    private:
        Accessor<pixel_cast> &input;
        Mask<float> &mask;
        const int size;

    public:
        GaussianFilterRow(IterationSpace<tmp_pixel_t> &iter, Accessor<pixel_t>
                &input, Mask<float> &mask, const int size) :
            Kernel(iter),
            input(input),
            mask(mask),
            size(size)
        { add_accessor(&input); }

        #ifdef USE_LAMBDA

        void kernel() {
            output() = TMP_PIXEL_CAST(convolve(mask, Reduce::SUM, [&] () -> float {
                    return mask() * input(mask);
                    }));
        }

        #else

        void kernel() {
            const int anchor = size >> 1;
            float sum = 0.0f;

            for (int xf = -anchor; xf<=anchor; ++xf) {
                sum += mask(xf, 0) * input(xf, 0);
            }

            output() = TMP_PIXEL_CAST(sum);
        }

        #endif
};

class GaussianFilterColumn: public Kernel<pixel_t> {
    private:
        Accessor<tmp_pixel_t> &input;
        Mask<float> &mask;
        const int size;

    public:
        GaussianFilterColumn(IterationSpace<pixel_t> &iter,
                Accessor<tmp_pixel_t> &input, Mask<float> &mask, const int size) :
            Kernel(iter),
            input(input),
            mask(mask),
            size(size)
        { add_accessor(&input); }

        #ifdef USE_LAMBDA

        void kernel() {
            output() = PIXEL_CAST(convolve(mask, Reduce::SUM, [&] () -> float {
                    return mask() * input(mask);
                    }) + 0.5f);
        }

        #else

        void kernel() {
            const int anchor = size >> 1;
            float sum = 0.5f;

            for (int yf = -anchor; yf<=anchor; ++yf) {
                sum += mask(0, yf) * input(0, yf);
            }

            output() = PIXEL_CAST(sum);
        }

        #endif
};

#endif

int main(int argc, const char *argv[]) {
    hipaccInitCUDA();
    
    const int size_x = SIZE_X;
    const int size_y = SIZE_Y;
    const double sigma1 = ((size_x-1)*0.5 - 1)*0.3 + 0.8;
    const double sigma2 = ((size_y-1)*0.5 - 1)*0.3 + 0.8;

    // filter coefficients
    #ifdef CONST_MASK

    // only filter kernel sizes 3x3, 5x5, and 7x7 implemented
    if (size_x != size_y || !(size_x == 3 || size_x == 5 || size_x == 7)) {
        std::cerr << "Wrong filter kernel size. Currently supported values: 3x3, 5x5, and 7x7!" << std::endl;
        exit(EXIT_FAILURE);
    }

    // convolution filter mask
    const float filter_x[1][SIZE_X] = {
        #if SIZE_X == 3
        { 0.238994f, 0.522011f, 0.238994f }
        #endif
        #if SIZE_X == 5
        { 0.070766f, 0.244460f, 0.369546f, 0.244460f, 0.070766f }
        #endif
        #if SIZE_X == 7
        { 0.028995f, 0.103818f, 0.223173f, 0.288026f, 0.223173f, 0.103818f, 0.028995f }
        #endif
    };

    const float filter_y[SIZE_Y][1] = {
        #if SIZE_Y == 3
        { 0.238994f }, { 0.522011f }, { 0.238994f }
        #endif
        #if SIZE_Y == 5
        { 0.070766f }, { 0.244460f }, { 0.369546f }, { 0.244460f }, { 0.070766f }
        #endif
        #if SIZE_Y == 7
        { 0.028995f }, { 0.103818f }, { 0.223173f }, { 0.288026f }, { 0.223173f }, { 0.103818f }, { 0.028995f }
        #endif
    };

    const float filter_xy[SIZE_Y][SIZE_X] = {
        #if SIZE_X == 3
        { 0.057118f, 0.124758f, 0.057118f },
        { 0.124758f, 0.272496f, 0.124758f },
        { 0.057118f, 0.124758f, 0.057118f }
        #endif
        #if SIZE_X == 5
        { 0.005008f, 0.017300f, 0.026151f, 0.017300f, 0.005008f },
        { 0.017300f, 0.059761f, 0.090339f, 0.059761f, 0.017300f },
        { 0.026151f, 0.090339f, 0.136565f, 0.090339f, 0.026151f },
        { 0.017300f, 0.059761f, 0.090339f, 0.059761f, 0.017300f },
        { 0.005008f, 0.017300f, 0.026151f, 0.017300f, 0.005008f }
        #endif
        #if SIZE_X == 7
        { 0.000841, 0.003010, 0.006471, 0.008351, 0.006471, 0.003010, 0.000841 },
        { 0.003010, 0.010778, 0.023169, 0.029902, 0.023169, 0.010778, 0.003010 },
        { 0.006471, 0.023169, 0.049806, 0.064280, 0.049806, 0.023169, 0.006471 },
        { 0.008351, 0.029902, 0.064280, 0.082959, 0.064280, 0.029902, 0.008351 },
        { 0.006471, 0.023169, 0.049806, 0.064280, 0.049806, 0.023169, 0.006471 },
        { 0.003010, 0.010778, 0.023169, 0.029902, 0.023169, 0.010778, 0.003010 },
        { 0.000841, 0.003010, 0.006471, 0.008351, 0.006471, 0.003010, 0.000841 }
        #endif
    };

    #else

    float filter_x[1][SIZE_X];
    float filter_y[SIZE_Y][1];
    float filter_xy[SIZE_Y][SIZE_X];

    double scale2X = -0.5/(sigma1*sigma1);
    double scale2Y = -0.5/(sigma2*sigma2);
    double sum_x = 0;
    double sum_y = 0;

    for (int i=0; i < size_x; ++i) {
        double x = i - (size_x-1)*0.5;
        double t = std::exp(scale2X*x*x);

        filter_x[0][i] = (float)t;
        sum_x += filter_x[0][i];
    }
    for (int i=0; i < size_y; ++i) {
        double x = i - (size_y-1)*0.5;
        double t = std::exp(scale2Y*x*x);

        filter_y[i][0] = (float)t;
        sum_y += filter_y[i][0];
    }

    sum_x = 1./sum_x;
    sum_y = 1./sum_y;
    for (int i=0; i < size_x; ++i) {
        filter_x[0][i] = (float)(filter_x[0][i]*sum_x);
    }
    for (int i=0; i < size_y; ++i) {
        filter_y[i][0] = (float)(filter_y[i][0]*sum_y);
    }

    for (int y=0; y < size_y; ++y) {
        for (int x=0; x < size_x; ++x) {
            filter_xy[y][x] = filter_x[0][x]*filter_y[y][0];
        }
    }

    #endif

    cv::Mat frame, frame_colored;

    frame_colored = cv::imread(argv[1], CV_LOAD_IMAGE_COLOR);
    cvtColor(frame_colored, frame, CV_BGR2GRAY);

    const int width = frame.cols;
    const int height = frame.rows;
    float timing = 0;

    // input and output image of width x height pixels
    HipaccImage input = hipaccCreateMemory<uchar>(NULL, width, height, 256);
    HipaccImage output = hipaccCreateMemory<uchar>(NULL, width, height, 256);
    HipaccImage tmp = hipaccCreateMemory<float>(NULL, width, height, 256);

    hipaccWriteMemory(input, frame.data);

    // filter mask
    hipaccWriteSymbol<float>((const void *)&_constmaskXY, (float *)filter_xy, 7, 7);
    
    

    HipaccAccessor iter_out(output);
    HipaccAccessor iter_tmp(tmp);

    #ifdef NO_SEP

    
    HipaccAccessor acc(input);
    

    hipacc_launch_info XY_info0(3, 3, iter_out, 8, 1);
    dim3 block0(32, 1);
    dim3 grid0(hipaccCalcGridFromBlock(XY_info0, block0));

    hipaccPrepareKernelLaunch(XY_info0, block0);
    hipaccConfigureCall(grid0, block0);

    size_t offset0 = 0;
    hipGetTextureReference(&_texinputXYRef, HIP_SYMBOL(&_texinputXY));
    hipaccBindTexture<uchar>(Linear1D, _texinputXYRef, acc.img);
    hipaccSetupArgument(&iter_out.img.mem, sizeof(uchar *), offset0);
    hipaccSetupArgument(&iter_out.width, sizeof(const int), offset0);
    hipaccSetupArgument(&iter_out.height, sizeof(const int), offset0);
    hipaccSetupArgument(&iter_out.img.stride, sizeof(const int), offset0);
    hipaccSetupArgument(&acc.width, sizeof(const int), offset0);
    hipaccSetupArgument(&acc.height, sizeof(const int), offset0);
    hipaccSetupArgument(&acc.img.stride, sizeof(const int), offset0);
    hipaccSetupArgument(&XY_info0.bh_start_left, sizeof(const int), offset0);
    hipaccSetupArgument(&XY_info0.bh_start_right, sizeof(const int), offset0);
    hipaccSetupArgument(&XY_info0.bh_start_top, sizeof(const int), offset0);
    hipaccSetupArgument(&XY_info0.bh_start_bottom, sizeof(const int), offset0);
    hipaccSetupArgument(&XY_info0.bh_fall_back, sizeof(const int), offset0);
    
    hipaccLaunchKernel((const void *)&cuGaussianFilterXYKernel, "cuGaussianFilterXYKernel", grid0, block0);
    timing = hipacc_last_kernel_timing();

    #else

    BoundaryCondition<pixel_t> cond_in(input, mask_x, Boundary::BH_MODE);
    Accessor<pixel_t> acc(cond_in);
    GaussianFilterRow X(iter_tmp, acc, mask_x, size_x);

    BoundaryCondition<tmp_pixel_t> cond_tmp(tmp, mask_y, Boundary::BH_MODE);
    Accessor<tmp_pixel_t> acc_tmp(cond_tmp);
    GaussianFilterColumn Y(iter_out, acc_tmp, mask_y, size_y);

    X.execute();
    timing = hipacc_last_kernel_timing();
    Y.execute();
    timing += hipacc_last_kernel_timing();

    #endif

    std::cerr << "Timing: " << timing << " ms, " << (width*height/timing)/1000 << " Mpixel/s" << std::endl;

    // OpenCV display image
    std::string outputfn;
    std::vector<int> compression_params;

    frame.data = hipaccReadMemory<uchar>(output);
    outputfn = argv[1];
    outputfn = outputfn.substr(0, outputfn.find_last_of(".")) + "-filtered.jpg";
    compression_params.push_back(CV_IMWRITE_JPEG_QUALITY);
    compression_params.push_back(100);

    try {
      cv::imwrite(outputfn, frame, compression_params);
    } catch(std::runtime_error &ex) {
      fprintf(stderr, "JPEG compression exception: %s\n", ex.what());
      return 1;
    }

    cv::namedWindow("Result", cv::WINDOW_NORMAL);
    cv::imshow("Result", frame);
    cv::waitKey(0);

    fprintf(stdout, "Done!\n");
    hipaccReleaseMemory<float>(tmp);
    hipaccReleaseMemory<uchar>(output);
    hipaccReleaseMemory<uchar>(input);
    return 0;
}
