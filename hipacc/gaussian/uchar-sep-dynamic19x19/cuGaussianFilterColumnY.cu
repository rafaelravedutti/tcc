#include "hip/hip_runtime.h"
#ifndef _CUGAUSSIANFILTERCOLUMNY_CU_
#define _CUGAUSSIANFILTERCOLUMNY_CU_

#include "hipacc_types.hpp"
#include "hipacc_math_functions.hpp"

texture<float, hipTextureType1D, hipReadModeElementType> _texinputY;
const textureReference *_texinputYRef;
__device__ __constant__ float _constmask_yY[19][1];


extern "C" {
__global__ __launch_bounds__ (32*2) void cuGaussianFilterColumnYKernel(uchar * __restrict__ iter, int iter_width, int iter_height, int iter_stride, int input_width, int input_height, int input_stride, int bh_start_right, int bh_start_top, int bh_start_bottom, int bh_fall_back) {
    const int gid_x = blockDim.x * blockIdx.x + threadIdx.x;
    const int gid_y = blockDim.y * blockIdx.y * 8 + threadIdx.y;
    float _smeminput[34][33] __attribute__((shared));
    if (bh_fall_back)
        goto BH_FB;
    if (blockIdx.y < bh_start_top)
        goto BH_T;
    if (blockIdx.y >= bh_start_bottom)
        goto BH_B;
    if (blockIdx.x >= bh_start_right)
        goto BH_R;
    goto BH_NO;
  BH_FB:
    {
        int _gid_y0 = gid_y + (-9);
        if (_gid_y0 >= input_height)
            _gid_y0 = input_height - 1;
        if (_gid_y0 < 0)
            _gid_y0 = 0;
        _smeminput[(int)threadIdx.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y0) * input_stride + gid_x);
        int _gid_y1 = gid_y + 1 * (int)blockDim.y + (-9);
        if (_gid_y1 >= input_height)
            _gid_y1 = input_height - 1;
        if (_gid_y1 < 0)
            _gid_y1 = 0;
        _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y1) * input_stride + gid_x);
        int _gid_y2 = gid_y + 2 * (int)blockDim.y + (-9);
        if (_gid_y2 >= input_height)
            _gid_y2 = input_height - 1;
        if (_gid_y2 < 0)
            _gid_y2 = 0;
        _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y2) * input_stride + gid_x);
        int _gid_y3 = gid_y + 3 * (int)blockDim.y + (-9);
        if (_gid_y3 >= input_height)
            _gid_y3 = input_height - 1;
        if (_gid_y3 < 0)
            _gid_y3 = 0;
        _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y3) * input_stride + gid_x);
        int _gid_y4 = gid_y + 4 * (int)blockDim.y + (-9);
        if (_gid_y4 >= input_height)
            _gid_y4 = input_height - 1;
        if (_gid_y4 < 0)
            _gid_y4 = 0;
        _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y4) * input_stride + gid_x);
        int _gid_y5 = gid_y + 5 * (int)blockDim.y + (-9);
        if (_gid_y5 >= input_height)
            _gid_y5 = input_height - 1;
        if (_gid_y5 < 0)
            _gid_y5 = 0;
        _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y5) * input_stride + gid_x);
        int _gid_y6 = gid_y + 6 * (int)blockDim.y + (-9);
        if (_gid_y6 >= input_height)
            _gid_y6 = input_height - 1;
        if (_gid_y6 < 0)
            _gid_y6 = 0;
        _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y6) * input_stride + gid_x);
        int _gid_y7 = gid_y + 7 * (int)blockDim.y + (-9);
        if (_gid_y7 >= input_height)
            _gid_y7 = input_height - 1;
        if (_gid_y7 < 0)
            _gid_y7 = 0;
        _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y7) * input_stride + gid_x);
        int _gid_y8 = gid_y + 8 * (int)blockDim.y + (-9);
        if (_gid_y8 >= input_height)
            _gid_y8 = input_height - 1;
        if (_gid_y8 < 0)
            _gid_y8 = 0;
        _smeminput[(int)threadIdx.y + 8 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y8) * input_stride + gid_x);
        int _gid_y9 = gid_y + 9 * (int)blockDim.y + (-9);
        if (_gid_y9 >= input_height)
            _gid_y9 = input_height - 1;
        if (_gid_y9 < 0)
            _gid_y9 = 0;
        _smeminput[(int)threadIdx.y + 9 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y9) * input_stride + gid_x);
        int _gid_y10 = gid_y + 10 * (int)blockDim.y + (-9);
        if (_gid_y10 >= input_height)
            _gid_y10 = input_height - 1;
        if (_gid_y10 < 0)
            _gid_y10 = 0;
        _smeminput[(int)threadIdx.y + 10 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y10) * input_stride + gid_x);
        int _gid_y11 = gid_y + 11 * (int)blockDim.y + (-9);
        if (_gid_y11 >= input_height)
            _gid_y11 = input_height - 1;
        if (_gid_y11 < 0)
            _gid_y11 = 0;
        _smeminput[(int)threadIdx.y + 11 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y11) * input_stride + gid_x);
        int _gid_y12 = gid_y + 12 * (int)blockDim.y + (-9);
        if (_gid_y12 >= input_height)
            _gid_y12 = input_height - 1;
        if (_gid_y12 < 0)
            _gid_y12 = 0;
        _smeminput[(int)threadIdx.y + 12 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y12) * input_stride + gid_x);
        int _gid_y13 = gid_y + 13 * (int)blockDim.y + (-9);
        if (_gid_y13 >= input_height)
            _gid_y13 = input_height - 1;
        if (_gid_y13 < 0)
            _gid_y13 = 0;
        _smeminput[(int)threadIdx.y + 13 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y13) * input_stride + gid_x);
        int _gid_y14 = gid_y + 14 * (int)blockDim.y + (-9);
        if (_gid_y14 >= input_height)
            _gid_y14 = input_height - 1;
        if (_gid_y14 < 0)
            _gid_y14 = 0;
        _smeminput[(int)threadIdx.y + 14 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y14) * input_stride + gid_x);
        int _gid_y15 = gid_y + 15 * (int)blockDim.y + (-9);
        if (_gid_y15 >= input_height)
            _gid_y15 = input_height - 1;
        if (_gid_y15 < 0)
            _gid_y15 = 0;
        _smeminput[(int)threadIdx.y + 15 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y15) * input_stride + gid_x);
        int _gid_y16 = gid_y + 16 * (int)blockDim.y + (-9);
        if (_gid_y16 >= input_height)
            _gid_y16 = input_height - 1;
        if (_gid_y16 < 0)
            _gid_y16 = 0;
        _smeminput[(int)threadIdx.y + 16 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y16) * input_stride + gid_x);
        __syncthreads();
        if (gid_x < iter_width) {
            if (gid_y < iter_height) {
                float _tmp17 = 0.F;
                {
                    _tmp17 += _constmask_yY[0][0] * _smeminput[(int)threadIdx.y + -9 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp17 += _constmask_yY[1][0] * _smeminput[(int)threadIdx.y + -8 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp17 += _constmask_yY[2][0] * _smeminput[(int)threadIdx.y + -7 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp17 += _constmask_yY[3][0] * _smeminput[(int)threadIdx.y + -6 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp17 += _constmask_yY[4][0] * _smeminput[(int)threadIdx.y + -5 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp17 += _constmask_yY[5][0] * _smeminput[(int)threadIdx.y + -4 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp17 += _constmask_yY[6][0] * _smeminput[(int)threadIdx.y + -3 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp17 += _constmask_yY[7][0] * _smeminput[(int)threadIdx.y + -2 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp17 += _constmask_yY[8][0] * _smeminput[(int)threadIdx.y + -1 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp17 += _constmask_yY[9][0] * _smeminput[(int)threadIdx.y + 0 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp17 += _constmask_yY[10][0] * _smeminput[(int)threadIdx.y + 1 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp17 += _constmask_yY[11][0] * _smeminput[(int)threadIdx.y + 2 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp17 += _constmask_yY[12][0] * _smeminput[(int)threadIdx.y + 3 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp17 += _constmask_yY[13][0] * _smeminput[(int)threadIdx.y + 4 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp17 += _constmask_yY[14][0] * _smeminput[(int)threadIdx.y + 5 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp17 += _constmask_yY[15][0] * _smeminput[(int)threadIdx.y + 6 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp17 += _constmask_yY[16][0] * _smeminput[(int)threadIdx.y + 7 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp17 += _constmask_yY[17][0] * _smeminput[(int)threadIdx.y + 8 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp17 += _constmask_yY[18][0] * _smeminput[(int)threadIdx.y + 9 + 9][(int)threadIdx.x + 0 + 0];
                }
                iter[(gid_y) * iter_stride + gid_x] = (uchar)(_tmp17 + 0.5F);
            }
        }
        if (gid_x < iter_width) {
            if (gid_y + 1 * (int)blockDim.y < iter_height) {
                float _tmp18 = 0.F;
                {
                    _tmp18 += _constmask_yY[0][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + -9 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp18 += _constmask_yY[1][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + -8 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp18 += _constmask_yY[2][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + -7 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp18 += _constmask_yY[3][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + -6 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp18 += _constmask_yY[4][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + -5 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp18 += _constmask_yY[5][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + -4 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp18 += _constmask_yY[6][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + -3 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp18 += _constmask_yY[7][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + -2 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp18 += _constmask_yY[8][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + -1 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp18 += _constmask_yY[9][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp18 += _constmask_yY[10][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 1 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp18 += _constmask_yY[11][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 2 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp18 += _constmask_yY[12][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 3 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp18 += _constmask_yY[13][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 4 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp18 += _constmask_yY[14][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 5 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp18 += _constmask_yY[15][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 6 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp18 += _constmask_yY[16][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 7 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp18 += _constmask_yY[17][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 8 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp18 += _constmask_yY[18][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 9 + 9][(int)threadIdx.x + 0 + 0];
                }
                iter[(gid_y + 1 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp18 + 0.5F);
            }
        }
        if (gid_x < iter_width) {
            if (gid_y + 2 * (int)blockDim.y < iter_height) {
                float _tmp19 = 0.F;
                {
                    _tmp19 += _constmask_yY[0][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + -9 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp19 += _constmask_yY[1][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + -8 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp19 += _constmask_yY[2][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + -7 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp19 += _constmask_yY[3][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + -6 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp19 += _constmask_yY[4][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + -5 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp19 += _constmask_yY[5][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + -4 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp19 += _constmask_yY[6][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + -3 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp19 += _constmask_yY[7][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + -2 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp19 += _constmask_yY[8][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + -1 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp19 += _constmask_yY[9][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp19 += _constmask_yY[10][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 1 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp19 += _constmask_yY[11][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 2 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp19 += _constmask_yY[12][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 3 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp19 += _constmask_yY[13][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 4 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp19 += _constmask_yY[14][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 5 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp19 += _constmask_yY[15][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 6 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp19 += _constmask_yY[16][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 7 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp19 += _constmask_yY[17][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 8 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp19 += _constmask_yY[18][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 9 + 9][(int)threadIdx.x + 0 + 0];
                }
                iter[(gid_y + 2 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp19 + 0.5F);
            }
        }
        if (gid_x < iter_width) {
            if (gid_y + 3 * (int)blockDim.y < iter_height) {
                float _tmp20 = 0.F;
                {
                    _tmp20 += _constmask_yY[0][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + -9 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp20 += _constmask_yY[1][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + -8 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp20 += _constmask_yY[2][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + -7 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp20 += _constmask_yY[3][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + -6 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp20 += _constmask_yY[4][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + -5 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp20 += _constmask_yY[5][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + -4 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp20 += _constmask_yY[6][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + -3 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp20 += _constmask_yY[7][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + -2 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp20 += _constmask_yY[8][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + -1 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp20 += _constmask_yY[9][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp20 += _constmask_yY[10][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 1 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp20 += _constmask_yY[11][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 2 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp20 += _constmask_yY[12][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 3 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp20 += _constmask_yY[13][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 4 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp20 += _constmask_yY[14][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 5 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp20 += _constmask_yY[15][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 6 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp20 += _constmask_yY[16][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 7 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp20 += _constmask_yY[17][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 8 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp20 += _constmask_yY[18][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 9 + 9][(int)threadIdx.x + 0 + 0];
                }
                iter[(gid_y + 3 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp20 + 0.5F);
            }
        }
        if (gid_x < iter_width) {
            if (gid_y + 4 * (int)blockDim.y < iter_height) {
                float _tmp21 = 0.F;
                {
                    _tmp21 += _constmask_yY[0][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + -9 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp21 += _constmask_yY[1][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + -8 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp21 += _constmask_yY[2][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + -7 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp21 += _constmask_yY[3][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + -6 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp21 += _constmask_yY[4][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + -5 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp21 += _constmask_yY[5][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + -4 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp21 += _constmask_yY[6][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + -3 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp21 += _constmask_yY[7][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + -2 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp21 += _constmask_yY[8][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + -1 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp21 += _constmask_yY[9][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp21 += _constmask_yY[10][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 1 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp21 += _constmask_yY[11][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 2 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp21 += _constmask_yY[12][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 3 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp21 += _constmask_yY[13][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 4 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp21 += _constmask_yY[14][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 5 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp21 += _constmask_yY[15][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 6 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp21 += _constmask_yY[16][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 7 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp21 += _constmask_yY[17][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 8 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp21 += _constmask_yY[18][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 9 + 9][(int)threadIdx.x + 0 + 0];
                }
                iter[(gid_y + 4 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp21 + 0.5F);
            }
        }
        if (gid_x < iter_width) {
            if (gid_y + 5 * (int)blockDim.y < iter_height) {
                float _tmp22 = 0.F;
                {
                    _tmp22 += _constmask_yY[0][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + -9 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp22 += _constmask_yY[1][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + -8 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp22 += _constmask_yY[2][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + -7 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp22 += _constmask_yY[3][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + -6 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp22 += _constmask_yY[4][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + -5 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp22 += _constmask_yY[5][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + -4 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp22 += _constmask_yY[6][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + -3 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp22 += _constmask_yY[7][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + -2 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp22 += _constmask_yY[8][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + -1 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp22 += _constmask_yY[9][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp22 += _constmask_yY[10][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 1 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp22 += _constmask_yY[11][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 2 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp22 += _constmask_yY[12][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 3 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp22 += _constmask_yY[13][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 4 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp22 += _constmask_yY[14][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 5 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp22 += _constmask_yY[15][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 6 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp22 += _constmask_yY[16][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 7 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp22 += _constmask_yY[17][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 8 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp22 += _constmask_yY[18][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 9 + 9][(int)threadIdx.x + 0 + 0];
                }
                iter[(gid_y + 5 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp22 + 0.5F);
            }
        }
        if (gid_x < iter_width) {
            if (gid_y + 6 * (int)blockDim.y < iter_height) {
                float _tmp23 = 0.F;
                {
                    _tmp23 += _constmask_yY[0][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + -9 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp23 += _constmask_yY[1][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + -8 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp23 += _constmask_yY[2][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + -7 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp23 += _constmask_yY[3][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + -6 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp23 += _constmask_yY[4][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + -5 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp23 += _constmask_yY[5][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + -4 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp23 += _constmask_yY[6][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + -3 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp23 += _constmask_yY[7][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + -2 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp23 += _constmask_yY[8][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + -1 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp23 += _constmask_yY[9][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp23 += _constmask_yY[10][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 1 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp23 += _constmask_yY[11][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 2 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp23 += _constmask_yY[12][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 3 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp23 += _constmask_yY[13][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 4 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp23 += _constmask_yY[14][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 5 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp23 += _constmask_yY[15][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 6 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp23 += _constmask_yY[16][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 7 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp23 += _constmask_yY[17][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 8 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp23 += _constmask_yY[18][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 9 + 9][(int)threadIdx.x + 0 + 0];
                }
                iter[(gid_y + 6 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp23 + 0.5F);
            }
        }
        if (gid_x < iter_width) {
            if (gid_y + 7 * (int)blockDim.y < iter_height) {
                float _tmp24 = 0.F;
                {
                    _tmp24 += _constmask_yY[0][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + -9 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp24 += _constmask_yY[1][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + -8 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp24 += _constmask_yY[2][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + -7 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp24 += _constmask_yY[3][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + -6 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp24 += _constmask_yY[4][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + -5 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp24 += _constmask_yY[5][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + -4 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp24 += _constmask_yY[6][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + -3 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp24 += _constmask_yY[7][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + -2 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp24 += _constmask_yY[8][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + -1 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp24 += _constmask_yY[9][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp24 += _constmask_yY[10][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 1 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp24 += _constmask_yY[11][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 2 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp24 += _constmask_yY[12][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 3 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp24 += _constmask_yY[13][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 4 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp24 += _constmask_yY[14][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 5 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp24 += _constmask_yY[15][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 6 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp24 += _constmask_yY[16][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 7 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp24 += _constmask_yY[17][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 8 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp24 += _constmask_yY[18][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 9 + 9][(int)threadIdx.x + 0 + 0];
                }
                iter[(gid_y + 7 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp24 + 0.5F);
            }
        }
    }
    goto BH_EXIT;
  BH_T:
    {
        int _gid_y25 = gid_y + (-9);
        if (_gid_y25 < 0)
            _gid_y25 = 0;
        _smeminput[(int)threadIdx.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y25) * input_stride + gid_x);
        int _gid_y26 = gid_y + 1 * (int)blockDim.y + (-9);
        if (_gid_y26 < 0)
            _gid_y26 = 0;
        _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y26) * input_stride + gid_x);
        int _gid_y27 = gid_y + 2 * (int)blockDim.y + (-9);
        if (_gid_y27 < 0)
            _gid_y27 = 0;
        _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y27) * input_stride + gid_x);
        int _gid_y28 = gid_y + 3 * (int)blockDim.y + (-9);
        if (_gid_y28 < 0)
            _gid_y28 = 0;
        _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y28) * input_stride + gid_x);
        int _gid_y29 = gid_y + 4 * (int)blockDim.y + (-9);
        if (_gid_y29 < 0)
            _gid_y29 = 0;
        _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y29) * input_stride + gid_x);
        int _gid_y30 = gid_y + 5 * (int)blockDim.y + (-9);
        if (_gid_y30 < 0)
            _gid_y30 = 0;
        _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y30) * input_stride + gid_x);
        int _gid_y31 = gid_y + 6 * (int)blockDim.y + (-9);
        if (_gid_y31 < 0)
            _gid_y31 = 0;
        _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y31) * input_stride + gid_x);
        int _gid_y32 = gid_y + 7 * (int)blockDim.y + (-9);
        if (_gid_y32 < 0)
            _gid_y32 = 0;
        _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y32) * input_stride + gid_x);
        int _gid_y33 = gid_y + 8 * (int)blockDim.y + (-9);
        if (_gid_y33 < 0)
            _gid_y33 = 0;
        _smeminput[(int)threadIdx.y + 8 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y33) * input_stride + gid_x);
        int _gid_y34 = gid_y + 9 * (int)blockDim.y + (-9);
        if (_gid_y34 < 0)
            _gid_y34 = 0;
        _smeminput[(int)threadIdx.y + 9 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y34) * input_stride + gid_x);
        int _gid_y35 = gid_y + 10 * (int)blockDim.y + (-9);
        if (_gid_y35 < 0)
            _gid_y35 = 0;
        _smeminput[(int)threadIdx.y + 10 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y35) * input_stride + gid_x);
        int _gid_y36 = gid_y + 11 * (int)blockDim.y + (-9);
        if (_gid_y36 < 0)
            _gid_y36 = 0;
        _smeminput[(int)threadIdx.y + 11 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y36) * input_stride + gid_x);
        int _gid_y37 = gid_y + 12 * (int)blockDim.y + (-9);
        if (_gid_y37 < 0)
            _gid_y37 = 0;
        _smeminput[(int)threadIdx.y + 12 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y37) * input_stride + gid_x);
        int _gid_y38 = gid_y + 13 * (int)blockDim.y + (-9);
        if (_gid_y38 < 0)
            _gid_y38 = 0;
        _smeminput[(int)threadIdx.y + 13 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y38) * input_stride + gid_x);
        int _gid_y39 = gid_y + 14 * (int)blockDim.y + (-9);
        if (_gid_y39 < 0)
            _gid_y39 = 0;
        _smeminput[(int)threadIdx.y + 14 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y39) * input_stride + gid_x);
        int _gid_y40 = gid_y + 15 * (int)blockDim.y + (-9);
        if (_gid_y40 < 0)
            _gid_y40 = 0;
        _smeminput[(int)threadIdx.y + 15 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y40) * input_stride + gid_x);
        int _gid_y41 = gid_y + 16 * (int)blockDim.y + (-9);
        if (_gid_y41 < 0)
            _gid_y41 = 0;
        _smeminput[(int)threadIdx.y + 16 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y41) * input_stride + gid_x);
        __syncthreads();
        if (gid_x < iter_width) {
            {
                float _tmp42 = 0.F;
                {
                    _tmp42 += _constmask_yY[0][0] * _smeminput[(int)threadIdx.y + -9 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp42 += _constmask_yY[1][0] * _smeminput[(int)threadIdx.y + -8 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp42 += _constmask_yY[2][0] * _smeminput[(int)threadIdx.y + -7 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp42 += _constmask_yY[3][0] * _smeminput[(int)threadIdx.y + -6 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp42 += _constmask_yY[4][0] * _smeminput[(int)threadIdx.y + -5 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp42 += _constmask_yY[5][0] * _smeminput[(int)threadIdx.y + -4 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp42 += _constmask_yY[6][0] * _smeminput[(int)threadIdx.y + -3 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp42 += _constmask_yY[7][0] * _smeminput[(int)threadIdx.y + -2 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp42 += _constmask_yY[8][0] * _smeminput[(int)threadIdx.y + -1 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp42 += _constmask_yY[9][0] * _smeminput[(int)threadIdx.y + 0 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp42 += _constmask_yY[10][0] * _smeminput[(int)threadIdx.y + 1 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp42 += _constmask_yY[11][0] * _smeminput[(int)threadIdx.y + 2 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp42 += _constmask_yY[12][0] * _smeminput[(int)threadIdx.y + 3 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp42 += _constmask_yY[13][0] * _smeminput[(int)threadIdx.y + 4 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp42 += _constmask_yY[14][0] * _smeminput[(int)threadIdx.y + 5 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp42 += _constmask_yY[15][0] * _smeminput[(int)threadIdx.y + 6 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp42 += _constmask_yY[16][0] * _smeminput[(int)threadIdx.y + 7 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp42 += _constmask_yY[17][0] * _smeminput[(int)threadIdx.y + 8 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp42 += _constmask_yY[18][0] * _smeminput[(int)threadIdx.y + 9 + 9][(int)threadIdx.x + 0 + 0];
                }
                iter[(gid_y) * iter_stride + gid_x] = (uchar)(_tmp42 + 0.5F);
            }
        }
        if (gid_x < iter_width) {
            {
                float _tmp43 = 0.F;
                {
                    _tmp43 += _constmask_yY[0][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + -9 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp43 += _constmask_yY[1][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + -8 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp43 += _constmask_yY[2][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + -7 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp43 += _constmask_yY[3][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + -6 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp43 += _constmask_yY[4][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + -5 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp43 += _constmask_yY[5][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + -4 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp43 += _constmask_yY[6][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + -3 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp43 += _constmask_yY[7][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + -2 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp43 += _constmask_yY[8][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + -1 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp43 += _constmask_yY[9][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp43 += _constmask_yY[10][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 1 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp43 += _constmask_yY[11][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 2 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp43 += _constmask_yY[12][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 3 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp43 += _constmask_yY[13][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 4 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp43 += _constmask_yY[14][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 5 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp43 += _constmask_yY[15][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 6 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp43 += _constmask_yY[16][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 7 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp43 += _constmask_yY[17][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 8 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp43 += _constmask_yY[18][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 9 + 9][(int)threadIdx.x + 0 + 0];
                }
                iter[(gid_y + 1 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp43 + 0.5F);
            }
        }
        if (gid_x < iter_width) {
            {
                float _tmp44 = 0.F;
                {
                    _tmp44 += _constmask_yY[0][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + -9 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp44 += _constmask_yY[1][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + -8 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp44 += _constmask_yY[2][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + -7 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp44 += _constmask_yY[3][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + -6 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp44 += _constmask_yY[4][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + -5 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp44 += _constmask_yY[5][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + -4 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp44 += _constmask_yY[6][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + -3 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp44 += _constmask_yY[7][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + -2 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp44 += _constmask_yY[8][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + -1 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp44 += _constmask_yY[9][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp44 += _constmask_yY[10][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 1 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp44 += _constmask_yY[11][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 2 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp44 += _constmask_yY[12][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 3 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp44 += _constmask_yY[13][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 4 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp44 += _constmask_yY[14][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 5 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp44 += _constmask_yY[15][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 6 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp44 += _constmask_yY[16][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 7 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp44 += _constmask_yY[17][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 8 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp44 += _constmask_yY[18][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 9 + 9][(int)threadIdx.x + 0 + 0];
                }
                iter[(gid_y + 2 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp44 + 0.5F);
            }
        }
        if (gid_x < iter_width) {
            {
                float _tmp45 = 0.F;
                {
                    _tmp45 += _constmask_yY[0][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + -9 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp45 += _constmask_yY[1][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + -8 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp45 += _constmask_yY[2][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + -7 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp45 += _constmask_yY[3][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + -6 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp45 += _constmask_yY[4][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + -5 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp45 += _constmask_yY[5][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + -4 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp45 += _constmask_yY[6][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + -3 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp45 += _constmask_yY[7][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + -2 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp45 += _constmask_yY[8][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + -1 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp45 += _constmask_yY[9][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp45 += _constmask_yY[10][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 1 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp45 += _constmask_yY[11][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 2 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp45 += _constmask_yY[12][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 3 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp45 += _constmask_yY[13][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 4 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp45 += _constmask_yY[14][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 5 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp45 += _constmask_yY[15][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 6 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp45 += _constmask_yY[16][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 7 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp45 += _constmask_yY[17][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 8 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp45 += _constmask_yY[18][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 9 + 9][(int)threadIdx.x + 0 + 0];
                }
                iter[(gid_y + 3 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp45 + 0.5F);
            }
        }
        if (gid_x < iter_width) {
            {
                float _tmp46 = 0.F;
                {
                    _tmp46 += _constmask_yY[0][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + -9 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp46 += _constmask_yY[1][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + -8 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp46 += _constmask_yY[2][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + -7 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp46 += _constmask_yY[3][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + -6 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp46 += _constmask_yY[4][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + -5 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp46 += _constmask_yY[5][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + -4 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp46 += _constmask_yY[6][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + -3 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp46 += _constmask_yY[7][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + -2 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp46 += _constmask_yY[8][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + -1 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp46 += _constmask_yY[9][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp46 += _constmask_yY[10][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 1 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp46 += _constmask_yY[11][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 2 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp46 += _constmask_yY[12][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 3 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp46 += _constmask_yY[13][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 4 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp46 += _constmask_yY[14][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 5 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp46 += _constmask_yY[15][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 6 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp46 += _constmask_yY[16][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 7 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp46 += _constmask_yY[17][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 8 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp46 += _constmask_yY[18][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 9 + 9][(int)threadIdx.x + 0 + 0];
                }
                iter[(gid_y + 4 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp46 + 0.5F);
            }
        }
        if (gid_x < iter_width) {
            {
                float _tmp47 = 0.F;
                {
                    _tmp47 += _constmask_yY[0][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + -9 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp47 += _constmask_yY[1][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + -8 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp47 += _constmask_yY[2][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + -7 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp47 += _constmask_yY[3][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + -6 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp47 += _constmask_yY[4][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + -5 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp47 += _constmask_yY[5][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + -4 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp47 += _constmask_yY[6][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + -3 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp47 += _constmask_yY[7][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + -2 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp47 += _constmask_yY[8][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + -1 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp47 += _constmask_yY[9][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp47 += _constmask_yY[10][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 1 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp47 += _constmask_yY[11][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 2 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp47 += _constmask_yY[12][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 3 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp47 += _constmask_yY[13][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 4 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp47 += _constmask_yY[14][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 5 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp47 += _constmask_yY[15][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 6 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp47 += _constmask_yY[16][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 7 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp47 += _constmask_yY[17][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 8 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp47 += _constmask_yY[18][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 9 + 9][(int)threadIdx.x + 0 + 0];
                }
                iter[(gid_y + 5 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp47 + 0.5F);
            }
        }
        if (gid_x < iter_width) {
            {
                float _tmp48 = 0.F;
                {
                    _tmp48 += _constmask_yY[0][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + -9 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp48 += _constmask_yY[1][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + -8 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp48 += _constmask_yY[2][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + -7 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp48 += _constmask_yY[3][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + -6 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp48 += _constmask_yY[4][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + -5 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp48 += _constmask_yY[5][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + -4 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp48 += _constmask_yY[6][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + -3 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp48 += _constmask_yY[7][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + -2 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp48 += _constmask_yY[8][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + -1 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp48 += _constmask_yY[9][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp48 += _constmask_yY[10][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 1 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp48 += _constmask_yY[11][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 2 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp48 += _constmask_yY[12][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 3 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp48 += _constmask_yY[13][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 4 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp48 += _constmask_yY[14][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 5 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp48 += _constmask_yY[15][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 6 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp48 += _constmask_yY[16][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 7 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp48 += _constmask_yY[17][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 8 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp48 += _constmask_yY[18][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 9 + 9][(int)threadIdx.x + 0 + 0];
                }
                iter[(gid_y + 6 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp48 + 0.5F);
            }
        }
        if (gid_x < iter_width) {
            {
                float _tmp49 = 0.F;
                {
                    _tmp49 += _constmask_yY[0][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + -9 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp49 += _constmask_yY[1][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + -8 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp49 += _constmask_yY[2][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + -7 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp49 += _constmask_yY[3][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + -6 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp49 += _constmask_yY[4][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + -5 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp49 += _constmask_yY[5][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + -4 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp49 += _constmask_yY[6][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + -3 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp49 += _constmask_yY[7][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + -2 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp49 += _constmask_yY[8][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + -1 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp49 += _constmask_yY[9][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp49 += _constmask_yY[10][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 1 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp49 += _constmask_yY[11][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 2 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp49 += _constmask_yY[12][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 3 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp49 += _constmask_yY[13][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 4 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp49 += _constmask_yY[14][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 5 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp49 += _constmask_yY[15][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 6 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp49 += _constmask_yY[16][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 7 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp49 += _constmask_yY[17][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 8 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp49 += _constmask_yY[18][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 9 + 9][(int)threadIdx.x + 0 + 0];
                }
                iter[(gid_y + 7 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp49 + 0.5F);
            }
        }
    }
    goto BH_EXIT;
  BH_B:
    {
        int _gid_y50 = gid_y + (-9);
        if (_gid_y50 >= input_height)
            _gid_y50 = input_height - 1;
        _smeminput[(int)threadIdx.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y50) * input_stride + gid_x);
        int _gid_y51 = gid_y + 1 * (int)blockDim.y + (-9);
        if (_gid_y51 >= input_height)
            _gid_y51 = input_height - 1;
        _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y51) * input_stride + gid_x);
        int _gid_y52 = gid_y + 2 * (int)blockDim.y + (-9);
        if (_gid_y52 >= input_height)
            _gid_y52 = input_height - 1;
        _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y52) * input_stride + gid_x);
        int _gid_y53 = gid_y + 3 * (int)blockDim.y + (-9);
        if (_gid_y53 >= input_height)
            _gid_y53 = input_height - 1;
        _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y53) * input_stride + gid_x);
        int _gid_y54 = gid_y + 4 * (int)blockDim.y + (-9);
        if (_gid_y54 >= input_height)
            _gid_y54 = input_height - 1;
        _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y54) * input_stride + gid_x);
        int _gid_y55 = gid_y + 5 * (int)blockDim.y + (-9);
        if (_gid_y55 >= input_height)
            _gid_y55 = input_height - 1;
        _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y55) * input_stride + gid_x);
        int _gid_y56 = gid_y + 6 * (int)blockDim.y + (-9);
        if (_gid_y56 >= input_height)
            _gid_y56 = input_height - 1;
        _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y56) * input_stride + gid_x);
        int _gid_y57 = gid_y + 7 * (int)blockDim.y + (-9);
        if (_gid_y57 >= input_height)
            _gid_y57 = input_height - 1;
        _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y57) * input_stride + gid_x);
        int _gid_y58 = gid_y + 8 * (int)blockDim.y + (-9);
        if (_gid_y58 >= input_height)
            _gid_y58 = input_height - 1;
        _smeminput[(int)threadIdx.y + 8 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y58) * input_stride + gid_x);
        int _gid_y59 = gid_y + 9 * (int)blockDim.y + (-9);
        if (_gid_y59 >= input_height)
            _gid_y59 = input_height - 1;
        _smeminput[(int)threadIdx.y + 9 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y59) * input_stride + gid_x);
        int _gid_y60 = gid_y + 10 * (int)blockDim.y + (-9);
        if (_gid_y60 >= input_height)
            _gid_y60 = input_height - 1;
        _smeminput[(int)threadIdx.y + 10 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y60) * input_stride + gid_x);
        int _gid_y61 = gid_y + 11 * (int)blockDim.y + (-9);
        if (_gid_y61 >= input_height)
            _gid_y61 = input_height - 1;
        _smeminput[(int)threadIdx.y + 11 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y61) * input_stride + gid_x);
        int _gid_y62 = gid_y + 12 * (int)blockDim.y + (-9);
        if (_gid_y62 >= input_height)
            _gid_y62 = input_height - 1;
        _smeminput[(int)threadIdx.y + 12 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y62) * input_stride + gid_x);
        int _gid_y63 = gid_y + 13 * (int)blockDim.y + (-9);
        if (_gid_y63 >= input_height)
            _gid_y63 = input_height - 1;
        _smeminput[(int)threadIdx.y + 13 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y63) * input_stride + gid_x);
        int _gid_y64 = gid_y + 14 * (int)blockDim.y + (-9);
        if (_gid_y64 >= input_height)
            _gid_y64 = input_height - 1;
        _smeminput[(int)threadIdx.y + 14 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y64) * input_stride + gid_x);
        int _gid_y65 = gid_y + 15 * (int)blockDim.y + (-9);
        if (_gid_y65 >= input_height)
            _gid_y65 = input_height - 1;
        _smeminput[(int)threadIdx.y + 15 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y65) * input_stride + gid_x);
        int _gid_y66 = gid_y + 16 * (int)blockDim.y + (-9);
        if (_gid_y66 >= input_height)
            _gid_y66 = input_height - 1;
        _smeminput[(int)threadIdx.y + 16 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y66) * input_stride + gid_x);
        __syncthreads();
        if (gid_x < iter_width) {
            if (gid_y < iter_height) {
                float _tmp67 = 0.F;
                {
                    _tmp67 += _constmask_yY[0][0] * _smeminput[(int)threadIdx.y + -9 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp67 += _constmask_yY[1][0] * _smeminput[(int)threadIdx.y + -8 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp67 += _constmask_yY[2][0] * _smeminput[(int)threadIdx.y + -7 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp67 += _constmask_yY[3][0] * _smeminput[(int)threadIdx.y + -6 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp67 += _constmask_yY[4][0] * _smeminput[(int)threadIdx.y + -5 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp67 += _constmask_yY[5][0] * _smeminput[(int)threadIdx.y + -4 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp67 += _constmask_yY[6][0] * _smeminput[(int)threadIdx.y + -3 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp67 += _constmask_yY[7][0] * _smeminput[(int)threadIdx.y + -2 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp67 += _constmask_yY[8][0] * _smeminput[(int)threadIdx.y + -1 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp67 += _constmask_yY[9][0] * _smeminput[(int)threadIdx.y + 0 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp67 += _constmask_yY[10][0] * _smeminput[(int)threadIdx.y + 1 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp67 += _constmask_yY[11][0] * _smeminput[(int)threadIdx.y + 2 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp67 += _constmask_yY[12][0] * _smeminput[(int)threadIdx.y + 3 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp67 += _constmask_yY[13][0] * _smeminput[(int)threadIdx.y + 4 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp67 += _constmask_yY[14][0] * _smeminput[(int)threadIdx.y + 5 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp67 += _constmask_yY[15][0] * _smeminput[(int)threadIdx.y + 6 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp67 += _constmask_yY[16][0] * _smeminput[(int)threadIdx.y + 7 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp67 += _constmask_yY[17][0] * _smeminput[(int)threadIdx.y + 8 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp67 += _constmask_yY[18][0] * _smeminput[(int)threadIdx.y + 9 + 9][(int)threadIdx.x + 0 + 0];
                }
                iter[(gid_y) * iter_stride + gid_x] = (uchar)(_tmp67 + 0.5F);
            }
        }
        if (gid_x < iter_width) {
            if (gid_y + 1 * (int)blockDim.y < iter_height) {
                float _tmp68 = 0.F;
                {
                    _tmp68 += _constmask_yY[0][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + -9 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp68 += _constmask_yY[1][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + -8 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp68 += _constmask_yY[2][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + -7 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp68 += _constmask_yY[3][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + -6 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp68 += _constmask_yY[4][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + -5 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp68 += _constmask_yY[5][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + -4 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp68 += _constmask_yY[6][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + -3 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp68 += _constmask_yY[7][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + -2 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp68 += _constmask_yY[8][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + -1 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp68 += _constmask_yY[9][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp68 += _constmask_yY[10][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 1 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp68 += _constmask_yY[11][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 2 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp68 += _constmask_yY[12][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 3 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp68 += _constmask_yY[13][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 4 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp68 += _constmask_yY[14][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 5 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp68 += _constmask_yY[15][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 6 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp68 += _constmask_yY[16][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 7 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp68 += _constmask_yY[17][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 8 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp68 += _constmask_yY[18][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 9 + 9][(int)threadIdx.x + 0 + 0];
                }
                iter[(gid_y + 1 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp68 + 0.5F);
            }
        }
        if (gid_x < iter_width) {
            if (gid_y + 2 * (int)blockDim.y < iter_height) {
                float _tmp69 = 0.F;
                {
                    _tmp69 += _constmask_yY[0][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + -9 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp69 += _constmask_yY[1][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + -8 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp69 += _constmask_yY[2][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + -7 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp69 += _constmask_yY[3][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + -6 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp69 += _constmask_yY[4][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + -5 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp69 += _constmask_yY[5][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + -4 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp69 += _constmask_yY[6][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + -3 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp69 += _constmask_yY[7][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + -2 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp69 += _constmask_yY[8][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + -1 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp69 += _constmask_yY[9][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp69 += _constmask_yY[10][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 1 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp69 += _constmask_yY[11][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 2 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp69 += _constmask_yY[12][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 3 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp69 += _constmask_yY[13][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 4 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp69 += _constmask_yY[14][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 5 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp69 += _constmask_yY[15][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 6 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp69 += _constmask_yY[16][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 7 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp69 += _constmask_yY[17][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 8 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp69 += _constmask_yY[18][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 9 + 9][(int)threadIdx.x + 0 + 0];
                }
                iter[(gid_y + 2 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp69 + 0.5F);
            }
        }
        if (gid_x < iter_width) {
            if (gid_y + 3 * (int)blockDim.y < iter_height) {
                float _tmp70 = 0.F;
                {
                    _tmp70 += _constmask_yY[0][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + -9 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp70 += _constmask_yY[1][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + -8 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp70 += _constmask_yY[2][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + -7 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp70 += _constmask_yY[3][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + -6 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp70 += _constmask_yY[4][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + -5 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp70 += _constmask_yY[5][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + -4 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp70 += _constmask_yY[6][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + -3 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp70 += _constmask_yY[7][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + -2 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp70 += _constmask_yY[8][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + -1 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp70 += _constmask_yY[9][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp70 += _constmask_yY[10][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 1 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp70 += _constmask_yY[11][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 2 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp70 += _constmask_yY[12][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 3 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp70 += _constmask_yY[13][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 4 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp70 += _constmask_yY[14][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 5 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp70 += _constmask_yY[15][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 6 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp70 += _constmask_yY[16][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 7 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp70 += _constmask_yY[17][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 8 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp70 += _constmask_yY[18][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 9 + 9][(int)threadIdx.x + 0 + 0];
                }
                iter[(gid_y + 3 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp70 + 0.5F);
            }
        }
        if (gid_x < iter_width) {
            if (gid_y + 4 * (int)blockDim.y < iter_height) {
                float _tmp71 = 0.F;
                {
                    _tmp71 += _constmask_yY[0][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + -9 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp71 += _constmask_yY[1][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + -8 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp71 += _constmask_yY[2][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + -7 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp71 += _constmask_yY[3][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + -6 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp71 += _constmask_yY[4][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + -5 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp71 += _constmask_yY[5][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + -4 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp71 += _constmask_yY[6][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + -3 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp71 += _constmask_yY[7][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + -2 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp71 += _constmask_yY[8][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + -1 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp71 += _constmask_yY[9][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp71 += _constmask_yY[10][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 1 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp71 += _constmask_yY[11][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 2 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp71 += _constmask_yY[12][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 3 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp71 += _constmask_yY[13][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 4 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp71 += _constmask_yY[14][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 5 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp71 += _constmask_yY[15][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 6 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp71 += _constmask_yY[16][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 7 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp71 += _constmask_yY[17][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 8 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp71 += _constmask_yY[18][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 9 + 9][(int)threadIdx.x + 0 + 0];
                }
                iter[(gid_y + 4 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp71 + 0.5F);
            }
        }
        if (gid_x < iter_width) {
            if (gid_y + 5 * (int)blockDim.y < iter_height) {
                float _tmp72 = 0.F;
                {
                    _tmp72 += _constmask_yY[0][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + -9 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp72 += _constmask_yY[1][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + -8 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp72 += _constmask_yY[2][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + -7 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp72 += _constmask_yY[3][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + -6 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp72 += _constmask_yY[4][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + -5 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp72 += _constmask_yY[5][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + -4 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp72 += _constmask_yY[6][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + -3 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp72 += _constmask_yY[7][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + -2 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp72 += _constmask_yY[8][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + -1 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp72 += _constmask_yY[9][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp72 += _constmask_yY[10][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 1 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp72 += _constmask_yY[11][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 2 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp72 += _constmask_yY[12][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 3 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp72 += _constmask_yY[13][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 4 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp72 += _constmask_yY[14][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 5 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp72 += _constmask_yY[15][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 6 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp72 += _constmask_yY[16][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 7 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp72 += _constmask_yY[17][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 8 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp72 += _constmask_yY[18][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 9 + 9][(int)threadIdx.x + 0 + 0];
                }
                iter[(gid_y + 5 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp72 + 0.5F);
            }
        }
        if (gid_x < iter_width) {
            if (gid_y + 6 * (int)blockDim.y < iter_height) {
                float _tmp73 = 0.F;
                {
                    _tmp73 += _constmask_yY[0][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + -9 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp73 += _constmask_yY[1][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + -8 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp73 += _constmask_yY[2][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + -7 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp73 += _constmask_yY[3][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + -6 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp73 += _constmask_yY[4][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + -5 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp73 += _constmask_yY[5][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + -4 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp73 += _constmask_yY[6][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + -3 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp73 += _constmask_yY[7][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + -2 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp73 += _constmask_yY[8][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + -1 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp73 += _constmask_yY[9][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp73 += _constmask_yY[10][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 1 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp73 += _constmask_yY[11][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 2 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp73 += _constmask_yY[12][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 3 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp73 += _constmask_yY[13][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 4 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp73 += _constmask_yY[14][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 5 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp73 += _constmask_yY[15][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 6 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp73 += _constmask_yY[16][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 7 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp73 += _constmask_yY[17][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 8 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp73 += _constmask_yY[18][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 9 + 9][(int)threadIdx.x + 0 + 0];
                }
                iter[(gid_y + 6 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp73 + 0.5F);
            }
        }
        if (gid_x < iter_width) {
            if (gid_y + 7 * (int)blockDim.y < iter_height) {
                float _tmp74 = 0.F;
                {
                    _tmp74 += _constmask_yY[0][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + -9 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp74 += _constmask_yY[1][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + -8 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp74 += _constmask_yY[2][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + -7 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp74 += _constmask_yY[3][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + -6 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp74 += _constmask_yY[4][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + -5 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp74 += _constmask_yY[5][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + -4 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp74 += _constmask_yY[6][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + -3 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp74 += _constmask_yY[7][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + -2 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp74 += _constmask_yY[8][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + -1 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp74 += _constmask_yY[9][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp74 += _constmask_yY[10][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 1 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp74 += _constmask_yY[11][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 2 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp74 += _constmask_yY[12][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 3 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp74 += _constmask_yY[13][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 4 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp74 += _constmask_yY[14][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 5 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp74 += _constmask_yY[15][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 6 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp74 += _constmask_yY[16][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 7 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp74 += _constmask_yY[17][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 8 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp74 += _constmask_yY[18][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 9 + 9][(int)threadIdx.x + 0 + 0];
                }
                iter[(gid_y + 7 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp74 + 0.5F);
            }
        }
    }
    goto BH_EXIT;
  BH_R:
    {
        int _gid_y75 = gid_y + (-9);
        _smeminput[(int)threadIdx.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y75) * input_stride + gid_x);
        int _gid_y76 = gid_y + 1 * (int)blockDim.y + (-9);
        _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y76) * input_stride + gid_x);
        int _gid_y77 = gid_y + 2 * (int)blockDim.y + (-9);
        _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y77) * input_stride + gid_x);
        int _gid_y78 = gid_y + 3 * (int)blockDim.y + (-9);
        _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y78) * input_stride + gid_x);
        int _gid_y79 = gid_y + 4 * (int)blockDim.y + (-9);
        _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y79) * input_stride + gid_x);
        int _gid_y80 = gid_y + 5 * (int)blockDim.y + (-9);
        _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y80) * input_stride + gid_x);
        int _gid_y81 = gid_y + 6 * (int)blockDim.y + (-9);
        _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y81) * input_stride + gid_x);
        int _gid_y82 = gid_y + 7 * (int)blockDim.y + (-9);
        _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y82) * input_stride + gid_x);
        int _gid_y83 = gid_y + 8 * (int)blockDim.y + (-9);
        _smeminput[(int)threadIdx.y + 8 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y83) * input_stride + gid_x);
        int _gid_y84 = gid_y + 9 * (int)blockDim.y + (-9);
        _smeminput[(int)threadIdx.y + 9 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y84) * input_stride + gid_x);
        int _gid_y85 = gid_y + 10 * (int)blockDim.y + (-9);
        _smeminput[(int)threadIdx.y + 10 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y85) * input_stride + gid_x);
        int _gid_y86 = gid_y + 11 * (int)blockDim.y + (-9);
        _smeminput[(int)threadIdx.y + 11 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y86) * input_stride + gid_x);
        int _gid_y87 = gid_y + 12 * (int)blockDim.y + (-9);
        _smeminput[(int)threadIdx.y + 12 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y87) * input_stride + gid_x);
        int _gid_y88 = gid_y + 13 * (int)blockDim.y + (-9);
        _smeminput[(int)threadIdx.y + 13 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y88) * input_stride + gid_x);
        int _gid_y89 = gid_y + 14 * (int)blockDim.y + (-9);
        _smeminput[(int)threadIdx.y + 14 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y89) * input_stride + gid_x);
        int _gid_y90 = gid_y + 15 * (int)blockDim.y + (-9);
        _smeminput[(int)threadIdx.y + 15 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y90) * input_stride + gid_x);
        int _gid_y91 = gid_y + 16 * (int)blockDim.y + (-9);
        _smeminput[(int)threadIdx.y + 16 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y91) * input_stride + gid_x);
        __syncthreads();
        if (gid_x < iter_width) {
            {
                float _tmp92 = 0.F;
                {
                    _tmp92 += _constmask_yY[0][0] * _smeminput[(int)threadIdx.y + -9 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp92 += _constmask_yY[1][0] * _smeminput[(int)threadIdx.y + -8 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp92 += _constmask_yY[2][0] * _smeminput[(int)threadIdx.y + -7 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp92 += _constmask_yY[3][0] * _smeminput[(int)threadIdx.y + -6 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp92 += _constmask_yY[4][0] * _smeminput[(int)threadIdx.y + -5 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp92 += _constmask_yY[5][0] * _smeminput[(int)threadIdx.y + -4 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp92 += _constmask_yY[6][0] * _smeminput[(int)threadIdx.y + -3 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp92 += _constmask_yY[7][0] * _smeminput[(int)threadIdx.y + -2 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp92 += _constmask_yY[8][0] * _smeminput[(int)threadIdx.y + -1 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp92 += _constmask_yY[9][0] * _smeminput[(int)threadIdx.y + 0 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp92 += _constmask_yY[10][0] * _smeminput[(int)threadIdx.y + 1 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp92 += _constmask_yY[11][0] * _smeminput[(int)threadIdx.y + 2 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp92 += _constmask_yY[12][0] * _smeminput[(int)threadIdx.y + 3 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp92 += _constmask_yY[13][0] * _smeminput[(int)threadIdx.y + 4 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp92 += _constmask_yY[14][0] * _smeminput[(int)threadIdx.y + 5 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp92 += _constmask_yY[15][0] * _smeminput[(int)threadIdx.y + 6 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp92 += _constmask_yY[16][0] * _smeminput[(int)threadIdx.y + 7 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp92 += _constmask_yY[17][0] * _smeminput[(int)threadIdx.y + 8 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp92 += _constmask_yY[18][0] * _smeminput[(int)threadIdx.y + 9 + 9][(int)threadIdx.x + 0 + 0];
                }
                iter[(gid_y) * iter_stride + gid_x] = (uchar)(_tmp92 + 0.5F);
            }
        }
        if (gid_x < iter_width) {
            {
                float _tmp93 = 0.F;
                {
                    _tmp93 += _constmask_yY[0][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + -9 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp93 += _constmask_yY[1][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + -8 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp93 += _constmask_yY[2][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + -7 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp93 += _constmask_yY[3][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + -6 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp93 += _constmask_yY[4][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + -5 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp93 += _constmask_yY[5][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + -4 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp93 += _constmask_yY[6][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + -3 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp93 += _constmask_yY[7][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + -2 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp93 += _constmask_yY[8][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + -1 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp93 += _constmask_yY[9][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp93 += _constmask_yY[10][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 1 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp93 += _constmask_yY[11][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 2 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp93 += _constmask_yY[12][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 3 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp93 += _constmask_yY[13][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 4 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp93 += _constmask_yY[14][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 5 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp93 += _constmask_yY[15][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 6 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp93 += _constmask_yY[16][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 7 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp93 += _constmask_yY[17][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 8 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp93 += _constmask_yY[18][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 9 + 9][(int)threadIdx.x + 0 + 0];
                }
                iter[(gid_y + 1 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp93 + 0.5F);
            }
        }
        if (gid_x < iter_width) {
            {
                float _tmp94 = 0.F;
                {
                    _tmp94 += _constmask_yY[0][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + -9 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp94 += _constmask_yY[1][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + -8 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp94 += _constmask_yY[2][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + -7 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp94 += _constmask_yY[3][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + -6 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp94 += _constmask_yY[4][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + -5 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp94 += _constmask_yY[5][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + -4 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp94 += _constmask_yY[6][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + -3 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp94 += _constmask_yY[7][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + -2 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp94 += _constmask_yY[8][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + -1 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp94 += _constmask_yY[9][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp94 += _constmask_yY[10][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 1 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp94 += _constmask_yY[11][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 2 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp94 += _constmask_yY[12][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 3 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp94 += _constmask_yY[13][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 4 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp94 += _constmask_yY[14][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 5 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp94 += _constmask_yY[15][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 6 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp94 += _constmask_yY[16][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 7 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp94 += _constmask_yY[17][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 8 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp94 += _constmask_yY[18][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 9 + 9][(int)threadIdx.x + 0 + 0];
                }
                iter[(gid_y + 2 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp94 + 0.5F);
            }
        }
        if (gid_x < iter_width) {
            {
                float _tmp95 = 0.F;
                {
                    _tmp95 += _constmask_yY[0][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + -9 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp95 += _constmask_yY[1][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + -8 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp95 += _constmask_yY[2][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + -7 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp95 += _constmask_yY[3][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + -6 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp95 += _constmask_yY[4][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + -5 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp95 += _constmask_yY[5][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + -4 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp95 += _constmask_yY[6][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + -3 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp95 += _constmask_yY[7][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + -2 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp95 += _constmask_yY[8][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + -1 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp95 += _constmask_yY[9][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp95 += _constmask_yY[10][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 1 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp95 += _constmask_yY[11][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 2 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp95 += _constmask_yY[12][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 3 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp95 += _constmask_yY[13][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 4 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp95 += _constmask_yY[14][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 5 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp95 += _constmask_yY[15][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 6 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp95 += _constmask_yY[16][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 7 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp95 += _constmask_yY[17][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 8 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp95 += _constmask_yY[18][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 9 + 9][(int)threadIdx.x + 0 + 0];
                }
                iter[(gid_y + 3 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp95 + 0.5F);
            }
        }
        if (gid_x < iter_width) {
            {
                float _tmp96 = 0.F;
                {
                    _tmp96 += _constmask_yY[0][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + -9 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp96 += _constmask_yY[1][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + -8 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp96 += _constmask_yY[2][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + -7 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp96 += _constmask_yY[3][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + -6 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp96 += _constmask_yY[4][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + -5 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp96 += _constmask_yY[5][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + -4 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp96 += _constmask_yY[6][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + -3 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp96 += _constmask_yY[7][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + -2 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp96 += _constmask_yY[8][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + -1 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp96 += _constmask_yY[9][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp96 += _constmask_yY[10][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 1 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp96 += _constmask_yY[11][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 2 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp96 += _constmask_yY[12][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 3 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp96 += _constmask_yY[13][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 4 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp96 += _constmask_yY[14][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 5 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp96 += _constmask_yY[15][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 6 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp96 += _constmask_yY[16][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 7 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp96 += _constmask_yY[17][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 8 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp96 += _constmask_yY[18][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 9 + 9][(int)threadIdx.x + 0 + 0];
                }
                iter[(gid_y + 4 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp96 + 0.5F);
            }
        }
        if (gid_x < iter_width) {
            {
                float _tmp97 = 0.F;
                {
                    _tmp97 += _constmask_yY[0][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + -9 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp97 += _constmask_yY[1][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + -8 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp97 += _constmask_yY[2][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + -7 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp97 += _constmask_yY[3][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + -6 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp97 += _constmask_yY[4][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + -5 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp97 += _constmask_yY[5][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + -4 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp97 += _constmask_yY[6][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + -3 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp97 += _constmask_yY[7][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + -2 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp97 += _constmask_yY[8][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + -1 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp97 += _constmask_yY[9][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp97 += _constmask_yY[10][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 1 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp97 += _constmask_yY[11][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 2 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp97 += _constmask_yY[12][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 3 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp97 += _constmask_yY[13][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 4 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp97 += _constmask_yY[14][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 5 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp97 += _constmask_yY[15][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 6 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp97 += _constmask_yY[16][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 7 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp97 += _constmask_yY[17][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 8 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp97 += _constmask_yY[18][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 9 + 9][(int)threadIdx.x + 0 + 0];
                }
                iter[(gid_y + 5 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp97 + 0.5F);
            }
        }
        if (gid_x < iter_width) {
            {
                float _tmp98 = 0.F;
                {
                    _tmp98 += _constmask_yY[0][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + -9 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp98 += _constmask_yY[1][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + -8 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp98 += _constmask_yY[2][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + -7 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp98 += _constmask_yY[3][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + -6 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp98 += _constmask_yY[4][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + -5 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp98 += _constmask_yY[5][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + -4 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp98 += _constmask_yY[6][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + -3 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp98 += _constmask_yY[7][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + -2 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp98 += _constmask_yY[8][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + -1 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp98 += _constmask_yY[9][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp98 += _constmask_yY[10][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 1 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp98 += _constmask_yY[11][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 2 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp98 += _constmask_yY[12][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 3 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp98 += _constmask_yY[13][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 4 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp98 += _constmask_yY[14][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 5 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp98 += _constmask_yY[15][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 6 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp98 += _constmask_yY[16][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 7 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp98 += _constmask_yY[17][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 8 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp98 += _constmask_yY[18][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 9 + 9][(int)threadIdx.x + 0 + 0];
                }
                iter[(gid_y + 6 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp98 + 0.5F);
            }
        }
        if (gid_x < iter_width) {
            {
                float _tmp99 = 0.F;
                {
                    _tmp99 += _constmask_yY[0][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + -9 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp99 += _constmask_yY[1][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + -8 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp99 += _constmask_yY[2][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + -7 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp99 += _constmask_yY[3][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + -6 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp99 += _constmask_yY[4][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + -5 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp99 += _constmask_yY[5][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + -4 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp99 += _constmask_yY[6][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + -3 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp99 += _constmask_yY[7][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + -2 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp99 += _constmask_yY[8][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + -1 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp99 += _constmask_yY[9][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp99 += _constmask_yY[10][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 1 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp99 += _constmask_yY[11][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 2 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp99 += _constmask_yY[12][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 3 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp99 += _constmask_yY[13][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 4 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp99 += _constmask_yY[14][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 5 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp99 += _constmask_yY[15][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 6 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp99 += _constmask_yY[16][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 7 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp99 += _constmask_yY[17][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 8 + 9][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp99 += _constmask_yY[18][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 9 + 9][(int)threadIdx.x + 0 + 0];
                }
                iter[(gid_y + 7 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp99 + 0.5F);
            }
        }
    }
    goto BH_EXIT;
  BH_NO:
    {
        _smeminput[(int)threadIdx.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (gid_y + (-9)) * input_stride + gid_x);
        _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (gid_y + 1 * (int)blockDim.y + (-9)) * input_stride + gid_x);
        _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (gid_y + 2 * (int)blockDim.y + (-9)) * input_stride + gid_x);
        _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (gid_y + 3 * (int)blockDim.y + (-9)) * input_stride + gid_x);
        _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (gid_y + 4 * (int)blockDim.y + (-9)) * input_stride + gid_x);
        _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (gid_y + 5 * (int)blockDim.y + (-9)) * input_stride + gid_x);
        _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (gid_y + 6 * (int)blockDim.y + (-9)) * input_stride + gid_x);
        _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (gid_y + 7 * (int)blockDim.y + (-9)) * input_stride + gid_x);
        _smeminput[(int)threadIdx.y + 8 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (gid_y + 8 * (int)blockDim.y + (-9)) * input_stride + gid_x);
        _smeminput[(int)threadIdx.y + 9 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (gid_y + 9 * (int)blockDim.y + (-9)) * input_stride + gid_x);
        _smeminput[(int)threadIdx.y + 10 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (gid_y + 10 * (int)blockDim.y + (-9)) * input_stride + gid_x);
        _smeminput[(int)threadIdx.y + 11 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (gid_y + 11 * (int)blockDim.y + (-9)) * input_stride + gid_x);
        _smeminput[(int)threadIdx.y + 12 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (gid_y + 12 * (int)blockDim.y + (-9)) * input_stride + gid_x);
        _smeminput[(int)threadIdx.y + 13 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (gid_y + 13 * (int)blockDim.y + (-9)) * input_stride + gid_x);
        _smeminput[(int)threadIdx.y + 14 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (gid_y + 14 * (int)blockDim.y + (-9)) * input_stride + gid_x);
        _smeminput[(int)threadIdx.y + 15 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (gid_y + 15 * (int)blockDim.y + (-9)) * input_stride + gid_x);
        _smeminput[(int)threadIdx.y + 16 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (gid_y + 16 * (int)blockDim.y + (-9)) * input_stride + gid_x);
        __syncthreads();
        {
            float _tmp100 = 0.F;
            {
                _tmp100 += _constmask_yY[0][0] * _smeminput[(int)threadIdx.y + -9 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp100 += _constmask_yY[1][0] * _smeminput[(int)threadIdx.y + -8 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp100 += _constmask_yY[2][0] * _smeminput[(int)threadIdx.y + -7 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp100 += _constmask_yY[3][0] * _smeminput[(int)threadIdx.y + -6 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp100 += _constmask_yY[4][0] * _smeminput[(int)threadIdx.y + -5 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp100 += _constmask_yY[5][0] * _smeminput[(int)threadIdx.y + -4 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp100 += _constmask_yY[6][0] * _smeminput[(int)threadIdx.y + -3 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp100 += _constmask_yY[7][0] * _smeminput[(int)threadIdx.y + -2 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp100 += _constmask_yY[8][0] * _smeminput[(int)threadIdx.y + -1 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp100 += _constmask_yY[9][0] * _smeminput[(int)threadIdx.y + 0 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp100 += _constmask_yY[10][0] * _smeminput[(int)threadIdx.y + 1 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp100 += _constmask_yY[11][0] * _smeminput[(int)threadIdx.y + 2 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp100 += _constmask_yY[12][0] * _smeminput[(int)threadIdx.y + 3 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp100 += _constmask_yY[13][0] * _smeminput[(int)threadIdx.y + 4 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp100 += _constmask_yY[14][0] * _smeminput[(int)threadIdx.y + 5 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp100 += _constmask_yY[15][0] * _smeminput[(int)threadIdx.y + 6 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp100 += _constmask_yY[16][0] * _smeminput[(int)threadIdx.y + 7 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp100 += _constmask_yY[17][0] * _smeminput[(int)threadIdx.y + 8 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp100 += _constmask_yY[18][0] * _smeminput[(int)threadIdx.y + 9 + 9][(int)threadIdx.x + 0 + 0];
            }
            iter[(gid_y) * iter_stride + gid_x] = (uchar)(_tmp100 + 0.5F);
        }
        {
            float _tmp101 = 0.F;
            {
                _tmp101 += _constmask_yY[0][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + -9 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp101 += _constmask_yY[1][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + -8 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp101 += _constmask_yY[2][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + -7 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp101 += _constmask_yY[3][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + -6 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp101 += _constmask_yY[4][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + -5 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp101 += _constmask_yY[5][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + -4 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp101 += _constmask_yY[6][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + -3 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp101 += _constmask_yY[7][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + -2 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp101 += _constmask_yY[8][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + -1 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp101 += _constmask_yY[9][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp101 += _constmask_yY[10][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 1 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp101 += _constmask_yY[11][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 2 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp101 += _constmask_yY[12][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 3 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp101 += _constmask_yY[13][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 4 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp101 += _constmask_yY[14][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 5 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp101 += _constmask_yY[15][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 6 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp101 += _constmask_yY[16][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 7 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp101 += _constmask_yY[17][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 8 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp101 += _constmask_yY[18][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 9 + 9][(int)threadIdx.x + 0 + 0];
            }
            iter[(gid_y + 1 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp101 + 0.5F);
        }
        {
            float _tmp102 = 0.F;
            {
                _tmp102 += _constmask_yY[0][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + -9 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp102 += _constmask_yY[1][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + -8 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp102 += _constmask_yY[2][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + -7 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp102 += _constmask_yY[3][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + -6 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp102 += _constmask_yY[4][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + -5 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp102 += _constmask_yY[5][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + -4 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp102 += _constmask_yY[6][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + -3 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp102 += _constmask_yY[7][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + -2 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp102 += _constmask_yY[8][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + -1 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp102 += _constmask_yY[9][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp102 += _constmask_yY[10][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 1 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp102 += _constmask_yY[11][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 2 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp102 += _constmask_yY[12][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 3 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp102 += _constmask_yY[13][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 4 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp102 += _constmask_yY[14][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 5 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp102 += _constmask_yY[15][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 6 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp102 += _constmask_yY[16][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 7 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp102 += _constmask_yY[17][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 8 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp102 += _constmask_yY[18][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 9 + 9][(int)threadIdx.x + 0 + 0];
            }
            iter[(gid_y + 2 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp102 + 0.5F);
        }
        {
            float _tmp103 = 0.F;
            {
                _tmp103 += _constmask_yY[0][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + -9 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp103 += _constmask_yY[1][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + -8 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp103 += _constmask_yY[2][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + -7 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp103 += _constmask_yY[3][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + -6 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp103 += _constmask_yY[4][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + -5 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp103 += _constmask_yY[5][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + -4 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp103 += _constmask_yY[6][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + -3 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp103 += _constmask_yY[7][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + -2 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp103 += _constmask_yY[8][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + -1 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp103 += _constmask_yY[9][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp103 += _constmask_yY[10][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 1 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp103 += _constmask_yY[11][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 2 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp103 += _constmask_yY[12][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 3 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp103 += _constmask_yY[13][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 4 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp103 += _constmask_yY[14][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 5 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp103 += _constmask_yY[15][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 6 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp103 += _constmask_yY[16][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 7 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp103 += _constmask_yY[17][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 8 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp103 += _constmask_yY[18][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 9 + 9][(int)threadIdx.x + 0 + 0];
            }
            iter[(gid_y + 3 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp103 + 0.5F);
        }
        {
            float _tmp104 = 0.F;
            {
                _tmp104 += _constmask_yY[0][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + -9 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp104 += _constmask_yY[1][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + -8 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp104 += _constmask_yY[2][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + -7 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp104 += _constmask_yY[3][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + -6 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp104 += _constmask_yY[4][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + -5 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp104 += _constmask_yY[5][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + -4 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp104 += _constmask_yY[6][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + -3 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp104 += _constmask_yY[7][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + -2 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp104 += _constmask_yY[8][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + -1 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp104 += _constmask_yY[9][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp104 += _constmask_yY[10][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 1 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp104 += _constmask_yY[11][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 2 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp104 += _constmask_yY[12][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 3 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp104 += _constmask_yY[13][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 4 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp104 += _constmask_yY[14][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 5 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp104 += _constmask_yY[15][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 6 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp104 += _constmask_yY[16][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 7 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp104 += _constmask_yY[17][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 8 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp104 += _constmask_yY[18][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 9 + 9][(int)threadIdx.x + 0 + 0];
            }
            iter[(gid_y + 4 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp104 + 0.5F);
        }
        {
            float _tmp105 = 0.F;
            {
                _tmp105 += _constmask_yY[0][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + -9 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp105 += _constmask_yY[1][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + -8 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp105 += _constmask_yY[2][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + -7 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp105 += _constmask_yY[3][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + -6 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp105 += _constmask_yY[4][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + -5 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp105 += _constmask_yY[5][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + -4 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp105 += _constmask_yY[6][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + -3 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp105 += _constmask_yY[7][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + -2 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp105 += _constmask_yY[8][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + -1 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp105 += _constmask_yY[9][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp105 += _constmask_yY[10][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 1 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp105 += _constmask_yY[11][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 2 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp105 += _constmask_yY[12][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 3 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp105 += _constmask_yY[13][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 4 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp105 += _constmask_yY[14][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 5 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp105 += _constmask_yY[15][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 6 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp105 += _constmask_yY[16][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 7 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp105 += _constmask_yY[17][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 8 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp105 += _constmask_yY[18][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 9 + 9][(int)threadIdx.x + 0 + 0];
            }
            iter[(gid_y + 5 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp105 + 0.5F);
        }
        {
            float _tmp106 = 0.F;
            {
                _tmp106 += _constmask_yY[0][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + -9 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp106 += _constmask_yY[1][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + -8 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp106 += _constmask_yY[2][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + -7 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp106 += _constmask_yY[3][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + -6 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp106 += _constmask_yY[4][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + -5 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp106 += _constmask_yY[5][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + -4 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp106 += _constmask_yY[6][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + -3 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp106 += _constmask_yY[7][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + -2 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp106 += _constmask_yY[8][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + -1 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp106 += _constmask_yY[9][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp106 += _constmask_yY[10][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 1 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp106 += _constmask_yY[11][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 2 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp106 += _constmask_yY[12][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 3 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp106 += _constmask_yY[13][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 4 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp106 += _constmask_yY[14][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 5 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp106 += _constmask_yY[15][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 6 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp106 += _constmask_yY[16][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 7 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp106 += _constmask_yY[17][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 8 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp106 += _constmask_yY[18][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 9 + 9][(int)threadIdx.x + 0 + 0];
            }
            iter[(gid_y + 6 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp106 + 0.5F);
        }
        {
            float _tmp107 = 0.F;
            {
                _tmp107 += _constmask_yY[0][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + -9 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp107 += _constmask_yY[1][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + -8 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp107 += _constmask_yY[2][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + -7 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp107 += _constmask_yY[3][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + -6 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp107 += _constmask_yY[4][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + -5 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp107 += _constmask_yY[5][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + -4 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp107 += _constmask_yY[6][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + -3 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp107 += _constmask_yY[7][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + -2 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp107 += _constmask_yY[8][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + -1 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp107 += _constmask_yY[9][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp107 += _constmask_yY[10][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 1 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp107 += _constmask_yY[11][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 2 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp107 += _constmask_yY[12][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 3 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp107 += _constmask_yY[13][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 4 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp107 += _constmask_yY[14][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 5 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp107 += _constmask_yY[15][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 6 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp107 += _constmask_yY[16][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 7 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp107 += _constmask_yY[17][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 8 + 9][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp107 += _constmask_yY[18][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 9 + 9][(int)threadIdx.x + 0 + 0];
            }
            iter[(gid_y + 7 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp107 + 0.5F);
        }
    }
    goto BH_EXIT;
  BH_EXIT:
    ;
}
}

#endif //_CUGAUSSIANFILTERCOLUMNY_CU_

