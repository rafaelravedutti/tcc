#include "hip/hip_runtime.h"
#ifndef _CUGAUSSIANFILTERROWX_CU_
#define _CUGAUSSIANFILTERROWX_CU_

#include "hipacc_types.hpp"
#include "hipacc_math_functions.hpp"

texture<uchar, hipTextureType1D, hipReadModeElementType> _texinputX;
const textureReference *_texinputXRef;
__device__ __constant__ float _constmask_xX[1][19];


extern "C" {
__global__ __launch_bounds__ (32*1) void cuGaussianFilterRowXKernel(float * __restrict__ iter, int iter_width, int iter_height, int iter_stride, int input_width, int input_height, int input_stride, int bh_start_left, int bh_start_right, int bh_start_bottom, int bh_fall_back) {
    const int gid_x = blockDim.x * blockIdx.x + threadIdx.x;
    const int gid_y = blockDim.y * blockIdx.y * 8 + threadIdx.y;
    uchar _smeminput[8][97] __attribute__((shared));
    if (bh_fall_back)
        goto BH_FB;
    if (blockIdx.y >= bh_start_bottom)
        goto BH_B;
    if (blockIdx.x >= bh_start_right)
        goto BH_R;
    if (blockIdx.x < bh_start_left)
        goto BH_L;
    goto BH_NO;
  BH_FB:
    {
        int _gid_x0 = gid_x + 0 * (int)blockDim.x - 32;
        if (_gid_x0 >= input_width)
            _gid_x0 = input_width - 1;
        if (_gid_x0 < 0)
            _gid_x0 = 0;
        _smeminput[(int)threadIdx.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y) * input_stride + _gid_x0);
        int _gid_x1 = gid_x + 1 * (int)blockDim.x - 32;
        if (_gid_x1 >= input_width)
            _gid_x1 = input_width - 1;
        if (_gid_x1 < 0)
            _gid_x1 = 0;
        _smeminput[(int)threadIdx.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y) * input_stride + _gid_x1);
        int _gid_x2 = gid_x + 2 * (int)blockDim.x - 32;
        if (_gid_x2 >= input_width)
            _gid_x2 = input_width - 1;
        if (_gid_x2 < 0)
            _gid_x2 = 0;
        _smeminput[(int)threadIdx.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y) * input_stride + _gid_x2);
        int _gid_x3 = gid_x + 0 * (int)blockDim.x - 32;
        if (_gid_x3 >= input_width)
            _gid_x3 = input_width - 1;
        if (_gid_x3 < 0)
            _gid_x3 = 0;
        _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 1 * (int)blockDim.y) * input_stride + _gid_x3);
        int _gid_x4 = gid_x + 1 * (int)blockDim.x - 32;
        if (_gid_x4 >= input_width)
            _gid_x4 = input_width - 1;
        if (_gid_x4 < 0)
            _gid_x4 = 0;
        _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 1 * (int)blockDim.y) * input_stride + _gid_x4);
        int _gid_x5 = gid_x + 2 * (int)blockDim.x - 32;
        if (_gid_x5 >= input_width)
            _gid_x5 = input_width - 1;
        if (_gid_x5 < 0)
            _gid_x5 = 0;
        _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 1 * (int)blockDim.y) * input_stride + _gid_x5);
        int _gid_x6 = gid_x + 0 * (int)blockDim.x - 32;
        if (_gid_x6 >= input_width)
            _gid_x6 = input_width - 1;
        if (_gid_x6 < 0)
            _gid_x6 = 0;
        _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 2 * (int)blockDim.y) * input_stride + _gid_x6);
        int _gid_x7 = gid_x + 1 * (int)blockDim.x - 32;
        if (_gid_x7 >= input_width)
            _gid_x7 = input_width - 1;
        if (_gid_x7 < 0)
            _gid_x7 = 0;
        _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 2 * (int)blockDim.y) * input_stride + _gid_x7);
        int _gid_x8 = gid_x + 2 * (int)blockDim.x - 32;
        if (_gid_x8 >= input_width)
            _gid_x8 = input_width - 1;
        if (_gid_x8 < 0)
            _gid_x8 = 0;
        _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 2 * (int)blockDim.y) * input_stride + _gid_x8);
        int _gid_x9 = gid_x + 0 * (int)blockDim.x - 32;
        if (_gid_x9 >= input_width)
            _gid_x9 = input_width - 1;
        if (_gid_x9 < 0)
            _gid_x9 = 0;
        _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 3 * (int)blockDim.y) * input_stride + _gid_x9);
        int _gid_x10 = gid_x + 1 * (int)blockDim.x - 32;
        if (_gid_x10 >= input_width)
            _gid_x10 = input_width - 1;
        if (_gid_x10 < 0)
            _gid_x10 = 0;
        _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 3 * (int)blockDim.y) * input_stride + _gid_x10);
        int _gid_x11 = gid_x + 2 * (int)blockDim.x - 32;
        if (_gid_x11 >= input_width)
            _gid_x11 = input_width - 1;
        if (_gid_x11 < 0)
            _gid_x11 = 0;
        _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 3 * (int)blockDim.y) * input_stride + _gid_x11);
        int _gid_x12 = gid_x + 0 * (int)blockDim.x - 32;
        if (_gid_x12 >= input_width)
            _gid_x12 = input_width - 1;
        if (_gid_x12 < 0)
            _gid_x12 = 0;
        _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 4 * (int)blockDim.y) * input_stride + _gid_x12);
        int _gid_x13 = gid_x + 1 * (int)blockDim.x - 32;
        if (_gid_x13 >= input_width)
            _gid_x13 = input_width - 1;
        if (_gid_x13 < 0)
            _gid_x13 = 0;
        _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 4 * (int)blockDim.y) * input_stride + _gid_x13);
        int _gid_x14 = gid_x + 2 * (int)blockDim.x - 32;
        if (_gid_x14 >= input_width)
            _gid_x14 = input_width - 1;
        if (_gid_x14 < 0)
            _gid_x14 = 0;
        _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 4 * (int)blockDim.y) * input_stride + _gid_x14);
        int _gid_x15 = gid_x + 0 * (int)blockDim.x - 32;
        if (_gid_x15 >= input_width)
            _gid_x15 = input_width - 1;
        if (_gid_x15 < 0)
            _gid_x15 = 0;
        _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 5 * (int)blockDim.y) * input_stride + _gid_x15);
        int _gid_x16 = gid_x + 1 * (int)blockDim.x - 32;
        if (_gid_x16 >= input_width)
            _gid_x16 = input_width - 1;
        if (_gid_x16 < 0)
            _gid_x16 = 0;
        _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 5 * (int)blockDim.y) * input_stride + _gid_x16);
        int _gid_x17 = gid_x + 2 * (int)blockDim.x - 32;
        if (_gid_x17 >= input_width)
            _gid_x17 = input_width - 1;
        if (_gid_x17 < 0)
            _gid_x17 = 0;
        _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 5 * (int)blockDim.y) * input_stride + _gid_x17);
        int _gid_x18 = gid_x + 0 * (int)blockDim.x - 32;
        if (_gid_x18 >= input_width)
            _gid_x18 = input_width - 1;
        if (_gid_x18 < 0)
            _gid_x18 = 0;
        _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 6 * (int)blockDim.y) * input_stride + _gid_x18);
        int _gid_x19 = gid_x + 1 * (int)blockDim.x - 32;
        if (_gid_x19 >= input_width)
            _gid_x19 = input_width - 1;
        if (_gid_x19 < 0)
            _gid_x19 = 0;
        _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 6 * (int)blockDim.y) * input_stride + _gid_x19);
        int _gid_x20 = gid_x + 2 * (int)blockDim.x - 32;
        if (_gid_x20 >= input_width)
            _gid_x20 = input_width - 1;
        if (_gid_x20 < 0)
            _gid_x20 = 0;
        _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 6 * (int)blockDim.y) * input_stride + _gid_x20);
        int _gid_x21 = gid_x + 0 * (int)blockDim.x - 32;
        if (_gid_x21 >= input_width)
            _gid_x21 = input_width - 1;
        if (_gid_x21 < 0)
            _gid_x21 = 0;
        _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 7 * (int)blockDim.y) * input_stride + _gid_x21);
        int _gid_x22 = gid_x + 1 * (int)blockDim.x - 32;
        if (_gid_x22 >= input_width)
            _gid_x22 = input_width - 1;
        if (_gid_x22 < 0)
            _gid_x22 = 0;
        _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 7 * (int)blockDim.y) * input_stride + _gid_x22);
        int _gid_x23 = gid_x + 2 * (int)blockDim.x - 32;
        if (_gid_x23 >= input_width)
            _gid_x23 = input_width - 1;
        if (_gid_x23 < 0)
            _gid_x23 = 0;
        _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 7 * (int)blockDim.y) * input_stride + _gid_x23);
        __syncthreads();
        if (gid_x < iter_width) {
            if (gid_y < iter_height) {
                float _tmp24 = 0.F;
                {
                    _tmp24 += _constmask_xX[0][0] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + -9 + 32];
                }
                {
                    _tmp24 += _constmask_xX[0][1] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + -8 + 32];
                }
                {
                    _tmp24 += _constmask_xX[0][2] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + -7 + 32];
                }
                {
                    _tmp24 += _constmask_xX[0][3] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + -6 + 32];
                }
                {
                    _tmp24 += _constmask_xX[0][4] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + -5 + 32];
                }
                {
                    _tmp24 += _constmask_xX[0][5] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + -4 + 32];
                }
                {
                    _tmp24 += _constmask_xX[0][6] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + -3 + 32];
                }
                {
                    _tmp24 += _constmask_xX[0][7] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + -2 + 32];
                }
                {
                    _tmp24 += _constmask_xX[0][8] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + -1 + 32];
                }
                {
                    _tmp24 += _constmask_xX[0][9] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + 0 + 32];
                }
                {
                    _tmp24 += _constmask_xX[0][10] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + 1 + 32];
                }
                {
                    _tmp24 += _constmask_xX[0][11] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + 2 + 32];
                }
                {
                    _tmp24 += _constmask_xX[0][12] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + 3 + 32];
                }
                {
                    _tmp24 += _constmask_xX[0][13] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + 4 + 32];
                }
                {
                    _tmp24 += _constmask_xX[0][14] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + 5 + 32];
                }
                {
                    _tmp24 += _constmask_xX[0][15] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + 6 + 32];
                }
                {
                    _tmp24 += _constmask_xX[0][16] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + 7 + 32];
                }
                {
                    _tmp24 += _constmask_xX[0][17] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + 8 + 32];
                }
                {
                    _tmp24 += _constmask_xX[0][18] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + 9 + 32];
                }
                iter[(gid_y) * iter_stride + gid_x] = (float)(_tmp24);
            }
        }
        if (gid_x < iter_width) {
            if (gid_y + 1 * (int)blockDim.y < iter_height) {
                float _tmp25 = 0.F;
                {
                    _tmp25 += _constmask_xX[0][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -9 + 32];
                }
                {
                    _tmp25 += _constmask_xX[0][1] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -8 + 32];
                }
                {
                    _tmp25 += _constmask_xX[0][2] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -7 + 32];
                }
                {
                    _tmp25 += _constmask_xX[0][3] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -6 + 32];
                }
                {
                    _tmp25 += _constmask_xX[0][4] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -5 + 32];
                }
                {
                    _tmp25 += _constmask_xX[0][5] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -4 + 32];
                }
                {
                    _tmp25 += _constmask_xX[0][6] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -3 + 32];
                }
                {
                    _tmp25 += _constmask_xX[0][7] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -2 + 32];
                }
                {
                    _tmp25 += _constmask_xX[0][8] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -1 + 32];
                }
                {
                    _tmp25 += _constmask_xX[0][9] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 0 + 32];
                }
                {
                    _tmp25 += _constmask_xX[0][10] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 1 + 32];
                }
                {
                    _tmp25 += _constmask_xX[0][11] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 2 + 32];
                }
                {
                    _tmp25 += _constmask_xX[0][12] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 3 + 32];
                }
                {
                    _tmp25 += _constmask_xX[0][13] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 4 + 32];
                }
                {
                    _tmp25 += _constmask_xX[0][14] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 5 + 32];
                }
                {
                    _tmp25 += _constmask_xX[0][15] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 6 + 32];
                }
                {
                    _tmp25 += _constmask_xX[0][16] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 7 + 32];
                }
                {
                    _tmp25 += _constmask_xX[0][17] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 8 + 32];
                }
                {
                    _tmp25 += _constmask_xX[0][18] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 9 + 32];
                }
                iter[(gid_y + 1 * (int)blockDim.y) * iter_stride + gid_x] = (float)(_tmp25);
            }
        }
        if (gid_x < iter_width) {
            if (gid_y + 2 * (int)blockDim.y < iter_height) {
                float _tmp26 = 0.F;
                {
                    _tmp26 += _constmask_xX[0][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -9 + 32];
                }
                {
                    _tmp26 += _constmask_xX[0][1] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -8 + 32];
                }
                {
                    _tmp26 += _constmask_xX[0][2] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -7 + 32];
                }
                {
                    _tmp26 += _constmask_xX[0][3] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -6 + 32];
                }
                {
                    _tmp26 += _constmask_xX[0][4] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -5 + 32];
                }
                {
                    _tmp26 += _constmask_xX[0][5] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -4 + 32];
                }
                {
                    _tmp26 += _constmask_xX[0][6] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -3 + 32];
                }
                {
                    _tmp26 += _constmask_xX[0][7] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -2 + 32];
                }
                {
                    _tmp26 += _constmask_xX[0][8] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -1 + 32];
                }
                {
                    _tmp26 += _constmask_xX[0][9] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 0 + 32];
                }
                {
                    _tmp26 += _constmask_xX[0][10] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 1 + 32];
                }
                {
                    _tmp26 += _constmask_xX[0][11] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 2 + 32];
                }
                {
                    _tmp26 += _constmask_xX[0][12] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 3 + 32];
                }
                {
                    _tmp26 += _constmask_xX[0][13] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 4 + 32];
                }
                {
                    _tmp26 += _constmask_xX[0][14] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 5 + 32];
                }
                {
                    _tmp26 += _constmask_xX[0][15] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 6 + 32];
                }
                {
                    _tmp26 += _constmask_xX[0][16] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 7 + 32];
                }
                {
                    _tmp26 += _constmask_xX[0][17] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 8 + 32];
                }
                {
                    _tmp26 += _constmask_xX[0][18] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 9 + 32];
                }
                iter[(gid_y + 2 * (int)blockDim.y) * iter_stride + gid_x] = (float)(_tmp26);
            }
        }
        if (gid_x < iter_width) {
            if (gid_y + 3 * (int)blockDim.y < iter_height) {
                float _tmp27 = 0.F;
                {
                    _tmp27 += _constmask_xX[0][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -9 + 32];
                }
                {
                    _tmp27 += _constmask_xX[0][1] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -8 + 32];
                }
                {
                    _tmp27 += _constmask_xX[0][2] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -7 + 32];
                }
                {
                    _tmp27 += _constmask_xX[0][3] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -6 + 32];
                }
                {
                    _tmp27 += _constmask_xX[0][4] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -5 + 32];
                }
                {
                    _tmp27 += _constmask_xX[0][5] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -4 + 32];
                }
                {
                    _tmp27 += _constmask_xX[0][6] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -3 + 32];
                }
                {
                    _tmp27 += _constmask_xX[0][7] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -2 + 32];
                }
                {
                    _tmp27 += _constmask_xX[0][8] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -1 + 32];
                }
                {
                    _tmp27 += _constmask_xX[0][9] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 0 + 32];
                }
                {
                    _tmp27 += _constmask_xX[0][10] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 1 + 32];
                }
                {
                    _tmp27 += _constmask_xX[0][11] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 2 + 32];
                }
                {
                    _tmp27 += _constmask_xX[0][12] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 3 + 32];
                }
                {
                    _tmp27 += _constmask_xX[0][13] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 4 + 32];
                }
                {
                    _tmp27 += _constmask_xX[0][14] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 5 + 32];
                }
                {
                    _tmp27 += _constmask_xX[0][15] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 6 + 32];
                }
                {
                    _tmp27 += _constmask_xX[0][16] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 7 + 32];
                }
                {
                    _tmp27 += _constmask_xX[0][17] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 8 + 32];
                }
                {
                    _tmp27 += _constmask_xX[0][18] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 9 + 32];
                }
                iter[(gid_y + 3 * (int)blockDim.y) * iter_stride + gid_x] = (float)(_tmp27);
            }
        }
        if (gid_x < iter_width) {
            if (gid_y + 4 * (int)blockDim.y < iter_height) {
                float _tmp28 = 0.F;
                {
                    _tmp28 += _constmask_xX[0][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -9 + 32];
                }
                {
                    _tmp28 += _constmask_xX[0][1] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -8 + 32];
                }
                {
                    _tmp28 += _constmask_xX[0][2] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -7 + 32];
                }
                {
                    _tmp28 += _constmask_xX[0][3] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -6 + 32];
                }
                {
                    _tmp28 += _constmask_xX[0][4] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -5 + 32];
                }
                {
                    _tmp28 += _constmask_xX[0][5] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -4 + 32];
                }
                {
                    _tmp28 += _constmask_xX[0][6] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -3 + 32];
                }
                {
                    _tmp28 += _constmask_xX[0][7] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -2 + 32];
                }
                {
                    _tmp28 += _constmask_xX[0][8] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -1 + 32];
                }
                {
                    _tmp28 += _constmask_xX[0][9] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 0 + 32];
                }
                {
                    _tmp28 += _constmask_xX[0][10] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 1 + 32];
                }
                {
                    _tmp28 += _constmask_xX[0][11] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 2 + 32];
                }
                {
                    _tmp28 += _constmask_xX[0][12] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 3 + 32];
                }
                {
                    _tmp28 += _constmask_xX[0][13] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 4 + 32];
                }
                {
                    _tmp28 += _constmask_xX[0][14] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 5 + 32];
                }
                {
                    _tmp28 += _constmask_xX[0][15] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 6 + 32];
                }
                {
                    _tmp28 += _constmask_xX[0][16] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 7 + 32];
                }
                {
                    _tmp28 += _constmask_xX[0][17] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 8 + 32];
                }
                {
                    _tmp28 += _constmask_xX[0][18] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 9 + 32];
                }
                iter[(gid_y + 4 * (int)blockDim.y) * iter_stride + gid_x] = (float)(_tmp28);
            }
        }
        if (gid_x < iter_width) {
            if (gid_y + 5 * (int)blockDim.y < iter_height) {
                float _tmp29 = 0.F;
                {
                    _tmp29 += _constmask_xX[0][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -9 + 32];
                }
                {
                    _tmp29 += _constmask_xX[0][1] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -8 + 32];
                }
                {
                    _tmp29 += _constmask_xX[0][2] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -7 + 32];
                }
                {
                    _tmp29 += _constmask_xX[0][3] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -6 + 32];
                }
                {
                    _tmp29 += _constmask_xX[0][4] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -5 + 32];
                }
                {
                    _tmp29 += _constmask_xX[0][5] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -4 + 32];
                }
                {
                    _tmp29 += _constmask_xX[0][6] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -3 + 32];
                }
                {
                    _tmp29 += _constmask_xX[0][7] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -2 + 32];
                }
                {
                    _tmp29 += _constmask_xX[0][8] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -1 + 32];
                }
                {
                    _tmp29 += _constmask_xX[0][9] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 0 + 32];
                }
                {
                    _tmp29 += _constmask_xX[0][10] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 1 + 32];
                }
                {
                    _tmp29 += _constmask_xX[0][11] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 2 + 32];
                }
                {
                    _tmp29 += _constmask_xX[0][12] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 3 + 32];
                }
                {
                    _tmp29 += _constmask_xX[0][13] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 4 + 32];
                }
                {
                    _tmp29 += _constmask_xX[0][14] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 5 + 32];
                }
                {
                    _tmp29 += _constmask_xX[0][15] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 6 + 32];
                }
                {
                    _tmp29 += _constmask_xX[0][16] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 7 + 32];
                }
                {
                    _tmp29 += _constmask_xX[0][17] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 8 + 32];
                }
                {
                    _tmp29 += _constmask_xX[0][18] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 9 + 32];
                }
                iter[(gid_y + 5 * (int)blockDim.y) * iter_stride + gid_x] = (float)(_tmp29);
            }
        }
        if (gid_x < iter_width) {
            if (gid_y + 6 * (int)blockDim.y < iter_height) {
                float _tmp30 = 0.F;
                {
                    _tmp30 += _constmask_xX[0][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -9 + 32];
                }
                {
                    _tmp30 += _constmask_xX[0][1] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -8 + 32];
                }
                {
                    _tmp30 += _constmask_xX[0][2] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -7 + 32];
                }
                {
                    _tmp30 += _constmask_xX[0][3] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -6 + 32];
                }
                {
                    _tmp30 += _constmask_xX[0][4] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -5 + 32];
                }
                {
                    _tmp30 += _constmask_xX[0][5] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -4 + 32];
                }
                {
                    _tmp30 += _constmask_xX[0][6] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -3 + 32];
                }
                {
                    _tmp30 += _constmask_xX[0][7] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -2 + 32];
                }
                {
                    _tmp30 += _constmask_xX[0][8] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -1 + 32];
                }
                {
                    _tmp30 += _constmask_xX[0][9] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 0 + 32];
                }
                {
                    _tmp30 += _constmask_xX[0][10] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 1 + 32];
                }
                {
                    _tmp30 += _constmask_xX[0][11] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 2 + 32];
                }
                {
                    _tmp30 += _constmask_xX[0][12] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 3 + 32];
                }
                {
                    _tmp30 += _constmask_xX[0][13] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 4 + 32];
                }
                {
                    _tmp30 += _constmask_xX[0][14] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 5 + 32];
                }
                {
                    _tmp30 += _constmask_xX[0][15] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 6 + 32];
                }
                {
                    _tmp30 += _constmask_xX[0][16] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 7 + 32];
                }
                {
                    _tmp30 += _constmask_xX[0][17] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 8 + 32];
                }
                {
                    _tmp30 += _constmask_xX[0][18] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 9 + 32];
                }
                iter[(gid_y + 6 * (int)blockDim.y) * iter_stride + gid_x] = (float)(_tmp30);
            }
        }
        if (gid_x < iter_width) {
            if (gid_y + 7 * (int)blockDim.y < iter_height) {
                float _tmp31 = 0.F;
                {
                    _tmp31 += _constmask_xX[0][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -9 + 32];
                }
                {
                    _tmp31 += _constmask_xX[0][1] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -8 + 32];
                }
                {
                    _tmp31 += _constmask_xX[0][2] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -7 + 32];
                }
                {
                    _tmp31 += _constmask_xX[0][3] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -6 + 32];
                }
                {
                    _tmp31 += _constmask_xX[0][4] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -5 + 32];
                }
                {
                    _tmp31 += _constmask_xX[0][5] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -4 + 32];
                }
                {
                    _tmp31 += _constmask_xX[0][6] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -3 + 32];
                }
                {
                    _tmp31 += _constmask_xX[0][7] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -2 + 32];
                }
                {
                    _tmp31 += _constmask_xX[0][8] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -1 + 32];
                }
                {
                    _tmp31 += _constmask_xX[0][9] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 0 + 32];
                }
                {
                    _tmp31 += _constmask_xX[0][10] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 1 + 32];
                }
                {
                    _tmp31 += _constmask_xX[0][11] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 2 + 32];
                }
                {
                    _tmp31 += _constmask_xX[0][12] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 3 + 32];
                }
                {
                    _tmp31 += _constmask_xX[0][13] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 4 + 32];
                }
                {
                    _tmp31 += _constmask_xX[0][14] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 5 + 32];
                }
                {
                    _tmp31 += _constmask_xX[0][15] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 6 + 32];
                }
                {
                    _tmp31 += _constmask_xX[0][16] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 7 + 32];
                }
                {
                    _tmp31 += _constmask_xX[0][17] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 8 + 32];
                }
                {
                    _tmp31 += _constmask_xX[0][18] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 9 + 32];
                }
                iter[(gid_y + 7 * (int)blockDim.y) * iter_stride + gid_x] = (float)(_tmp31);
            }
        }
    }
    goto BH_EXIT;
  BH_B:
    {
        int _gid_x32 = gid_x + 0 * (int)blockDim.x - 32;
        _smeminput[(int)threadIdx.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y) * input_stride + _gid_x32);
        int _gid_x33 = gid_x + 1 * (int)blockDim.x - 32;
        _smeminput[(int)threadIdx.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y) * input_stride + _gid_x33);
        int _gid_x34 = gid_x + 2 * (int)blockDim.x - 32;
        _smeminput[(int)threadIdx.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y) * input_stride + _gid_x34);
        int _gid_x35 = gid_x + 0 * (int)blockDim.x - 32;
        _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 1 * (int)blockDim.y) * input_stride + _gid_x35);
        int _gid_x36 = gid_x + 1 * (int)blockDim.x - 32;
        _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 1 * (int)blockDim.y) * input_stride + _gid_x36);
        int _gid_x37 = gid_x + 2 * (int)blockDim.x - 32;
        _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 1 * (int)blockDim.y) * input_stride + _gid_x37);
        int _gid_x38 = gid_x + 0 * (int)blockDim.x - 32;
        _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 2 * (int)blockDim.y) * input_stride + _gid_x38);
        int _gid_x39 = gid_x + 1 * (int)blockDim.x - 32;
        _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 2 * (int)blockDim.y) * input_stride + _gid_x39);
        int _gid_x40 = gid_x + 2 * (int)blockDim.x - 32;
        _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 2 * (int)blockDim.y) * input_stride + _gid_x40);
        int _gid_x41 = gid_x + 0 * (int)blockDim.x - 32;
        _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 3 * (int)blockDim.y) * input_stride + _gid_x41);
        int _gid_x42 = gid_x + 1 * (int)blockDim.x - 32;
        _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 3 * (int)blockDim.y) * input_stride + _gid_x42);
        int _gid_x43 = gid_x + 2 * (int)blockDim.x - 32;
        _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 3 * (int)blockDim.y) * input_stride + _gid_x43);
        int _gid_x44 = gid_x + 0 * (int)blockDim.x - 32;
        _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 4 * (int)blockDim.y) * input_stride + _gid_x44);
        int _gid_x45 = gid_x + 1 * (int)blockDim.x - 32;
        _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 4 * (int)blockDim.y) * input_stride + _gid_x45);
        int _gid_x46 = gid_x + 2 * (int)blockDim.x - 32;
        _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 4 * (int)blockDim.y) * input_stride + _gid_x46);
        int _gid_x47 = gid_x + 0 * (int)blockDim.x - 32;
        _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 5 * (int)blockDim.y) * input_stride + _gid_x47);
        int _gid_x48 = gid_x + 1 * (int)blockDim.x - 32;
        _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 5 * (int)blockDim.y) * input_stride + _gid_x48);
        int _gid_x49 = gid_x + 2 * (int)blockDim.x - 32;
        _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 5 * (int)blockDim.y) * input_stride + _gid_x49);
        int _gid_x50 = gid_x + 0 * (int)blockDim.x - 32;
        _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 6 * (int)blockDim.y) * input_stride + _gid_x50);
        int _gid_x51 = gid_x + 1 * (int)blockDim.x - 32;
        _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 6 * (int)blockDim.y) * input_stride + _gid_x51);
        int _gid_x52 = gid_x + 2 * (int)blockDim.x - 32;
        _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 6 * (int)blockDim.y) * input_stride + _gid_x52);
        int _gid_x53 = gid_x + 0 * (int)blockDim.x - 32;
        _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 7 * (int)blockDim.y) * input_stride + _gid_x53);
        int _gid_x54 = gid_x + 1 * (int)blockDim.x - 32;
        _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 7 * (int)blockDim.y) * input_stride + _gid_x54);
        int _gid_x55 = gid_x + 2 * (int)blockDim.x - 32;
        _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 7 * (int)blockDim.y) * input_stride + _gid_x55);
        __syncthreads();
        if (gid_y < iter_height) {
            float _tmp56 = 0.F;
            {
                _tmp56 += _constmask_xX[0][0] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + -9 + 32];
            }
            {
                _tmp56 += _constmask_xX[0][1] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + -8 + 32];
            }
            {
                _tmp56 += _constmask_xX[0][2] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + -7 + 32];
            }
            {
                _tmp56 += _constmask_xX[0][3] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + -6 + 32];
            }
            {
                _tmp56 += _constmask_xX[0][4] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + -5 + 32];
            }
            {
                _tmp56 += _constmask_xX[0][5] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + -4 + 32];
            }
            {
                _tmp56 += _constmask_xX[0][6] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + -3 + 32];
            }
            {
                _tmp56 += _constmask_xX[0][7] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + -2 + 32];
            }
            {
                _tmp56 += _constmask_xX[0][8] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + -1 + 32];
            }
            {
                _tmp56 += _constmask_xX[0][9] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + 0 + 32];
            }
            {
                _tmp56 += _constmask_xX[0][10] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + 1 + 32];
            }
            {
                _tmp56 += _constmask_xX[0][11] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + 2 + 32];
            }
            {
                _tmp56 += _constmask_xX[0][12] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + 3 + 32];
            }
            {
                _tmp56 += _constmask_xX[0][13] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + 4 + 32];
            }
            {
                _tmp56 += _constmask_xX[0][14] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + 5 + 32];
            }
            {
                _tmp56 += _constmask_xX[0][15] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + 6 + 32];
            }
            {
                _tmp56 += _constmask_xX[0][16] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + 7 + 32];
            }
            {
                _tmp56 += _constmask_xX[0][17] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + 8 + 32];
            }
            {
                _tmp56 += _constmask_xX[0][18] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + 9 + 32];
            }
            iter[(gid_y) * iter_stride + gid_x] = (float)(_tmp56);
        }
        if (gid_y + 1 * (int)blockDim.y < iter_height) {
            float _tmp57 = 0.F;
            {
                _tmp57 += _constmask_xX[0][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -9 + 32];
            }
            {
                _tmp57 += _constmask_xX[0][1] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -8 + 32];
            }
            {
                _tmp57 += _constmask_xX[0][2] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -7 + 32];
            }
            {
                _tmp57 += _constmask_xX[0][3] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -6 + 32];
            }
            {
                _tmp57 += _constmask_xX[0][4] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -5 + 32];
            }
            {
                _tmp57 += _constmask_xX[0][5] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -4 + 32];
            }
            {
                _tmp57 += _constmask_xX[0][6] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -3 + 32];
            }
            {
                _tmp57 += _constmask_xX[0][7] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -2 + 32];
            }
            {
                _tmp57 += _constmask_xX[0][8] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -1 + 32];
            }
            {
                _tmp57 += _constmask_xX[0][9] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 0 + 32];
            }
            {
                _tmp57 += _constmask_xX[0][10] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 1 + 32];
            }
            {
                _tmp57 += _constmask_xX[0][11] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 2 + 32];
            }
            {
                _tmp57 += _constmask_xX[0][12] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 3 + 32];
            }
            {
                _tmp57 += _constmask_xX[0][13] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 4 + 32];
            }
            {
                _tmp57 += _constmask_xX[0][14] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 5 + 32];
            }
            {
                _tmp57 += _constmask_xX[0][15] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 6 + 32];
            }
            {
                _tmp57 += _constmask_xX[0][16] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 7 + 32];
            }
            {
                _tmp57 += _constmask_xX[0][17] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 8 + 32];
            }
            {
                _tmp57 += _constmask_xX[0][18] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 9 + 32];
            }
            iter[(gid_y + 1 * (int)blockDim.y) * iter_stride + gid_x] = (float)(_tmp57);
        }
        if (gid_y + 2 * (int)blockDim.y < iter_height) {
            float _tmp58 = 0.F;
            {
                _tmp58 += _constmask_xX[0][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -9 + 32];
            }
            {
                _tmp58 += _constmask_xX[0][1] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -8 + 32];
            }
            {
                _tmp58 += _constmask_xX[0][2] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -7 + 32];
            }
            {
                _tmp58 += _constmask_xX[0][3] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -6 + 32];
            }
            {
                _tmp58 += _constmask_xX[0][4] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -5 + 32];
            }
            {
                _tmp58 += _constmask_xX[0][5] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -4 + 32];
            }
            {
                _tmp58 += _constmask_xX[0][6] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -3 + 32];
            }
            {
                _tmp58 += _constmask_xX[0][7] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -2 + 32];
            }
            {
                _tmp58 += _constmask_xX[0][8] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -1 + 32];
            }
            {
                _tmp58 += _constmask_xX[0][9] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 0 + 32];
            }
            {
                _tmp58 += _constmask_xX[0][10] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 1 + 32];
            }
            {
                _tmp58 += _constmask_xX[0][11] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 2 + 32];
            }
            {
                _tmp58 += _constmask_xX[0][12] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 3 + 32];
            }
            {
                _tmp58 += _constmask_xX[0][13] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 4 + 32];
            }
            {
                _tmp58 += _constmask_xX[0][14] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 5 + 32];
            }
            {
                _tmp58 += _constmask_xX[0][15] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 6 + 32];
            }
            {
                _tmp58 += _constmask_xX[0][16] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 7 + 32];
            }
            {
                _tmp58 += _constmask_xX[0][17] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 8 + 32];
            }
            {
                _tmp58 += _constmask_xX[0][18] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 9 + 32];
            }
            iter[(gid_y + 2 * (int)blockDim.y) * iter_stride + gid_x] = (float)(_tmp58);
        }
        if (gid_y + 3 * (int)blockDim.y < iter_height) {
            float _tmp59 = 0.F;
            {
                _tmp59 += _constmask_xX[0][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -9 + 32];
            }
            {
                _tmp59 += _constmask_xX[0][1] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -8 + 32];
            }
            {
                _tmp59 += _constmask_xX[0][2] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -7 + 32];
            }
            {
                _tmp59 += _constmask_xX[0][3] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -6 + 32];
            }
            {
                _tmp59 += _constmask_xX[0][4] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -5 + 32];
            }
            {
                _tmp59 += _constmask_xX[0][5] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -4 + 32];
            }
            {
                _tmp59 += _constmask_xX[0][6] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -3 + 32];
            }
            {
                _tmp59 += _constmask_xX[0][7] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -2 + 32];
            }
            {
                _tmp59 += _constmask_xX[0][8] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -1 + 32];
            }
            {
                _tmp59 += _constmask_xX[0][9] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 0 + 32];
            }
            {
                _tmp59 += _constmask_xX[0][10] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 1 + 32];
            }
            {
                _tmp59 += _constmask_xX[0][11] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 2 + 32];
            }
            {
                _tmp59 += _constmask_xX[0][12] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 3 + 32];
            }
            {
                _tmp59 += _constmask_xX[0][13] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 4 + 32];
            }
            {
                _tmp59 += _constmask_xX[0][14] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 5 + 32];
            }
            {
                _tmp59 += _constmask_xX[0][15] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 6 + 32];
            }
            {
                _tmp59 += _constmask_xX[0][16] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 7 + 32];
            }
            {
                _tmp59 += _constmask_xX[0][17] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 8 + 32];
            }
            {
                _tmp59 += _constmask_xX[0][18] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 9 + 32];
            }
            iter[(gid_y + 3 * (int)blockDim.y) * iter_stride + gid_x] = (float)(_tmp59);
        }
        if (gid_y + 4 * (int)blockDim.y < iter_height) {
            float _tmp60 = 0.F;
            {
                _tmp60 += _constmask_xX[0][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -9 + 32];
            }
            {
                _tmp60 += _constmask_xX[0][1] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -8 + 32];
            }
            {
                _tmp60 += _constmask_xX[0][2] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -7 + 32];
            }
            {
                _tmp60 += _constmask_xX[0][3] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -6 + 32];
            }
            {
                _tmp60 += _constmask_xX[0][4] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -5 + 32];
            }
            {
                _tmp60 += _constmask_xX[0][5] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -4 + 32];
            }
            {
                _tmp60 += _constmask_xX[0][6] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -3 + 32];
            }
            {
                _tmp60 += _constmask_xX[0][7] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -2 + 32];
            }
            {
                _tmp60 += _constmask_xX[0][8] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -1 + 32];
            }
            {
                _tmp60 += _constmask_xX[0][9] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 0 + 32];
            }
            {
                _tmp60 += _constmask_xX[0][10] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 1 + 32];
            }
            {
                _tmp60 += _constmask_xX[0][11] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 2 + 32];
            }
            {
                _tmp60 += _constmask_xX[0][12] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 3 + 32];
            }
            {
                _tmp60 += _constmask_xX[0][13] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 4 + 32];
            }
            {
                _tmp60 += _constmask_xX[0][14] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 5 + 32];
            }
            {
                _tmp60 += _constmask_xX[0][15] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 6 + 32];
            }
            {
                _tmp60 += _constmask_xX[0][16] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 7 + 32];
            }
            {
                _tmp60 += _constmask_xX[0][17] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 8 + 32];
            }
            {
                _tmp60 += _constmask_xX[0][18] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 9 + 32];
            }
            iter[(gid_y + 4 * (int)blockDim.y) * iter_stride + gid_x] = (float)(_tmp60);
        }
        if (gid_y + 5 * (int)blockDim.y < iter_height) {
            float _tmp61 = 0.F;
            {
                _tmp61 += _constmask_xX[0][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -9 + 32];
            }
            {
                _tmp61 += _constmask_xX[0][1] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -8 + 32];
            }
            {
                _tmp61 += _constmask_xX[0][2] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -7 + 32];
            }
            {
                _tmp61 += _constmask_xX[0][3] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -6 + 32];
            }
            {
                _tmp61 += _constmask_xX[0][4] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -5 + 32];
            }
            {
                _tmp61 += _constmask_xX[0][5] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -4 + 32];
            }
            {
                _tmp61 += _constmask_xX[0][6] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -3 + 32];
            }
            {
                _tmp61 += _constmask_xX[0][7] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -2 + 32];
            }
            {
                _tmp61 += _constmask_xX[0][8] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -1 + 32];
            }
            {
                _tmp61 += _constmask_xX[0][9] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 0 + 32];
            }
            {
                _tmp61 += _constmask_xX[0][10] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 1 + 32];
            }
            {
                _tmp61 += _constmask_xX[0][11] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 2 + 32];
            }
            {
                _tmp61 += _constmask_xX[0][12] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 3 + 32];
            }
            {
                _tmp61 += _constmask_xX[0][13] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 4 + 32];
            }
            {
                _tmp61 += _constmask_xX[0][14] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 5 + 32];
            }
            {
                _tmp61 += _constmask_xX[0][15] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 6 + 32];
            }
            {
                _tmp61 += _constmask_xX[0][16] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 7 + 32];
            }
            {
                _tmp61 += _constmask_xX[0][17] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 8 + 32];
            }
            {
                _tmp61 += _constmask_xX[0][18] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 9 + 32];
            }
            iter[(gid_y + 5 * (int)blockDim.y) * iter_stride + gid_x] = (float)(_tmp61);
        }
        if (gid_y + 6 * (int)blockDim.y < iter_height) {
            float _tmp62 = 0.F;
            {
                _tmp62 += _constmask_xX[0][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -9 + 32];
            }
            {
                _tmp62 += _constmask_xX[0][1] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -8 + 32];
            }
            {
                _tmp62 += _constmask_xX[0][2] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -7 + 32];
            }
            {
                _tmp62 += _constmask_xX[0][3] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -6 + 32];
            }
            {
                _tmp62 += _constmask_xX[0][4] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -5 + 32];
            }
            {
                _tmp62 += _constmask_xX[0][5] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -4 + 32];
            }
            {
                _tmp62 += _constmask_xX[0][6] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -3 + 32];
            }
            {
                _tmp62 += _constmask_xX[0][7] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -2 + 32];
            }
            {
                _tmp62 += _constmask_xX[0][8] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -1 + 32];
            }
            {
                _tmp62 += _constmask_xX[0][9] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 0 + 32];
            }
            {
                _tmp62 += _constmask_xX[0][10] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 1 + 32];
            }
            {
                _tmp62 += _constmask_xX[0][11] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 2 + 32];
            }
            {
                _tmp62 += _constmask_xX[0][12] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 3 + 32];
            }
            {
                _tmp62 += _constmask_xX[0][13] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 4 + 32];
            }
            {
                _tmp62 += _constmask_xX[0][14] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 5 + 32];
            }
            {
                _tmp62 += _constmask_xX[0][15] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 6 + 32];
            }
            {
                _tmp62 += _constmask_xX[0][16] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 7 + 32];
            }
            {
                _tmp62 += _constmask_xX[0][17] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 8 + 32];
            }
            {
                _tmp62 += _constmask_xX[0][18] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 9 + 32];
            }
            iter[(gid_y + 6 * (int)blockDim.y) * iter_stride + gid_x] = (float)(_tmp62);
        }
        if (gid_y + 7 * (int)blockDim.y < iter_height) {
            float _tmp63 = 0.F;
            {
                _tmp63 += _constmask_xX[0][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -9 + 32];
            }
            {
                _tmp63 += _constmask_xX[0][1] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -8 + 32];
            }
            {
                _tmp63 += _constmask_xX[0][2] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -7 + 32];
            }
            {
                _tmp63 += _constmask_xX[0][3] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -6 + 32];
            }
            {
                _tmp63 += _constmask_xX[0][4] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -5 + 32];
            }
            {
                _tmp63 += _constmask_xX[0][5] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -4 + 32];
            }
            {
                _tmp63 += _constmask_xX[0][6] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -3 + 32];
            }
            {
                _tmp63 += _constmask_xX[0][7] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -2 + 32];
            }
            {
                _tmp63 += _constmask_xX[0][8] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -1 + 32];
            }
            {
                _tmp63 += _constmask_xX[0][9] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 0 + 32];
            }
            {
                _tmp63 += _constmask_xX[0][10] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 1 + 32];
            }
            {
                _tmp63 += _constmask_xX[0][11] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 2 + 32];
            }
            {
                _tmp63 += _constmask_xX[0][12] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 3 + 32];
            }
            {
                _tmp63 += _constmask_xX[0][13] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 4 + 32];
            }
            {
                _tmp63 += _constmask_xX[0][14] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 5 + 32];
            }
            {
                _tmp63 += _constmask_xX[0][15] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 6 + 32];
            }
            {
                _tmp63 += _constmask_xX[0][16] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 7 + 32];
            }
            {
                _tmp63 += _constmask_xX[0][17] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 8 + 32];
            }
            {
                _tmp63 += _constmask_xX[0][18] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 9 + 32];
            }
            iter[(gid_y + 7 * (int)blockDim.y) * iter_stride + gid_x] = (float)(_tmp63);
        }
    }
    goto BH_EXIT;
  BH_R:
    {
        int _gid_x64 = gid_x + 0 * (int)blockDim.x - 32;
        if (_gid_x64 >= input_width)
            _gid_x64 = input_width - 1;
        _smeminput[(int)threadIdx.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y) * input_stride + _gid_x64);
        int _gid_x65 = gid_x + 1 * (int)blockDim.x - 32;
        if (_gid_x65 >= input_width)
            _gid_x65 = input_width - 1;
        _smeminput[(int)threadIdx.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y) * input_stride + _gid_x65);
        int _gid_x66 = gid_x + 2 * (int)blockDim.x - 32;
        if (_gid_x66 >= input_width)
            _gid_x66 = input_width - 1;
        _smeminput[(int)threadIdx.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y) * input_stride + _gid_x66);
        int _gid_x67 = gid_x + 0 * (int)blockDim.x - 32;
        if (_gid_x67 >= input_width)
            _gid_x67 = input_width - 1;
        _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 1 * (int)blockDim.y) * input_stride + _gid_x67);
        int _gid_x68 = gid_x + 1 * (int)blockDim.x - 32;
        if (_gid_x68 >= input_width)
            _gid_x68 = input_width - 1;
        _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 1 * (int)blockDim.y) * input_stride + _gid_x68);
        int _gid_x69 = gid_x + 2 * (int)blockDim.x - 32;
        if (_gid_x69 >= input_width)
            _gid_x69 = input_width - 1;
        _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 1 * (int)blockDim.y) * input_stride + _gid_x69);
        int _gid_x70 = gid_x + 0 * (int)blockDim.x - 32;
        if (_gid_x70 >= input_width)
            _gid_x70 = input_width - 1;
        _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 2 * (int)blockDim.y) * input_stride + _gid_x70);
        int _gid_x71 = gid_x + 1 * (int)blockDim.x - 32;
        if (_gid_x71 >= input_width)
            _gid_x71 = input_width - 1;
        _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 2 * (int)blockDim.y) * input_stride + _gid_x71);
        int _gid_x72 = gid_x + 2 * (int)blockDim.x - 32;
        if (_gid_x72 >= input_width)
            _gid_x72 = input_width - 1;
        _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 2 * (int)blockDim.y) * input_stride + _gid_x72);
        int _gid_x73 = gid_x + 0 * (int)blockDim.x - 32;
        if (_gid_x73 >= input_width)
            _gid_x73 = input_width - 1;
        _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 3 * (int)blockDim.y) * input_stride + _gid_x73);
        int _gid_x74 = gid_x + 1 * (int)blockDim.x - 32;
        if (_gid_x74 >= input_width)
            _gid_x74 = input_width - 1;
        _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 3 * (int)blockDim.y) * input_stride + _gid_x74);
        int _gid_x75 = gid_x + 2 * (int)blockDim.x - 32;
        if (_gid_x75 >= input_width)
            _gid_x75 = input_width - 1;
        _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 3 * (int)blockDim.y) * input_stride + _gid_x75);
        int _gid_x76 = gid_x + 0 * (int)blockDim.x - 32;
        if (_gid_x76 >= input_width)
            _gid_x76 = input_width - 1;
        _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 4 * (int)blockDim.y) * input_stride + _gid_x76);
        int _gid_x77 = gid_x + 1 * (int)blockDim.x - 32;
        if (_gid_x77 >= input_width)
            _gid_x77 = input_width - 1;
        _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 4 * (int)blockDim.y) * input_stride + _gid_x77);
        int _gid_x78 = gid_x + 2 * (int)blockDim.x - 32;
        if (_gid_x78 >= input_width)
            _gid_x78 = input_width - 1;
        _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 4 * (int)blockDim.y) * input_stride + _gid_x78);
        int _gid_x79 = gid_x + 0 * (int)blockDim.x - 32;
        if (_gid_x79 >= input_width)
            _gid_x79 = input_width - 1;
        _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 5 * (int)blockDim.y) * input_stride + _gid_x79);
        int _gid_x80 = gid_x + 1 * (int)blockDim.x - 32;
        if (_gid_x80 >= input_width)
            _gid_x80 = input_width - 1;
        _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 5 * (int)blockDim.y) * input_stride + _gid_x80);
        int _gid_x81 = gid_x + 2 * (int)blockDim.x - 32;
        if (_gid_x81 >= input_width)
            _gid_x81 = input_width - 1;
        _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 5 * (int)blockDim.y) * input_stride + _gid_x81);
        int _gid_x82 = gid_x + 0 * (int)blockDim.x - 32;
        if (_gid_x82 >= input_width)
            _gid_x82 = input_width - 1;
        _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 6 * (int)blockDim.y) * input_stride + _gid_x82);
        int _gid_x83 = gid_x + 1 * (int)blockDim.x - 32;
        if (_gid_x83 >= input_width)
            _gid_x83 = input_width - 1;
        _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 6 * (int)blockDim.y) * input_stride + _gid_x83);
        int _gid_x84 = gid_x + 2 * (int)blockDim.x - 32;
        if (_gid_x84 >= input_width)
            _gid_x84 = input_width - 1;
        _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 6 * (int)blockDim.y) * input_stride + _gid_x84);
        int _gid_x85 = gid_x + 0 * (int)blockDim.x - 32;
        if (_gid_x85 >= input_width)
            _gid_x85 = input_width - 1;
        _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 7 * (int)blockDim.y) * input_stride + _gid_x85);
        int _gid_x86 = gid_x + 1 * (int)blockDim.x - 32;
        if (_gid_x86 >= input_width)
            _gid_x86 = input_width - 1;
        _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 7 * (int)blockDim.y) * input_stride + _gid_x86);
        int _gid_x87 = gid_x + 2 * (int)blockDim.x - 32;
        if (_gid_x87 >= input_width)
            _gid_x87 = input_width - 1;
        _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 7 * (int)blockDim.y) * input_stride + _gid_x87);
        __syncthreads();
        if (gid_x < iter_width) {
            if (gid_y < iter_height) {
                float _tmp88 = 0.F;
                {
                    _tmp88 += _constmask_xX[0][0] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + -9 + 32];
                }
                {
                    _tmp88 += _constmask_xX[0][1] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + -8 + 32];
                }
                {
                    _tmp88 += _constmask_xX[0][2] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + -7 + 32];
                }
                {
                    _tmp88 += _constmask_xX[0][3] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + -6 + 32];
                }
                {
                    _tmp88 += _constmask_xX[0][4] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + -5 + 32];
                }
                {
                    _tmp88 += _constmask_xX[0][5] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + -4 + 32];
                }
                {
                    _tmp88 += _constmask_xX[0][6] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + -3 + 32];
                }
                {
                    _tmp88 += _constmask_xX[0][7] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + -2 + 32];
                }
                {
                    _tmp88 += _constmask_xX[0][8] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + -1 + 32];
                }
                {
                    _tmp88 += _constmask_xX[0][9] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + 0 + 32];
                }
                {
                    _tmp88 += _constmask_xX[0][10] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + 1 + 32];
                }
                {
                    _tmp88 += _constmask_xX[0][11] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + 2 + 32];
                }
                {
                    _tmp88 += _constmask_xX[0][12] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + 3 + 32];
                }
                {
                    _tmp88 += _constmask_xX[0][13] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + 4 + 32];
                }
                {
                    _tmp88 += _constmask_xX[0][14] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + 5 + 32];
                }
                {
                    _tmp88 += _constmask_xX[0][15] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + 6 + 32];
                }
                {
                    _tmp88 += _constmask_xX[0][16] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + 7 + 32];
                }
                {
                    _tmp88 += _constmask_xX[0][17] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + 8 + 32];
                }
                {
                    _tmp88 += _constmask_xX[0][18] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + 9 + 32];
                }
                iter[(gid_y) * iter_stride + gid_x] = (float)(_tmp88);
            }
        }
        if (gid_x < iter_width) {
            if (gid_y + 1 * (int)blockDim.y < iter_height) {
                float _tmp89 = 0.F;
                {
                    _tmp89 += _constmask_xX[0][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -9 + 32];
                }
                {
                    _tmp89 += _constmask_xX[0][1] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -8 + 32];
                }
                {
                    _tmp89 += _constmask_xX[0][2] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -7 + 32];
                }
                {
                    _tmp89 += _constmask_xX[0][3] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -6 + 32];
                }
                {
                    _tmp89 += _constmask_xX[0][4] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -5 + 32];
                }
                {
                    _tmp89 += _constmask_xX[0][5] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -4 + 32];
                }
                {
                    _tmp89 += _constmask_xX[0][6] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -3 + 32];
                }
                {
                    _tmp89 += _constmask_xX[0][7] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -2 + 32];
                }
                {
                    _tmp89 += _constmask_xX[0][8] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -1 + 32];
                }
                {
                    _tmp89 += _constmask_xX[0][9] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 0 + 32];
                }
                {
                    _tmp89 += _constmask_xX[0][10] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 1 + 32];
                }
                {
                    _tmp89 += _constmask_xX[0][11] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 2 + 32];
                }
                {
                    _tmp89 += _constmask_xX[0][12] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 3 + 32];
                }
                {
                    _tmp89 += _constmask_xX[0][13] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 4 + 32];
                }
                {
                    _tmp89 += _constmask_xX[0][14] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 5 + 32];
                }
                {
                    _tmp89 += _constmask_xX[0][15] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 6 + 32];
                }
                {
                    _tmp89 += _constmask_xX[0][16] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 7 + 32];
                }
                {
                    _tmp89 += _constmask_xX[0][17] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 8 + 32];
                }
                {
                    _tmp89 += _constmask_xX[0][18] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 9 + 32];
                }
                iter[(gid_y + 1 * (int)blockDim.y) * iter_stride + gid_x] = (float)(_tmp89);
            }
        }
        if (gid_x < iter_width) {
            if (gid_y + 2 * (int)blockDim.y < iter_height) {
                float _tmp90 = 0.F;
                {
                    _tmp90 += _constmask_xX[0][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -9 + 32];
                }
                {
                    _tmp90 += _constmask_xX[0][1] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -8 + 32];
                }
                {
                    _tmp90 += _constmask_xX[0][2] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -7 + 32];
                }
                {
                    _tmp90 += _constmask_xX[0][3] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -6 + 32];
                }
                {
                    _tmp90 += _constmask_xX[0][4] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -5 + 32];
                }
                {
                    _tmp90 += _constmask_xX[0][5] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -4 + 32];
                }
                {
                    _tmp90 += _constmask_xX[0][6] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -3 + 32];
                }
                {
                    _tmp90 += _constmask_xX[0][7] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -2 + 32];
                }
                {
                    _tmp90 += _constmask_xX[0][8] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -1 + 32];
                }
                {
                    _tmp90 += _constmask_xX[0][9] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 0 + 32];
                }
                {
                    _tmp90 += _constmask_xX[0][10] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 1 + 32];
                }
                {
                    _tmp90 += _constmask_xX[0][11] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 2 + 32];
                }
                {
                    _tmp90 += _constmask_xX[0][12] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 3 + 32];
                }
                {
                    _tmp90 += _constmask_xX[0][13] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 4 + 32];
                }
                {
                    _tmp90 += _constmask_xX[0][14] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 5 + 32];
                }
                {
                    _tmp90 += _constmask_xX[0][15] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 6 + 32];
                }
                {
                    _tmp90 += _constmask_xX[0][16] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 7 + 32];
                }
                {
                    _tmp90 += _constmask_xX[0][17] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 8 + 32];
                }
                {
                    _tmp90 += _constmask_xX[0][18] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 9 + 32];
                }
                iter[(gid_y + 2 * (int)blockDim.y) * iter_stride + gid_x] = (float)(_tmp90);
            }
        }
        if (gid_x < iter_width) {
            if (gid_y + 3 * (int)blockDim.y < iter_height) {
                float _tmp91 = 0.F;
                {
                    _tmp91 += _constmask_xX[0][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -9 + 32];
                }
                {
                    _tmp91 += _constmask_xX[0][1] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -8 + 32];
                }
                {
                    _tmp91 += _constmask_xX[0][2] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -7 + 32];
                }
                {
                    _tmp91 += _constmask_xX[0][3] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -6 + 32];
                }
                {
                    _tmp91 += _constmask_xX[0][4] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -5 + 32];
                }
                {
                    _tmp91 += _constmask_xX[0][5] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -4 + 32];
                }
                {
                    _tmp91 += _constmask_xX[0][6] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -3 + 32];
                }
                {
                    _tmp91 += _constmask_xX[0][7] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -2 + 32];
                }
                {
                    _tmp91 += _constmask_xX[0][8] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -1 + 32];
                }
                {
                    _tmp91 += _constmask_xX[0][9] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 0 + 32];
                }
                {
                    _tmp91 += _constmask_xX[0][10] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 1 + 32];
                }
                {
                    _tmp91 += _constmask_xX[0][11] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 2 + 32];
                }
                {
                    _tmp91 += _constmask_xX[0][12] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 3 + 32];
                }
                {
                    _tmp91 += _constmask_xX[0][13] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 4 + 32];
                }
                {
                    _tmp91 += _constmask_xX[0][14] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 5 + 32];
                }
                {
                    _tmp91 += _constmask_xX[0][15] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 6 + 32];
                }
                {
                    _tmp91 += _constmask_xX[0][16] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 7 + 32];
                }
                {
                    _tmp91 += _constmask_xX[0][17] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 8 + 32];
                }
                {
                    _tmp91 += _constmask_xX[0][18] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 9 + 32];
                }
                iter[(gid_y + 3 * (int)blockDim.y) * iter_stride + gid_x] = (float)(_tmp91);
            }
        }
        if (gid_x < iter_width) {
            if (gid_y + 4 * (int)blockDim.y < iter_height) {
                float _tmp92 = 0.F;
                {
                    _tmp92 += _constmask_xX[0][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -9 + 32];
                }
                {
                    _tmp92 += _constmask_xX[0][1] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -8 + 32];
                }
                {
                    _tmp92 += _constmask_xX[0][2] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -7 + 32];
                }
                {
                    _tmp92 += _constmask_xX[0][3] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -6 + 32];
                }
                {
                    _tmp92 += _constmask_xX[0][4] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -5 + 32];
                }
                {
                    _tmp92 += _constmask_xX[0][5] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -4 + 32];
                }
                {
                    _tmp92 += _constmask_xX[0][6] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -3 + 32];
                }
                {
                    _tmp92 += _constmask_xX[0][7] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -2 + 32];
                }
                {
                    _tmp92 += _constmask_xX[0][8] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -1 + 32];
                }
                {
                    _tmp92 += _constmask_xX[0][9] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 0 + 32];
                }
                {
                    _tmp92 += _constmask_xX[0][10] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 1 + 32];
                }
                {
                    _tmp92 += _constmask_xX[0][11] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 2 + 32];
                }
                {
                    _tmp92 += _constmask_xX[0][12] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 3 + 32];
                }
                {
                    _tmp92 += _constmask_xX[0][13] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 4 + 32];
                }
                {
                    _tmp92 += _constmask_xX[0][14] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 5 + 32];
                }
                {
                    _tmp92 += _constmask_xX[0][15] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 6 + 32];
                }
                {
                    _tmp92 += _constmask_xX[0][16] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 7 + 32];
                }
                {
                    _tmp92 += _constmask_xX[0][17] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 8 + 32];
                }
                {
                    _tmp92 += _constmask_xX[0][18] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 9 + 32];
                }
                iter[(gid_y + 4 * (int)blockDim.y) * iter_stride + gid_x] = (float)(_tmp92);
            }
        }
        if (gid_x < iter_width) {
            if (gid_y + 5 * (int)blockDim.y < iter_height) {
                float _tmp93 = 0.F;
                {
                    _tmp93 += _constmask_xX[0][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -9 + 32];
                }
                {
                    _tmp93 += _constmask_xX[0][1] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -8 + 32];
                }
                {
                    _tmp93 += _constmask_xX[0][2] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -7 + 32];
                }
                {
                    _tmp93 += _constmask_xX[0][3] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -6 + 32];
                }
                {
                    _tmp93 += _constmask_xX[0][4] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -5 + 32];
                }
                {
                    _tmp93 += _constmask_xX[0][5] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -4 + 32];
                }
                {
                    _tmp93 += _constmask_xX[0][6] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -3 + 32];
                }
                {
                    _tmp93 += _constmask_xX[0][7] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -2 + 32];
                }
                {
                    _tmp93 += _constmask_xX[0][8] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -1 + 32];
                }
                {
                    _tmp93 += _constmask_xX[0][9] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 0 + 32];
                }
                {
                    _tmp93 += _constmask_xX[0][10] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 1 + 32];
                }
                {
                    _tmp93 += _constmask_xX[0][11] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 2 + 32];
                }
                {
                    _tmp93 += _constmask_xX[0][12] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 3 + 32];
                }
                {
                    _tmp93 += _constmask_xX[0][13] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 4 + 32];
                }
                {
                    _tmp93 += _constmask_xX[0][14] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 5 + 32];
                }
                {
                    _tmp93 += _constmask_xX[0][15] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 6 + 32];
                }
                {
                    _tmp93 += _constmask_xX[0][16] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 7 + 32];
                }
                {
                    _tmp93 += _constmask_xX[0][17] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 8 + 32];
                }
                {
                    _tmp93 += _constmask_xX[0][18] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 9 + 32];
                }
                iter[(gid_y + 5 * (int)blockDim.y) * iter_stride + gid_x] = (float)(_tmp93);
            }
        }
        if (gid_x < iter_width) {
            if (gid_y + 6 * (int)blockDim.y < iter_height) {
                float _tmp94 = 0.F;
                {
                    _tmp94 += _constmask_xX[0][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -9 + 32];
                }
                {
                    _tmp94 += _constmask_xX[0][1] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -8 + 32];
                }
                {
                    _tmp94 += _constmask_xX[0][2] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -7 + 32];
                }
                {
                    _tmp94 += _constmask_xX[0][3] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -6 + 32];
                }
                {
                    _tmp94 += _constmask_xX[0][4] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -5 + 32];
                }
                {
                    _tmp94 += _constmask_xX[0][5] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -4 + 32];
                }
                {
                    _tmp94 += _constmask_xX[0][6] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -3 + 32];
                }
                {
                    _tmp94 += _constmask_xX[0][7] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -2 + 32];
                }
                {
                    _tmp94 += _constmask_xX[0][8] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -1 + 32];
                }
                {
                    _tmp94 += _constmask_xX[0][9] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 0 + 32];
                }
                {
                    _tmp94 += _constmask_xX[0][10] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 1 + 32];
                }
                {
                    _tmp94 += _constmask_xX[0][11] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 2 + 32];
                }
                {
                    _tmp94 += _constmask_xX[0][12] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 3 + 32];
                }
                {
                    _tmp94 += _constmask_xX[0][13] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 4 + 32];
                }
                {
                    _tmp94 += _constmask_xX[0][14] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 5 + 32];
                }
                {
                    _tmp94 += _constmask_xX[0][15] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 6 + 32];
                }
                {
                    _tmp94 += _constmask_xX[0][16] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 7 + 32];
                }
                {
                    _tmp94 += _constmask_xX[0][17] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 8 + 32];
                }
                {
                    _tmp94 += _constmask_xX[0][18] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 9 + 32];
                }
                iter[(gid_y + 6 * (int)blockDim.y) * iter_stride + gid_x] = (float)(_tmp94);
            }
        }
        if (gid_x < iter_width) {
            if (gid_y + 7 * (int)blockDim.y < iter_height) {
                float _tmp95 = 0.F;
                {
                    _tmp95 += _constmask_xX[0][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -9 + 32];
                }
                {
                    _tmp95 += _constmask_xX[0][1] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -8 + 32];
                }
                {
                    _tmp95 += _constmask_xX[0][2] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -7 + 32];
                }
                {
                    _tmp95 += _constmask_xX[0][3] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -6 + 32];
                }
                {
                    _tmp95 += _constmask_xX[0][4] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -5 + 32];
                }
                {
                    _tmp95 += _constmask_xX[0][5] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -4 + 32];
                }
                {
                    _tmp95 += _constmask_xX[0][6] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -3 + 32];
                }
                {
                    _tmp95 += _constmask_xX[0][7] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -2 + 32];
                }
                {
                    _tmp95 += _constmask_xX[0][8] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -1 + 32];
                }
                {
                    _tmp95 += _constmask_xX[0][9] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 0 + 32];
                }
                {
                    _tmp95 += _constmask_xX[0][10] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 1 + 32];
                }
                {
                    _tmp95 += _constmask_xX[0][11] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 2 + 32];
                }
                {
                    _tmp95 += _constmask_xX[0][12] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 3 + 32];
                }
                {
                    _tmp95 += _constmask_xX[0][13] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 4 + 32];
                }
                {
                    _tmp95 += _constmask_xX[0][14] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 5 + 32];
                }
                {
                    _tmp95 += _constmask_xX[0][15] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 6 + 32];
                }
                {
                    _tmp95 += _constmask_xX[0][16] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 7 + 32];
                }
                {
                    _tmp95 += _constmask_xX[0][17] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 8 + 32];
                }
                {
                    _tmp95 += _constmask_xX[0][18] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 9 + 32];
                }
                iter[(gid_y + 7 * (int)blockDim.y) * iter_stride + gid_x] = (float)(_tmp95);
            }
        }
    }
    goto BH_EXIT;
  BH_L:
    {
        int _gid_x96 = gid_x + 0 * (int)blockDim.x - 32;
        if (_gid_x96 < 0)
            _gid_x96 = 0;
        _smeminput[(int)threadIdx.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y) * input_stride + _gid_x96);
        int _gid_x97 = gid_x + 1 * (int)blockDim.x - 32;
        if (_gid_x97 < 0)
            _gid_x97 = 0;
        _smeminput[(int)threadIdx.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y) * input_stride + _gid_x97);
        int _gid_x98 = gid_x + 2 * (int)blockDim.x - 32;
        if (_gid_x98 < 0)
            _gid_x98 = 0;
        _smeminput[(int)threadIdx.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y) * input_stride + _gid_x98);
        int _gid_x99 = gid_x + 0 * (int)blockDim.x - 32;
        if (_gid_x99 < 0)
            _gid_x99 = 0;
        _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 1 * (int)blockDim.y) * input_stride + _gid_x99);
        int _gid_x100 = gid_x + 1 * (int)blockDim.x - 32;
        if (_gid_x100 < 0)
            _gid_x100 = 0;
        _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 1 * (int)blockDim.y) * input_stride + _gid_x100);
        int _gid_x101 = gid_x + 2 * (int)blockDim.x - 32;
        if (_gid_x101 < 0)
            _gid_x101 = 0;
        _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 1 * (int)blockDim.y) * input_stride + _gid_x101);
        int _gid_x102 = gid_x + 0 * (int)blockDim.x - 32;
        if (_gid_x102 < 0)
            _gid_x102 = 0;
        _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 2 * (int)blockDim.y) * input_stride + _gid_x102);
        int _gid_x103 = gid_x + 1 * (int)blockDim.x - 32;
        if (_gid_x103 < 0)
            _gid_x103 = 0;
        _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 2 * (int)blockDim.y) * input_stride + _gid_x103);
        int _gid_x104 = gid_x + 2 * (int)blockDim.x - 32;
        if (_gid_x104 < 0)
            _gid_x104 = 0;
        _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 2 * (int)blockDim.y) * input_stride + _gid_x104);
        int _gid_x105 = gid_x + 0 * (int)blockDim.x - 32;
        if (_gid_x105 < 0)
            _gid_x105 = 0;
        _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 3 * (int)blockDim.y) * input_stride + _gid_x105);
        int _gid_x106 = gid_x + 1 * (int)blockDim.x - 32;
        if (_gid_x106 < 0)
            _gid_x106 = 0;
        _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 3 * (int)blockDim.y) * input_stride + _gid_x106);
        int _gid_x107 = gid_x + 2 * (int)blockDim.x - 32;
        if (_gid_x107 < 0)
            _gid_x107 = 0;
        _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 3 * (int)blockDim.y) * input_stride + _gid_x107);
        int _gid_x108 = gid_x + 0 * (int)blockDim.x - 32;
        if (_gid_x108 < 0)
            _gid_x108 = 0;
        _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 4 * (int)blockDim.y) * input_stride + _gid_x108);
        int _gid_x109 = gid_x + 1 * (int)blockDim.x - 32;
        if (_gid_x109 < 0)
            _gid_x109 = 0;
        _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 4 * (int)blockDim.y) * input_stride + _gid_x109);
        int _gid_x110 = gid_x + 2 * (int)blockDim.x - 32;
        if (_gid_x110 < 0)
            _gid_x110 = 0;
        _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 4 * (int)blockDim.y) * input_stride + _gid_x110);
        int _gid_x111 = gid_x + 0 * (int)blockDim.x - 32;
        if (_gid_x111 < 0)
            _gid_x111 = 0;
        _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 5 * (int)blockDim.y) * input_stride + _gid_x111);
        int _gid_x112 = gid_x + 1 * (int)blockDim.x - 32;
        if (_gid_x112 < 0)
            _gid_x112 = 0;
        _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 5 * (int)blockDim.y) * input_stride + _gid_x112);
        int _gid_x113 = gid_x + 2 * (int)blockDim.x - 32;
        if (_gid_x113 < 0)
            _gid_x113 = 0;
        _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 5 * (int)blockDim.y) * input_stride + _gid_x113);
        int _gid_x114 = gid_x + 0 * (int)blockDim.x - 32;
        if (_gid_x114 < 0)
            _gid_x114 = 0;
        _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 6 * (int)blockDim.y) * input_stride + _gid_x114);
        int _gid_x115 = gid_x + 1 * (int)blockDim.x - 32;
        if (_gid_x115 < 0)
            _gid_x115 = 0;
        _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 6 * (int)blockDim.y) * input_stride + _gid_x115);
        int _gid_x116 = gid_x + 2 * (int)blockDim.x - 32;
        if (_gid_x116 < 0)
            _gid_x116 = 0;
        _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 6 * (int)blockDim.y) * input_stride + _gid_x116);
        int _gid_x117 = gid_x + 0 * (int)blockDim.x - 32;
        if (_gid_x117 < 0)
            _gid_x117 = 0;
        _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 7 * (int)blockDim.y) * input_stride + _gid_x117);
        int _gid_x118 = gid_x + 1 * (int)blockDim.x - 32;
        if (_gid_x118 < 0)
            _gid_x118 = 0;
        _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 7 * (int)blockDim.y) * input_stride + _gid_x118);
        int _gid_x119 = gid_x + 2 * (int)blockDim.x - 32;
        if (_gid_x119 < 0)
            _gid_x119 = 0;
        _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 7 * (int)blockDim.y) * input_stride + _gid_x119);
        __syncthreads();
        if (gid_y < iter_height) {
            float _tmp120 = 0.F;
            {
                _tmp120 += _constmask_xX[0][0] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + -9 + 32];
            }
            {
                _tmp120 += _constmask_xX[0][1] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + -8 + 32];
            }
            {
                _tmp120 += _constmask_xX[0][2] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + -7 + 32];
            }
            {
                _tmp120 += _constmask_xX[0][3] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + -6 + 32];
            }
            {
                _tmp120 += _constmask_xX[0][4] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + -5 + 32];
            }
            {
                _tmp120 += _constmask_xX[0][5] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + -4 + 32];
            }
            {
                _tmp120 += _constmask_xX[0][6] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + -3 + 32];
            }
            {
                _tmp120 += _constmask_xX[0][7] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + -2 + 32];
            }
            {
                _tmp120 += _constmask_xX[0][8] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + -1 + 32];
            }
            {
                _tmp120 += _constmask_xX[0][9] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + 0 + 32];
            }
            {
                _tmp120 += _constmask_xX[0][10] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + 1 + 32];
            }
            {
                _tmp120 += _constmask_xX[0][11] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + 2 + 32];
            }
            {
                _tmp120 += _constmask_xX[0][12] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + 3 + 32];
            }
            {
                _tmp120 += _constmask_xX[0][13] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + 4 + 32];
            }
            {
                _tmp120 += _constmask_xX[0][14] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + 5 + 32];
            }
            {
                _tmp120 += _constmask_xX[0][15] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + 6 + 32];
            }
            {
                _tmp120 += _constmask_xX[0][16] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + 7 + 32];
            }
            {
                _tmp120 += _constmask_xX[0][17] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + 8 + 32];
            }
            {
                _tmp120 += _constmask_xX[0][18] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + 9 + 32];
            }
            iter[(gid_y) * iter_stride + gid_x] = (float)(_tmp120);
        }
        if (gid_y + 1 * (int)blockDim.y < iter_height) {
            float _tmp121 = 0.F;
            {
                _tmp121 += _constmask_xX[0][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -9 + 32];
            }
            {
                _tmp121 += _constmask_xX[0][1] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -8 + 32];
            }
            {
                _tmp121 += _constmask_xX[0][2] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -7 + 32];
            }
            {
                _tmp121 += _constmask_xX[0][3] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -6 + 32];
            }
            {
                _tmp121 += _constmask_xX[0][4] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -5 + 32];
            }
            {
                _tmp121 += _constmask_xX[0][5] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -4 + 32];
            }
            {
                _tmp121 += _constmask_xX[0][6] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -3 + 32];
            }
            {
                _tmp121 += _constmask_xX[0][7] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -2 + 32];
            }
            {
                _tmp121 += _constmask_xX[0][8] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -1 + 32];
            }
            {
                _tmp121 += _constmask_xX[0][9] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 0 + 32];
            }
            {
                _tmp121 += _constmask_xX[0][10] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 1 + 32];
            }
            {
                _tmp121 += _constmask_xX[0][11] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 2 + 32];
            }
            {
                _tmp121 += _constmask_xX[0][12] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 3 + 32];
            }
            {
                _tmp121 += _constmask_xX[0][13] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 4 + 32];
            }
            {
                _tmp121 += _constmask_xX[0][14] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 5 + 32];
            }
            {
                _tmp121 += _constmask_xX[0][15] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 6 + 32];
            }
            {
                _tmp121 += _constmask_xX[0][16] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 7 + 32];
            }
            {
                _tmp121 += _constmask_xX[0][17] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 8 + 32];
            }
            {
                _tmp121 += _constmask_xX[0][18] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 9 + 32];
            }
            iter[(gid_y + 1 * (int)blockDim.y) * iter_stride + gid_x] = (float)(_tmp121);
        }
        if (gid_y + 2 * (int)blockDim.y < iter_height) {
            float _tmp122 = 0.F;
            {
                _tmp122 += _constmask_xX[0][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -9 + 32];
            }
            {
                _tmp122 += _constmask_xX[0][1] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -8 + 32];
            }
            {
                _tmp122 += _constmask_xX[0][2] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -7 + 32];
            }
            {
                _tmp122 += _constmask_xX[0][3] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -6 + 32];
            }
            {
                _tmp122 += _constmask_xX[0][4] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -5 + 32];
            }
            {
                _tmp122 += _constmask_xX[0][5] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -4 + 32];
            }
            {
                _tmp122 += _constmask_xX[0][6] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -3 + 32];
            }
            {
                _tmp122 += _constmask_xX[0][7] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -2 + 32];
            }
            {
                _tmp122 += _constmask_xX[0][8] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -1 + 32];
            }
            {
                _tmp122 += _constmask_xX[0][9] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 0 + 32];
            }
            {
                _tmp122 += _constmask_xX[0][10] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 1 + 32];
            }
            {
                _tmp122 += _constmask_xX[0][11] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 2 + 32];
            }
            {
                _tmp122 += _constmask_xX[0][12] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 3 + 32];
            }
            {
                _tmp122 += _constmask_xX[0][13] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 4 + 32];
            }
            {
                _tmp122 += _constmask_xX[0][14] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 5 + 32];
            }
            {
                _tmp122 += _constmask_xX[0][15] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 6 + 32];
            }
            {
                _tmp122 += _constmask_xX[0][16] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 7 + 32];
            }
            {
                _tmp122 += _constmask_xX[0][17] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 8 + 32];
            }
            {
                _tmp122 += _constmask_xX[0][18] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 9 + 32];
            }
            iter[(gid_y + 2 * (int)blockDim.y) * iter_stride + gid_x] = (float)(_tmp122);
        }
        if (gid_y + 3 * (int)blockDim.y < iter_height) {
            float _tmp123 = 0.F;
            {
                _tmp123 += _constmask_xX[0][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -9 + 32];
            }
            {
                _tmp123 += _constmask_xX[0][1] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -8 + 32];
            }
            {
                _tmp123 += _constmask_xX[0][2] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -7 + 32];
            }
            {
                _tmp123 += _constmask_xX[0][3] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -6 + 32];
            }
            {
                _tmp123 += _constmask_xX[0][4] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -5 + 32];
            }
            {
                _tmp123 += _constmask_xX[0][5] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -4 + 32];
            }
            {
                _tmp123 += _constmask_xX[0][6] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -3 + 32];
            }
            {
                _tmp123 += _constmask_xX[0][7] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -2 + 32];
            }
            {
                _tmp123 += _constmask_xX[0][8] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -1 + 32];
            }
            {
                _tmp123 += _constmask_xX[0][9] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 0 + 32];
            }
            {
                _tmp123 += _constmask_xX[0][10] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 1 + 32];
            }
            {
                _tmp123 += _constmask_xX[0][11] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 2 + 32];
            }
            {
                _tmp123 += _constmask_xX[0][12] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 3 + 32];
            }
            {
                _tmp123 += _constmask_xX[0][13] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 4 + 32];
            }
            {
                _tmp123 += _constmask_xX[0][14] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 5 + 32];
            }
            {
                _tmp123 += _constmask_xX[0][15] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 6 + 32];
            }
            {
                _tmp123 += _constmask_xX[0][16] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 7 + 32];
            }
            {
                _tmp123 += _constmask_xX[0][17] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 8 + 32];
            }
            {
                _tmp123 += _constmask_xX[0][18] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 9 + 32];
            }
            iter[(gid_y + 3 * (int)blockDim.y) * iter_stride + gid_x] = (float)(_tmp123);
        }
        if (gid_y + 4 * (int)blockDim.y < iter_height) {
            float _tmp124 = 0.F;
            {
                _tmp124 += _constmask_xX[0][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -9 + 32];
            }
            {
                _tmp124 += _constmask_xX[0][1] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -8 + 32];
            }
            {
                _tmp124 += _constmask_xX[0][2] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -7 + 32];
            }
            {
                _tmp124 += _constmask_xX[0][3] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -6 + 32];
            }
            {
                _tmp124 += _constmask_xX[0][4] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -5 + 32];
            }
            {
                _tmp124 += _constmask_xX[0][5] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -4 + 32];
            }
            {
                _tmp124 += _constmask_xX[0][6] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -3 + 32];
            }
            {
                _tmp124 += _constmask_xX[0][7] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -2 + 32];
            }
            {
                _tmp124 += _constmask_xX[0][8] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -1 + 32];
            }
            {
                _tmp124 += _constmask_xX[0][9] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 0 + 32];
            }
            {
                _tmp124 += _constmask_xX[0][10] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 1 + 32];
            }
            {
                _tmp124 += _constmask_xX[0][11] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 2 + 32];
            }
            {
                _tmp124 += _constmask_xX[0][12] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 3 + 32];
            }
            {
                _tmp124 += _constmask_xX[0][13] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 4 + 32];
            }
            {
                _tmp124 += _constmask_xX[0][14] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 5 + 32];
            }
            {
                _tmp124 += _constmask_xX[0][15] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 6 + 32];
            }
            {
                _tmp124 += _constmask_xX[0][16] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 7 + 32];
            }
            {
                _tmp124 += _constmask_xX[0][17] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 8 + 32];
            }
            {
                _tmp124 += _constmask_xX[0][18] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 9 + 32];
            }
            iter[(gid_y + 4 * (int)blockDim.y) * iter_stride + gid_x] = (float)(_tmp124);
        }
        if (gid_y + 5 * (int)blockDim.y < iter_height) {
            float _tmp125 = 0.F;
            {
                _tmp125 += _constmask_xX[0][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -9 + 32];
            }
            {
                _tmp125 += _constmask_xX[0][1] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -8 + 32];
            }
            {
                _tmp125 += _constmask_xX[0][2] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -7 + 32];
            }
            {
                _tmp125 += _constmask_xX[0][3] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -6 + 32];
            }
            {
                _tmp125 += _constmask_xX[0][4] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -5 + 32];
            }
            {
                _tmp125 += _constmask_xX[0][5] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -4 + 32];
            }
            {
                _tmp125 += _constmask_xX[0][6] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -3 + 32];
            }
            {
                _tmp125 += _constmask_xX[0][7] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -2 + 32];
            }
            {
                _tmp125 += _constmask_xX[0][8] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -1 + 32];
            }
            {
                _tmp125 += _constmask_xX[0][9] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 0 + 32];
            }
            {
                _tmp125 += _constmask_xX[0][10] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 1 + 32];
            }
            {
                _tmp125 += _constmask_xX[0][11] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 2 + 32];
            }
            {
                _tmp125 += _constmask_xX[0][12] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 3 + 32];
            }
            {
                _tmp125 += _constmask_xX[0][13] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 4 + 32];
            }
            {
                _tmp125 += _constmask_xX[0][14] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 5 + 32];
            }
            {
                _tmp125 += _constmask_xX[0][15] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 6 + 32];
            }
            {
                _tmp125 += _constmask_xX[0][16] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 7 + 32];
            }
            {
                _tmp125 += _constmask_xX[0][17] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 8 + 32];
            }
            {
                _tmp125 += _constmask_xX[0][18] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 9 + 32];
            }
            iter[(gid_y + 5 * (int)blockDim.y) * iter_stride + gid_x] = (float)(_tmp125);
        }
        if (gid_y + 6 * (int)blockDim.y < iter_height) {
            float _tmp126 = 0.F;
            {
                _tmp126 += _constmask_xX[0][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -9 + 32];
            }
            {
                _tmp126 += _constmask_xX[0][1] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -8 + 32];
            }
            {
                _tmp126 += _constmask_xX[0][2] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -7 + 32];
            }
            {
                _tmp126 += _constmask_xX[0][3] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -6 + 32];
            }
            {
                _tmp126 += _constmask_xX[0][4] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -5 + 32];
            }
            {
                _tmp126 += _constmask_xX[0][5] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -4 + 32];
            }
            {
                _tmp126 += _constmask_xX[0][6] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -3 + 32];
            }
            {
                _tmp126 += _constmask_xX[0][7] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -2 + 32];
            }
            {
                _tmp126 += _constmask_xX[0][8] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -1 + 32];
            }
            {
                _tmp126 += _constmask_xX[0][9] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 0 + 32];
            }
            {
                _tmp126 += _constmask_xX[0][10] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 1 + 32];
            }
            {
                _tmp126 += _constmask_xX[0][11] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 2 + 32];
            }
            {
                _tmp126 += _constmask_xX[0][12] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 3 + 32];
            }
            {
                _tmp126 += _constmask_xX[0][13] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 4 + 32];
            }
            {
                _tmp126 += _constmask_xX[0][14] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 5 + 32];
            }
            {
                _tmp126 += _constmask_xX[0][15] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 6 + 32];
            }
            {
                _tmp126 += _constmask_xX[0][16] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 7 + 32];
            }
            {
                _tmp126 += _constmask_xX[0][17] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 8 + 32];
            }
            {
                _tmp126 += _constmask_xX[0][18] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 9 + 32];
            }
            iter[(gid_y + 6 * (int)blockDim.y) * iter_stride + gid_x] = (float)(_tmp126);
        }
        if (gid_y + 7 * (int)blockDim.y < iter_height) {
            float _tmp127 = 0.F;
            {
                _tmp127 += _constmask_xX[0][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -9 + 32];
            }
            {
                _tmp127 += _constmask_xX[0][1] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -8 + 32];
            }
            {
                _tmp127 += _constmask_xX[0][2] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -7 + 32];
            }
            {
                _tmp127 += _constmask_xX[0][3] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -6 + 32];
            }
            {
                _tmp127 += _constmask_xX[0][4] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -5 + 32];
            }
            {
                _tmp127 += _constmask_xX[0][5] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -4 + 32];
            }
            {
                _tmp127 += _constmask_xX[0][6] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -3 + 32];
            }
            {
                _tmp127 += _constmask_xX[0][7] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -2 + 32];
            }
            {
                _tmp127 += _constmask_xX[0][8] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -1 + 32];
            }
            {
                _tmp127 += _constmask_xX[0][9] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 0 + 32];
            }
            {
                _tmp127 += _constmask_xX[0][10] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 1 + 32];
            }
            {
                _tmp127 += _constmask_xX[0][11] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 2 + 32];
            }
            {
                _tmp127 += _constmask_xX[0][12] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 3 + 32];
            }
            {
                _tmp127 += _constmask_xX[0][13] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 4 + 32];
            }
            {
                _tmp127 += _constmask_xX[0][14] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 5 + 32];
            }
            {
                _tmp127 += _constmask_xX[0][15] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 6 + 32];
            }
            {
                _tmp127 += _constmask_xX[0][16] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 7 + 32];
            }
            {
                _tmp127 += _constmask_xX[0][17] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 8 + 32];
            }
            {
                _tmp127 += _constmask_xX[0][18] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 9 + 32];
            }
            iter[(gid_y + 7 * (int)blockDim.y) * iter_stride + gid_x] = (float)(_tmp127);
        }
    }
    goto BH_EXIT;
  BH_NO:
    {
        _smeminput[(int)threadIdx.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y) * input_stride + gid_x + 0 * (int)blockDim.x - 32);
        _smeminput[(int)threadIdx.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y) * input_stride + gid_x + 1 * (int)blockDim.x - 32);
        _smeminput[(int)threadIdx.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y) * input_stride + gid_x + 2 * (int)blockDim.x - 32);
        _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 1 * (int)blockDim.y) * input_stride + gid_x + 0 * (int)blockDim.x - 32);
        _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 1 * (int)blockDim.y) * input_stride + gid_x + 1 * (int)blockDim.x - 32);
        _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 1 * (int)blockDim.y) * input_stride + gid_x + 2 * (int)blockDim.x - 32);
        _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 2 * (int)blockDim.y) * input_stride + gid_x + 0 * (int)blockDim.x - 32);
        _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 2 * (int)blockDim.y) * input_stride + gid_x + 1 * (int)blockDim.x - 32);
        _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 2 * (int)blockDim.y) * input_stride + gid_x + 2 * (int)blockDim.x - 32);
        _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 3 * (int)blockDim.y) * input_stride + gid_x + 0 * (int)blockDim.x - 32);
        _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 3 * (int)blockDim.y) * input_stride + gid_x + 1 * (int)blockDim.x - 32);
        _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 3 * (int)blockDim.y) * input_stride + gid_x + 2 * (int)blockDim.x - 32);
        _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 4 * (int)blockDim.y) * input_stride + gid_x + 0 * (int)blockDim.x - 32);
        _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 4 * (int)blockDim.y) * input_stride + gid_x + 1 * (int)blockDim.x - 32);
        _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 4 * (int)blockDim.y) * input_stride + gid_x + 2 * (int)blockDim.x - 32);
        _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 5 * (int)blockDim.y) * input_stride + gid_x + 0 * (int)blockDim.x - 32);
        _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 5 * (int)blockDim.y) * input_stride + gid_x + 1 * (int)blockDim.x - 32);
        _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 5 * (int)blockDim.y) * input_stride + gid_x + 2 * (int)blockDim.x - 32);
        _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 6 * (int)blockDim.y) * input_stride + gid_x + 0 * (int)blockDim.x - 32);
        _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 6 * (int)blockDim.y) * input_stride + gid_x + 1 * (int)blockDim.x - 32);
        _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 6 * (int)blockDim.y) * input_stride + gid_x + 2 * (int)blockDim.x - 32);
        _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y][(int)threadIdx.x + 0 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 7 * (int)blockDim.y) * input_stride + gid_x + 0 * (int)blockDim.x - 32);
        _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y][(int)threadIdx.x + 1 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 7 * (int)blockDim.y) * input_stride + gid_x + 1 * (int)blockDim.x - 32);
        _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y][(int)threadIdx.x + 2 * (int)blockDim.x] = tex1Dfetch(_texinputX, (gid_y + 7 * (int)blockDim.y) * input_stride + gid_x + 2 * (int)blockDim.x - 32);
        __syncthreads();
        {
            float _tmp128 = 0.F;
            {
                _tmp128 += _constmask_xX[0][0] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + -9 + 32];
            }
            {
                _tmp128 += _constmask_xX[0][1] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + -8 + 32];
            }
            {
                _tmp128 += _constmask_xX[0][2] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + -7 + 32];
            }
            {
                _tmp128 += _constmask_xX[0][3] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + -6 + 32];
            }
            {
                _tmp128 += _constmask_xX[0][4] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + -5 + 32];
            }
            {
                _tmp128 += _constmask_xX[0][5] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + -4 + 32];
            }
            {
                _tmp128 += _constmask_xX[0][6] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + -3 + 32];
            }
            {
                _tmp128 += _constmask_xX[0][7] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + -2 + 32];
            }
            {
                _tmp128 += _constmask_xX[0][8] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + -1 + 32];
            }
            {
                _tmp128 += _constmask_xX[0][9] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + 0 + 32];
            }
            {
                _tmp128 += _constmask_xX[0][10] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + 1 + 32];
            }
            {
                _tmp128 += _constmask_xX[0][11] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + 2 + 32];
            }
            {
                _tmp128 += _constmask_xX[0][12] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + 3 + 32];
            }
            {
                _tmp128 += _constmask_xX[0][13] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + 4 + 32];
            }
            {
                _tmp128 += _constmask_xX[0][14] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + 5 + 32];
            }
            {
                _tmp128 += _constmask_xX[0][15] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + 6 + 32];
            }
            {
                _tmp128 += _constmask_xX[0][16] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + 7 + 32];
            }
            {
                _tmp128 += _constmask_xX[0][17] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + 8 + 32];
            }
            {
                _tmp128 += _constmask_xX[0][18] * _smeminput[(int)threadIdx.y + 0 + 0][(int)threadIdx.x + 9 + 32];
            }
            iter[(gid_y) * iter_stride + gid_x] = (float)(_tmp128);
        }
        {
            float _tmp129 = 0.F;
            {
                _tmp129 += _constmask_xX[0][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -9 + 32];
            }
            {
                _tmp129 += _constmask_xX[0][1] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -8 + 32];
            }
            {
                _tmp129 += _constmask_xX[0][2] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -7 + 32];
            }
            {
                _tmp129 += _constmask_xX[0][3] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -6 + 32];
            }
            {
                _tmp129 += _constmask_xX[0][4] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -5 + 32];
            }
            {
                _tmp129 += _constmask_xX[0][5] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -4 + 32];
            }
            {
                _tmp129 += _constmask_xX[0][6] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -3 + 32];
            }
            {
                _tmp129 += _constmask_xX[0][7] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -2 + 32];
            }
            {
                _tmp129 += _constmask_xX[0][8] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -1 + 32];
            }
            {
                _tmp129 += _constmask_xX[0][9] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 0 + 32];
            }
            {
                _tmp129 += _constmask_xX[0][10] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 1 + 32];
            }
            {
                _tmp129 += _constmask_xX[0][11] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 2 + 32];
            }
            {
                _tmp129 += _constmask_xX[0][12] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 3 + 32];
            }
            {
                _tmp129 += _constmask_xX[0][13] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 4 + 32];
            }
            {
                _tmp129 += _constmask_xX[0][14] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 5 + 32];
            }
            {
                _tmp129 += _constmask_xX[0][15] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 6 + 32];
            }
            {
                _tmp129 += _constmask_xX[0][16] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 7 + 32];
            }
            {
                _tmp129 += _constmask_xX[0][17] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 8 + 32];
            }
            {
                _tmp129 += _constmask_xX[0][18] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 9 + 32];
            }
            iter[(gid_y + 1 * (int)blockDim.y) * iter_stride + gid_x] = (float)(_tmp129);
        }
        {
            float _tmp130 = 0.F;
            {
                _tmp130 += _constmask_xX[0][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -9 + 32];
            }
            {
                _tmp130 += _constmask_xX[0][1] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -8 + 32];
            }
            {
                _tmp130 += _constmask_xX[0][2] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -7 + 32];
            }
            {
                _tmp130 += _constmask_xX[0][3] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -6 + 32];
            }
            {
                _tmp130 += _constmask_xX[0][4] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -5 + 32];
            }
            {
                _tmp130 += _constmask_xX[0][5] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -4 + 32];
            }
            {
                _tmp130 += _constmask_xX[0][6] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -3 + 32];
            }
            {
                _tmp130 += _constmask_xX[0][7] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -2 + 32];
            }
            {
                _tmp130 += _constmask_xX[0][8] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -1 + 32];
            }
            {
                _tmp130 += _constmask_xX[0][9] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 0 + 32];
            }
            {
                _tmp130 += _constmask_xX[0][10] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 1 + 32];
            }
            {
                _tmp130 += _constmask_xX[0][11] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 2 + 32];
            }
            {
                _tmp130 += _constmask_xX[0][12] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 3 + 32];
            }
            {
                _tmp130 += _constmask_xX[0][13] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 4 + 32];
            }
            {
                _tmp130 += _constmask_xX[0][14] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 5 + 32];
            }
            {
                _tmp130 += _constmask_xX[0][15] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 6 + 32];
            }
            {
                _tmp130 += _constmask_xX[0][16] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 7 + 32];
            }
            {
                _tmp130 += _constmask_xX[0][17] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 8 + 32];
            }
            {
                _tmp130 += _constmask_xX[0][18] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 9 + 32];
            }
            iter[(gid_y + 2 * (int)blockDim.y) * iter_stride + gid_x] = (float)(_tmp130);
        }
        {
            float _tmp131 = 0.F;
            {
                _tmp131 += _constmask_xX[0][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -9 + 32];
            }
            {
                _tmp131 += _constmask_xX[0][1] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -8 + 32];
            }
            {
                _tmp131 += _constmask_xX[0][2] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -7 + 32];
            }
            {
                _tmp131 += _constmask_xX[0][3] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -6 + 32];
            }
            {
                _tmp131 += _constmask_xX[0][4] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -5 + 32];
            }
            {
                _tmp131 += _constmask_xX[0][5] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -4 + 32];
            }
            {
                _tmp131 += _constmask_xX[0][6] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -3 + 32];
            }
            {
                _tmp131 += _constmask_xX[0][7] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -2 + 32];
            }
            {
                _tmp131 += _constmask_xX[0][8] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -1 + 32];
            }
            {
                _tmp131 += _constmask_xX[0][9] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 0 + 32];
            }
            {
                _tmp131 += _constmask_xX[0][10] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 1 + 32];
            }
            {
                _tmp131 += _constmask_xX[0][11] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 2 + 32];
            }
            {
                _tmp131 += _constmask_xX[0][12] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 3 + 32];
            }
            {
                _tmp131 += _constmask_xX[0][13] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 4 + 32];
            }
            {
                _tmp131 += _constmask_xX[0][14] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 5 + 32];
            }
            {
                _tmp131 += _constmask_xX[0][15] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 6 + 32];
            }
            {
                _tmp131 += _constmask_xX[0][16] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 7 + 32];
            }
            {
                _tmp131 += _constmask_xX[0][17] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 8 + 32];
            }
            {
                _tmp131 += _constmask_xX[0][18] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 9 + 32];
            }
            iter[(gid_y + 3 * (int)blockDim.y) * iter_stride + gid_x] = (float)(_tmp131);
        }
        {
            float _tmp132 = 0.F;
            {
                _tmp132 += _constmask_xX[0][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -9 + 32];
            }
            {
                _tmp132 += _constmask_xX[0][1] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -8 + 32];
            }
            {
                _tmp132 += _constmask_xX[0][2] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -7 + 32];
            }
            {
                _tmp132 += _constmask_xX[0][3] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -6 + 32];
            }
            {
                _tmp132 += _constmask_xX[0][4] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -5 + 32];
            }
            {
                _tmp132 += _constmask_xX[0][5] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -4 + 32];
            }
            {
                _tmp132 += _constmask_xX[0][6] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -3 + 32];
            }
            {
                _tmp132 += _constmask_xX[0][7] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -2 + 32];
            }
            {
                _tmp132 += _constmask_xX[0][8] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -1 + 32];
            }
            {
                _tmp132 += _constmask_xX[0][9] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 0 + 32];
            }
            {
                _tmp132 += _constmask_xX[0][10] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 1 + 32];
            }
            {
                _tmp132 += _constmask_xX[0][11] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 2 + 32];
            }
            {
                _tmp132 += _constmask_xX[0][12] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 3 + 32];
            }
            {
                _tmp132 += _constmask_xX[0][13] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 4 + 32];
            }
            {
                _tmp132 += _constmask_xX[0][14] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 5 + 32];
            }
            {
                _tmp132 += _constmask_xX[0][15] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 6 + 32];
            }
            {
                _tmp132 += _constmask_xX[0][16] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 7 + 32];
            }
            {
                _tmp132 += _constmask_xX[0][17] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 8 + 32];
            }
            {
                _tmp132 += _constmask_xX[0][18] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 9 + 32];
            }
            iter[(gid_y + 4 * (int)blockDim.y) * iter_stride + gid_x] = (float)(_tmp132);
        }
        {
            float _tmp133 = 0.F;
            {
                _tmp133 += _constmask_xX[0][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -9 + 32];
            }
            {
                _tmp133 += _constmask_xX[0][1] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -8 + 32];
            }
            {
                _tmp133 += _constmask_xX[0][2] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -7 + 32];
            }
            {
                _tmp133 += _constmask_xX[0][3] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -6 + 32];
            }
            {
                _tmp133 += _constmask_xX[0][4] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -5 + 32];
            }
            {
                _tmp133 += _constmask_xX[0][5] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -4 + 32];
            }
            {
                _tmp133 += _constmask_xX[0][6] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -3 + 32];
            }
            {
                _tmp133 += _constmask_xX[0][7] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -2 + 32];
            }
            {
                _tmp133 += _constmask_xX[0][8] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -1 + 32];
            }
            {
                _tmp133 += _constmask_xX[0][9] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 0 + 32];
            }
            {
                _tmp133 += _constmask_xX[0][10] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 1 + 32];
            }
            {
                _tmp133 += _constmask_xX[0][11] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 2 + 32];
            }
            {
                _tmp133 += _constmask_xX[0][12] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 3 + 32];
            }
            {
                _tmp133 += _constmask_xX[0][13] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 4 + 32];
            }
            {
                _tmp133 += _constmask_xX[0][14] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 5 + 32];
            }
            {
                _tmp133 += _constmask_xX[0][15] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 6 + 32];
            }
            {
                _tmp133 += _constmask_xX[0][16] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 7 + 32];
            }
            {
                _tmp133 += _constmask_xX[0][17] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 8 + 32];
            }
            {
                _tmp133 += _constmask_xX[0][18] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 9 + 32];
            }
            iter[(gid_y + 5 * (int)blockDim.y) * iter_stride + gid_x] = (float)(_tmp133);
        }
        {
            float _tmp134 = 0.F;
            {
                _tmp134 += _constmask_xX[0][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -9 + 32];
            }
            {
                _tmp134 += _constmask_xX[0][1] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -8 + 32];
            }
            {
                _tmp134 += _constmask_xX[0][2] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -7 + 32];
            }
            {
                _tmp134 += _constmask_xX[0][3] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -6 + 32];
            }
            {
                _tmp134 += _constmask_xX[0][4] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -5 + 32];
            }
            {
                _tmp134 += _constmask_xX[0][5] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -4 + 32];
            }
            {
                _tmp134 += _constmask_xX[0][6] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -3 + 32];
            }
            {
                _tmp134 += _constmask_xX[0][7] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -2 + 32];
            }
            {
                _tmp134 += _constmask_xX[0][8] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -1 + 32];
            }
            {
                _tmp134 += _constmask_xX[0][9] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 0 + 32];
            }
            {
                _tmp134 += _constmask_xX[0][10] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 1 + 32];
            }
            {
                _tmp134 += _constmask_xX[0][11] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 2 + 32];
            }
            {
                _tmp134 += _constmask_xX[0][12] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 3 + 32];
            }
            {
                _tmp134 += _constmask_xX[0][13] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 4 + 32];
            }
            {
                _tmp134 += _constmask_xX[0][14] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 5 + 32];
            }
            {
                _tmp134 += _constmask_xX[0][15] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 6 + 32];
            }
            {
                _tmp134 += _constmask_xX[0][16] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 7 + 32];
            }
            {
                _tmp134 += _constmask_xX[0][17] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 8 + 32];
            }
            {
                _tmp134 += _constmask_xX[0][18] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 9 + 32];
            }
            iter[(gid_y + 6 * (int)blockDim.y) * iter_stride + gid_x] = (float)(_tmp134);
        }
        {
            float _tmp135 = 0.F;
            {
                _tmp135 += _constmask_xX[0][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -9 + 32];
            }
            {
                _tmp135 += _constmask_xX[0][1] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -8 + 32];
            }
            {
                _tmp135 += _constmask_xX[0][2] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -7 + 32];
            }
            {
                _tmp135 += _constmask_xX[0][3] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -6 + 32];
            }
            {
                _tmp135 += _constmask_xX[0][4] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -5 + 32];
            }
            {
                _tmp135 += _constmask_xX[0][5] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -4 + 32];
            }
            {
                _tmp135 += _constmask_xX[0][6] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -3 + 32];
            }
            {
                _tmp135 += _constmask_xX[0][7] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -2 + 32];
            }
            {
                _tmp135 += _constmask_xX[0][8] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + -1 + 32];
            }
            {
                _tmp135 += _constmask_xX[0][9] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 0 + 32];
            }
            {
                _tmp135 += _constmask_xX[0][10] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 1 + 32];
            }
            {
                _tmp135 += _constmask_xX[0][11] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 2 + 32];
            }
            {
                _tmp135 += _constmask_xX[0][12] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 3 + 32];
            }
            {
                _tmp135 += _constmask_xX[0][13] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 4 + 32];
            }
            {
                _tmp135 += _constmask_xX[0][14] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 5 + 32];
            }
            {
                _tmp135 += _constmask_xX[0][15] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 6 + 32];
            }
            {
                _tmp135 += _constmask_xX[0][16] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 7 + 32];
            }
            {
                _tmp135 += _constmask_xX[0][17] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 8 + 32];
            }
            {
                _tmp135 += _constmask_xX[0][18] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 0][(int)threadIdx.x + 9 + 32];
            }
            iter[(gid_y + 7 * (int)blockDim.y) * iter_stride + gid_x] = (float)(_tmp135);
        }
    }
    goto BH_EXIT;
  BH_EXIT:
    ;
}
}

#endif //_CUGAUSSIANFILTERROWX_CU_

