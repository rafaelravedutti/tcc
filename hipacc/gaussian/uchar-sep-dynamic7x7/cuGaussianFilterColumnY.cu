#include "hip/hip_runtime.h"
#ifndef _CUGAUSSIANFILTERCOLUMNY_CU_
#define _CUGAUSSIANFILTERCOLUMNY_CU_

#include "hipacc_types.hpp"
#include "hipacc_math_functions.hpp"

texture<float, hipTextureType1D, hipReadModeElementType> _texinputY;
const textureReference *_texinputYRef;
__device__ __constant__ float _constmask_yY[7][1];


extern "C" {
__global__ __launch_bounds__ (32*1) void cuGaussianFilterColumnYKernel(uchar * __restrict__ iter, int iter_width, int iter_height, int iter_stride, int input_width, int input_height, int input_stride, int bh_start_right, int bh_start_top, int bh_start_bottom, int bh_fall_back) {
    const int gid_x = blockDim.x * blockIdx.x + threadIdx.x;
    const int gid_y = blockDim.y * blockIdx.y * 8 + threadIdx.y;
    float _smeminput[14][33] __attribute__((shared));
    if (bh_fall_back)
        goto BH_FB;
    if (blockIdx.y < bh_start_top)
        goto BH_T;
    if (blockIdx.y >= bh_start_bottom)
        goto BH_B;
    if (blockIdx.x >= bh_start_right)
        goto BH_R;
    goto BH_NO;
  BH_FB:
    {
        int _gid_y0 = gid_y + (-3);
        if (_gid_y0 >= input_height)
            _gid_y0 = input_height - 1;
        if (_gid_y0 < 0)
            _gid_y0 = 0;
        _smeminput[(int)threadIdx.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y0) * input_stride + gid_x);
        int _gid_y1 = gid_y + 1 * (int)blockDim.y + (-3);
        if (_gid_y1 >= input_height)
            _gid_y1 = input_height - 1;
        if (_gid_y1 < 0)
            _gid_y1 = 0;
        _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y1) * input_stride + gid_x);
        int _gid_y2 = gid_y + 2 * (int)blockDim.y + (-3);
        if (_gid_y2 >= input_height)
            _gid_y2 = input_height - 1;
        if (_gid_y2 < 0)
            _gid_y2 = 0;
        _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y2) * input_stride + gid_x);
        int _gid_y3 = gid_y + 3 * (int)blockDim.y + (-3);
        if (_gid_y3 >= input_height)
            _gid_y3 = input_height - 1;
        if (_gid_y3 < 0)
            _gid_y3 = 0;
        _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y3) * input_stride + gid_x);
        int _gid_y4 = gid_y + 4 * (int)blockDim.y + (-3);
        if (_gid_y4 >= input_height)
            _gid_y4 = input_height - 1;
        if (_gid_y4 < 0)
            _gid_y4 = 0;
        _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y4) * input_stride + gid_x);
        int _gid_y5 = gid_y + 5 * (int)blockDim.y + (-3);
        if (_gid_y5 >= input_height)
            _gid_y5 = input_height - 1;
        if (_gid_y5 < 0)
            _gid_y5 = 0;
        _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y5) * input_stride + gid_x);
        int _gid_y6 = gid_y + 6 * (int)blockDim.y + (-3);
        if (_gid_y6 >= input_height)
            _gid_y6 = input_height - 1;
        if (_gid_y6 < 0)
            _gid_y6 = 0;
        _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y6) * input_stride + gid_x);
        int _gid_y7 = gid_y + 7 * (int)blockDim.y + (-3);
        if (_gid_y7 >= input_height)
            _gid_y7 = input_height - 1;
        if (_gid_y7 < 0)
            _gid_y7 = 0;
        _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y7) * input_stride + gid_x);
        int _gid_y8 = gid_y + 8 * (int)blockDim.y + (-3);
        if (_gid_y8 >= input_height)
            _gid_y8 = input_height - 1;
        if (_gid_y8 < 0)
            _gid_y8 = 0;
        _smeminput[(int)threadIdx.y + 8 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y8) * input_stride + gid_x);
        int _gid_y9 = gid_y + 9 * (int)blockDim.y + (-3);
        if (_gid_y9 >= input_height)
            _gid_y9 = input_height - 1;
        if (_gid_y9 < 0)
            _gid_y9 = 0;
        _smeminput[(int)threadIdx.y + 9 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y9) * input_stride + gid_x);
        int _gid_y10 = gid_y + 10 * (int)blockDim.y + (-3);
        if (_gid_y10 >= input_height)
            _gid_y10 = input_height - 1;
        if (_gid_y10 < 0)
            _gid_y10 = 0;
        _smeminput[(int)threadIdx.y + 10 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y10) * input_stride + gid_x);
        int _gid_y11 = gid_y + 11 * (int)blockDim.y + (-3);
        if (_gid_y11 >= input_height)
            _gid_y11 = input_height - 1;
        if (_gid_y11 < 0)
            _gid_y11 = 0;
        _smeminput[(int)threadIdx.y + 11 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y11) * input_stride + gid_x);
        int _gid_y12 = gid_y + 12 * (int)blockDim.y + (-3);
        if (_gid_y12 >= input_height)
            _gid_y12 = input_height - 1;
        if (_gid_y12 < 0)
            _gid_y12 = 0;
        _smeminput[(int)threadIdx.y + 12 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y12) * input_stride + gid_x);
        int _gid_y13 = gid_y + 13 * (int)blockDim.y + (-3);
        if (_gid_y13 >= input_height)
            _gid_y13 = input_height - 1;
        if (_gid_y13 < 0)
            _gid_y13 = 0;
        _smeminput[(int)threadIdx.y + 13 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y13) * input_stride + gid_x);
        __syncthreads();
        if (gid_x < iter_width) {
            if (gid_y < iter_height) {
                float _tmp14 = 0.F;
                {
                    _tmp14 += _constmask_yY[0][0] * _smeminput[(int)threadIdx.y + -3 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp14 += _constmask_yY[1][0] * _smeminput[(int)threadIdx.y + -2 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp14 += _constmask_yY[2][0] * _smeminput[(int)threadIdx.y + -1 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp14 += _constmask_yY[3][0] * _smeminput[(int)threadIdx.y + 0 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp14 += _constmask_yY[4][0] * _smeminput[(int)threadIdx.y + 1 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp14 += _constmask_yY[5][0] * _smeminput[(int)threadIdx.y + 2 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp14 += _constmask_yY[6][0] * _smeminput[(int)threadIdx.y + 3 + 3][(int)threadIdx.x + 0 + 0];
                }
                iter[(gid_y) * iter_stride + gid_x] = (uchar)(_tmp14 + 0.5F);
            }
        }
        if (gid_x < iter_width) {
            if (gid_y + 1 * (int)blockDim.y < iter_height) {
                float _tmp15 = 0.F;
                {
                    _tmp15 += _constmask_yY[0][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + -3 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp15 += _constmask_yY[1][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + -2 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp15 += _constmask_yY[2][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + -1 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp15 += _constmask_yY[3][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp15 += _constmask_yY[4][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 1 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp15 += _constmask_yY[5][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 2 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp15 += _constmask_yY[6][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 3 + 3][(int)threadIdx.x + 0 + 0];
                }
                iter[(gid_y + 1 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp15 + 0.5F);
            }
        }
        if (gid_x < iter_width) {
            if (gid_y + 2 * (int)blockDim.y < iter_height) {
                float _tmp16 = 0.F;
                {
                    _tmp16 += _constmask_yY[0][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + -3 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp16 += _constmask_yY[1][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + -2 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp16 += _constmask_yY[2][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + -1 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp16 += _constmask_yY[3][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp16 += _constmask_yY[4][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 1 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp16 += _constmask_yY[5][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 2 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp16 += _constmask_yY[6][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 3 + 3][(int)threadIdx.x + 0 + 0];
                }
                iter[(gid_y + 2 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp16 + 0.5F);
            }
        }
        if (gid_x < iter_width) {
            if (gid_y + 3 * (int)blockDim.y < iter_height) {
                float _tmp17 = 0.F;
                {
                    _tmp17 += _constmask_yY[0][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + -3 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp17 += _constmask_yY[1][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + -2 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp17 += _constmask_yY[2][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + -1 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp17 += _constmask_yY[3][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp17 += _constmask_yY[4][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 1 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp17 += _constmask_yY[5][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 2 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp17 += _constmask_yY[6][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 3 + 3][(int)threadIdx.x + 0 + 0];
                }
                iter[(gid_y + 3 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp17 + 0.5F);
            }
        }
        if (gid_x < iter_width) {
            if (gid_y + 4 * (int)blockDim.y < iter_height) {
                float _tmp18 = 0.F;
                {
                    _tmp18 += _constmask_yY[0][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + -3 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp18 += _constmask_yY[1][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + -2 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp18 += _constmask_yY[2][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + -1 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp18 += _constmask_yY[3][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp18 += _constmask_yY[4][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 1 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp18 += _constmask_yY[5][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 2 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp18 += _constmask_yY[6][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 3 + 3][(int)threadIdx.x + 0 + 0];
                }
                iter[(gid_y + 4 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp18 + 0.5F);
            }
        }
        if (gid_x < iter_width) {
            if (gid_y + 5 * (int)blockDim.y < iter_height) {
                float _tmp19 = 0.F;
                {
                    _tmp19 += _constmask_yY[0][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + -3 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp19 += _constmask_yY[1][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + -2 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp19 += _constmask_yY[2][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + -1 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp19 += _constmask_yY[3][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp19 += _constmask_yY[4][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 1 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp19 += _constmask_yY[5][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 2 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp19 += _constmask_yY[6][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 3 + 3][(int)threadIdx.x + 0 + 0];
                }
                iter[(gid_y + 5 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp19 + 0.5F);
            }
        }
        if (gid_x < iter_width) {
            if (gid_y + 6 * (int)blockDim.y < iter_height) {
                float _tmp20 = 0.F;
                {
                    _tmp20 += _constmask_yY[0][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + -3 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp20 += _constmask_yY[1][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + -2 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp20 += _constmask_yY[2][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + -1 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp20 += _constmask_yY[3][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp20 += _constmask_yY[4][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 1 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp20 += _constmask_yY[5][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 2 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp20 += _constmask_yY[6][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 3 + 3][(int)threadIdx.x + 0 + 0];
                }
                iter[(gid_y + 6 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp20 + 0.5F);
            }
        }
        if (gid_x < iter_width) {
            if (gid_y + 7 * (int)blockDim.y < iter_height) {
                float _tmp21 = 0.F;
                {
                    _tmp21 += _constmask_yY[0][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + -3 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp21 += _constmask_yY[1][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + -2 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp21 += _constmask_yY[2][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + -1 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp21 += _constmask_yY[3][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp21 += _constmask_yY[4][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 1 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp21 += _constmask_yY[5][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 2 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp21 += _constmask_yY[6][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 3 + 3][(int)threadIdx.x + 0 + 0];
                }
                iter[(gid_y + 7 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp21 + 0.5F);
            }
        }
    }
    goto BH_EXIT;
  BH_T:
    {
        int _gid_y22 = gid_y + (-3);
        if (_gid_y22 < 0)
            _gid_y22 = 0;
        _smeminput[(int)threadIdx.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y22) * input_stride + gid_x);
        int _gid_y23 = gid_y + 1 * (int)blockDim.y + (-3);
        if (_gid_y23 < 0)
            _gid_y23 = 0;
        _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y23) * input_stride + gid_x);
        int _gid_y24 = gid_y + 2 * (int)blockDim.y + (-3);
        if (_gid_y24 < 0)
            _gid_y24 = 0;
        _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y24) * input_stride + gid_x);
        int _gid_y25 = gid_y + 3 * (int)blockDim.y + (-3);
        if (_gid_y25 < 0)
            _gid_y25 = 0;
        _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y25) * input_stride + gid_x);
        int _gid_y26 = gid_y + 4 * (int)blockDim.y + (-3);
        if (_gid_y26 < 0)
            _gid_y26 = 0;
        _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y26) * input_stride + gid_x);
        int _gid_y27 = gid_y + 5 * (int)blockDim.y + (-3);
        if (_gid_y27 < 0)
            _gid_y27 = 0;
        _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y27) * input_stride + gid_x);
        int _gid_y28 = gid_y + 6 * (int)blockDim.y + (-3);
        if (_gid_y28 < 0)
            _gid_y28 = 0;
        _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y28) * input_stride + gid_x);
        int _gid_y29 = gid_y + 7 * (int)blockDim.y + (-3);
        if (_gid_y29 < 0)
            _gid_y29 = 0;
        _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y29) * input_stride + gid_x);
        int _gid_y30 = gid_y + 8 * (int)blockDim.y + (-3);
        if (_gid_y30 < 0)
            _gid_y30 = 0;
        _smeminput[(int)threadIdx.y + 8 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y30) * input_stride + gid_x);
        int _gid_y31 = gid_y + 9 * (int)blockDim.y + (-3);
        if (_gid_y31 < 0)
            _gid_y31 = 0;
        _smeminput[(int)threadIdx.y + 9 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y31) * input_stride + gid_x);
        int _gid_y32 = gid_y + 10 * (int)blockDim.y + (-3);
        if (_gid_y32 < 0)
            _gid_y32 = 0;
        _smeminput[(int)threadIdx.y + 10 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y32) * input_stride + gid_x);
        int _gid_y33 = gid_y + 11 * (int)blockDim.y + (-3);
        if (_gid_y33 < 0)
            _gid_y33 = 0;
        _smeminput[(int)threadIdx.y + 11 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y33) * input_stride + gid_x);
        int _gid_y34 = gid_y + 12 * (int)blockDim.y + (-3);
        if (_gid_y34 < 0)
            _gid_y34 = 0;
        _smeminput[(int)threadIdx.y + 12 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y34) * input_stride + gid_x);
        int _gid_y35 = gid_y + 13 * (int)blockDim.y + (-3);
        if (_gid_y35 < 0)
            _gid_y35 = 0;
        _smeminput[(int)threadIdx.y + 13 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y35) * input_stride + gid_x);
        __syncthreads();
        if (gid_x < iter_width) {
            {
                float _tmp36 = 0.F;
                {
                    _tmp36 += _constmask_yY[0][0] * _smeminput[(int)threadIdx.y + -3 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp36 += _constmask_yY[1][0] * _smeminput[(int)threadIdx.y + -2 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp36 += _constmask_yY[2][0] * _smeminput[(int)threadIdx.y + -1 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp36 += _constmask_yY[3][0] * _smeminput[(int)threadIdx.y + 0 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp36 += _constmask_yY[4][0] * _smeminput[(int)threadIdx.y + 1 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp36 += _constmask_yY[5][0] * _smeminput[(int)threadIdx.y + 2 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp36 += _constmask_yY[6][0] * _smeminput[(int)threadIdx.y + 3 + 3][(int)threadIdx.x + 0 + 0];
                }
                iter[(gid_y) * iter_stride + gid_x] = (uchar)(_tmp36 + 0.5F);
            }
        }
        if (gid_x < iter_width) {
            {
                float _tmp37 = 0.F;
                {
                    _tmp37 += _constmask_yY[0][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + -3 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp37 += _constmask_yY[1][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + -2 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp37 += _constmask_yY[2][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + -1 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp37 += _constmask_yY[3][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp37 += _constmask_yY[4][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 1 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp37 += _constmask_yY[5][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 2 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp37 += _constmask_yY[6][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 3 + 3][(int)threadIdx.x + 0 + 0];
                }
                iter[(gid_y + 1 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp37 + 0.5F);
            }
        }
        if (gid_x < iter_width) {
            {
                float _tmp38 = 0.F;
                {
                    _tmp38 += _constmask_yY[0][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + -3 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp38 += _constmask_yY[1][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + -2 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp38 += _constmask_yY[2][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + -1 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp38 += _constmask_yY[3][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp38 += _constmask_yY[4][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 1 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp38 += _constmask_yY[5][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 2 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp38 += _constmask_yY[6][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 3 + 3][(int)threadIdx.x + 0 + 0];
                }
                iter[(gid_y + 2 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp38 + 0.5F);
            }
        }
        if (gid_x < iter_width) {
            {
                float _tmp39 = 0.F;
                {
                    _tmp39 += _constmask_yY[0][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + -3 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp39 += _constmask_yY[1][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + -2 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp39 += _constmask_yY[2][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + -1 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp39 += _constmask_yY[3][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp39 += _constmask_yY[4][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 1 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp39 += _constmask_yY[5][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 2 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp39 += _constmask_yY[6][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 3 + 3][(int)threadIdx.x + 0 + 0];
                }
                iter[(gid_y + 3 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp39 + 0.5F);
            }
        }
        if (gid_x < iter_width) {
            {
                float _tmp40 = 0.F;
                {
                    _tmp40 += _constmask_yY[0][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + -3 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp40 += _constmask_yY[1][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + -2 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp40 += _constmask_yY[2][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + -1 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp40 += _constmask_yY[3][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp40 += _constmask_yY[4][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 1 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp40 += _constmask_yY[5][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 2 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp40 += _constmask_yY[6][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 3 + 3][(int)threadIdx.x + 0 + 0];
                }
                iter[(gid_y + 4 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp40 + 0.5F);
            }
        }
        if (gid_x < iter_width) {
            {
                float _tmp41 = 0.F;
                {
                    _tmp41 += _constmask_yY[0][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + -3 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp41 += _constmask_yY[1][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + -2 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp41 += _constmask_yY[2][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + -1 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp41 += _constmask_yY[3][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp41 += _constmask_yY[4][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 1 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp41 += _constmask_yY[5][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 2 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp41 += _constmask_yY[6][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 3 + 3][(int)threadIdx.x + 0 + 0];
                }
                iter[(gid_y + 5 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp41 + 0.5F);
            }
        }
        if (gid_x < iter_width) {
            {
                float _tmp42 = 0.F;
                {
                    _tmp42 += _constmask_yY[0][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + -3 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp42 += _constmask_yY[1][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + -2 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp42 += _constmask_yY[2][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + -1 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp42 += _constmask_yY[3][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp42 += _constmask_yY[4][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 1 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp42 += _constmask_yY[5][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 2 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp42 += _constmask_yY[6][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 3 + 3][(int)threadIdx.x + 0 + 0];
                }
                iter[(gid_y + 6 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp42 + 0.5F);
            }
        }
        if (gid_x < iter_width) {
            {
                float _tmp43 = 0.F;
                {
                    _tmp43 += _constmask_yY[0][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + -3 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp43 += _constmask_yY[1][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + -2 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp43 += _constmask_yY[2][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + -1 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp43 += _constmask_yY[3][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp43 += _constmask_yY[4][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 1 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp43 += _constmask_yY[5][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 2 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp43 += _constmask_yY[6][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 3 + 3][(int)threadIdx.x + 0 + 0];
                }
                iter[(gid_y + 7 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp43 + 0.5F);
            }
        }
    }
    goto BH_EXIT;
  BH_B:
    {
        int _gid_y44 = gid_y + (-3);
        if (_gid_y44 >= input_height)
            _gid_y44 = input_height - 1;
        _smeminput[(int)threadIdx.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y44) * input_stride + gid_x);
        int _gid_y45 = gid_y + 1 * (int)blockDim.y + (-3);
        if (_gid_y45 >= input_height)
            _gid_y45 = input_height - 1;
        _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y45) * input_stride + gid_x);
        int _gid_y46 = gid_y + 2 * (int)blockDim.y + (-3);
        if (_gid_y46 >= input_height)
            _gid_y46 = input_height - 1;
        _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y46) * input_stride + gid_x);
        int _gid_y47 = gid_y + 3 * (int)blockDim.y + (-3);
        if (_gid_y47 >= input_height)
            _gid_y47 = input_height - 1;
        _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y47) * input_stride + gid_x);
        int _gid_y48 = gid_y + 4 * (int)blockDim.y + (-3);
        if (_gid_y48 >= input_height)
            _gid_y48 = input_height - 1;
        _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y48) * input_stride + gid_x);
        int _gid_y49 = gid_y + 5 * (int)blockDim.y + (-3);
        if (_gid_y49 >= input_height)
            _gid_y49 = input_height - 1;
        _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y49) * input_stride + gid_x);
        int _gid_y50 = gid_y + 6 * (int)blockDim.y + (-3);
        if (_gid_y50 >= input_height)
            _gid_y50 = input_height - 1;
        _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y50) * input_stride + gid_x);
        int _gid_y51 = gid_y + 7 * (int)blockDim.y + (-3);
        if (_gid_y51 >= input_height)
            _gid_y51 = input_height - 1;
        _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y51) * input_stride + gid_x);
        int _gid_y52 = gid_y + 8 * (int)blockDim.y + (-3);
        if (_gid_y52 >= input_height)
            _gid_y52 = input_height - 1;
        _smeminput[(int)threadIdx.y + 8 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y52) * input_stride + gid_x);
        int _gid_y53 = gid_y + 9 * (int)blockDim.y + (-3);
        if (_gid_y53 >= input_height)
            _gid_y53 = input_height - 1;
        _smeminput[(int)threadIdx.y + 9 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y53) * input_stride + gid_x);
        int _gid_y54 = gid_y + 10 * (int)blockDim.y + (-3);
        if (_gid_y54 >= input_height)
            _gid_y54 = input_height - 1;
        _smeminput[(int)threadIdx.y + 10 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y54) * input_stride + gid_x);
        int _gid_y55 = gid_y + 11 * (int)blockDim.y + (-3);
        if (_gid_y55 >= input_height)
            _gid_y55 = input_height - 1;
        _smeminput[(int)threadIdx.y + 11 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y55) * input_stride + gid_x);
        int _gid_y56 = gid_y + 12 * (int)blockDim.y + (-3);
        if (_gid_y56 >= input_height)
            _gid_y56 = input_height - 1;
        _smeminput[(int)threadIdx.y + 12 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y56) * input_stride + gid_x);
        int _gid_y57 = gid_y + 13 * (int)blockDim.y + (-3);
        if (_gid_y57 >= input_height)
            _gid_y57 = input_height - 1;
        _smeminput[(int)threadIdx.y + 13 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y57) * input_stride + gid_x);
        __syncthreads();
        if (gid_x < iter_width) {
            if (gid_y < iter_height) {
                float _tmp58 = 0.F;
                {
                    _tmp58 += _constmask_yY[0][0] * _smeminput[(int)threadIdx.y + -3 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp58 += _constmask_yY[1][0] * _smeminput[(int)threadIdx.y + -2 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp58 += _constmask_yY[2][0] * _smeminput[(int)threadIdx.y + -1 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp58 += _constmask_yY[3][0] * _smeminput[(int)threadIdx.y + 0 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp58 += _constmask_yY[4][0] * _smeminput[(int)threadIdx.y + 1 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp58 += _constmask_yY[5][0] * _smeminput[(int)threadIdx.y + 2 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp58 += _constmask_yY[6][0] * _smeminput[(int)threadIdx.y + 3 + 3][(int)threadIdx.x + 0 + 0];
                }
                iter[(gid_y) * iter_stride + gid_x] = (uchar)(_tmp58 + 0.5F);
            }
        }
        if (gid_x < iter_width) {
            if (gid_y + 1 * (int)blockDim.y < iter_height) {
                float _tmp59 = 0.F;
                {
                    _tmp59 += _constmask_yY[0][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + -3 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp59 += _constmask_yY[1][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + -2 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp59 += _constmask_yY[2][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + -1 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp59 += _constmask_yY[3][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp59 += _constmask_yY[4][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 1 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp59 += _constmask_yY[5][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 2 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp59 += _constmask_yY[6][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 3 + 3][(int)threadIdx.x + 0 + 0];
                }
                iter[(gid_y + 1 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp59 + 0.5F);
            }
        }
        if (gid_x < iter_width) {
            if (gid_y + 2 * (int)blockDim.y < iter_height) {
                float _tmp60 = 0.F;
                {
                    _tmp60 += _constmask_yY[0][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + -3 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp60 += _constmask_yY[1][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + -2 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp60 += _constmask_yY[2][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + -1 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp60 += _constmask_yY[3][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp60 += _constmask_yY[4][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 1 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp60 += _constmask_yY[5][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 2 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp60 += _constmask_yY[6][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 3 + 3][(int)threadIdx.x + 0 + 0];
                }
                iter[(gid_y + 2 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp60 + 0.5F);
            }
        }
        if (gid_x < iter_width) {
            if (gid_y + 3 * (int)blockDim.y < iter_height) {
                float _tmp61 = 0.F;
                {
                    _tmp61 += _constmask_yY[0][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + -3 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp61 += _constmask_yY[1][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + -2 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp61 += _constmask_yY[2][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + -1 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp61 += _constmask_yY[3][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp61 += _constmask_yY[4][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 1 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp61 += _constmask_yY[5][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 2 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp61 += _constmask_yY[6][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 3 + 3][(int)threadIdx.x + 0 + 0];
                }
                iter[(gid_y + 3 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp61 + 0.5F);
            }
        }
        if (gid_x < iter_width) {
            if (gid_y + 4 * (int)blockDim.y < iter_height) {
                float _tmp62 = 0.F;
                {
                    _tmp62 += _constmask_yY[0][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + -3 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp62 += _constmask_yY[1][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + -2 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp62 += _constmask_yY[2][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + -1 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp62 += _constmask_yY[3][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp62 += _constmask_yY[4][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 1 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp62 += _constmask_yY[5][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 2 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp62 += _constmask_yY[6][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 3 + 3][(int)threadIdx.x + 0 + 0];
                }
                iter[(gid_y + 4 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp62 + 0.5F);
            }
        }
        if (gid_x < iter_width) {
            if (gid_y + 5 * (int)blockDim.y < iter_height) {
                float _tmp63 = 0.F;
                {
                    _tmp63 += _constmask_yY[0][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + -3 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp63 += _constmask_yY[1][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + -2 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp63 += _constmask_yY[2][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + -1 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp63 += _constmask_yY[3][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp63 += _constmask_yY[4][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 1 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp63 += _constmask_yY[5][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 2 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp63 += _constmask_yY[6][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 3 + 3][(int)threadIdx.x + 0 + 0];
                }
                iter[(gid_y + 5 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp63 + 0.5F);
            }
        }
        if (gid_x < iter_width) {
            if (gid_y + 6 * (int)blockDim.y < iter_height) {
                float _tmp64 = 0.F;
                {
                    _tmp64 += _constmask_yY[0][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + -3 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp64 += _constmask_yY[1][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + -2 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp64 += _constmask_yY[2][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + -1 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp64 += _constmask_yY[3][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp64 += _constmask_yY[4][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 1 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp64 += _constmask_yY[5][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 2 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp64 += _constmask_yY[6][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 3 + 3][(int)threadIdx.x + 0 + 0];
                }
                iter[(gid_y + 6 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp64 + 0.5F);
            }
        }
        if (gid_x < iter_width) {
            if (gid_y + 7 * (int)blockDim.y < iter_height) {
                float _tmp65 = 0.F;
                {
                    _tmp65 += _constmask_yY[0][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + -3 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp65 += _constmask_yY[1][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + -2 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp65 += _constmask_yY[2][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + -1 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp65 += _constmask_yY[3][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp65 += _constmask_yY[4][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 1 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp65 += _constmask_yY[5][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 2 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp65 += _constmask_yY[6][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 3 + 3][(int)threadIdx.x + 0 + 0];
                }
                iter[(gid_y + 7 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp65 + 0.5F);
            }
        }
    }
    goto BH_EXIT;
  BH_R:
    {
        int _gid_y66 = gid_y + (-3);
        _smeminput[(int)threadIdx.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y66) * input_stride + gid_x);
        int _gid_y67 = gid_y + 1 * (int)blockDim.y + (-3);
        _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y67) * input_stride + gid_x);
        int _gid_y68 = gid_y + 2 * (int)blockDim.y + (-3);
        _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y68) * input_stride + gid_x);
        int _gid_y69 = gid_y + 3 * (int)blockDim.y + (-3);
        _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y69) * input_stride + gid_x);
        int _gid_y70 = gid_y + 4 * (int)blockDim.y + (-3);
        _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y70) * input_stride + gid_x);
        int _gid_y71 = gid_y + 5 * (int)blockDim.y + (-3);
        _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y71) * input_stride + gid_x);
        int _gid_y72 = gid_y + 6 * (int)blockDim.y + (-3);
        _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y72) * input_stride + gid_x);
        int _gid_y73 = gid_y + 7 * (int)blockDim.y + (-3);
        _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y73) * input_stride + gid_x);
        int _gid_y74 = gid_y + 8 * (int)blockDim.y + (-3);
        _smeminput[(int)threadIdx.y + 8 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y74) * input_stride + gid_x);
        int _gid_y75 = gid_y + 9 * (int)blockDim.y + (-3);
        _smeminput[(int)threadIdx.y + 9 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y75) * input_stride + gid_x);
        int _gid_y76 = gid_y + 10 * (int)blockDim.y + (-3);
        _smeminput[(int)threadIdx.y + 10 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y76) * input_stride + gid_x);
        int _gid_y77 = gid_y + 11 * (int)blockDim.y + (-3);
        _smeminput[(int)threadIdx.y + 11 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y77) * input_stride + gid_x);
        int _gid_y78 = gid_y + 12 * (int)blockDim.y + (-3);
        _smeminput[(int)threadIdx.y + 12 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y78) * input_stride + gid_x);
        int _gid_y79 = gid_y + 13 * (int)blockDim.y + (-3);
        _smeminput[(int)threadIdx.y + 13 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (_gid_y79) * input_stride + gid_x);
        __syncthreads();
        if (gid_x < iter_width) {
            {
                float _tmp80 = 0.F;
                {
                    _tmp80 += _constmask_yY[0][0] * _smeminput[(int)threadIdx.y + -3 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp80 += _constmask_yY[1][0] * _smeminput[(int)threadIdx.y + -2 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp80 += _constmask_yY[2][0] * _smeminput[(int)threadIdx.y + -1 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp80 += _constmask_yY[3][0] * _smeminput[(int)threadIdx.y + 0 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp80 += _constmask_yY[4][0] * _smeminput[(int)threadIdx.y + 1 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp80 += _constmask_yY[5][0] * _smeminput[(int)threadIdx.y + 2 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp80 += _constmask_yY[6][0] * _smeminput[(int)threadIdx.y + 3 + 3][(int)threadIdx.x + 0 + 0];
                }
                iter[(gid_y) * iter_stride + gid_x] = (uchar)(_tmp80 + 0.5F);
            }
        }
        if (gid_x < iter_width) {
            {
                float _tmp81 = 0.F;
                {
                    _tmp81 += _constmask_yY[0][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + -3 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp81 += _constmask_yY[1][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + -2 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp81 += _constmask_yY[2][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + -1 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp81 += _constmask_yY[3][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp81 += _constmask_yY[4][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 1 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp81 += _constmask_yY[5][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 2 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp81 += _constmask_yY[6][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 3 + 3][(int)threadIdx.x + 0 + 0];
                }
                iter[(gid_y + 1 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp81 + 0.5F);
            }
        }
        if (gid_x < iter_width) {
            {
                float _tmp82 = 0.F;
                {
                    _tmp82 += _constmask_yY[0][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + -3 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp82 += _constmask_yY[1][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + -2 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp82 += _constmask_yY[2][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + -1 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp82 += _constmask_yY[3][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp82 += _constmask_yY[4][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 1 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp82 += _constmask_yY[5][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 2 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp82 += _constmask_yY[6][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 3 + 3][(int)threadIdx.x + 0 + 0];
                }
                iter[(gid_y + 2 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp82 + 0.5F);
            }
        }
        if (gid_x < iter_width) {
            {
                float _tmp83 = 0.F;
                {
                    _tmp83 += _constmask_yY[0][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + -3 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp83 += _constmask_yY[1][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + -2 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp83 += _constmask_yY[2][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + -1 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp83 += _constmask_yY[3][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp83 += _constmask_yY[4][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 1 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp83 += _constmask_yY[5][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 2 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp83 += _constmask_yY[6][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 3 + 3][(int)threadIdx.x + 0 + 0];
                }
                iter[(gid_y + 3 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp83 + 0.5F);
            }
        }
        if (gid_x < iter_width) {
            {
                float _tmp84 = 0.F;
                {
                    _tmp84 += _constmask_yY[0][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + -3 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp84 += _constmask_yY[1][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + -2 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp84 += _constmask_yY[2][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + -1 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp84 += _constmask_yY[3][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp84 += _constmask_yY[4][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 1 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp84 += _constmask_yY[5][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 2 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp84 += _constmask_yY[6][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 3 + 3][(int)threadIdx.x + 0 + 0];
                }
                iter[(gid_y + 4 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp84 + 0.5F);
            }
        }
        if (gid_x < iter_width) {
            {
                float _tmp85 = 0.F;
                {
                    _tmp85 += _constmask_yY[0][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + -3 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp85 += _constmask_yY[1][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + -2 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp85 += _constmask_yY[2][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + -1 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp85 += _constmask_yY[3][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp85 += _constmask_yY[4][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 1 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp85 += _constmask_yY[5][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 2 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp85 += _constmask_yY[6][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 3 + 3][(int)threadIdx.x + 0 + 0];
                }
                iter[(gid_y + 5 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp85 + 0.5F);
            }
        }
        if (gid_x < iter_width) {
            {
                float _tmp86 = 0.F;
                {
                    _tmp86 += _constmask_yY[0][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + -3 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp86 += _constmask_yY[1][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + -2 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp86 += _constmask_yY[2][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + -1 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp86 += _constmask_yY[3][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp86 += _constmask_yY[4][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 1 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp86 += _constmask_yY[5][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 2 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp86 += _constmask_yY[6][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 3 + 3][(int)threadIdx.x + 0 + 0];
                }
                iter[(gid_y + 6 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp86 + 0.5F);
            }
        }
        if (gid_x < iter_width) {
            {
                float _tmp87 = 0.F;
                {
                    _tmp87 += _constmask_yY[0][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + -3 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp87 += _constmask_yY[1][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + -2 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp87 += _constmask_yY[2][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + -1 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp87 += _constmask_yY[3][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp87 += _constmask_yY[4][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 1 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp87 += _constmask_yY[5][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 2 + 3][(int)threadIdx.x + 0 + 0];
                }
                {
                    _tmp87 += _constmask_yY[6][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 3 + 3][(int)threadIdx.x + 0 + 0];
                }
                iter[(gid_y + 7 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp87 + 0.5F);
            }
        }
    }
    goto BH_EXIT;
  BH_NO:
    {
        _smeminput[(int)threadIdx.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (gid_y + (-3)) * input_stride + gid_x);
        _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (gid_y + 1 * (int)blockDim.y + (-3)) * input_stride + gid_x);
        _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (gid_y + 2 * (int)blockDim.y + (-3)) * input_stride + gid_x);
        _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (gid_y + 3 * (int)blockDim.y + (-3)) * input_stride + gid_x);
        _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (gid_y + 4 * (int)blockDim.y + (-3)) * input_stride + gid_x);
        _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (gid_y + 5 * (int)blockDim.y + (-3)) * input_stride + gid_x);
        _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (gid_y + 6 * (int)blockDim.y + (-3)) * input_stride + gid_x);
        _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (gid_y + 7 * (int)blockDim.y + (-3)) * input_stride + gid_x);
        _smeminput[(int)threadIdx.y + 8 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (gid_y + 8 * (int)blockDim.y + (-3)) * input_stride + gid_x);
        _smeminput[(int)threadIdx.y + 9 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (gid_y + 9 * (int)blockDim.y + (-3)) * input_stride + gid_x);
        _smeminput[(int)threadIdx.y + 10 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (gid_y + 10 * (int)blockDim.y + (-3)) * input_stride + gid_x);
        _smeminput[(int)threadIdx.y + 11 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (gid_y + 11 * (int)blockDim.y + (-3)) * input_stride + gid_x);
        _smeminput[(int)threadIdx.y + 12 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (gid_y + 12 * (int)blockDim.y + (-3)) * input_stride + gid_x);
        _smeminput[(int)threadIdx.y + 13 * (int)blockDim.y][(int)threadIdx.x] = tex1Dfetch(_texinputY, (gid_y + 13 * (int)blockDim.y + (-3)) * input_stride + gid_x);
        __syncthreads();
        {
            float _tmp88 = 0.F;
            {
                _tmp88 += _constmask_yY[0][0] * _smeminput[(int)threadIdx.y + -3 + 3][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp88 += _constmask_yY[1][0] * _smeminput[(int)threadIdx.y + -2 + 3][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp88 += _constmask_yY[2][0] * _smeminput[(int)threadIdx.y + -1 + 3][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp88 += _constmask_yY[3][0] * _smeminput[(int)threadIdx.y + 0 + 3][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp88 += _constmask_yY[4][0] * _smeminput[(int)threadIdx.y + 1 + 3][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp88 += _constmask_yY[5][0] * _smeminput[(int)threadIdx.y + 2 + 3][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp88 += _constmask_yY[6][0] * _smeminput[(int)threadIdx.y + 3 + 3][(int)threadIdx.x + 0 + 0];
            }
            iter[(gid_y) * iter_stride + gid_x] = (uchar)(_tmp88 + 0.5F);
        }
        {
            float _tmp89 = 0.F;
            {
                _tmp89 += _constmask_yY[0][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + -3 + 3][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp89 += _constmask_yY[1][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + -2 + 3][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp89 += _constmask_yY[2][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + -1 + 3][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp89 += _constmask_yY[3][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 0 + 3][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp89 += _constmask_yY[4][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 1 + 3][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp89 += _constmask_yY[5][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 2 + 3][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp89 += _constmask_yY[6][0] * _smeminput[(int)threadIdx.y + 1 * (int)blockDim.y + 3 + 3][(int)threadIdx.x + 0 + 0];
            }
            iter[(gid_y + 1 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp89 + 0.5F);
        }
        {
            float _tmp90 = 0.F;
            {
                _tmp90 += _constmask_yY[0][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + -3 + 3][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp90 += _constmask_yY[1][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + -2 + 3][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp90 += _constmask_yY[2][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + -1 + 3][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp90 += _constmask_yY[3][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 0 + 3][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp90 += _constmask_yY[4][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 1 + 3][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp90 += _constmask_yY[5][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 2 + 3][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp90 += _constmask_yY[6][0] * _smeminput[(int)threadIdx.y + 2 * (int)blockDim.y + 3 + 3][(int)threadIdx.x + 0 + 0];
            }
            iter[(gid_y + 2 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp90 + 0.5F);
        }
        {
            float _tmp91 = 0.F;
            {
                _tmp91 += _constmask_yY[0][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + -3 + 3][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp91 += _constmask_yY[1][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + -2 + 3][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp91 += _constmask_yY[2][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + -1 + 3][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp91 += _constmask_yY[3][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 0 + 3][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp91 += _constmask_yY[4][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 1 + 3][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp91 += _constmask_yY[5][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 2 + 3][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp91 += _constmask_yY[6][0] * _smeminput[(int)threadIdx.y + 3 * (int)blockDim.y + 3 + 3][(int)threadIdx.x + 0 + 0];
            }
            iter[(gid_y + 3 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp91 + 0.5F);
        }
        {
            float _tmp92 = 0.F;
            {
                _tmp92 += _constmask_yY[0][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + -3 + 3][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp92 += _constmask_yY[1][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + -2 + 3][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp92 += _constmask_yY[2][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + -1 + 3][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp92 += _constmask_yY[3][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 0 + 3][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp92 += _constmask_yY[4][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 1 + 3][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp92 += _constmask_yY[5][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 2 + 3][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp92 += _constmask_yY[6][0] * _smeminput[(int)threadIdx.y + 4 * (int)blockDim.y + 3 + 3][(int)threadIdx.x + 0 + 0];
            }
            iter[(gid_y + 4 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp92 + 0.5F);
        }
        {
            float _tmp93 = 0.F;
            {
                _tmp93 += _constmask_yY[0][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + -3 + 3][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp93 += _constmask_yY[1][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + -2 + 3][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp93 += _constmask_yY[2][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + -1 + 3][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp93 += _constmask_yY[3][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 0 + 3][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp93 += _constmask_yY[4][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 1 + 3][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp93 += _constmask_yY[5][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 2 + 3][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp93 += _constmask_yY[6][0] * _smeminput[(int)threadIdx.y + 5 * (int)blockDim.y + 3 + 3][(int)threadIdx.x + 0 + 0];
            }
            iter[(gid_y + 5 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp93 + 0.5F);
        }
        {
            float _tmp94 = 0.F;
            {
                _tmp94 += _constmask_yY[0][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + -3 + 3][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp94 += _constmask_yY[1][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + -2 + 3][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp94 += _constmask_yY[2][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + -1 + 3][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp94 += _constmask_yY[3][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 0 + 3][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp94 += _constmask_yY[4][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 1 + 3][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp94 += _constmask_yY[5][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 2 + 3][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp94 += _constmask_yY[6][0] * _smeminput[(int)threadIdx.y + 6 * (int)blockDim.y + 3 + 3][(int)threadIdx.x + 0 + 0];
            }
            iter[(gid_y + 6 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp94 + 0.5F);
        }
        {
            float _tmp95 = 0.F;
            {
                _tmp95 += _constmask_yY[0][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + -3 + 3][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp95 += _constmask_yY[1][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + -2 + 3][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp95 += _constmask_yY[2][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + -1 + 3][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp95 += _constmask_yY[3][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 0 + 3][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp95 += _constmask_yY[4][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 1 + 3][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp95 += _constmask_yY[5][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 2 + 3][(int)threadIdx.x + 0 + 0];
            }
            {
                _tmp95 += _constmask_yY[6][0] * _smeminput[(int)threadIdx.y + 7 * (int)blockDim.y + 3 + 3][(int)threadIdx.x + 0 + 0];
            }
            iter[(gid_y + 7 * (int)blockDim.y) * iter_stride + gid_x] = (uchar)(_tmp95 + 0.5F);
        }
    }
    goto BH_EXIT;
  BH_EXIT:
    ;
}
}

#endif //_CUGAUSSIANFILTERCOLUMNY_CU_

