#include "hipacc_cu.hpp"

#include "cuSobelFilterS.cu"
#include <algorithm>
#include <cmath>
#include <cstdlib>
#include <iostream>
#include <vector>
#include <sys/time.h>
#include <opencv2/opencv.hpp>
//---

#ifndef BH_MODE
#  define BH_MODE           CLAMP
#endif

#define PIXEL_CAST(a)       (pixel_t)(a)
#define TMP_PIXEL_CAST(a)   (tmp_pixel_t)(a)

#define pixel_t             uchar
#define tmp_pixel_t         float

#define USE_LAMBDA


// Obtêm o timestamp
double time_ms () {
  struct timeval tv;
  gettimeofday (&tv, NULL);
  return ((double)(tv.tv_sec) * 1e+3 + (double)(tv.tv_usec) * 1e-3);
}

/* Kernel do filtro Sobel */

int main(int argc, const char *argv[]) {
    hipaccInitCUDA();
    
    cv::Mat frame;                        /* Matriz OpenCV */
    std::string outputfn;                 /* Nome do arquivo resultado */
    std::vector<int> compression_params;  /* Vetor de parâmetros de compressão JPEG */
    float timing = 0;                     /* Tempo */
    unsigned int width, height;           /* Dimensões da imagem */

    /* Verifica parâmetros */
    if(argc < 2) {
      std::cerr << "Uso: " << argv[0] << " [imagem de entrada]" << std::endl;
      return 0;
    }

    /* Coeficientes de filtros */
    float filter_x[3][3] = {
      {-1,   0,   1},
      {-2,   0,   2},
      {-1,   0,   1}
    };

    float filter_y[3][3] = {
      {-1,  -2,  -1},
      { 0,   0,   0},
      { 1,   2,   1}
    };

    /* Carrega imagem do arquivo e converte-a para escala de cinza */
    frame = cv::imread(argv[1], CV_LOAD_IMAGE_COLOR);
    cvtColor(frame, frame, CV_BGR2GRAY);

    /* Define dimensões da imagem */
    width = frame.cols;
    height = frame.rows;

    /* Imagens de entrada e saída HIPACC */
    HipaccImage input = hipaccCreateMemory<uchar>(NULL, width, height, 256);
    HipaccImage output = hipaccCreateMemory<uchar>(NULL, width, height, 256);

    /* Atribui o buffer de entrada da imagem à imagem de entrada do HIPACC */
    hipaccWriteMemory(input, frame.data);

    /* Define as máscaras usando as matrizes de coeficientes */
    hipaccWriteSymbol<float>((const void *)&_constmask_xS, (float *)filter_x, 3, 3);
    hipaccWriteSymbol<float>((const void *)&_constmask_yS, (float *)filter_y, 3, 3);

    /* Espaço de iteração da imagem de saída */
    HipaccAccessor iter_out(output);

    /* Condições de fronteira da imagem */
    
    HipaccAccessor acc(input);

    /* Especificação e execução do kernel */
    
    hipacc_launch_info S_info0(1, 1, iter_out, 8, 1);
    dim3 block0(32, 1);
    dim3 grid0(hipaccCalcGridFromBlock(S_info0, block0));

    hipaccPrepareKernelLaunch(S_info0, block0);
    hipaccConfigureCall(grid0, block0);

    size_t offset0 = 0;
    hipGetTextureReference(&_texinputSRef, HIP_SYMBOL(&_texinputS));
    hipaccBindTexture<uchar>(Linear1D, _texinputSRef, acc.img);
    hipaccSetupArgument(&iter_out.img.mem, sizeof(uchar *), offset0);
    hipaccSetupArgument(&iter_out.width, sizeof(const int), offset0);
    hipaccSetupArgument(&iter_out.height, sizeof(const int), offset0);
    hipaccSetupArgument(&iter_out.img.stride, sizeof(const int), offset0);
    hipaccSetupArgument(&acc.width, sizeof(const int), offset0);
    hipaccSetupArgument(&acc.height, sizeof(const int), offset0);
    hipaccSetupArgument(&acc.img.stride, sizeof(const int), offset0);
    hipaccSetupArgument(&S_info0.bh_start_left, sizeof(const int), offset0);
    hipaccSetupArgument(&S_info0.bh_start_right, sizeof(const int), offset0);
    hipaccSetupArgument(&S_info0.bh_start_top, sizeof(const int), offset0);
    hipaccSetupArgument(&S_info0.bh_start_bottom, sizeof(const int), offset0);
    hipaccSetupArgument(&S_info0.bh_fall_back, sizeof(const int), offset0);
    
    hipaccLaunchKernel((const void *)&cuSobelFilterSKernel, "cuSobelFilterSKernel", grid0, block0);

    /* Calcula tempo de execução */
    timing = hipacc_last_kernel_timing();

    /* Exibe resultados */
    std::cerr << "Timing: " << timing << " ms, " << (width*height/timing)/1000 << " Mpixel/s" << std::endl;

    /* Carrega o resultado da imagem HIPACC para a matriz OpenCV */
    frame.data = hipaccReadMemory<uchar>(output);

    /* Gera o nome do arquivo de saída */
    outputfn = argv[1];
    outputfn = outputfn.substr(0, outputfn.find_last_of(".")) + "-filtered.jpg";

    /* Parâmetros de compressão/qualidade da imagem JPEG */
    compression_params.push_back(CV_IMWRITE_JPEG_QUALITY);
    compression_params.push_back(100);

    /* Escreve a imagem OpenCV */
    try {
      cv::imwrite(outputfn, frame, compression_params);
    } catch(std::runtime_error &ex) {
      std::cerr << "JPEG compression exception: " << ex.what() << std::endl;
      return 1;
    }

    /* Cria uma nova janela do OpenCV */
    cv::namedWindow("Result", cv::WINDOW_NORMAL);

    /* Exibe a imagem resultado na mesma */
    cv::imshow("Result", frame);

    /* Aguarda o usuário pressionar alguma tecla */
    cv::waitKey(0);

    fprintf(stdout, "Done!\n");
    hipaccReleaseMemory<uchar>(input);
    hipaccReleaseMemory<uchar>(output);
    return 0;
}
